/*
 * This file is part of the GROMACS molecular simulation package.
 *
 * Copyright (c) 2012,2014,2015,2016,2017, by the GROMACS development team, led by
 * Mark Abraham, David van der Spoel, Berk Hess, and Erik Lindahl,
 * and including many others, as listed in the AUTHORS file in the
 * top-level source directory and at http://www.gromacs.org.
 *
 * GROMACS is free software; you can redistribute it and/or
 * modify it under the terms of the GNU Lesser General Public License
 * as published by the Free Software Foundation; either version 2.1
 * of the License, or (at your option) any later version.
 *
 * GROMACS is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
 * Lesser General Public License for more details.
 *
 * You should have received a copy of the GNU Lesser General Public
 * License along with GROMACS; if not, see
 * http://www.gnu.org/licenses, or write to the Free Software Foundation,
 * Inc., 51 Franklin Street, Fifth Floor, Boston, MA  02110-1301  USA.
 *
 * If you want to redistribute modifications to GROMACS, please
 * consider that scientific software is very special. Version
 * control is crucial - bugs must be traceable. We will be happy to
 * consider code for inclusion in the official distribution, but
 * derived work must not be called official GROMACS. Details are found
 * in the README & COPYING files - if they are missing, get the
 * official version at http://www.gromacs.org.
 *
 * To help us fund GROMACS development, we humbly ask that you cite
 * the research papers on the package. Check out http://www.gromacs.org.
 */

#include "gmxpre.h"

#include "cudautils.cuh"

#include <cassert>
#include <cstdlib>

#include "gromacs/gpu_utils/cuda_arch_utils.cuh"
#include "gromacs/utility/smalloc.h"

/*** Generic CUDA data operation wrappers ***/

/*! Launches synchronous or asynchronous host to device memory copy.
 *
 *  The copy is launched in stream s or if not specified, in stream 0.
 */
static int cu_copy_D2H_generic(void * h_dest, void * d_src, size_t bytes,
                               bool bAsync = false, hipStream_t s = 0)
{
    hipError_t stat;

    if (h_dest == NULL || d_src == NULL || bytes == 0)
    {
        return -1;
    }

    if (bAsync)
    {
        stat = hipMemcpyAsync(h_dest, d_src, bytes, hipMemcpyDeviceToHost, s);
        CU_RET_ERR(stat, "DtoH hipMemcpyAsync failed");

    }
    else
    {
        stat = hipMemcpy(h_dest, d_src, bytes, hipMemcpyDeviceToHost);
        CU_RET_ERR(stat, "DtoH hipMemcpy failed");
    }

    return 0;
}

int cu_copy_D2H(void * h_dest, void * d_src, size_t bytes)
{
    return cu_copy_D2H_generic(h_dest, d_src, bytes, false);
}

/*!
 *  The copy is launched in stream s or if not specified, in stream 0.
 */
int cu_copy_D2H_async(void * h_dest, void * d_src, size_t bytes, hipStream_t s = 0)
{
    return cu_copy_D2H_generic(h_dest, d_src, bytes, true, s);
}

/*! Launches synchronous or asynchronous device to host memory copy.
 *
 *  The copy is launched in stream s or if not specified, in stream 0.
 */
static int cu_copy_H2D_generic(void * d_dest, void * h_src, size_t bytes,
                               bool bAsync = false, hipStream_t s = 0)
{
    hipError_t stat;

    if (d_dest == NULL || h_src == NULL || bytes == 0)
    {
        return -1;
    }

    if (bAsync)
    {
        stat = hipMemcpyAsync(d_dest, h_src, bytes, hipMemcpyHostToDevice, s);
        CU_RET_ERR(stat, "HtoD hipMemcpyAsync failed");
    }
    else
    {
        stat = hipMemcpy(d_dest, h_src, bytes, hipMemcpyHostToDevice);
        CU_RET_ERR(stat, "HtoD hipMemcpy failed");
    }

    return 0;
}

int cu_copy_H2D(void * d_dest, void * h_src, size_t bytes)
{
    return cu_copy_H2D_generic(d_dest, h_src, bytes, false);
}

/*!
 *  The copy is launched in stream s or if not specified, in stream 0.
 */
int cu_copy_H2D_async(void * d_dest, void * h_src, size_t bytes, hipStream_t s = 0)
{
    return cu_copy_H2D_generic(d_dest, h_src, bytes, true, s);
}

float cu_event_elapsed(hipEvent_t start, hipEvent_t end)
{
    float       t = 0.0;
    hipError_t stat;

    stat = hipEventElapsedTime(&t, start, end);
    CU_RET_ERR(stat, "hipEventElapsedTime failed in cu_event_elapsed");

    return t;
}

int cu_wait_event(hipEvent_t e)
{
    hipError_t s;

    s = hipEventSynchronize(e);
    CU_RET_ERR(s, "hipEventSynchronize failed in cu_wait_event");

    return 0;
}

/*!
 *  If time != NULL it also calculates the time elapsed between start and end and
 *  return this is milliseconds.
 */
int cu_wait_event_time(hipEvent_t end, hipEvent_t start, float *time)
{
    hipError_t s;

    s = hipEventSynchronize(end);
    CU_RET_ERR(s, "hipEventSynchronize failed in cu_wait_event");

    if (time)
    {
        *time = cu_event_elapsed(start, end);
    }

    return 0;
}

/**** Operation on buffered arrays (arrays with "over-allocation" in gmx wording) *****/

/*!
 * If the pointers to the size variables are NULL no resetting happens.
 */
void cu_free_buffered(void *d_ptr, int *n, int *nalloc)
{
    hipError_t stat;

    if (d_ptr)
    {
        stat = hipFree(d_ptr);
        CU_RET_ERR(stat, "hipFree failed");
    }

    if (n)
    {
        *n = -1;
    }

    if (nalloc)
    {
        *nalloc = -1;
    }
}

/*!
 *  Reallocation of the memory pointed by d_ptr and copying of the data from
 *  the location pointed by h_src host-side pointer is done. Allocation is
 *  buffered and therefore freeing is only needed if the previously allocated
 *  space is not enough.
 *  The H2D copy is launched in stream s and can be done synchronously or
 *  asynchronously (the default is the latter).
 */
void cu_realloc_buffered(void **d_dest, void *h_src,
                         size_t type_size,
                         int *curr_size, int *curr_alloc_size,
                         int req_size,
                         hipStream_t s,
                         bool bAsync = true)
{
    hipError_t stat;

    if (d_dest == NULL || req_size < 0)
    {
        return;
    }

    /* reallocate only if the data does not fit = allocation size is smaller
       than the current requested size */
    if (req_size > *curr_alloc_size)
    {
        /* only free if the array has already been initialized */
        if (*curr_alloc_size >= 0)
        {
            cu_free_buffered(*d_dest, curr_size, curr_alloc_size);
        }

        *curr_alloc_size = over_alloc_large(req_size);

        stat = hipMalloc(d_dest, *curr_alloc_size * type_size);
        CU_RET_ERR(stat, "hipMalloc failed in cu_free_buffered");
    }

    /* size could have changed without actual reallocation */
    *curr_size = req_size;

    /* upload to device */
    if (h_src)
    {
        if (bAsync)
        {
            cu_copy_H2D_async(*d_dest, h_src, *curr_size * type_size, s);
        }
        else
        {
            cu_copy_H2D(*d_dest, h_src,  *curr_size * type_size);
        }
    }
}

bool use_texobj(const gmx_device_info_t *dev_info)
{
    assert(!c_disableCudaTextures);
    /* Only device CC >= 3.0 (Kepler and later) support texture objects */
    return (dev_info->prop.major >= 3);
}

/*! \brief Set up texture object for an array of type T.
 *
 * Set up texture object for an array of type T and bind it to the device memory
 * \p d_ptr points to.
 *
 * \tparam[in] T        Raw data type
 * \param[out] texObj   texture object to initialize
 * \param[in]  d_ptr    pointer to device global memory to bind \p texObj to
 * \param[in]  sizeInBytes  size of memory area to bind \p texObj to
 */
template <typename T>
static void setup1DTexture(hipTextureObject_t &texObj,
                           void                *d_ptr,
                           size_t               sizeInBytes)
{
    assert(!c_disableCudaTextures);

    hipError_t      stat;
    hipResourceDesc rd;
    hipTextureDesc  td;

    memset(&rd, 0, sizeof(rd));
    rd.resType                = hipResourceTypeLinear;
    rd.res.linear.devPtr      = d_ptr;
    rd.res.linear.desc        = hipCreateChannelDesc<T>();
    rd.res.linear.sizeInBytes = sizeInBytes;

    memset(&td, 0, sizeof(td));
    td.readMode                 = hipReadModeElementType;
    stat = hipCreateTextureObject(&texObj, &rd, &td, NULL);
    CU_RET_ERR(stat, "hipCreateTextureObject failed");
}

/*! \brief Set up texture reference for an array of type T.
 *
 * Set up texture object for an array of type T and bind it to the device memory
 * \p d_ptr points to.
 *
 * \tparam[in] T        Raw data type
 * \param[out] texObj   texture reference to initialize
 * \param[in]  d_ptr    pointer to device global memory to bind \p texObj to
 * \param[in]  sizeInBytes  size of memory area to bind \p texObj to
 */
template <typename T>
static void setup1DTexture(const struct texture<T, 1, hipReadModeElementType> *texRef,
                           const void                                          *d_ptr,
                           size_t                                              sizeInBytes)
{
    assert(!c_disableCudaTextures);

    hipError_t           stat;
    hipChannelFormatDesc cd;

    cd   = hipCreateChannelDesc<T>();
    stat = hipBindTexture(nullptr, texRef, d_ptr, &cd, sizeInBytes);
    CU_RET_ERR(stat, "hipBindTexture failed");
}

template <typename T>
void initParamLookupTable(T                        * &d_ptr,
                          hipTextureObject_t       &texObj,
                          const struct texture<T, 1, hipReadModeElementType> *texRef,
                          const T                   *h_ptr,
                          int                        numElem,
                          const gmx_device_info_t   *devInfo)
{
    const size_t sizeInBytes = numElem * sizeof(*d_ptr);
    hipError_t  stat        = hipMalloc((void **)&d_ptr, sizeInBytes);
    CU_RET_ERR(stat, "hipMalloc failed in initParamLookupTable");
    cu_copy_H2D(d_ptr, (void *)h_ptr, sizeInBytes);

    if (!c_disableCudaTextures)
    {
        if (use_texobj(devInfo))
        {
            setup1DTexture<T>(texObj, d_ptr, sizeInBytes);
        }
        else
        {
            setup1DTexture<T>(texRef, d_ptr, sizeInBytes);
        }
    }
}

//! Add explicit instantiations of initParamLookupTable() here as needed
template void initParamLookupTable<float>(float * &, hipTextureObject_t &, const texture<float, 1, hipReadModeElementType> *, const float *, int, const gmx_device_info_t *);
