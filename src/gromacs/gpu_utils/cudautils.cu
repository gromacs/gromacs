/*
 * This file is part of the GROMACS molecular simulation package.
 *
 * Copyright (c) 2012,2014,2015,2016,2017, by the GROMACS development team, led by
 * Mark Abraham, David van der Spoel, Berk Hess, and Erik Lindahl,
 * and including many others, as listed in the AUTHORS file in the
 * top-level source directory and at http://www.gromacs.org.
 *
 * GROMACS is free software; you can redistribute it and/or
 * modify it under the terms of the GNU Lesser General Public License
 * as published by the Free Software Foundation; either version 2.1
 * of the License, or (at your option) any later version.
 *
 * GROMACS is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
 * Lesser General Public License for more details.
 *
 * You should have received a copy of the GNU Lesser General Public
 * License along with GROMACS; if not, see
 * http://www.gnu.org/licenses, or write to the Free Software Foundation,
 * Inc., 51 Franklin Street, Fifth Floor, Boston, MA  02110-1301  USA.
 *
 * If you want to redistribute modifications to GROMACS, please
 * consider that scientific software is very special. Version
 * control is crucial - bugs must be traceable. We will be happy to
 * consider code for inclusion in the official distribution, but
 * derived work must not be called official GROMACS. Details are found
 * in the README & COPYING files - if they are missing, get the
 * official version at http://www.gromacs.org.
 *
 * To help us fund GROMACS development, we humbly ask that you cite
 * the research papers on the package. Check out http://www.gromacs.org.
 */

#include "gmxpre.h"

#include "cudautils.cuh"

#include <cassert>
#include <cstdlib>

#include "gromacs/gpu_utils/cuda_arch_utils.cuh"
#include "gromacs/gpu_utils/gpu_utils.h"
#include "gromacs/utility/gmxassert.h"
#include "gromacs/utility/smalloc.h"

/*** Generic CUDA data operation wrappers ***/

// TODO: template on transferKind to avoid runtime conditionals
int cu_copy_D2H(void *h_dest, void *d_src, size_t bytes,
                GpuApiCallBehavior transferKind, hipStream_t s = 0)
{
    hipError_t stat;

    if (h_dest == NULL || d_src == NULL || bytes == 0)
    {
        return -1;
    }

    switch (transferKind)
    {
        case GpuApiCallBehavior::Async:
            GMX_ASSERT(isHostMemoryPinned(h_dest), "Destination buffer was not pinned for CUDA");
            stat = hipMemcpyAsync(h_dest, d_src, bytes, hipMemcpyDeviceToHost, s);
            CU_RET_ERR(stat, "DtoH hipMemcpyAsync failed");
            break;

        case GpuApiCallBehavior::Sync:
            stat = hipMemcpy(h_dest, d_src, bytes, hipMemcpyDeviceToHost);
            CU_RET_ERR(stat, "DtoH hipMemcpy failed");
            break;

        default:
            throw;
    }

    return 0;
}

int cu_copy_D2H_sync(void * h_dest, void * d_src, size_t bytes)
{
    return cu_copy_D2H(h_dest, d_src, bytes, GpuApiCallBehavior::Sync);
}

/*!
 *  The copy is launched in stream s or if not specified, in stream 0.
 */
int cu_copy_D2H_async(void * h_dest, void * d_src, size_t bytes, hipStream_t s = 0)
{
    return cu_copy_D2H(h_dest, d_src, bytes, GpuApiCallBehavior::Async, s);
}

// TODO: template on transferKind to avoid runtime conditionals
int cu_copy_H2D(void *d_dest, void *h_src, size_t bytes,
                GpuApiCallBehavior transferKind, hipStream_t s = 0)
{
    hipError_t stat;

    if (d_dest == NULL || h_src == NULL || bytes == 0)
    {
        return -1;
    }

    switch (transferKind)
    {
        case GpuApiCallBehavior::Async:
            GMX_ASSERT(isHostMemoryPinned(h_src), "Source buffer was not pinned for CUDA");
            stat = hipMemcpyAsync(d_dest, h_src, bytes, hipMemcpyHostToDevice, s);
            CU_RET_ERR(stat, "HtoD hipMemcpyAsync failed");
            break;

        case GpuApiCallBehavior::Sync:
            stat = hipMemcpy(d_dest, h_src, bytes, hipMemcpyHostToDevice);
            CU_RET_ERR(stat, "HtoD hipMemcpy failed");
            break;

        default:
            throw;
    }

    return 0;
}

int cu_copy_H2D_sync(void * d_dest, void * h_src, size_t bytes)
{
    return cu_copy_H2D(d_dest, h_src, bytes, GpuApiCallBehavior::Sync);
}

/*!
 *  The copy is launched in stream s or if not specified, in stream 0.
 */
int cu_copy_H2D_async(void * d_dest, void * h_src, size_t bytes, hipStream_t s = 0)
{
    return cu_copy_H2D(d_dest, h_src, bytes, GpuApiCallBehavior::Async, s);
}

/**** Operation on buffered arrays (arrays with "over-allocation" in gmx wording) *****/

/*!
 * If the pointers to the size variables are NULL no resetting happens.
 */
void cu_free_buffered(void *d_ptr, int *n, int *nalloc)
{
    hipError_t stat;

    if (d_ptr)
    {
        stat = hipFree(d_ptr);
        CU_RET_ERR(stat, "hipFree failed");
    }

    if (n)
    {
        *n = -1;
    }

    if (nalloc)
    {
        *nalloc = -1;
    }
}

/*!
 *  Reallocation of the memory pointed by d_ptr and copying of the data from
 *  the location pointed by h_src host-side pointer is done. Allocation is
 *  buffered and therefore freeing is only needed if the previously allocated
 *  space is not enough.
 *  The H2D copy is launched in stream s and can be done synchronously or
 *  asynchronously (the default is the latter).
 */
void cu_realloc_buffered(void **d_dest, void *h_src,
                         size_t type_size,
                         int *curr_size, int *curr_alloc_size,
                         int req_size,
                         hipStream_t s,
                         bool bAsync = true)
{
    hipError_t stat;

    if (d_dest == NULL || req_size < 0)
    {
        return;
    }

    /* reallocate only if the data does not fit = allocation size is smaller
       than the current requested size */
    if (req_size > *curr_alloc_size)
    {
        /* only free if the array has already been initialized */
        if (*curr_alloc_size >= 0)
        {
            cu_free_buffered(*d_dest, curr_size, curr_alloc_size);
        }

        *curr_alloc_size = over_alloc_large(req_size);

        stat = hipMalloc(d_dest, *curr_alloc_size * type_size);
        CU_RET_ERR(stat, "hipMalloc failed in cu_free_buffered");
    }

    /* size could have changed without actual reallocation */
    *curr_size = req_size;

    /* upload to device */
    if (h_src)
    {
        if (bAsync)
        {
            cu_copy_H2D_async(*d_dest, h_src, *curr_size * type_size, s);
        }
        else
        {
            cu_copy_H2D_sync(*d_dest, h_src,  *curr_size * type_size);
        }
    }
}

/*! \brief Return whether texture objects are used on this device.
 *
 * \param[in]   pointer to the GPU device info structure to inspect for texture objects support
 * \return      true if texture objects are used on this device
 */
static inline bool use_texobj(const gmx_device_info_t *dev_info)
{
    assert(!c_disableCudaTextures);
    /* Only device CC >= 3.0 (Kepler and later) support texture objects */
    return (dev_info->prop.major >= 3);
}

/*! \brief Set up texture object for an array of type T.
 *
 * Set up texture object for an array of type T and bind it to the device memory
 * \p d_ptr points to.
 *
 * \tparam[in] T        Raw data type
 * \param[out] texObj   texture object to initialize
 * \param[in]  d_ptr    pointer to device global memory to bind \p texObj to
 * \param[in]  sizeInBytes  size of memory area to bind \p texObj to
 */
template <typename T>
static void setup1DTexture(hipTextureObject_t &texObj,
                           void                *d_ptr,
                           size_t               sizeInBytes)
{
    assert(!c_disableCudaTextures);

    hipError_t      stat;
    hipResourceDesc rd;
    hipTextureDesc  td;

    memset(&rd, 0, sizeof(rd));
    rd.resType                = hipResourceTypeLinear;
    rd.res.linear.devPtr      = d_ptr;
    rd.res.linear.desc        = hipCreateChannelDesc<T>();
    rd.res.linear.sizeInBytes = sizeInBytes;

    memset(&td, 0, sizeof(td));
    td.readMode                 = hipReadModeElementType;
    stat = hipCreateTextureObject(&texObj, &rd, &td, NULL);
    CU_RET_ERR(stat, "hipCreateTextureObject failed");
}

/*! \brief Set up texture reference for an array of type T.
 *
 * Set up texture object for an array of type T and bind it to the device memory
 * \p d_ptr points to.
 *
 * \tparam[in] T        Raw data type
 * \param[out] texObj   texture reference to initialize
 * \param[in]  d_ptr    pointer to device global memory to bind \p texObj to
 * \param[in]  sizeInBytes  size of memory area to bind \p texObj to
 */
template <typename T>
static void setup1DTexture(const struct texture<T, 1, hipReadModeElementType> *texRef,
                           const void                                          *d_ptr,
                           size_t                                              sizeInBytes)
{
    assert(!c_disableCudaTextures);

    hipError_t           stat;
    hipChannelFormatDesc cd;

    cd   = hipCreateChannelDesc<T>();
    stat = hipBindTexture(nullptr, texRef, d_ptr, &cd, sizeInBytes);
    CU_RET_ERR(stat, "hipBindTexture failed");
}

template <typename T>
void initParamLookupTable(T                        * &d_ptr,
                          hipTextureObject_t       &texObj,
                          const struct texture<T, 1, hipReadModeElementType> *texRef,
                          const T                   *h_ptr,
                          int                        numElem,
                          const gmx_device_info_t   *devInfo)
{
    const size_t sizeInBytes = numElem * sizeof(*d_ptr);
    hipError_t  stat        = hipMalloc((void **)&d_ptr, sizeInBytes);
    CU_RET_ERR(stat, "hipMalloc failed in initParamLookupTable");
    cu_copy_H2D_sync(d_ptr, (void *)h_ptr, sizeInBytes);

    if (!c_disableCudaTextures)
    {
        if (use_texobj(devInfo))
        {
            setup1DTexture<T>(texObj, d_ptr, sizeInBytes);
        }
        else
        {
            setup1DTexture<T>(texRef, d_ptr, sizeInBytes);
        }
    }
}

template <typename T>
void destroyParamLookupTable(T                       *d_ptr,
                             hipTextureObject_t      texObj,
                             const struct texture<T, 1, hipReadModeElementType> *texRef,
                             const gmx_device_info_t *devInfo)
{
    if (!c_disableCudaTextures)
    {
        if (use_texobj(devInfo))
        {
            CU_RET_ERR(hipDestroyTextureObject(texObj), "hipDestroyTextureObject on texObj failed");
        }
        else
        {
            CU_RET_ERR(hipUnbindTexture(texRef), "hipUnbindTexture on texRef failed");
        }
    }
    CU_RET_ERR(hipFree(d_ptr), "hipFree failed");
}

/*! \brief Add explicit instantiations of init/destroyParamLookupTable() here as needed.
 * One should also verify that the result of hipCreateChannelDesc<T>() during texture setup
 * looks reasonable, when instantiating the templates for new types - just in case.
 */
template void initParamLookupTable<float>(float * &, hipTextureObject_t &, const texture<float, 1, hipReadModeElementType> *, const float *, int, const gmx_device_info_t *);
template void destroyParamLookupTable<float>(float *, hipTextureObject_t, const texture<float, 1, hipReadModeElementType> *, const gmx_device_info_t *);
template void initParamLookupTable<int>(int * &, hipTextureObject_t &, const texture<int, 1, hipReadModeElementType> *, const int *, int, const gmx_device_info_t *);
template void destroyParamLookupTable<int>(int *, hipTextureObject_t, const texture<int, 1, hipReadModeElementType> *, const gmx_device_info_t *);
