#include "hip/hip_runtime.h"
/*
 * This file is part of the GROMACS molecular simulation package.
 *
 * Copyright (c) 2010,2011,2012,2013,2014,2015,2016, by the GROMACS development team, led by
 * Mark Abraham, David van der Spoel, Berk Hess, and Erik Lindahl,
 * and including many others, as listed in the AUTHORS file in the
 * top-level source directory and at http://www.gromacs.org.
 *
 * GROMACS is free software; you can redistribute it and/or
 * modify it under the terms of the GNU Lesser General Public License
 * as published by the Free Software Foundation; either version 2.1
 * of the License, or (at your option) any later version.
 *
 * GROMACS is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
 * Lesser General Public License for more details.
 *
 * You should have received a copy of the GNU Lesser General Public
 * License along with GROMACS; if not, see
 * http://www.gnu.org/licenses, or write to the Free Software Foundation,
 * Inc., 51 Franklin Street, Fifth Floor, Boston, MA  02110-1301  USA.
 *
 * If you want to redistribute modifications to GROMACS, please
 * consider that scientific software is very special. Version
 * control is crucial - bugs must be traceable. We will be happy to
 * consider code for inclusion in the official distribution, but
 * derived work must not be called official GROMACS. Details are found
 * in the README & COPYING files - if they are missing, get the
 * official version at http://www.gromacs.org.
 *
 * To help us fund GROMACS development, we humbly ask that you cite
 * the research papers on the package. Check out http://www.gromacs.org.
 */
/*! \file
 *  \brief Define functions for detection and initialization for CUDA devices.
 *
 *  \author Szilard Pall <pall.szilard@gmail.com>
 */

#include "gmxpre.h"

#include "gpu_utils.h"

#include <assert.h>
#include <stdio.h>
#include <stdlib.h>

#include <string>

#include <hip/hip_runtime_api.h>

#include "gromacs/gpu_utils/cudautils.cuh"
#include "gromacs/gpu_utils/pmalloc_cuda.h"
#include "gromacs/hardware/gpu_hw_info.h"
#include "gromacs/utility/basedefinitions.h"
#include "gromacs/utility/cstringutil.h"
#include "gromacs/utility/smalloc.h"
#include "gromacs/utility/stringutil.h"

/*! \internal \brief
 * Max number of devices supported by CUDA (for consistency checking).
 *
 * In reality it is 16 with CUDA <=v5.0, but let's stay on the safe side.
 */
static int  cuda_max_device_count = 32;

static bool cudaProfilerRun      = ((getenv("NVPROF_ID") != NULL));

/** Dummy kernel used for sanity checking. */
__global__ void k_dummy_test()
{
}


/*!
 * \brief Runs GPU sanity checks.
 *
 * Runs a series of checks to determine that the given GPU and underlying CUDA
 * driver/runtime functions properly.
 * Returns properties of a device with given ID or the one that has
 * already been initialized earlier in the case if of \dev_id == -1.
 *
 * \param[in]  dev_id      the device ID of the GPU or -1 if the device has already been initialized
 * \param[out] dev_prop    pointer to the structure in which the device properties will be returned
 * \returns                0 if the device looks OK
 *
 * TODO: introduce errors codes and handle errors more smoothly.
 */
static int do_sanity_checks(int dev_id, hipDeviceProp_t *dev_prop)
{
    hipError_t cu_err;
    int         dev_count, id;

    cu_err = hipGetDeviceCount(&dev_count);
    if (cu_err != hipSuccess)
    {
        fprintf(stderr, "Error %d while querying device count: %s\n", cu_err,
                hipGetErrorString(cu_err));
        return -1;
    }

    /* no CUDA compatible device at all */
    if (dev_count == 0)
    {
        return -1;
    }

    /* things might go horribly wrong if cudart is not compatible with the driver */
    if (dev_count < 0 || dev_count > cuda_max_device_count)
    {
        return -1;
    }

    if (dev_id == -1) /* device already selected let's not destroy the context */
    {
        cu_err = hipGetDevice(&id);
        if (cu_err != hipSuccess)
        {
            fprintf(stderr, "Error %d while querying device id: %s\n", cu_err,
                    hipGetErrorString(cu_err));
            return -1;
        }
    }
    else
    {
        id = dev_id;
        if (id > dev_count - 1) /* pfff there's no such device */
        {
            fprintf(stderr, "The requested device with id %d does not seem to exist (device count=%d)\n",
                    dev_id, dev_count);
            return -1;
        }
    }

    memset(dev_prop, 0, sizeof(hipDeviceProp_t));
    cu_err = hipGetDeviceProperties(dev_prop, id);
    if (cu_err != hipSuccess)
    {
        fprintf(stderr, "Error %d while querying device properties: %s\n", cu_err,
                hipGetErrorString(cu_err));
        return -1;
    }

    /* both major & minor is 9999 if no CUDA capable devices are present */
    if (dev_prop->major == 9999 && dev_prop->minor == 9999)
    {
        return -1;
    }
    /* we don't care about emulation mode */
    if (dev_prop->major == 0)
    {
        return -1;
    }

    if (id != -1)
    {
        cu_err = hipSetDevice(id);
        if (cu_err != hipSuccess)
        {
            fprintf(stderr, "Error %d while switching to device #%d: %s\n",
                    cu_err, id, hipGetErrorString(cu_err));
            return -1;
        }
    }

    /* try to execute a dummy kernel */
    k_dummy_test<<< 1, 512>>> ();
    if (hipDeviceSynchronize() != hipSuccess)
    {
        return -1;
    }

    /* destroy context if we created one */
    if (id != -1)
    {
        cu_err = hipDeviceReset();
        CU_RET_ERR(cu_err, "hipDeviceReset failed");
    }

    return 0;
}

gmx_bool init_gpu(int mygpu, std::string *errorMessage,
                  std::string *logMessage,
                  const struct gmx_gpu_info_t *gpu_info,
                  const struct gmx_gpu_opt_t *gpu_opt)
{
    hipError_t stat;
    char        sbuf[STRLEN];
    int         gpuid;

    assert(gpu_info);
    assert(errorMessage);
    assert(logMessage);

    if (mygpu < 0 || mygpu >= gpu_opt->n_dev_use)
    {
        sprintf(sbuf, "Trying to initialize an non-existent GPU: "
                "there are %d %s-selected GPU(s), but #%d was requested.",
                gpu_opt->n_dev_use, gpu_opt->bUserSet ? "user" : "auto", mygpu);
        gmx_incons(sbuf);
    }

    gpuid = gpu_info->gpu_dev[gpu_opt->dev_use[mygpu]].id;

    stat = hipSetDevice(gpuid);
    if (stat != hipSuccess)
    {
        errorMessage->assign(hipGetErrorString(stat));
        return false;
    }

    if (debug)
    {
        fprintf(stderr, "Initialized GPU ID #%d: %s\n", gpuid, gpu_info->gpu_dev[gpuid].prop.name);
    }

    try
    {
        gmx_device_info_t *device = &gpu_info->gpu_dev[gpuid];
        device->nvml.setup(device->prop, logMessage);
        if (!device->nvml.getClocksCanBeChanged())
        {
            return true;
        }
        device->nvml.changeClocks(logMessage);
    }
    catch (const gmx::NvmlException &e)
    {
        // Hardware or software failure associated with using NVML is
        // something we can just warn about before moving on, no need
        // for stopping the simulation.
    }
    // TODO In principle, we should call
    // GMX_CATCH_ALL_AND_EXIT_WITH_FATAL_ERROR here, because the rest
    // of init_gpu isn't exception safe, but that doesn't work until
    // we can use C++11 in host-side CUDA code. In practice, only
    // std::bad_alloc might leak, so this is not a big deal. However,
    // we do catch it appropriately in the caller.

    return true;
}

gmx_bool free_cuda_gpu(
        int gmx_unused mygpu, char *result_str,
        const gmx_gpu_info_t gmx_unused *gpu_info,
        const gmx_gpu_opt_t gmx_unused *gpu_opt
        )
{
    assert(result_str);

    if (debug)
    {
        int         gpuid;
        hipError_t stat = hipGetDevice(&gpuid);
        CU_RET_ERR(stat, "hipGetDevice failed");
        fprintf(stderr, "Cleaning up context on GPU ID #%d\n", gpuid);
    }

    int gpuid = gpu_opt ? gpu_opt->dev_use[mygpu] : -1;
    if (gpuid != -1)
    {
        gpu_info->gpu_dev[gpuid].nvml.resetClocks();
    }

    hipError_t stat = hipDeviceReset();
    strncpy(result_str, hipGetErrorString(stat), STRLEN);
    return (stat == hipSuccess);
}

/*! \brief Returns true if the gpu characterized by the device properties is
 *  supported by the native gpu acceleration.
 *
 * \param[in] dev_prop  the CUDA device properties of the gpus to test.
 * \returns             true if the GPU properties passed indicate a compatible
 *                      GPU, otherwise false.
 */
static bool is_gmx_supported_gpu(const hipDeviceProp_t *dev_prop)
{
    return (dev_prop->major >= 2);
}

/*! \brief Helper function that checks whether a given GPU status indicates compatible GPU.
 *
 * \param[in] stat  GPU status.
 * \returns         true if the provided status is egpuCompatible, otherwise false.
 */
static bool is_compatible_gpu(int stat)
{
    return (stat == egpuCompatible);
}

/*! \brief Checks if a GPU with a given ID is supported by the native GROMACS acceleration.
 *
 *  Returns a status value which indicates compatibility or one of the following
 *  errors: incompatibility, insistence, or insanity (=unexpected behavior).
 *  It also returns the respective device's properties in \dev_prop (if applicable).
 *
 *  \param[in]  dev_id   the ID of the GPU to check.
 *  \param[out] dev_prop the CUDA device properties of the device checked.
 *  \returns             the status of the requested device
 */
static int is_gmx_supported_gpu_id(int dev_id, hipDeviceProp_t *dev_prop)
{
    hipError_t stat;
    int         ndev;

    stat = hipGetDeviceCount(&ndev);
    if (stat != hipSuccess)
    {
        return egpuInsane;
    }

    if (dev_id > ndev - 1)
    {
        return egpuNonexistent;
    }

    /* TODO: currently we do not make a distinction between the type of errors
     * that can appear during sanity checks. This needs to be improved, e.g if
     * the dummy test kernel fails to execute with a "device busy message" we
     * should appropriately report that the device is busy instead of insane.
     */
    if (do_sanity_checks(dev_id, dev_prop) == 0)
    {
        if (is_gmx_supported_gpu(dev_prop))
        {
            return egpuCompatible;
        }
        else
        {
            return egpuIncompatible;
        }
    }
    else
    {
        return egpuInsane;
    }
}


int detect_gpus(gmx_gpu_info_t *gpu_info, char *err_str)
{
    int                i, ndev, checkres, retval;
    hipError_t        stat;
    hipDeviceProp_t     prop;
    gmx_device_info_t *devs;

    assert(gpu_info);
    assert(err_str);

    gpu_info->n_dev_compatible = 0;

    ndev    = 0;
    devs    = NULL;

    stat = hipGetDeviceCount(&ndev);
    if (stat != hipSuccess)
    {
        const char *s;

        /* hipGetDeviceCount failed which means that there is something
         * wrong with the machine: driver-runtime mismatch, all GPUs being
         * busy in exclusive mode, or some other condition which should
         * result in us issuing a warning a falling back to CPUs. */
        retval = -1;
        s      = hipGetErrorString(stat);
        strncpy(err_str, s, STRLEN*sizeof(err_str[0]));
    }
    else
    {
        snew(devs, ndev);
        for (i = 0; i < ndev; i++)
        {
            checkres = is_gmx_supported_gpu_id(i, &prop);

            devs[i].id   = i;
            devs[i].prop = prop;
            devs[i].stat = checkres;

            if (checkres == egpuCompatible)
            {
                gpu_info->n_dev_compatible++;
            }
        }
        retval = 0;
    }

    gpu_info->n_dev   = ndev;
    gpu_info->gpu_dev = devs;

    return retval;
}

void pick_compatible_gpus(const gmx_gpu_info_t *gpu_info,
                          gmx_gpu_opt_t        *gpu_opt)
{
    int  i, ncompat;
    int *compat;

    assert(gpu_info);
    /* gpu_dev/n_dev have to be either NULL/0 or not (NULL/0) */
    assert((gpu_info->n_dev != 0 ? 0 : 1) ^ (gpu_info->gpu_dev == NULL ? 0 : 1));

    snew(compat, gpu_info->n_dev);
    ncompat = 0;
    for (i = 0; i < gpu_info->n_dev; i++)
    {
        if (is_compatible_gpu(gpu_info->gpu_dev[i].stat))
        {
            ncompat++;
            compat[ncompat - 1] = i;
        }
    }

    gpu_opt->n_dev_compatible = ncompat;
    snew(gpu_opt->dev_compatible, ncompat);
    memcpy(gpu_opt->dev_compatible, compat, ncompat*sizeof(*compat));
    sfree(compat);
}

gmx_bool check_selected_gpus(int                  *checkres,
                             const gmx_gpu_info_t *gpu_info,
                             gmx_gpu_opt_t        *gpu_opt)
{
    int  i, id;
    bool bAllOk;

    assert(checkres);
    assert(gpu_info);
    assert(gpu_opt->n_dev_use >= 0);

    if (gpu_opt->n_dev_use == 0)
    {
        return TRUE;
    }

    assert(gpu_opt->dev_use);

    /* we will assume that all GPUs requested are valid IDs,
       otherwise we'll bail anyways */

    bAllOk = true;
    for (i = 0; i < gpu_opt->n_dev_use; i++)
    {
        id = gpu_opt->dev_use[i];

        /* devices are stored in increasing order of IDs in gpu_dev */
        gpu_opt->dev_use[i] = id;

        checkres[i] = (id >= gpu_info->n_dev) ?
            egpuNonexistent : gpu_info->gpu_dev[id].stat;

        bAllOk = bAllOk && is_compatible_gpu(checkres[i]);
    }

    return bAllOk;
}

void free_gpu_info(const gmx_gpu_info_t *gpu_info)
{
    if (gpu_info == NULL)
    {
        return;
    }

    sfree(gpu_info->gpu_dev);
}

void get_gpu_device_info_string(char *s, const gmx_gpu_info_t *gpu_info, int index)
{
    assert(s);
    assert(gpu_info);

    if (index < 0 && index >= gpu_info->n_dev)
    {
        return;
    }

    gmx_device_info_t *dinfo = &gpu_info->gpu_dev[index];

    bool               bGpuExists =
        dinfo->stat == egpuCompatible ||
        dinfo->stat == egpuIncompatible;

    if (!bGpuExists)
    {
        sprintf(s, "#%d: %s, stat: %s",
                dinfo->id, "N/A",
                gpu_detect_res_str[dinfo->stat]);
    }
    else
    {
        sprintf(s, "#%d: NVIDIA %s, compute cap.: %d.%d, ECC: %3s, stat: %s",
                dinfo->id, dinfo->prop.name,
                dinfo->prop.major, dinfo->prop.minor,
                dinfo->prop.ECCEnabled ? "yes" : " no",
                gpu_detect_res_str[dinfo->stat]);
    }
}

int get_gpu_device_id(const gmx_gpu_info_t *gpu_info,
                      const gmx_gpu_opt_t  *gpu_opt,
                      int                   idx)
{
    assert(gpu_info);
    assert(gpu_opt);
    assert(idx >= 0 && idx < gpu_opt->n_dev_use);

    return gpu_info->gpu_dev[gpu_opt->dev_use[idx]].id;
}

int get_current_cuda_gpu_device_id(void)
{
    int gpuid;
    CU_RET_ERR(hipGetDevice(&gpuid), "hipGetDevice failed");

    return gpuid;
}

size_t sizeof_gpu_dev_info(void)
{
    return sizeof(gmx_device_info_t);
}

void gpu_set_host_malloc_and_free(bool               bUseGpuKernels,
                                  gmx_host_alloc_t **nb_alloc,
                                  gmx_host_free_t  **nb_free)
{
    if (bUseGpuKernels)
    {
        *nb_alloc = &pmalloc;
        *nb_free  = &pfree;
    }
    else
    {
        *nb_alloc = NULL;
        *nb_free  = NULL;
    }
}

void startGpuProfiler(void)
{
    /* The NVPROF_ID environment variable is set by nvprof and indicates that
       mdrun is executed in the CUDA profiler.
       If nvprof was run is with "--profile-from-start off", the profiler will
       be started here. This way we can avoid tracing the CUDA events from the
       first part of the run. Starting the profiler again does nothing.
     */
    if (cudaProfilerRun)
    {
        hipError_t stat;
        stat = hipProfilerStart();
        CU_RET_ERR(stat, "hipProfilerStart failed");
    }
}

void stopGpuProfiler(void)
{
    /* Stopping the nvidia here allows us to eliminate the subsequent
       API calls from the trace, e.g. uninitialization and cleanup. */
    if (cudaProfilerRun)
    {
        hipError_t stat;
        stat = hipProfilerStop();
        CU_RET_ERR(stat, "hipProfilerStop failed");
    }
}

void resetGpuProfiler(void)
{
    /* With CUDA <=7.5 the profiler can't be properly reset; we can only start
     *  the profiling here (can't stop it) which will achieve the desired effect if
     *  the run was started with the profiling disabled.
     *
     * TODO: add a stop (or replace it with reset) when this will work correctly in CUDA.
     * stopGpuProfiler();
     */
    if (cudaProfilerRun)
    {
        startGpuProfiler();
    }
}
