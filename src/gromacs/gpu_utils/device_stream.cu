/*
 * This file is part of the GROMACS molecular simulation package.
 *
 * Copyright 2020- The GROMACS Authors
 * and the project initiators Erik Lindahl, Berk Hess and David van der Spoel.
 * Consult the AUTHORS/COPYING files and https://www.gromacs.org for details.
 *
 * GROMACS is free software; you can redistribute it and/or
 * modify it under the terms of the GNU Lesser General Public License
 * as published by the Free Software Foundation; either version 2.1
 * of the License, or (at your option) any later version.
 *
 * GROMACS is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
 * Lesser General Public License for more details.
 *
 * You should have received a copy of the GNU Lesser General Public
 * License along with GROMACS; if not, see
 * https://www.gnu.org/licenses, or write to the Free Software Foundation,
 * Inc., 51 Franklin Street, Fifth Floor, Boston, MA  02110-1301  USA.
 *
 * If you want to redistribute modifications to GROMACS, please
 * consider that scientific software is very special. Version
 * control is crucial - bugs must be traceable. We will be happy to
 * consider code for inclusion in the official distribution, but
 * derived work must not be called official GROMACS. Details are found
 * in the README & COPYING files - if they are missing, get the
 * official version at https://www.gromacs.org.
 *
 * To help us fund GROMACS development, we humbly ask that you cite
 * the research papers on the package. Check out https://www.gromacs.org.
 */
/*! \internal \file
 *
 * \brief Implements the DeviceStream for CUDA.
 *
 * \author Artem Zhmurov <zhmurov@gmail.com>
 *
 * \ingroup module_gpu_utils
 */
#include "gmxpre.h"

#include "device_stream.h"

#include <cstdio>

#include "gromacs/gpu_utils/cudautils.cuh"
#include "gromacs/utility/exceptions.h"
#include "gromacs/utility/gmxassert.h"
#include "gromacs/utility/stringutil.h"

DeviceStream::DeviceStream(const DeviceContext& /* deviceContext */,
                           DeviceStreamPriority priority,
                           const bool /* useTiming */)
{
    hipError_t stat;
    // Note that the device we're running on does not have to
    // support priorities, because we are querying the priority
    // range, which in that case will be a single value.
    int lowestPriority;
    int highestPriority;
    stat = hipDeviceGetStreamPriorityRange(&lowestPriority, &highestPriority);
    gmx::checkDeviceError(stat, "Could not query CUDA stream priority range.");
    int middlePriority = (lowestPriority + highestPriority) / 2;

    if (priority == DeviceStreamPriority::Low)
    {
        stat = hipStreamCreateWithPriority(&stream_, hipStreamDefault, lowestPriority);
        gmx::checkDeviceError(stat, "Could not create CUDA stream with low priority.");
    }
    else if (priority == DeviceStreamPriority::Normal)
    {
        stat = hipStreamCreateWithPriority(&stream_, hipStreamDefault, middlePriority);
        gmx::checkDeviceError(stat, "Could not create CUDA streami with middle pirority.");
    }
    else if (priority == DeviceStreamPriority::High)
    {
        stat = hipStreamCreateWithPriority(&stream_, hipStreamDefault, highestPriority);
        gmx::checkDeviceError(stat, "Could not create CUDA stream with high priority.");
    }
}

DeviceStream::~DeviceStream()
{
    if (isValid())
    {
        hipError_t stat = hipStreamDestroy(stream_);
        if (stat != hipSuccess)
        {
            // Don't throw in the destructor, just print a warning
            std::fprintf(stderr,
                         "Failed to release CUDA stream. %s\n",
                         gmx::getDeviceErrorString(stat).c_str());
        }
        stream_ = nullptr;
    }
}

hipStream_t DeviceStream::stream() const
{
    return stream_;
}

bool DeviceStream::isValid() const
{
    return (stream_ != nullptr);
}

void DeviceStream::synchronize() const
{
    hipError_t stat = hipStreamSynchronize(stream_);
    GMX_RELEASE_ASSERT(stat == hipSuccess,
                       ("hipStreamSynchronize failed. " + gmx::getDeviceErrorString(stat)).c_str());
}

void issueClFlushInStream(const DeviceStream& /*deviceStream*/) {}
