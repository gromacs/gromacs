#include "hip/hip_runtime.h"
/*
 * This file is part of the GROMACS molecular simulation package.
 *
 * Copyright (c) 2018,2019, by the GROMACS development team, led by
 * Mark Abraham, David van der Spoel, Berk Hess, and Erik Lindahl,
 * and including many others, as listed in the AUTHORS file in the
 * top-level source directory and at http://www.gromacs.org.
 *
 * GROMACS is free software; you can redistribute it and/or
 * modify it under the terms of the GNU Lesser General Public License
 * as published by the Free Software Foundation; either version 2.1
 * of the License, or (at your option) any later version.
 *
 * GROMACS is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
 * Lesser General Public License for more details.
 *
 * You should have received a copy of the GNU Lesser General Public
 * License along with GROMACS; if not, see
 * http://www.gnu.org/licenses, or write to the Free Software Foundation,
 * Inc., 51 Franklin Street, Fifth Floor, Boston, MA  02110-1301  USA.
 *
 * If you want to redistribute modifications to GROMACS, please
 * consider that scientific software is very special. Version
 * control is crucial - bugs must be traceable. We will be happy to
 * consider code for inclusion in the official distribution, but
 * derived work must not be called official GROMACS. Details are found
 * in the README & COPYING files - if they are missing, get the
 * official version at http://www.gromacs.org.
 *
 * To help us fund GROMACS development, we humbly ask that you cite
 * the research papers on the package. Check out http://www.gromacs.org.
 */
/*! \internal \file
 *
 * \brief Implements CUDA bonded functionality
 *
 * \author Jon Vincent <jvincent@nvidia.com>
 * \author Magnus Lundborg <lundborg.magnus@gmail.com>
 * \author Berk Hess <hess@kth.se>
 * \author Szilárd Páll <pall.szilard@gmail.com>
 * \author Alan Gray <alang@nvidia.com>
 * \author Mark Abraham <mark.j.abraham@gmail.com>
 *
 * \ingroup module_listed_forces
 */

#include "gmxpre.h"

#include <cassert>

#include <hip/hip_math_constants.h>

#include "gromacs/gpu_utils/cudautils.cuh"
#include "gromacs/gpu_utils/gpu_vec.cuh"
#include "gromacs/listed_forces/gpubonded.h"
#include "gromacs/math/units.h"
#include "gromacs/mdlib/force_flags.h"
#include "gromacs/mdtypes/forcerec.h"
#include "gromacs/mdtypes/simulation_workload.h"
#include "gromacs/pbcutil/pbc.h"
#include "gromacs/pbcutil/pbc_aiuc_cuda.cuh"
#include "gromacs/utility/gmxassert.h"

#include "gpubonded_impl.h"

#if defined(_MSVC)
#    include <limits>
#endif

// CUDA threads per block
#define TPB_BONDED 256

/*-------------------------------- CUDA kernels-------------------------------- */
/*------------------------------------------------------------------------------*/

#define CUDA_DEG2RAD_F (HIP_PI_F / 180.0f)

/*---------------- BONDED CUDA kernels--------------*/

/* Harmonic */
__device__ __forceinline__ static void
           harmonic_gpu(const float kA, const float xA, const float x, float* V, float* F)
{
    constexpr float half = 0.5f;
    float           dx, dx2;

    dx  = x - xA;
    dx2 = dx * dx;

    *F = -kA * dx;
    *V = half * kA * dx2;
}

template<bool calcVir, bool calcEner>
__device__ void bonds_gpu(const int       i,
                          float*          vtot_loc,
                          const int       numBonds,
                          const t_iatom   d_forceatoms[],
                          const t_iparams d_forceparams[],
                          const float4    gm_xq[],
                          fvec            gm_f[],
                          fvec            sm_fShiftLoc[],
                          const PbcAiuc   pbcAiuc)
{
    if (i < numBonds)
    {
        int3 bondData = *(int3*)(d_forceatoms + 3 * i);
        int  type     = bondData.x;
        int  ai       = bondData.y;
        int  aj       = bondData.z;

        /* dx = xi - xj, corrected for periodic boundary conditions. */
        fvec dx;
        int  ki = pbcDxAiuc<calcVir>(pbcAiuc, gm_xq[ai], gm_xq[aj], dx);

        float dr2 = iprod_gpu(dx, dx);
        float dr  = sqrt(dr2);

        float vbond;
        float fbond;
        harmonic_gpu(d_forceparams[type].harmonic.krA, d_forceparams[type].harmonic.rA, dr, &vbond, &fbond);

        if (calcEner)
        {
            *vtot_loc += vbond;
        }

        if (dr2 != 0.0f)
        {
            fbond *= rsqrtf(dr2);

#pragma unroll
            for (int m = 0; m < DIM; m++)
            {
                float fij = fbond * dx[m];
                atomicAdd(&gm_f[ai][m], fij);
                atomicAdd(&gm_f[aj][m], -fij);
                if (calcVir && ki != CENTRAL)
                {
                    atomicAdd(&sm_fShiftLoc[ki][m], fij);
                    atomicAdd(&sm_fShiftLoc[CENTRAL][m], -fij);
                }
            }
        }
    }
}

template<bool returnShift>
__device__ __forceinline__ static float bond_angle_gpu(const float4   xi,
                                                       const float4   xj,
                                                       const float4   xk,
                                                       const PbcAiuc& pbcAiuc,
                                                       fvec           r_ij,
                                                       fvec           r_kj,
                                                       float*         costh,
                                                       int*           t1,
                                                       int*           t2)
/* Return value is the angle between the bonds i-j and j-k */
{
    *t1 = pbcDxAiuc<returnShift>(pbcAiuc, xi, xj, r_ij);
    *t2 = pbcDxAiuc<returnShift>(pbcAiuc, xk, xj, r_kj);

    *costh   = cos_angle_gpu(r_ij, r_kj);
    float th = acosf(*costh);

    return th;
}

template<bool calcVir, bool calcEner>
__device__ void angles_gpu(const int       i,
                           float*          vtot_loc,
                           const int       numBonds,
                           const t_iatom   d_forceatoms[],
                           const t_iparams d_forceparams[],
                           const float4    gm_xq[],
                           fvec            gm_f[],
                           fvec            sm_fShiftLoc[],
                           const PbcAiuc   pbcAiuc)
{
    if (i < numBonds)
    {
        int4 angleData = *(int4*)(d_forceatoms + 4 * i);
        int  type      = angleData.x;
        int  ai        = angleData.y;
        int  aj        = angleData.z;
        int  ak        = angleData.w;

        fvec  r_ij;
        fvec  r_kj;
        float cos_theta;
        int   t1;
        int   t2;
        float theta = bond_angle_gpu<calcVir>(gm_xq[ai], gm_xq[aj], gm_xq[ak], pbcAiuc, r_ij, r_kj,
                                              &cos_theta, &t1, &t2);

        float va;
        float dVdt;
        harmonic_gpu(d_forceparams[type].harmonic.krA,
                     d_forceparams[type].harmonic.rA * CUDA_DEG2RAD_F, theta, &va, &dVdt);

        if (calcEner)
        {
            *vtot_loc += va;
        }

        float cos_theta2 = cos_theta * cos_theta;
        if (cos_theta2 < 1.0f)
        {
            float st    = dVdt * rsqrtf(1.0f - cos_theta2);
            float sth   = st * cos_theta;
            float nrij2 = iprod_gpu(r_ij, r_ij);
            float nrkj2 = iprod_gpu(r_kj, r_kj);

            float nrij_1 = rsqrtf(nrij2);
            float nrkj_1 = rsqrtf(nrkj2);

            float cik = st * nrij_1 * nrkj_1;
            float cii = sth * nrij_1 * nrij_1;
            float ckk = sth * nrkj_1 * nrkj_1;

            fvec f_i;
            fvec f_k;
            fvec f_j;
#pragma unroll
            for (int m = 0; m < DIM; m++)
            {
                f_i[m] = -(cik * r_kj[m] - cii * r_ij[m]);
                f_k[m] = -(cik * r_ij[m] - ckk * r_kj[m]);
                f_j[m] = -f_i[m] - f_k[m];
                atomicAdd(&gm_f[ai][m], f_i[m]);
                atomicAdd(&gm_f[aj][m], f_j[m]);
                atomicAdd(&gm_f[ak][m], f_k[m]);
                if (calcVir)
                {
                    atomicAdd(&sm_fShiftLoc[t1][m], f_i[m]);
                    atomicAdd(&sm_fShiftLoc[CENTRAL][m], f_j[m]);
                    atomicAdd(&sm_fShiftLoc[t2][m], f_k[m]);
                }
            }
        }
    }
}

template<bool calcVir, bool calcEner>
__device__ void urey_bradley_gpu(const int       i,
                                 float*          vtot_loc,
                                 const int       numBonds,
                                 const t_iatom   d_forceatoms[],
                                 const t_iparams d_forceparams[],
                                 const float4    gm_xq[],
                                 fvec            gm_f[],
                                 fvec            sm_fShiftLoc[],
                                 const PbcAiuc   pbcAiuc)
{
    if (i < numBonds)
    {
        int4 ubData = *(int4*)(d_forceatoms + 4 * i);
        int  type   = ubData.x;
        int  ai     = ubData.y;
        int  aj     = ubData.z;
        int  ak     = ubData.w;

        float th0A = d_forceparams[type].u_b.thetaA * CUDA_DEG2RAD_F;
        float kthA = d_forceparams[type].u_b.kthetaA;
        float r13A = d_forceparams[type].u_b.r13A;
        float kUBA = d_forceparams[type].u_b.kUBA;

        fvec  r_ij;
        fvec  r_kj;
        float cos_theta;
        int   t1;
        int   t2;
        float theta = bond_angle_gpu<calcVir>(gm_xq[ai], gm_xq[aj], gm_xq[ak], pbcAiuc, r_ij, r_kj,
                                              &cos_theta, &t1, &t2);

        float va;
        float dVdt;
        harmonic_gpu(kthA, th0A, theta, &va, &dVdt);

        if (calcEner)
        {
            *vtot_loc += va;
        }

        fvec r_ik;
        int  ki = pbcDxAiuc<calcVir>(pbcAiuc, gm_xq[ai], gm_xq[ak], r_ik);

        float dr2 = iprod_gpu(r_ik, r_ik);
        float dr  = dr2 * rsqrtf(dr2);

        float vbond;
        float fbond;
        harmonic_gpu(kUBA, r13A, dr, &vbond, &fbond);

        float cos_theta2 = cos_theta * cos_theta;
        if (cos_theta2 < 1.0f)
        {
            float st  = dVdt * rsqrtf(1.0f - cos_theta2);
            float sth = st * cos_theta;

            float nrkj2 = iprod_gpu(r_kj, r_kj);
            float nrij2 = iprod_gpu(r_ij, r_ij);

            float cik = st * rsqrtf(nrkj2 * nrij2);
            float cii = sth / nrij2;
            float ckk = sth / nrkj2;

            fvec f_i;
            fvec f_j;
            fvec f_k;
#pragma unroll
            for (int m = 0; m < DIM; m++)
            {
                f_i[m] = -(cik * r_kj[m] - cii * r_ij[m]);
                f_k[m] = -(cik * r_ij[m] - ckk * r_kj[m]);
                f_j[m] = -f_i[m] - f_k[m];
                atomicAdd(&gm_f[ai][m], f_i[m]);
                atomicAdd(&gm_f[aj][m], f_j[m]);
                atomicAdd(&gm_f[ak][m], f_k[m]);
                if (calcVir)
                {
                    atomicAdd(&sm_fShiftLoc[t1][m], f_i[m]);
                    atomicAdd(&sm_fShiftLoc[CENTRAL][m], f_j[m]);
                    atomicAdd(&sm_fShiftLoc[t2][m], f_k[m]);
                }
            }
        }

        /* Time for the bond calculations */
        if (dr2 != 0.0f)
        {
            if (calcEner)
            {
                *vtot_loc += vbond;
            }

            fbond *= rsqrtf(dr2);

#pragma unroll
            for (int m = 0; m < DIM; m++)
            {
                float fik = fbond * r_ik[m];
                atomicAdd(&gm_f[ai][m], fik);
                atomicAdd(&gm_f[ak][m], -fik);

                if (calcVir && ki != CENTRAL)
                {
                    atomicAdd(&sm_fShiftLoc[ki][m], fik);
                    atomicAdd(&sm_fShiftLoc[CENTRAL][m], -fik);
                }
            }
        }
    }
}

template<bool returnShift, typename T>
__device__ __forceinline__ static float dih_angle_gpu(const T        xi,
                                                      const T        xj,
                                                      const T        xk,
                                                      const T        xl,
                                                      const PbcAiuc& pbcAiuc,
                                                      fvec           r_ij,
                                                      fvec           r_kj,
                                                      fvec           r_kl,
                                                      fvec           m,
                                                      fvec           n,
                                                      int*           t1,
                                                      int*           t2,
                                                      int*           t3)
{
    *t1 = pbcDxAiuc<returnShift>(pbcAiuc, xi, xj, r_ij);
    *t2 = pbcDxAiuc<returnShift>(pbcAiuc, xk, xj, r_kj);
    *t3 = pbcDxAiuc<returnShift>(pbcAiuc, xk, xl, r_kl);

    cprod_gpu(r_ij, r_kj, m);
    cprod_gpu(r_kj, r_kl, n);
    float phi  = gmx_angle_gpu(m, n);
    float ipr  = iprod_gpu(r_ij, n);
    float sign = (ipr < 0.0f) ? -1.0f : 1.0f;
    phi        = sign * phi;

    return phi;
}


__device__ __forceinline__ static void
           dopdihs_gpu(const float cpA, const float phiA, const int mult, const float phi, float* v, float* f)
{
    float mdphi, sdphi;

    mdphi = mult * phi - phiA * CUDA_DEG2RAD_F;
    sdphi = sinf(mdphi);
    *v    = cpA * (1.0f + cosf(mdphi));
    *f    = -cpA * mult * sdphi;
}

template<bool calcVir>
__device__ static void do_dih_fup_gpu(const int      i,
                                      const int      j,
                                      const int      k,
                                      const int      l,
                                      const float    ddphi,
                                      const fvec     r_ij,
                                      const fvec     r_kj,
                                      const fvec     r_kl,
                                      const fvec     m,
                                      const fvec     n,
                                      fvec           gm_f[],
                                      fvec           sm_fShiftLoc[],
                                      const PbcAiuc& pbcAiuc,
                                      const float4   gm_xq[],
                                      const int      t1,
                                      const int      t2,
                                      const int gmx_unused t3)
{
    float iprm  = iprod_gpu(m, m);
    float iprn  = iprod_gpu(n, n);
    float nrkj2 = iprod_gpu(r_kj, r_kj);
    float toler = nrkj2 * GMX_REAL_EPS;
    if ((iprm > toler) && (iprn > toler))
    {
        float nrkj_1 = rsqrtf(nrkj2); // replacing std::invsqrt call
        float nrkj_2 = nrkj_1 * nrkj_1;
        float nrkj   = nrkj2 * nrkj_1;
        float a      = -ddphi * nrkj / iprm;
        fvec  f_i;
        svmul_gpu(a, m, f_i);
        float b = ddphi * nrkj / iprn;
        fvec  f_l;
        svmul_gpu(b, n, f_l);
        float p = iprod_gpu(r_ij, r_kj);
        p *= nrkj_2;
        float q = iprod_gpu(r_kl, r_kj);
        q *= nrkj_2;
        fvec uvec;
        svmul_gpu(p, f_i, uvec);
        fvec vvec;
        svmul_gpu(q, f_l, vvec);
        fvec svec;
        fvec_sub_gpu(uvec, vvec, svec);
        fvec f_j;
        fvec_sub_gpu(f_i, svec, f_j);
        fvec f_k;
        fvec_add_gpu(f_l, svec, f_k);
#pragma unroll
        for (int m = 0; (m < DIM); m++)
        {
            atomicAdd(&gm_f[i][m], f_i[m]);
            atomicAdd(&gm_f[j][m], -f_j[m]);
            atomicAdd(&gm_f[k][m], -f_k[m]);
            atomicAdd(&gm_f[l][m], f_l[m]);
        }

        if (calcVir)
        {
            fvec dx_jl;
            int  t3 = pbcDxAiuc<calcVir>(pbcAiuc, gm_xq[l], gm_xq[j], dx_jl);

#pragma unroll
            for (int m = 0; (m < DIM); m++)
            {
                atomicAdd(&sm_fShiftLoc[t1][m], f_i[m]);
                atomicAdd(&sm_fShiftLoc[CENTRAL][m], -f_j[m]);
                atomicAdd(&sm_fShiftLoc[t2][m], -f_k[m]);
                atomicAdd(&sm_fShiftLoc[t3][m], f_l[m]);
            }
        }
    }
}

template<bool calcVir, bool calcEner>
__device__ void pdihs_gpu(const int       i,
                          float*          vtot_loc,
                          const int       numBonds,
                          const t_iatom   d_forceatoms[],
                          const t_iparams d_forceparams[],
                          const float4    gm_xq[],
                          fvec            gm_f[],
                          fvec            sm_fShiftLoc[],
                          const PbcAiuc   pbcAiuc)
{
    if (i < numBonds)
    {
        int type = d_forceatoms[5 * i];
        int ai   = d_forceatoms[5 * i + 1];
        int aj   = d_forceatoms[5 * i + 2];
        int ak   = d_forceatoms[5 * i + 3];
        int al   = d_forceatoms[5 * i + 4];

        fvec  r_ij;
        fvec  r_kj;
        fvec  r_kl;
        fvec  m;
        fvec  n;
        int   t1;
        int   t2;
        int   t3;
        float phi = dih_angle_gpu<calcVir>(gm_xq[ai], gm_xq[aj], gm_xq[ak], gm_xq[al], pbcAiuc,
                                           r_ij, r_kj, r_kl, m, n, &t1, &t2, &t3);

        float vpd;
        float ddphi;
        dopdihs_gpu(d_forceparams[type].pdihs.cpA, d_forceparams[type].pdihs.phiA,
                    d_forceparams[type].pdihs.mult, phi, &vpd, &ddphi);

        if (calcEner)
        {
            *vtot_loc += vpd;
        }

        do_dih_fup_gpu<calcVir>(ai, aj, ak, al, ddphi, r_ij, r_kj, r_kl, m, n, gm_f, sm_fShiftLoc,
                                pbcAiuc, gm_xq, t1, t2, t3);
    }
}

template<bool calcVir, bool calcEner>
__device__ void rbdihs_gpu(const int       i,
                           float*          vtot_loc,
                           const int       numBonds,
                           const t_iatom   d_forceatoms[],
                           const t_iparams d_forceparams[],
                           const float4    gm_xq[],
                           fvec            gm_f[],
                           fvec            sm_fShiftLoc[],
                           const PbcAiuc   pbcAiuc)
{
    constexpr float c0 = 0.0f, c1 = 1.0f, c2 = 2.0f, c3 = 3.0f, c4 = 4.0f, c5 = 5.0f;

    if (i < numBonds)
    {
        int type = d_forceatoms[5 * i];
        int ai   = d_forceatoms[5 * i + 1];
        int aj   = d_forceatoms[5 * i + 2];
        int ak   = d_forceatoms[5 * i + 3];
        int al   = d_forceatoms[5 * i + 4];

        fvec  r_ij;
        fvec  r_kj;
        fvec  r_kl;
        fvec  m;
        fvec  n;
        int   t1;
        int   t2;
        int   t3;
        float phi = dih_angle_gpu<calcVir>(gm_xq[ai], gm_xq[aj], gm_xq[ak], gm_xq[al], pbcAiuc,
                                           r_ij, r_kj, r_kl, m, n, &t1, &t2, &t3);

        /* Change to polymer convention */
        if (phi < c0)
        {
            phi += HIP_PI_F;
        }
        else
        {
            phi -= HIP_PI_F;
        }
        float cos_phi = cosf(phi);
        /* Beware of accuracy loss, cannot use 1-sqrt(cos^2) ! */
        float sin_phi = sinf(phi);

        float parm[NR_RBDIHS];
        for (int j = 0; j < NR_RBDIHS; j++)
        {
            parm[j] = d_forceparams[type].rbdihs.rbcA[j];
        }
        /* Calculate cosine powers */
        /* Calculate the energy */
        /* Calculate the derivative */
        float v      = parm[0];
        float ddphi  = c0;
        float cosfac = c1;

        float rbp = parm[1];
        ddphi += rbp * cosfac;
        cosfac *= cos_phi;
        if (calcEner)
        {
            v += cosfac * rbp;
        }
        rbp = parm[2];
        ddphi += c2 * rbp * cosfac;
        cosfac *= cos_phi;
        if (calcEner)
        {
            v += cosfac * rbp;
        }
        rbp = parm[3];
        ddphi += c3 * rbp * cosfac;
        cosfac *= cos_phi;
        if (calcEner)
        {
            v += cosfac * rbp;
        }
        rbp = parm[4];
        ddphi += c4 * rbp * cosfac;
        cosfac *= cos_phi;
        if (calcEner)
        {
            v += cosfac * rbp;
        }
        rbp = parm[5];
        ddphi += c5 * rbp * cosfac;
        cosfac *= cos_phi;
        if (calcEner)
        {
            v += cosfac * rbp;
        }

        ddphi = -ddphi * sin_phi;

        do_dih_fup_gpu<calcVir>(ai, aj, ak, al, ddphi, r_ij, r_kj, r_kl, m, n, gm_f, sm_fShiftLoc,
                                pbcAiuc, gm_xq, t1, t2, t3);
        if (calcEner)
        {
            *vtot_loc += v;
        }
    }
}

__device__ __forceinline__ static void make_dp_periodic_gpu(float* dp)
{
    /* dp cannot be outside (-pi,pi) */
    if (*dp >= HIP_PI_F)
    {
        *dp -= 2.0f * HIP_PI_F;
    }
    else if (*dp < -HIP_PI_F)
    {
        *dp += 2.0f * HIP_PI_F;
    }
}

template<bool calcVir, bool calcEner>
__device__ void idihs_gpu(const int       i,
                          float*          vtot_loc,
                          const int       numBonds,
                          const t_iatom   d_forceatoms[],
                          const t_iparams d_forceparams[],
                          const float4    gm_xq[],
                          fvec            gm_f[],
                          fvec            sm_fShiftLoc[],
                          const PbcAiuc   pbcAiuc)
{
    if (i < numBonds)
    {
        int type = d_forceatoms[5 * i];
        int ai   = d_forceatoms[5 * i + 1];
        int aj   = d_forceatoms[5 * i + 2];
        int ak   = d_forceatoms[5 * i + 3];
        int al   = d_forceatoms[5 * i + 4];

        fvec  r_ij;
        fvec  r_kj;
        fvec  r_kl;
        fvec  m;
        fvec  n;
        int   t1;
        int   t2;
        int   t3;
        float phi = dih_angle_gpu<calcVir>(gm_xq[ai], gm_xq[aj], gm_xq[ak], gm_xq[al], pbcAiuc,
                                           r_ij, r_kj, r_kl, m, n, &t1, &t2, &t3);

        /* phi can jump if phi0 is close to Pi/-Pi, which will cause huge
         * force changes if we just apply a normal harmonic.
         * Instead, we first calculate phi-phi0 and take it modulo (-Pi,Pi).
         * This means we will never have the periodicity problem, unless
         * the dihedral is Pi away from phiO, which is very unlikely due to
         * the potential.
         */
        float kA = d_forceparams[type].harmonic.krA;
        float pA = d_forceparams[type].harmonic.rA;

        float phi0 = pA * CUDA_DEG2RAD_F;

        float dp = phi - phi0;

        make_dp_periodic_gpu(&dp);

        float ddphi = -kA * dp;

        do_dih_fup_gpu<calcVir>(ai, aj, ak, al, -ddphi, r_ij, r_kj, r_kl, m, n, gm_f, sm_fShiftLoc,
                                pbcAiuc, gm_xq, t1, t2, t3);

        if (calcEner)
        {
            *vtot_loc += -0.5f * ddphi * dp;
        }
    }
}

template<bool calcVir, bool calcEner>
__device__ void pairs_gpu(const int       i,
                          const int       numBonds,
                          const t_iatom   d_forceatoms[],
                          const t_iparams iparams[],
                          const float4    gm_xq[],
                          fvec            gm_f[],
                          fvec            sm_fShiftLoc[],
                          const PbcAiuc   pbcAiuc,
                          const float     scale_factor,
                          float*          vtotVdw_loc,
                          float*          vtotElec_loc)
{
    if (i < numBonds)
    {
        int3 pairData = *(int3*)(d_forceatoms + 3 * i);
        int  type     = pairData.x;
        int  ai       = pairData.y;
        int  aj       = pairData.z;

        float qq  = gm_xq[ai].w * gm_xq[aj].w;
        float c6  = iparams[type].lj14.c6A;
        float c12 = iparams[type].lj14.c12A;

        /* Do we need to apply full periodic boundary conditions? */
        fvec dr;
        int  fshift_index = pbcDxAiuc<calcVir>(pbcAiuc, gm_xq[ai], gm_xq[aj], dr);

        float r2    = norm2_gpu(dr);
        float rinv  = rsqrtf(r2);
        float rinv2 = rinv * rinv;
        float rinv6 = rinv2 * rinv2 * rinv2;

        /* Calculate the Coulomb force * r */
        float velec = scale_factor * qq * rinv;

        /* Calculate the LJ force * r and add it to the Coulomb part */
        float fr = (12.0f * c12 * rinv6 - 6.0f * c6) * rinv6 + velec;

        float finvr = fr * rinv2;
        fvec  f;
        svmul_gpu(finvr, dr, f);

        /* Add the forces */
#pragma unroll
        for (int m = 0; m < DIM; m++)
        {
            atomicAdd(&gm_f[ai][m], f[m]);
            atomicAdd(&gm_f[aj][m], -f[m]);
            if (calcVir && fshift_index != CENTRAL)
            {
                atomicAdd(&sm_fShiftLoc[fshift_index][m], f[m]);
                atomicAdd(&sm_fShiftLoc[CENTRAL][m], -f[m]);
            }
        }

        if (calcEner)
        {
            *vtotVdw_loc += (c12 * rinv6 - c6) * rinv6;
            *vtotElec_loc += velec;
        }
    }
}

namespace gmx
{

template<bool calcVir, bool calcEner>
__global__ void exec_kernel_gpu(BondedCudaKernelParameters kernelParams)
{
    assert(blockDim.y == 1 && blockDim.z == 1);
    const int  tid          = blockIdx.x * blockDim.x + threadIdx.x;
    float      vtot_loc     = 0;
    float      vtotVdw_loc  = 0;
    float      vtotElec_loc = 0;
    __shared__ fvec sm_fShiftLoc[SHIFTS];

    if (calcVir)
    {
        if (threadIdx.x < SHIFTS)
        {
            sm_fShiftLoc[threadIdx.x][XX] = 0.0f;
            sm_fShiftLoc[threadIdx.x][YY] = 0.0f;
            sm_fShiftLoc[threadIdx.x][ZZ] = 0.0f;
        }
        __syncthreads();
    }

    int  fType;
    bool threadComputedPotential = false;
#pragma unroll
    for (int j = 0; j < numFTypesOnGpu; j++)
    {
        if (tid >= kernelParams.fTypeRangeStart[j] && tid <= kernelParams.fTypeRangeEnd[j])
        {
            const int      numBonds = kernelParams.numFTypeBonds[j];
            int            fTypeTid = tid - kernelParams.fTypeRangeStart[j];
            const t_iatom* iatoms   = kernelParams.d_iatoms[j];
            fType                   = kernelParams.fTypesOnGpu[j];
            if (calcEner)
            {
                threadComputedPotential = true;
            }

            switch (fType)
            {
                case F_BONDS:
                    bonds_gpu<calcVir, calcEner>(fTypeTid, &vtot_loc, numBonds, iatoms,
                                                 kernelParams.d_forceParams, kernelParams.d_xq,
                                                 kernelParams.d_f, sm_fShiftLoc, kernelParams.pbcAiuc);
                    break;
                case F_ANGLES:
                    angles_gpu<calcVir, calcEner>(
                            fTypeTid, &vtot_loc, numBonds, iatoms, kernelParams.d_forceParams,
                            kernelParams.d_xq, kernelParams.d_f, sm_fShiftLoc, kernelParams.pbcAiuc);
                    break;
                case F_UREY_BRADLEY:
                    urey_bradley_gpu<calcVir, calcEner>(
                            fTypeTid, &vtot_loc, numBonds, iatoms, kernelParams.d_forceParams,
                            kernelParams.d_xq, kernelParams.d_f, sm_fShiftLoc, kernelParams.pbcAiuc);
                    break;
                case F_PDIHS:
                case F_PIDIHS:
                    pdihs_gpu<calcVir, calcEner>(fTypeTid, &vtot_loc, numBonds, iatoms,
                                                 kernelParams.d_forceParams, kernelParams.d_xq,
                                                 kernelParams.d_f, sm_fShiftLoc, kernelParams.pbcAiuc);
                    break;
                case F_RBDIHS:
                    rbdihs_gpu<calcVir, calcEner>(
                            fTypeTid, &vtot_loc, numBonds, iatoms, kernelParams.d_forceParams,
                            kernelParams.d_xq, kernelParams.d_f, sm_fShiftLoc, kernelParams.pbcAiuc);
                    break;
                case F_IDIHS:
                    idihs_gpu<calcVir, calcEner>(fTypeTid, &vtot_loc, numBonds, iatoms,
                                                 kernelParams.d_forceParams, kernelParams.d_xq,
                                                 kernelParams.d_f, sm_fShiftLoc, kernelParams.pbcAiuc);
                    break;
                case F_LJ14:
                    pairs_gpu<calcVir, calcEner>(fTypeTid, numBonds, iatoms, kernelParams.d_forceParams,
                                                 kernelParams.d_xq, kernelParams.d_f, sm_fShiftLoc,
                                                 kernelParams.pbcAiuc, kernelParams.scaleFactor,
                                                 &vtotVdw_loc, &vtotElec_loc);
                    break;
            }
            break;
        }
    }

    if (threadComputedPotential)
    {
        float* vtotVdw  = kernelParams.d_vTot + F_LJ14;
        float* vtotElec = kernelParams.d_vTot + F_COUL14;
        atomicAdd(kernelParams.d_vTot + fType, vtot_loc);
        atomicAdd(vtotVdw, vtotVdw_loc);
        atomicAdd(vtotElec, vtotElec_loc);
    }
    /* Accumulate shift vectors from shared memory to global memory on the first SHIFTS threads of the block. */
    if (calcVir)
    {
        __syncthreads();
        if (threadIdx.x < SHIFTS)
        {
            fvec_inc_atomic(kernelParams.d_fShift[threadIdx.x], sm_fShiftLoc[threadIdx.x]);
        }
    }
}


/*-------------------------------- End CUDA kernels-----------------------------*/


template<bool calcVir, bool calcEner>
void GpuBonded::Impl::launchKernel(const t_forcerec* fr, const matrix box)
{
    GMX_ASSERT(haveInteractions_,
               "Cannot launch bonded GPU kernels unless bonded GPU work was scheduled");
    static_assert(TPB_BONDED >= SHIFTS,
                  "TPB_BONDED must be >= SHIFTS for the virial kernel (calcVir=true)");

    PbcAiuc pbcAiuc;
    setPbcAiuc(fr->bMolPBC ? ePBC2npbcdim(fr->ePBC) : 0, box, &pbcAiuc);

    int fTypeRangeEnd = kernelParams_.fTypeRangeEnd[numFTypesOnGpu - 1];

    if (fTypeRangeEnd < 0)
    {
        return;
    }

    KernelLaunchConfig config;
    config.blockSize[0] = TPB_BONDED;
    config.blockSize[1] = 1;
    config.blockSize[2] = 1;
    config.gridSize[0]  = (fTypeRangeEnd + TPB_BONDED) / TPB_BONDED;
    config.gridSize[1]  = 1;
    config.gridSize[2]  = 1;
    config.stream       = stream_;

    auto kernelPtr            = exec_kernel_gpu<calcVir, calcEner>;
    kernelParams_.scaleFactor = fr->ic->epsfac * fr->fudgeQQ;
    kernelParams_.pbcAiuc     = pbcAiuc;

    const auto kernelArgs = prepareGpuKernelArguments(kernelPtr, config, &kernelParams_);

    launchGpuKernel(kernelPtr, config, nullptr, "exec_kernel_gpu<calcVir, calcEner>", kernelArgs);
}

void GpuBonded::launchKernel(const t_forcerec* fr, const gmx::StepWorkload& stepWork, const matrix box)
{
    if (stepWork.computeEnergy)
    {
        // When we need the energy, we also need the virial
        impl_->launchKernel<true, true>(fr, box);
    }
    else if (stepWork.computeVirial)
    {
        impl_->launchKernel<true, false>(fr, box);
    }
    else
    {
        impl_->launchKernel<false, false>(fr, box);
    }
}

} // namespace gmx
