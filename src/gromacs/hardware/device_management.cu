#include "hip/hip_runtime.h"
/*
 * This file is part of the GROMACS molecular simulation package.
 *
 * Copyright 2012- The GROMACS Authors
 * and the project initiators Erik Lindahl, Berk Hess and David van der Spoel.
 * Consult the AUTHORS/COPYING files and https://www.gromacs.org for details.
 *
 * GROMACS is free software; you can redistribute it and/or
 * modify it under the terms of the GNU Lesser General Public License
 * as published by the Free Software Foundation; either version 2.1
 * of the License, or (at your option) any later version.
 *
 * GROMACS is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
 * Lesser General Public License for more details.
 *
 * You should have received a copy of the GNU Lesser General Public
 * License along with GROMACS; if not, see
 * https://www.gnu.org/licenses, or write to the Free Software Foundation,
 * Inc., 51 Franklin Street, Fifth Floor, Boston, MA  02110-1301  USA.
 *
 * If you want to redistribute modifications to GROMACS, please
 * consider that scientific software is very special. Version
 * control is crucial - bugs must be traceable. We will be happy to
 * consider code for inclusion in the official distribution, but
 * derived work must not be called official GROMACS. Details are found
 * in the README & COPYING files - if they are missing, get the
 * official version at https://www.gromacs.org.
 *
 * To help us fund GROMACS development, we humbly ask that you cite
 * the research papers on the package. Check out https://www.gromacs.org.
 */
/*! \internal \file
 *  \brief Defines the CUDA implementations of the device management.
 *
 *  \author Anca Hamuraru <anca@streamcomputing.eu>
 *  \author Dimitrios Karkoulis <dimitris.karkoulis@gmail.com>
 *  \author Teemu Virolainen <teemu@streamcomputing.eu>
 *  \author Mark Abraham <mark.j.abraham@gmail.com>
 *  \author Szilárd Páll <pall.szilard@gmail.com>
 *  \author Artem Zhmurov <zhmurov@gmail.com>
 *
 * \ingroup module_hardware
 */
#include "gmxpre.h"

#include "device_management.h"

#include <cassert>

#include "gromacs/gpu_utils/cudautils.cuh"
#include "gromacs/gpu_utils/device_context.h"
#include "gromacs/gpu_utils/device_stream.h"
#include "gromacs/utility/exceptions.h"
#include "gromacs/utility/logger.h"
#include "gromacs/utility/mpiinfo.h"
#include "gromacs/utility/programcontext.h"
#include "gromacs/utility/smalloc.h"
#include "gromacs/utility/stringutil.h"

#include "device_information.h"

/*! \internal \brief
 * Max number of devices supported by CUDA (for consistency checking).
 *
 * In reality it is 16 with CUDA <=v5.0, but let's stay on the safe side.
 */
static const int c_cudaMaxDeviceCount = 32;

/** Dummy kernel used for sanity checking. */
static __global__ void dummy_kernel() {}

void warnWhenDeviceNotTargeted(const gmx::MDLogger& mdlog, const DeviceInformation& deviceInfo)
{
    if (deviceInfo.status != DeviceStatus::DeviceNotTargeted)
    {
        return;
    }
    gmx::TextLineWrapper wrapper;
    wrapper.settings().setLineLength(80);
    GMX_LOG(mdlog.warning)
            .asParagraph()
            .appendText(wrapper.wrapToString(gmx::formatString(
                    "WARNING: The %s binary does not include support for the CUDA architecture of "
                    "the GPU ID #%d (compute capability %d.%d) detected during detection. "
                    "By default, GROMACS supports all architectures of compute "
                    "capability >= 3.5, so your GPU "
                    "might be rare, or some architectures were disabled in the build. "
                    "Consult the install guide for how to use the GMX_CUDA_TARGET_SM and "
                    "GMX_CUDA_TARGET_COMPUTE CMake variables to add this architecture.",
                    gmx::getProgramContext().displayName(),
                    deviceInfo.id,
                    deviceInfo.prop.major,
                    deviceInfo.prop.minor)));
}

/*! \brief Runs GPU compatibility and sanity checks on the indicated device.
 *
 * Runs a series of checks to determine that the given GPU and underlying CUDA
 * driver/runtime functions properly.
 *
 *  As the error handling only permits returning the state of the GPU, this function
 *  does not clear the CUDA runtime API status allowing the caller to inspect the error
 *  upon return. Note that this also means it is the caller's responsibility to
 *  reset the CUDA runtime state.
 *
 * \todo Currently we do not make a distinction between the type of errors
 *       that can appear during functionality checks. This needs to be improved,
 *       e.g if the dummy test kernel fails to execute with a "device busy message"
 *       we should appropriately report that the device is busy instead of NonFunctional.
 *
 * \todo Introduce errors codes and handle errors more smoothly.
 *
 *
 * \param[in]  deviceInfo  Device information on the device to check.
 * \returns                The status enumeration value for the checked device:
 */
static DeviceStatus checkDeviceStatus(const DeviceInformation& deviceInfo)
{
    hipError_t cu_err;

    // Is the generation of the device supported?
    if (deviceInfo.prop.major < 3)
    {
        return DeviceStatus::Incompatible;
    }

    /* both major & minor is 9999 if no CUDA capable devices are present */
    if (deviceInfo.prop.major == 9999 && deviceInfo.prop.minor == 9999)
    {
        return DeviceStatus::NonFunctional;
    }
    /* we don't care about emulation mode */
    if (deviceInfo.prop.major == 0)
    {
        return DeviceStatus::NonFunctional;
    }

    cu_err = hipSetDevice(deviceInfo.id);
    if (cu_err != hipSuccess)
    {
        fprintf(stderr,
                "Error while switching to device #%d. %s\n",
                deviceInfo.id,
                gmx::getDeviceErrorString(cu_err).c_str());
        return DeviceStatus::NonFunctional;
    }

    hipFuncAttributes attributes;
    cu_err = hipFuncGetAttributes(&attributes, reinterpret_cast<const void*>(dummy_kernel));

    if (cu_err == hipErrorInvalidDeviceFunction)
    {
        // Clear the error from attempting to compile the kernel
        hipGetLastError();
        return DeviceStatus::DeviceNotTargeted;
    }

    // Avoid triggering an error if GPU devices are in exclusive or prohibited mode;
    // it is enough to check for cudaErrorDevicesUnavailable only here because
    // if we encounter it that will happen in above hipFuncGetAttributes.
    if (cu_err == cudaErrorDevicesUnavailable)
    {
        return DeviceStatus::Unavailable;
    }
    else if (cu_err != hipSuccess)
    {
        return DeviceStatus::NonFunctional;
    }

    /* try to execute a dummy kernel */
    try
    {
        KernelLaunchConfig config;
        config.blockSize[0]                = 512;
        const auto          dummyArguments = prepareGpuKernelArguments(dummy_kernel, config);
        const DeviceContext deviceContext(deviceInfo);
        const DeviceStream  deviceStream(deviceContext, DeviceStreamPriority::Normal, false);
        launchGpuKernel(dummy_kernel, config, deviceStream, nullptr, "Dummy kernel", dummyArguments);
    }
    catch (gmx::GromacsException& ex)
    {
        // launchGpuKernel error is not fatal and should continue with marking the device bad
        fprintf(stderr,
                "Error occurred while running dummy kernel sanity check on device #%d:\n %s\n",
                deviceInfo.id,
                formatExceptionMessageToString(ex).c_str());
        return DeviceStatus::NonFunctional;
    }

    if (hipDeviceSynchronize() != hipSuccess)
    {
        return DeviceStatus::NonFunctional;
    }

    // Skip context teardown when using CUDA-aware MPI because this can lead to
    // corruption and a crash in MPI when when mdrunner is invoked multiple times
    // in the same process in gmxapi or mdrun integration tests. Ref #3952
    const bool haveDetectedOrForcedCudaAwareMpi =
            (gmx::checkMpiCudaAwareSupport() == gmx::GpuAwareMpiStatus::Supported
             || gmx::checkMpiCudaAwareSupport() == gmx::GpuAwareMpiStatus::Forced);
    if (!haveDetectedOrForcedCudaAwareMpi)
    {
        cu_err = hipDeviceReset();
        CU_RET_ERR(cu_err, "hipDeviceReset failed");
    }

    return DeviceStatus::Compatible;
}

bool isDeviceDetectionFunctional(std::string* errorMessage)
{
    hipError_t stat;
    int         driverVersion = -1;
    stat                      = hipDriverGetVersion(&driverVersion);
    GMX_ASSERT(stat != hipErrorInvalidValue,
               "An impossible null pointer was passed to hipDriverGetVersion");
    GMX_RELEASE_ASSERT(stat == hipSuccess,
                       ("An unexpected value was returned from hipDriverGetVersion. "
                        + gmx::getDeviceErrorString(stat))
                               .c_str());
    bool foundDriver = (driverVersion > 0);
    if (!foundDriver)
    {
        // Can't detect GPUs if there is no driver
        if (errorMessage != nullptr)
        {
            errorMessage->assign("No valid CUDA driver found");
        }
        return false;
    }

    int numDevices;
    stat = hipGetDeviceCount(&numDevices);
    if (stat != hipSuccess)
    {
        if (errorMessage != nullptr)
        {
            /* hipGetDeviceCount failed which means that there is
             * something wrong with the machine: driver-runtime
             * mismatch, all GPUs being busy in exclusive mode,
             * invalid CUDA_VISIBLE_DEVICES, or some other condition
             * which should result in GROMACS issuing at least a
             * warning. */
            errorMessage->assign(hipGetErrorString(stat));
        }

        // Consume the error now that we have prepared to handle
        // it. This stops it reappearing next time we check for
        // errors. Note that if CUDA_VISIBLE_DEVICES does not contain
        // valid devices, then hipGetLastError returns the
        // (undocumented) hipErrorNoDevice, but this should not be a
        // problem as there should be no future CUDA API calls.
        // NVIDIA bug report #2038718 has been filed.
        hipGetLastError();
        // Can't detect GPUs
        return false;
    }

    // We don't actually use numDevices here, that's not the job of
    // this function.
    return true;
}

std::vector<std::unique_ptr<DeviceInformation>> findDevices()
{
    int         numDevices;
    hipError_t stat = hipGetDeviceCount(&numDevices);
    gmx::checkDeviceError(stat,
                          "Invalid call of findDevices() when CUDA API returned an error, perhaps "
                          "canPerformDeviceDetection() was not called appropriately beforehand.");

    /* things might go horribly wrong if cudart is not compatible with the driver */
    numDevices = std::min(numDevices, c_cudaMaxDeviceCount);

    // We expect to start device support/sanity checks with a clean runtime error state
    gmx::ensureNoPendingDeviceError("Trying to find available CUDA devices.");

    const gmx::GpuAwareMpiStatus gpuAwareMpiStatus =
            GMX_LIB_MPI ? gmx::checkMpiCudaAwareSupport() : gmx::GpuAwareMpiStatus::NotSupported;

    std::vector<std::unique_ptr<DeviceInformation>> deviceInfoList(numDevices);
    for (int i = 0; i < numDevices; i++)
    {
        hipDeviceProp_t prop;
        memset(&prop, 0, sizeof(hipDeviceProp_t));
        stat = hipGetDeviceProperties(&prop, i);

        deviceInfoList[i]               = std::make_unique<DeviceInformation>();
        deviceInfoList[i]->id           = i;
        deviceInfoList[i]->prop         = prop;
        deviceInfoList[i]->deviceVendor = DeviceVendor::Nvidia;

        deviceInfoList[i]->supportedSubGroupSizes.push_back(32);

        deviceInfoList[i]->gpuAwareMpiStatus = gpuAwareMpiStatus;

        const DeviceStatus checkResult = (stat != hipSuccess) ? DeviceStatus::NonFunctional
                                                               : checkDeviceStatus(*deviceInfoList[i]);

        deviceInfoList[i]->status = checkResult;

        if (checkResult != DeviceStatus::Compatible)
        {
            // TODO:
            //  - we inspect the CUDA API state to retrieve and record any
            //    errors that occurred during is_gmx_supported_gpu_id() here,
            //    but this would be more elegant done within is_gmx_supported_gpu_id()
            //    and only return a string with the error if one was encountered.
            //  - we'll be reporting without rank information which is not ideal.
            //  - we'll end up warning also in cases where users would already
            //    get an error before mdrun aborts.
            //
            // Here we also clear the CUDA API error state so potential
            // errors during sanity checks don't propagate.
            const std::string errorMessage = gmx::formatString(
                    "An error occurred while sanity checking device #%d.", deviceInfoList[i]->id);
            gmx::ensureNoPendingDeviceError(errorMessage);
        }
    }

    stat = hipPeekAtLastError();
    GMX_RELEASE_ASSERT(
            stat == hipSuccess,
            ("We promise to return with clean CUDA state, but non-success state encountered. "
             + gmx::getDeviceErrorString(stat))
                    .c_str());

    return deviceInfoList;
}

void setActiveDevice(const DeviceInformation& deviceInfo)
{
    int         deviceId = deviceInfo.id;
    hipError_t stat;

    stat = hipSetDevice(deviceId);
    if (stat != hipSuccess)
    {
        auto message = gmx::formatString("Failed to initialize GPU #%d", deviceId);
        CU_RET_ERR(stat, message);
    }

    if (debug)
    {
        fprintf(stderr, "Initialized GPU ID #%d: %s\n", deviceId, deviceInfo.prop.name);
    }
}

void releaseDevice()
{
    hipError_t stat;

    int gpuid;
    stat = hipGetDevice(&gpuid);
    if (stat == hipSuccess)
    {
        if (debug)
        {
            fprintf(stderr, "Cleaning up context on GPU ID #%d.\n", gpuid);
        }

        stat = hipDeviceReset();
        if (stat != hipSuccess)
        {
            gmx_warning("Failed to free GPU #%d. %s", gpuid, gmx::getDeviceErrorString(stat).c_str());
        }
    }
}

std::string getDeviceInformationString(const DeviceInformation& deviceInfo)
{
    bool gpuExists = (deviceInfo.status != DeviceStatus::Nonexistent
                      && deviceInfo.status != DeviceStatus::NonFunctional);

    if (!gpuExists)
    {
        return gmx::formatString(
                "#%d: %s, stat: %s", deviceInfo.id, "N/A", c_deviceStateString[deviceInfo.status]);
    }
    else
    {
        return gmx::formatString("#%d: NVIDIA %s, compute cap.: %d.%d, ECC: %3s, stat: %s",
                                 deviceInfo.id,
                                 deviceInfo.prop.name,
                                 deviceInfo.prop.major,
                                 deviceInfo.prop.minor,
                                 deviceInfo.prop.ECCEnabled ? "yes" : " no",
                                 c_deviceStateString[deviceInfo.status]);
    }
}

void doubleCheckGpuAwareMpiWillWork(const DeviceInformation& /* deviceInfo */) {}

gmx::PairlistType getDeviceSpecificGpuPairlistLayout(const DeviceInformation& /* deviceInfo */)
{
    // only one kernel type supported for CUDA devices.
    return gmx::PairlistType::Hierarchical8x8x8;
}
