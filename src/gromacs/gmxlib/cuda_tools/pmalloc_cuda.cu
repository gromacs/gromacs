/*
 * This file is part of the GROMACS molecular simulation package.
 *
 * Copyright (c) 2012,2014, by the GROMACS development team, led by
 * Mark Abraham, David van der Spoel, Berk Hess, and Erik Lindahl,
 * and including many others, as listed in the AUTHORS file in the
 * top-level source directory and at http://www.gromacs.org.
 *
 * GROMACS is free software; you can redistribute it and/or
 * modify it under the terms of the GNU Lesser General Public License
 * as published by the Free Software Foundation; either version 2.1
 * of the License, or (at your option) any later version.
 *
 * GROMACS is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
 * Lesser General Public License for more details.
 *
 * You should have received a copy of the GNU Lesser General Public
 * License along with GROMACS; if not, see
 * http://www.gnu.org/licenses, or write to the Free Software Foundation,
 * Inc., 51 Franklin Street, Fifth Floor, Boston, MA  02110-1301  USA.
 *
 * If you want to redistribute modifications to GROMACS, please
 * consider that scientific software is very special. Version
 * control is crucial - bugs must be traceable. We will be happy to
 * consider code for inclusion in the official distribution, but
 * derived work must not be called official GROMACS. Details are found
 * in the README & COPYING files - if they are missing, get the
 * official version at http://www.gromacs.org.
 *
 * To help us fund GROMACS development, we humbly ask that you cite
 * the research papers on the package. Check out http://www.gromacs.org.
 */

#include <stdlib.h>

#include "cudautils.cuh"
#include "pmalloc_cuda.h"

/*! Allocates nbytes of page-locked memory.
 *  This memory should always be freed using pfree (or with the page-locked
 *  free functions provied by the CUDA library).
 */
void pmalloc(void **h_ptr, size_t nbytes)
{
    hipError_t stat;
    char        strbuf[STRLEN];
    int         flag = hipHostMallocDefault;

    if (nbytes == 0)
    {
        *h_ptr = NULL;
        return;
    }

    CU_CHECK_PREV_ERR();

    stat = hipHostMalloc(h_ptr, nbytes, flag);
    sprintf(strbuf, "hipHostMalloc of size %d bytes failed", (int)nbytes);
    CU_RET_ERR(stat, strbuf);
}

/*! Allocates nbytes of page-locked memory with write-combining.
 *  This memory should always be freed using pfree (or with the page-locked
 *  free functions provied by the CUDA library).
 */
void pmalloc_wc(void **h_ptr, size_t nbytes)
{
    hipError_t stat;
    char        strbuf[STRLEN];
    int         flag = hipHostMallocDefault || hipHostMallocWriteCombined;

    if (nbytes == 0)
    {
        *h_ptr = NULL;
        return;
    }

    CU_CHECK_PREV_ERR();

    stat = hipHostMalloc(h_ptr, nbytes, flag);
    sprintf(strbuf, "hipHostMalloc of size %d bytes failed", (int)nbytes);
    CU_RET_ERR(stat, strbuf);
}

/*! Frees page locked memory allocated with pmalloc.
 *  This function can safely be called also with a pointer to a page-locked
 *  memory allocated directly with CUDA API calls.
 */
void pfree(void *h_ptr)
{
    hipError_t stat;

    if (h_ptr == NULL)
    {
        return;
    }

    CU_CHECK_PREV_ERR();

    stat = hipHostFree(h_ptr);
    CU_RET_ERR(stat, "hipHostFree failed");
}
