#include "hip/hip_runtime.h"
/*
 * This file is part of the GROMACS molecular simulation package.
 *
 * Copyright 2012- The GROMACS Authors
 * and the project initiators Erik Lindahl, Berk Hess and David van der Spoel.
 * Consult the AUTHORS/COPYING files and https://www.gromacs.org for details.
 *
 * GROMACS is free software; you can redistribute it and/or
 * modify it under the terms of the GNU Lesser General Public License
 * as published by the Free Software Foundation; either version 2.1
 * of the License, or (at your option) any later version.
 *
 * GROMACS is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
 * Lesser General Public License for more details.
 *
 * You should have received a copy of the GNU Lesser General Public
 * License along with GROMACS; if not, see
 * https://www.gnu.org/licenses, or write to the Free Software Foundation,
 * Inc., 51 Franklin Street, Fifth Floor, Boston, MA  02110-1301  USA.
 *
 * If you want to redistribute modifications to GROMACS, please
 * consider that scientific software is very special. Version
 * control is crucial - bugs must be traceable. We will be happy to
 * consider code for inclusion in the official distribution, but
 * derived work must not be called official GROMACS. Details are found
 * in the README & COPYING files - if they are missing, get the
 * official version at https://www.gromacs.org.
 *
 * To help us fund GROMACS development, we humbly ask that you cite
 * the research papers on the package. Check out https://www.gromacs.org.
 */
/*! \file
 *  \brief Define CUDA implementation of nbnxn_gpu.h
 *
 *  \author Szilard Pall <pall.szilard@gmail.com>
 */
#include "gmxpre.h"

#include "config.h"

#include <cassert>
#include <cstdlib>

#include <cub/device/device_scan.cuh>

#include "gromacs/nbnxm/gpu_types_common.h"
#include "gromacs/nbnxm/nbnxm_gpu.h"

#if defined(_MSVC)
#    include <limits>
#endif


#include "gromacs/gpu_utils/devicebuffer.h"
#include "gromacs/gpu_utils/gpu_utils.h"
#include "gromacs/gpu_utils/gpueventsynchronizer.h"
#include "gromacs/gpu_utils/typecasts_cuda_hip.h"
#include "gromacs/gpu_utils/vectype_ops_cuda.h"
#include "gromacs/hardware/device_information.h"
#include "gromacs/mdtypes/simulation_workload.h"
#include "gromacs/nbnxm/atomdata.h"
#include "gromacs/nbnxm/gpu_common.h"
#include "gromacs/nbnxm/gpu_common_utils.h"
#include "gromacs/nbnxm/gpu_data_mgmt.h"
#include "gromacs/nbnxm/grid.h"
#include "gromacs/nbnxm/nbnxm.h"
#include "gromacs/nbnxm/nbnxm_gpu_data_mgmt.h"
#include "gromacs/nbnxm/pairlist.h"
#include "gromacs/timing/gpu_timing.h"
#include "gromacs/utility/cstringutil.h"
#include "gromacs/utility/gmxassert.h"

#include "nbnxm_cuda.h"
#include "nbnxm_cuda_kernel_utils.cuh"
#include "nbnxm_cuda_types.h"

/***** The kernel declarations/definitions come here *****/


/* Top-level kernel declaration generation: will generate through multiple
 * inclusion the following flavors for all kernel declarations:
 * - force-only output;
 * - force and energy output;
 * - force-only with pair list pruning;
 * - force and energy output with pair list pruning.
 */
#define FUNCTION_DECLARATION_ONLY
/** Force only **/
#include "nbnxm_cuda_kernels.cuh"
/** Force & energy **/
#define CALC_ENERGIES
#include "nbnxm_cuda_kernels.cuh"
#undef CALC_ENERGIES

/*** Pair-list pruning kernels ***/
/** Force only **/
#define PRUNE_NBL
#include "nbnxm_cuda_kernels.cuh"
/** Force & energy **/
#define CALC_ENERGIES
#include "nbnxm_cuda_kernels.cuh"
#undef CALC_ENERGIES
#undef PRUNE_NBL

/* Prune-only kernels */
#include "nbnxm_cuda_kernel_pruneonly.cuh"
#undef FUNCTION_DECLARATION_ONLY

/* Now generate the function definitions if we are using a single compilation unit. */
#if GMX_CUDA_NB_SINGLE_COMPILATION_UNIT
#    include "nbnxm_cuda_kernel_F_noprune.cu"
#    include "nbnxm_cuda_kernel_F_prune.cu"
#    include "nbnxm_cuda_kernel_VF_noprune.cu"
#    include "nbnxm_cuda_kernel_VF_prune.cu"
#    include "nbnxm_cuda_kernel_pruneonly.cu"
#endif /* GMX_CUDA_NB_SINGLE_COMPILATION_UNIT */

#include "nbnxm_cuda_kernel_sci_sort.cuh"

namespace gmx
{

/*! Nonbonded kernel function pointer type */
typedef void (*nbnxn_cu_kfunc_ptr_t)(const NBAtomDataGpu, const NBParamGpu, const GpuPairlist, bool);

/*********************************/

/*! Returns the number of blocks to be used for the nonbonded GPU kernel. */
static inline int calc_nb_kernel_nblock(int nwork_units, const DeviceInformation* deviceInfo)
{
    int max_grid_x_size;

    assert(deviceInfo);
    /* CUDA does not accept grid dimension of 0 (which can happen e.g. with an
       empty domain) and that case should be handled before this point. */
    assert(nwork_units > 0);

    max_grid_x_size = deviceInfo->prop.maxGridSize[0];

    /* do we exceed the grid x dimension limit? */
    if (nwork_units > max_grid_x_size)
    {
        gmx_fatal(FARGS,
                  "Watch out, the input system is too large to simulate!\n"
                  "The number of nonbonded work units (=number of super-clusters) exceeds the"
                  "maximum grid size in x dimension (%d > %d)!",
                  nwork_units,
                  max_grid_x_size);
    }

    return nwork_units;
}


/* Constant arrays listing all kernel function pointers and enabling selection
   of a kernel in an elegant manner. */

/*! Pointers to the non-bonded kernels organized in 2-dim arrays by:
 *  electrostatics and VDW type.
 *
 *  Note that the row- and column-order of function pointers has to match the
 *  order of corresponding enumerated electrostatics and vdw types, resp.,
 *  defined in nbnxn_cuda_types.h.
 */

/*! Force-only kernel function pointers. */
static const nbnxn_cu_kfunc_ptr_t nb_kfunc_noener_noprune_ptr[c_numElecTypes][c_numVdwTypes] = {
    { nbnxn_kernel_ElecCut_VdwLJ_F_cuda,
      nbnxn_kernel_ElecCut_VdwLJCombGeom_F_cuda,
      nbnxn_kernel_ElecCut_VdwLJCombLB_F_cuda,
      nbnxn_kernel_ElecCut_VdwLJFsw_F_cuda,
      nbnxn_kernel_ElecCut_VdwLJPsw_F_cuda,
      nbnxn_kernel_ElecCut_VdwLJEwCombGeom_F_cuda,
      nbnxn_kernel_ElecCut_VdwLJEwCombLB_F_cuda },
    { nbnxn_kernel_ElecRF_VdwLJ_F_cuda,
      nbnxn_kernel_ElecRF_VdwLJCombGeom_F_cuda,
      nbnxn_kernel_ElecRF_VdwLJCombLB_F_cuda,
      nbnxn_kernel_ElecRF_VdwLJFsw_F_cuda,
      nbnxn_kernel_ElecRF_VdwLJPsw_F_cuda,
      nbnxn_kernel_ElecRF_VdwLJEwCombGeom_F_cuda,
      nbnxn_kernel_ElecRF_VdwLJEwCombLB_F_cuda },
    { nbnxn_kernel_ElecEwQSTab_VdwLJ_F_cuda,
      nbnxn_kernel_ElecEwQSTab_VdwLJCombGeom_F_cuda,
      nbnxn_kernel_ElecEwQSTab_VdwLJCombLB_F_cuda,
      nbnxn_kernel_ElecEwQSTab_VdwLJFsw_F_cuda,
      nbnxn_kernel_ElecEwQSTab_VdwLJPsw_F_cuda,
      nbnxn_kernel_ElecEwQSTab_VdwLJEwCombGeom_F_cuda,
      nbnxn_kernel_ElecEwQSTab_VdwLJEwCombLB_F_cuda },
    { nbnxn_kernel_ElecEwQSTabTwinCut_VdwLJ_F_cuda,
      nbnxn_kernel_ElecEwQSTabTwinCut_VdwLJCombGeom_F_cuda,
      nbnxn_kernel_ElecEwQSTabTwinCut_VdwLJCombLB_F_cuda,
      nbnxn_kernel_ElecEwQSTabTwinCut_VdwLJFsw_F_cuda,
      nbnxn_kernel_ElecEwQSTabTwinCut_VdwLJPsw_F_cuda,
      nbnxn_kernel_ElecEwQSTabTwinCut_VdwLJEwCombGeom_F_cuda,
      nbnxn_kernel_ElecEwQSTabTwinCut_VdwLJEwCombLB_F_cuda },
    { nbnxn_kernel_ElecEw_VdwLJ_F_cuda,
      nbnxn_kernel_ElecEw_VdwLJCombGeom_F_cuda,
      nbnxn_kernel_ElecEw_VdwLJCombLB_F_cuda,
      nbnxn_kernel_ElecEw_VdwLJFsw_F_cuda,
      nbnxn_kernel_ElecEw_VdwLJPsw_F_cuda,
      nbnxn_kernel_ElecEw_VdwLJEwCombGeom_F_cuda,
      nbnxn_kernel_ElecEw_VdwLJEwCombLB_F_cuda },
    { nbnxn_kernel_ElecEwTwinCut_VdwLJ_F_cuda,
      nbnxn_kernel_ElecEwTwinCut_VdwLJCombGeom_F_cuda,
      nbnxn_kernel_ElecEwTwinCut_VdwLJCombLB_F_cuda,
      nbnxn_kernel_ElecEwTwinCut_VdwLJFsw_F_cuda,
      nbnxn_kernel_ElecEwTwinCut_VdwLJPsw_F_cuda,
      nbnxn_kernel_ElecEwTwinCut_VdwLJEwCombGeom_F_cuda,
      nbnxn_kernel_ElecEwTwinCut_VdwLJEwCombLB_F_cuda }
};

/*! Force + energy kernel function pointers. */
static const nbnxn_cu_kfunc_ptr_t nb_kfunc_ener_noprune_ptr[c_numElecTypes][c_numVdwTypes] = {
    { nbnxn_kernel_ElecCut_VdwLJ_VF_cuda,
      nbnxn_kernel_ElecCut_VdwLJCombGeom_VF_cuda,
      nbnxn_kernel_ElecCut_VdwLJCombLB_VF_cuda,
      nbnxn_kernel_ElecCut_VdwLJFsw_VF_cuda,
      nbnxn_kernel_ElecCut_VdwLJPsw_VF_cuda,
      nbnxn_kernel_ElecCut_VdwLJEwCombGeom_VF_cuda,
      nbnxn_kernel_ElecCut_VdwLJEwCombLB_VF_cuda },
    { nbnxn_kernel_ElecRF_VdwLJ_VF_cuda,
      nbnxn_kernel_ElecRF_VdwLJCombGeom_VF_cuda,
      nbnxn_kernel_ElecRF_VdwLJCombLB_VF_cuda,
      nbnxn_kernel_ElecRF_VdwLJFsw_VF_cuda,
      nbnxn_kernel_ElecRF_VdwLJPsw_VF_cuda,
      nbnxn_kernel_ElecRF_VdwLJEwCombGeom_VF_cuda,
      nbnxn_kernel_ElecRF_VdwLJEwCombLB_VF_cuda },
    { nbnxn_kernel_ElecEwQSTab_VdwLJ_VF_cuda,
      nbnxn_kernel_ElecEwQSTab_VdwLJCombGeom_VF_cuda,
      nbnxn_kernel_ElecEwQSTab_VdwLJCombLB_VF_cuda,
      nbnxn_kernel_ElecEwQSTab_VdwLJFsw_VF_cuda,
      nbnxn_kernel_ElecEwQSTab_VdwLJPsw_VF_cuda,
      nbnxn_kernel_ElecEwQSTab_VdwLJEwCombGeom_VF_cuda,
      nbnxn_kernel_ElecEwQSTab_VdwLJEwCombLB_VF_cuda },
    { nbnxn_kernel_ElecEwQSTabTwinCut_VdwLJ_VF_cuda,
      nbnxn_kernel_ElecEwQSTabTwinCut_VdwLJCombGeom_VF_cuda,
      nbnxn_kernel_ElecEwQSTabTwinCut_VdwLJCombLB_VF_cuda,
      nbnxn_kernel_ElecEwQSTabTwinCut_VdwLJFsw_VF_cuda,
      nbnxn_kernel_ElecEwQSTabTwinCut_VdwLJPsw_VF_cuda,
      nbnxn_kernel_ElecEwQSTabTwinCut_VdwLJEwCombGeom_VF_cuda,
      nbnxn_kernel_ElecEwQSTabTwinCut_VdwLJEwCombLB_VF_cuda },
    { nbnxn_kernel_ElecEw_VdwLJ_VF_cuda,
      nbnxn_kernel_ElecEw_VdwLJCombGeom_VF_cuda,
      nbnxn_kernel_ElecEw_VdwLJCombLB_VF_cuda,
      nbnxn_kernel_ElecEw_VdwLJFsw_VF_cuda,
      nbnxn_kernel_ElecEw_VdwLJPsw_VF_cuda,
      nbnxn_kernel_ElecEw_VdwLJEwCombGeom_VF_cuda,
      nbnxn_kernel_ElecEw_VdwLJEwCombLB_VF_cuda },
    { nbnxn_kernel_ElecEwTwinCut_VdwLJ_VF_cuda,
      nbnxn_kernel_ElecEwTwinCut_VdwLJCombGeom_VF_cuda,
      nbnxn_kernel_ElecEwTwinCut_VdwLJCombLB_VF_cuda,
      nbnxn_kernel_ElecEwTwinCut_VdwLJFsw_VF_cuda,
      nbnxn_kernel_ElecEwTwinCut_VdwLJPsw_VF_cuda,
      nbnxn_kernel_ElecEwTwinCut_VdwLJEwCombGeom_VF_cuda,
      nbnxn_kernel_ElecEwTwinCut_VdwLJEwCombLB_VF_cuda }
};

/*! Force + pruning kernel function pointers. */
static const nbnxn_cu_kfunc_ptr_t nb_kfunc_noener_prune_ptr[c_numElecTypes][c_numVdwTypes] = {
    { nbnxn_kernel_ElecCut_VdwLJ_F_prune_cuda,
      nbnxn_kernel_ElecCut_VdwLJCombGeom_F_prune_cuda,
      nbnxn_kernel_ElecCut_VdwLJCombLB_F_prune_cuda,
      nbnxn_kernel_ElecCut_VdwLJFsw_F_prune_cuda,
      nbnxn_kernel_ElecCut_VdwLJPsw_F_prune_cuda,
      nbnxn_kernel_ElecCut_VdwLJEwCombGeom_F_prune_cuda,
      nbnxn_kernel_ElecCut_VdwLJEwCombLB_F_prune_cuda },
    { nbnxn_kernel_ElecRF_VdwLJ_F_prune_cuda,
      nbnxn_kernel_ElecRF_VdwLJCombGeom_F_prune_cuda,
      nbnxn_kernel_ElecRF_VdwLJCombLB_F_prune_cuda,
      nbnxn_kernel_ElecRF_VdwLJFsw_F_prune_cuda,
      nbnxn_kernel_ElecRF_VdwLJPsw_F_prune_cuda,
      nbnxn_kernel_ElecRF_VdwLJEwCombGeom_F_prune_cuda,
      nbnxn_kernel_ElecRF_VdwLJEwCombLB_F_prune_cuda },
    { nbnxn_kernel_ElecEwQSTab_VdwLJ_F_prune_cuda,
      nbnxn_kernel_ElecEwQSTab_VdwLJCombGeom_F_prune_cuda,
      nbnxn_kernel_ElecEwQSTab_VdwLJCombLB_F_prune_cuda,
      nbnxn_kernel_ElecEwQSTab_VdwLJFsw_F_prune_cuda,
      nbnxn_kernel_ElecEwQSTab_VdwLJPsw_F_prune_cuda,
      nbnxn_kernel_ElecEwQSTab_VdwLJEwCombGeom_F_prune_cuda,
      nbnxn_kernel_ElecEwQSTab_VdwLJEwCombLB_F_prune_cuda },
    { nbnxn_kernel_ElecEwQSTabTwinCut_VdwLJ_F_prune_cuda,
      nbnxn_kernel_ElecEwQSTabTwinCut_VdwLJCombGeom_F_prune_cuda,
      nbnxn_kernel_ElecEwQSTabTwinCut_VdwLJCombLB_F_prune_cuda,
      nbnxn_kernel_ElecEwQSTabTwinCut_VdwLJFsw_F_prune_cuda,
      nbnxn_kernel_ElecEwQSTabTwinCut_VdwLJPsw_F_prune_cuda,
      nbnxn_kernel_ElecEwQSTabTwinCut_VdwLJEwCombGeom_F_prune_cuda,
      nbnxn_kernel_ElecEwQSTabTwinCut_VdwLJEwCombLB_F_prune_cuda },
    { nbnxn_kernel_ElecEw_VdwLJ_F_prune_cuda,
      nbnxn_kernel_ElecEw_VdwLJCombGeom_F_prune_cuda,
      nbnxn_kernel_ElecEw_VdwLJCombLB_F_prune_cuda,
      nbnxn_kernel_ElecEw_VdwLJFsw_F_prune_cuda,
      nbnxn_kernel_ElecEw_VdwLJPsw_F_prune_cuda,
      nbnxn_kernel_ElecEw_VdwLJEwCombGeom_F_prune_cuda,
      nbnxn_kernel_ElecEw_VdwLJEwCombLB_F_prune_cuda },
    { nbnxn_kernel_ElecEwTwinCut_VdwLJ_F_prune_cuda,
      nbnxn_kernel_ElecEwTwinCut_VdwLJCombGeom_F_prune_cuda,
      nbnxn_kernel_ElecEwTwinCut_VdwLJCombLB_F_prune_cuda,
      nbnxn_kernel_ElecEwTwinCut_VdwLJFsw_F_prune_cuda,
      nbnxn_kernel_ElecEwTwinCut_VdwLJPsw_F_prune_cuda,
      nbnxn_kernel_ElecEwTwinCut_VdwLJEwCombGeom_F_prune_cuda,
      nbnxn_kernel_ElecEwTwinCut_VdwLJEwCombLB_F_prune_cuda }
};

/*! Force + energy + pruning kernel function pointers. */
static const nbnxn_cu_kfunc_ptr_t nb_kfunc_ener_prune_ptr[c_numElecTypes][c_numVdwTypes] = {
    { nbnxn_kernel_ElecCut_VdwLJ_VF_prune_cuda,
      nbnxn_kernel_ElecCut_VdwLJCombGeom_VF_prune_cuda,
      nbnxn_kernel_ElecCut_VdwLJCombLB_VF_prune_cuda,
      nbnxn_kernel_ElecCut_VdwLJFsw_VF_prune_cuda,
      nbnxn_kernel_ElecCut_VdwLJPsw_VF_prune_cuda,
      nbnxn_kernel_ElecCut_VdwLJEwCombGeom_VF_prune_cuda,
      nbnxn_kernel_ElecCut_VdwLJEwCombLB_VF_prune_cuda },
    { nbnxn_kernel_ElecRF_VdwLJ_VF_prune_cuda,
      nbnxn_kernel_ElecRF_VdwLJCombGeom_VF_prune_cuda,
      nbnxn_kernel_ElecRF_VdwLJCombLB_VF_prune_cuda,
      nbnxn_kernel_ElecRF_VdwLJFsw_VF_prune_cuda,
      nbnxn_kernel_ElecRF_VdwLJPsw_VF_prune_cuda,
      nbnxn_kernel_ElecRF_VdwLJEwCombGeom_VF_prune_cuda,
      nbnxn_kernel_ElecRF_VdwLJEwCombLB_VF_prune_cuda },
    { nbnxn_kernel_ElecEwQSTab_VdwLJ_VF_prune_cuda,
      nbnxn_kernel_ElecEwQSTab_VdwLJCombGeom_VF_prune_cuda,
      nbnxn_kernel_ElecEwQSTab_VdwLJCombLB_VF_prune_cuda,
      nbnxn_kernel_ElecEwQSTab_VdwLJFsw_VF_prune_cuda,
      nbnxn_kernel_ElecEwQSTab_VdwLJPsw_VF_prune_cuda,
      nbnxn_kernel_ElecEwQSTab_VdwLJEwCombGeom_VF_prune_cuda,
      nbnxn_kernel_ElecEwQSTab_VdwLJEwCombLB_VF_prune_cuda },
    { nbnxn_kernel_ElecEwQSTabTwinCut_VdwLJ_VF_prune_cuda,
      nbnxn_kernel_ElecEwQSTabTwinCut_VdwLJCombGeom_VF_prune_cuda,
      nbnxn_kernel_ElecEwQSTabTwinCut_VdwLJCombLB_VF_prune_cuda,
      nbnxn_kernel_ElecEwQSTabTwinCut_VdwLJFsw_VF_prune_cuda,
      nbnxn_kernel_ElecEwQSTabTwinCut_VdwLJPsw_VF_prune_cuda,
      nbnxn_kernel_ElecEwQSTabTwinCut_VdwLJEwCombGeom_VF_prune_cuda,
      nbnxn_kernel_ElecEwQSTabTwinCut_VdwLJEwCombLB_VF_prune_cuda },
    { nbnxn_kernel_ElecEw_VdwLJ_VF_prune_cuda,
      nbnxn_kernel_ElecEw_VdwLJCombGeom_VF_prune_cuda,
      nbnxn_kernel_ElecEw_VdwLJCombLB_VF_prune_cuda,
      nbnxn_kernel_ElecEw_VdwLJFsw_VF_prune_cuda,
      nbnxn_kernel_ElecEw_VdwLJPsw_VF_prune_cuda,
      nbnxn_kernel_ElecEw_VdwLJEwCombGeom_VF_prune_cuda,
      nbnxn_kernel_ElecEw_VdwLJEwCombLB_VF_prune_cuda },
    { nbnxn_kernel_ElecEwTwinCut_VdwLJ_VF_prune_cuda,
      nbnxn_kernel_ElecEwTwinCut_VdwLJCombGeom_VF_prune_cuda,
      nbnxn_kernel_ElecEwTwinCut_VdwLJCombLB_VF_prune_cuda,
      nbnxn_kernel_ElecEwTwinCut_VdwLJFsw_VF_prune_cuda,
      nbnxn_kernel_ElecEwTwinCut_VdwLJPsw_VF_prune_cuda,
      nbnxn_kernel_ElecEwTwinCut_VdwLJEwCombGeom_VF_prune_cuda,
      nbnxn_kernel_ElecEwTwinCut_VdwLJEwCombLB_VF_prune_cuda }
};

/*! Return a pointer to the kernel version to be executed at the current step. */
static inline nbnxn_cu_kfunc_ptr_t select_nbnxn_kernel(enum ElecType elecType,
                                                       enum VdwType  vdwType,
                                                       bool          bDoEne,
                                                       bool          bDoPrune,
                                                       const DeviceInformation gmx_unused* deviceInfo)
{
    const int elecTypeIdx = static_cast<int>(elecType);
    const int vdwTypeIdx  = static_cast<int>(vdwType);

    GMX_ASSERT(elecTypeIdx < c_numElecTypes,
               "The electrostatics type requested is not implemented in the CUDA kernels.");
    GMX_ASSERT(vdwTypeIdx < c_numVdwTypes,
               "The VdW type requested is not implemented in the CUDA kernels.");

    /* assert assumptions made by the kernels */
    GMX_ASSERT(c_clusterSize * c_clusterSize / c_clusterSplitSize == deviceInfo->prop.warpSize,
               "The CUDA kernels require the "
               "cluster_size_i*cluster_size_j/nbnxn_gpu_clusterpair_split to match the warp size "
               "of the architecture targeted.");

    if (bDoEne)
    {
        if (bDoPrune)
        {
            return nb_kfunc_ener_prune_ptr[elecTypeIdx][vdwTypeIdx];
        }
        else
        {
            return nb_kfunc_ener_noprune_ptr[elecTypeIdx][vdwTypeIdx];
        }
    }
    else
    {
        if (bDoPrune)
        {
            return nb_kfunc_noener_prune_ptr[elecTypeIdx][vdwTypeIdx];
        }
        else
        {
            return nb_kfunc_noener_noprune_ptr[elecTypeIdx][vdwTypeIdx];
        }
    }
}

/*! \brief Calculates the amount of shared memory required by the nonbonded kernel in use. */
static inline int calc_shmem_required_nonbonded(const int                           num_threads_z,
                                                const DeviceInformation gmx_unused* deviceInfo,
                                                const NBParamGpu*                   nbp)
{
    int shmem;

    assert(deviceInfo);

    /* size of shmem (force-buffers/xq/atom type preloading) */
    /* NOTE: with the default kernel on sm3.0 we need shmem only for pre-loading */
    /* i-atom x+q in shared memory */
    shmem = c_superClusterSize * c_clusterSize * sizeof(float4);
    /* cj in shared memory, for each warp separately */
    shmem += num_threads_z * c_clusterSplitSize * c_jGroupSize * sizeof(int);

    if (nbp->vdwType == VdwType::CutCombGeom || nbp->vdwType == VdwType::CutCombLB)
    {
        /* i-atom LJ combination parameters in shared memory */
        shmem += c_superClusterSize * c_clusterSize * sizeof(float2);
    }
    else
    {
        /* i-atom types in shared memory */
        shmem += c_superClusterSize * c_clusterSize * sizeof(int);
    }
    /* for reducing prunedPairListCount over all warps in the block, to be used in plist sorting */
    shmem += 1 * sizeof(int);

    return shmem;
}


/*! \brief Calculates the amount of shared memory required by the nonbonded kernel in use.
 *
 * Take counts prepared in combined prune and interaction kernel and use them to sort plist.
 * Note that this sorted list is not available in the combined prune and interaction kernel
 * itself, which causes a performance degredation of 1-10% for that initial call */
static inline void gpuLaunchKernelSciSort(GpuPairlist* plist, const DeviceStream& deviceStream)
{
    performExclusiveScan(plist->sorting.nscanTemporary, plist->sorting.scanTemporary, plist, deviceStream);

    KernelLaunchConfig configSortSci;
    configSortSci.blockSize[0]     = c_sciSortingThreadsPerBlock;
    configSortSci.blockSize[1]     = 1;
    configSortSci.blockSize[2]     = 1;
    configSortSci.gridSize[0]      = gmx::divideRoundUp(plist->numSci, c_sciSortingThreadsPerBlock);
    configSortSci.sharedMemorySize = 0;

    const auto kernelSciSort = nbnxnKernelBucketSciSort;

    const auto kernelSciSortArgs = prepareGpuKernelArguments(kernelSciSort, configSortSci, plist);

    launchGpuKernel(kernelSciSort, configSortSci, deviceStream, nullptr, "nbnxn_kernel_sci_sort", kernelSciSortArgs);
}


/*! As we execute nonbonded workload in separate streams, before launching
   the kernel we need to make sure that he following operations have completed:
   - atomdata allocation and related H2D transfers (every nstlist step);
   - pair list H2D transfer (every nstlist step);
   - shift vector H2D transfer (every nstlist step);
   - force (+shift force and energy) output clearing (every step).

   These operations are issued in the local stream at the beginning of the step
   and therefore always complete before the local kernel launch. The non-local
   kernel is launched after the local on the same device/context hence it is
   inherently scheduled after the operations in the local stream (including the
   above "misc_ops") on pre-GK110 devices with single hardware queue, but on later
   devices with multiple hardware queues the dependency needs to be enforced.
   We use the misc_ops_and_local_H2D_done event to record the point where
   the local x+q H2D (and all preceding) tasks are complete and synchronize
   with this event in the non-local stream before launching the non-bonded kernel.
 */
void gpu_launch_kernel(NbnxmGpu* nb, const gmx::StepWorkload& stepWork, const InteractionLocality iloc)
{
    NBAtomDataGpu*      adat         = nb->atdat;
    NBParamGpu*         nbp          = nb->nbparam;
    auto*               plist        = nb->plist[iloc].get();
    GpuTimers*          timers       = nb->timers;
    const DeviceStream& deviceStream = *nb->deviceStreams[iloc];

    bool bDoTime = nb->bDoTime;

    /* Don't launch the non-local kernel if there is no work to do.
       Doing the same for the local kernel is more complicated, since the
       local part of the force array also depends on the non-local kernel.
       So to avoid complicating the code and to reduce the risk of bugs,
       we always call the local kernel, and later (not in
       this function) the stream wait, local f copyback and the f buffer
       clearing. All these operations, except for the local interaction kernel,
       are needed for the non-local interactions. The skip of the local kernel
       call is taken care of later in this function. */
    if (canSkipNonbondedWork(*nb, iloc))
    {
        plist->haveFreshList = false;

        return;
    }

    if (nbp->useDynamicPruning && plist->haveFreshList)
    {
        /* Prunes for rlistOuter and rlistInner, sets plist->haveFreshList=false
           (TODO: ATM that's the way the timing accounting can distinguish between
           separate prune kernel and combined force+prune, maybe we need a better way?).
         */
        gpu_launch_kernel_pruneonly(nb, iloc, 1);
    }

    if (plist->numSci == 0)
    {
        /* Don't launch an empty local kernel (not allowed with CUDA) */
        return;
    }

    /* beginning of timed nonbonded calculation section */
    if (bDoTime)
    {
        timers->interaction[iloc].nb_k.openTimingRegion(deviceStream);
    }

    /* Kernel launch config:
     * - The thread block dimensions match the size of i-clusters, j-clusters,
     *   and j-cluster concurrency, in x, y, and z, respectively.
     * - The 1D block-grid contains as many blocks as super-clusters.
     */
    int num_threads_z = 1;
    if (nb->deviceContext_->deviceInfo().prop.major == 3 && nb->deviceContext_->deviceInfo().prop.minor == 7)
    {
        num_threads_z = 2;
    }
    int nblock = calc_nb_kernel_nblock(plist->numSci, &nb->deviceContext_->deviceInfo());


    KernelLaunchConfig config;
    config.blockSize[0] = c_clusterSize;
    config.blockSize[1] = c_clusterSize;
    config.blockSize[2] = num_threads_z;
    config.gridSize[0]  = nblock;
    config.sharedMemorySize =
            calc_shmem_required_nonbonded(num_threads_z, &nb->deviceContext_->deviceInfo(), nbp);

    if (debug)
    {
        fprintf(debug,
                "Non-bonded GPU launch configuration:\n\tThread block: %zux%zux%zu\n\t"
                "\tGrid: %zux%zu\n\t#Super-clusters/clusters: %d/%d (%d)\n"
                "\tShMem: %zu\n",
                config.blockSize[0],
                config.blockSize[1],
                config.blockSize[2],
                config.gridSize[0],
                config.gridSize[1],
                plist->numSci * c_superClusterSize,
                c_superClusterSize,
                plist->numAtomsPerCluster,
                config.sharedMemorySize);
    }

    auto* timingEvent = bDoTime ? timers->interaction[iloc].nb_k.fetchNextEvent() : nullptr;

    /* Whether we need to call a combined prune and interaction kernel or just an interaction
     * kernel. bDoPrune being true implies we are not using dynamic pruning and are in the first
     * call to the interaction kernel after a neighbour list step */
    bool       bDoPrune = (plist->haveFreshList && !nb->timers->interaction[iloc].didPrune);
    const auto kernel   = select_nbnxn_kernel(
            nbp->elecType, nbp->vdwType, stepWork.computeEnergy, bDoPrune, &nb->deviceContext_->deviceInfo());
    const auto kernelArgs =
            prepareGpuKernelArguments(kernel, config, adat, nbp, plist, &stepWork.computeVirial);
    launchGpuKernel(kernel, config, deviceStream, timingEvent, "k_calc_nb", kernelArgs);

    if (bDoPrune)
    {
        gpuLaunchKernelSciSort(plist, deviceStream);
    }


    if (bDoTime)
    {
        timers->interaction[iloc].nb_k.closeTimingRegion(deviceStream);
    }

    if (GMX_NATIVE_WINDOWS)
    {
        /* Windows: force flushing WDDM queue */
        hipStreamQuery(deviceStream.stream());
    }
}

/*! Calculates the amount of shared memory required by the CUDA kernel in use. */
static inline int calc_shmem_required_prune(const int num_threads_z, const DeviceInformation* deviceInfo)
{
    const int  archMajor = deviceInfo->prop.major;
    const bool preloadCj = archMajor < 7;
    int        shmem;

    /* i-atom x in shared memory */
    shmem = c_superClusterSize * c_clusterSize * sizeof(float4);
    if (preloadCj)
    {
        /* cj in shared memory, for each warp separately */
        shmem += num_threads_z * c_clusterSplitSize * c_jGroupSize * sizeof(int);
    }
    /* add 1 int for pruned pair count */
    shmem += sizeof(int);

    return shmem;
}

void gpu_launch_kernel_pruneonly(NbnxmGpu* nb, const InteractionLocality iloc, const int numParts)
{
    NBAtomDataGpu*      adat         = nb->atdat;
    NBParamGpu*         nbp          = nb->nbparam;
    auto*               plist        = nb->plist[iloc].get();
    GpuTimers*          timers       = nb->timers;
    const DeviceStream& deviceStream = *nb->deviceStreams[iloc];

    bool bDoTime = nb->bDoTime;

    if (plist->haveFreshList)
    {
        GMX_ASSERT(numParts == 1, "With first pruning we expect 1 part");

        /* Set rollingPruningNumParts to signal that it is not set */
        plist->rollingPruningNumParts = 0;
    }
    else
    {
        if (plist->rollingPruningNumParts == 0)
        {
            plist->rollingPruningNumParts = numParts;
        }
        else
        {
            GMX_ASSERT(numParts == plist->rollingPruningNumParts,
                       "It is not allowed to change numParts in between list generation steps");
        }
    }

    /* Compute the max number of list entries to prune across all passes
     * Note that the actual number for a specific pass will be computed inside the kernel.
     * Also note that this CUDA implementation (parts tracking on device) differs from the
     * other backends (parts tracking on host, passed as kernel argument).
     */
    const int numSciInPartMax = (plist->numSci + numParts - 1) / numParts;

    /* Don't launch the kernel if there is no work to do (not allowed with CUDA) */
    if (numSciInPartMax <= 0)
    {
        plist->haveFreshList = false;

        return;
    }

    GpuRegionTimer* timer = nullptr;
    if (bDoTime)
    {
        timer = &(plist->haveFreshList ? timers->interaction[iloc].prune_k
                                       : timers->interaction[iloc].rollingPrune_k);
    }

    /* beginning of timed prune calculation section */
    if (bDoTime)
    {
        timer->openTimingRegion(deviceStream);
    }

    /* Kernel launch config:
     * - The thread block dimensions match the size of i-clusters, j-clusters,
     *   and j-cluster concurrency, in x, y, and z, respectively.
     * - The 1D block-grid contains as many blocks as super-clusters.
     */
    int num_threads_z = c_pruneKernelJPackedConcurrency;
    int nblock        = calc_nb_kernel_nblock(numSciInPartMax, &nb->deviceContext_->deviceInfo());

    KernelLaunchConfig config;
    config.blockSize[0] = c_clusterSize;
    config.blockSize[1] = c_clusterSize;
    config.blockSize[2] = num_threads_z;
    config.gridSize[0]  = nblock;
    config.sharedMemorySize =
            calc_shmem_required_prune(num_threads_z, &nb->deviceContext_->deviceInfo());

    if (debug)
    {
        fprintf(debug,
                "Pruning GPU kernel launch configuration:\n\tThread block: %zux%zux%zu\n\t"
                "\tGrid: %zux%zu\n\t#Super-clusters/clusters: %d/%d (%d)\n"
                "\tShMem: %zu\n",
                config.blockSize[0],
                config.blockSize[1],
                config.blockSize[2],
                config.gridSize[0],
                config.gridSize[1],
                numSciInPartMax * c_superClusterSize,
                c_superClusterSize,
                plist->numAtomsPerCluster,
                config.sharedMemorySize);
    }

    auto*          timingEvent  = bDoTime ? timer->fetchNextEvent() : nullptr;
    constexpr char kernelName[] = "k_pruneonly";
    const auto     kernel =
            plist->haveFreshList ? nbnxn_kernel_prune_cuda<true> : nbnxn_kernel_prune_cuda<false>;
    const auto kernelArgs = prepareGpuKernelArguments(kernel, config, adat, nbp, plist, &numParts);
    launchGpuKernel(kernel, config, deviceStream, timingEvent, kernelName, kernelArgs);

    if (plist->haveFreshList)
    {
        gpuLaunchKernelSciSort(plist, deviceStream);
    }

    /* TODO: consider a more elegant way to track which kernel has been called
       (combined or separate 1st pass prune, rolling prune). */
    if (plist->haveFreshList)
    {
        plist->haveFreshList = false;
        /* Mark that pruning has been done */
        nb->timers->interaction[iloc].didPrune = true;
    }
    else
    {
        /* Mark that rolling pruning has been done */
        nb->timers->interaction[iloc].didRollingPrune = true;
    }

    if (bDoTime)
    {
        timer->closeTimingRegion(deviceStream);
    }

    if (GMX_NATIVE_WINDOWS)
    {
        /* Windows: force flushing WDDM queue */
        hipStreamQuery(deviceStream.stream());
    }
}

void cuda_set_cacheconfig()
{
    hipError_t stat;

    for (int i = 0; i < c_numElecTypes; i++)
    {
        for (int j = 0; j < c_numVdwTypes; j++)
        {
            /* Default kernel 32/32 kB Shared/L1 */
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(nb_kfunc_ener_prune_ptr[i][j]), hipFuncCachePreferEqual);
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(nb_kfunc_ener_noprune_ptr[i][j]), hipFuncCachePreferEqual);
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(nb_kfunc_noener_prune_ptr[i][j]), hipFuncCachePreferEqual);
            stat = hipFuncSetCacheConfig(reinterpret_cast<const void*>(nb_kfunc_noener_noprune_ptr[i][j]), hipFuncCachePreferEqual);
            CU_RET_ERR(stat, "hipFuncSetCacheConfig failed");
        }
    }
}

} // namespace gmx
