/*
 * This file is part of the GROMACS molecular simulation package.
 *
 * Copyright 2012- The GROMACS Authors
 * and the project initiators Erik Lindahl, Berk Hess and David van der Spoel.
 * Consult the AUTHORS/COPYING files and https://www.gromacs.org for details.
 *
 * GROMACS is free software; you can redistribute it and/or
 * modify it under the terms of the GNU Lesser General Public License
 * as published by the Free Software Foundation; either version 2.1
 * of the License, or (at your option) any later version.
 *
 * GROMACS is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
 * Lesser General Public License for more details.
 *
 * You should have received a copy of the GNU Lesser General Public
 * License along with GROMACS; if not, see
 * https://www.gnu.org/licenses, or write to the Free Software Foundation,
 * Inc., 51 Franklin Street, Fifth Floor, Boston, MA  02110-1301  USA.
 *
 * If you want to redistribute modifications to GROMACS, please
 * consider that scientific software is very special. Version
 * control is crucial - bugs must be traceable. We will be happy to
 * consider code for inclusion in the official distribution, but
 * derived work must not be called official GROMACS. Details are found
 * in the README & COPYING files - if they are missing, get the
 * official version at https://www.gromacs.org.
 *
 * To help us fund GROMACS development, we humbly ask that you cite
 * the research papers on the package. Check out https://www.gromacs.org.
 */
/*! \file
 *  \brief Define CUDA implementation of nbnxn_gpu_data_mgmt.h
 *
 *  \author Szilard Pall <pall.szilard@gmail.com>
 */
#include "gmxpre.h"

#include <cassert>
#include <cstdarg>
#include <cstdio>
#include <cstdlib>

// TODO We would like to move this down, but the way NbnxmGpu
//      is currently declared means this has to be before gpu_types.h
#include "nbnxm_cuda_types.h"

// TODO Remove this comment when the above order issue is resolved
#include <cub/device/device_scan.cuh>

#include "gromacs/gpu_utils/cudautils.cuh"
#include "gromacs/gpu_utils/device_context.h"
#include "gromacs/gpu_utils/gpu_utils.h"
#include "gromacs/gpu_utils/gpueventsynchronizer.h"
#include "gromacs/hardware/device_information.h"
#include "gromacs/hardware/device_management.h"
#include "gromacs/math/vectypes.h"
#include "gromacs/mdlib/force_flags.h"
#include "gromacs/mdtypes/interaction_const.h"
#include "gromacs/mdtypes/md_enums.h"
#include "gromacs/nbnxm/atomdata.h"
#include "gromacs/nbnxm/gpu_data_mgmt.h"
#include "gromacs/nbnxm/gpu_types_common.h"
#include "gromacs/nbnxm/gridset.h"
#include "gromacs/nbnxm/nbnxm.h"
#include "gromacs/nbnxm/nbnxm_gpu.h"
#include "gromacs/nbnxm/nbnxm_gpu_data_mgmt.h"
#include "gromacs/nbnxm/pairlistsets.h"
#include "gromacs/pbcutil/ishift.h"
#include "gromacs/timing/gpu_timing.h"
#include "gromacs/utility/basedefinitions.h"
#include "gromacs/utility/cstringutil.h"
#include "gromacs/utility/fatalerror.h"
#include "gromacs/utility/real.h"
#include "gromacs/utility/smalloc.h"

#include "nbnxm_cuda.h"
/* Required to stop gcc emitting multiple definition warnings as hip/hip_fp16.h, which is included by
 * device_scan.cuh, doesn't undef __WSB_DEPRECATION_MESSAGE and this is later redefined in
 * device_atomic_functions.h used by nbnxm_cuda_types.h. Seen in cuda 10 and 11 with gcc-11. */
#undef __WSB_DEPRECATION_MESSAGE

namespace gmx
{

/* This is a heuristically determined parameter for the Kepler
 * and Maxwell architectures for the minimum size of ci lists by multiplying
 * this constant with the # of multiprocessors on the current device.
 * Since the maximum number of blocks per multiprocessor is 16, the ideal
 * count for small systems is 32 or 48 blocks per multiprocessor. Because
 * there is a bit of fluctuations in the generated block counts, we use
 * a target of 44 instead of the ideal value of 48.
 */

#if GMX_PTX_ARCH <= 700
static const unsigned int gpu_min_ci_balanced_factor = 44;
#else
/* Updated benchmarking on Ampere, Ada, Hopper shows the ideal count is
 * between 61 and 83 depending on chip */
static const unsigned int gpu_min_ci_balanced_factor = 61;
#endif


void gpu_init_platform_specific(NbnxmGpu* /* nb */)
{
    /* set the kernel type for the current GPU */
    /* pick L1 cache configuration */
    cuda_set_cacheconfig();
}

void gpu_free_platform_specific(NbnxmGpu* /* nb */)
{
    // Nothing specific in CUDA
}

int gpu_min_ci_balanced(NbnxmGpu* nb)
{
    return nb != nullptr ? gpu_min_ci_balanced_factor * nb->deviceContext_->deviceInfo().prop.multiProcessorCount
                         : 0;
}

namespace
{

template<PairlistType pairlistType>
size_t cudaCubWrapper(size_t                     temporaryBufferSize,
                      char*                      temporaryBuffer,
                      GpuPairlist<pairlistType>* d_plist,
                      const DeviceStream&        deviceStream)
{
    size_t size = temporaryBufferSize;
    hipcub::DeviceScan::ExclusiveSum(temporaryBuffer,
                                  size,
                                  d_plist->sorting.sciHistogram,
                                  d_plist->sorting.sciOffset,
                                  c_sciHistogramSize,
                                  deviceStream.stream());
    return size;
}

} // namespace

template<PairlistType pairlistType>
size_t getExclusiveScanWorkingArraySize(GpuPairlist<pairlistType>* plist, const DeviceStream& deviceStream)
{
    return cudaCubWrapper(0, nullptr, plist, deviceStream);
}

template<PairlistType pairlistType>
void performExclusiveScan(size_t                     temporaryBufferSize,
                          char*                      temporaryBuffer,
                          GpuPairlist<pairlistType>* plist,
                          const DeviceStream&        deviceStream)
{
    std::ignore = cudaCubWrapper(temporaryBufferSize, temporaryBuffer, plist, deviceStream);
}

template size_t getExclusiveScanWorkingArraySize<PairlistType::Hierarchical8x8x8>(
        GpuPairlist<PairlistType::Hierarchical8x8x8>* plist,
        const DeviceStream&                           deviceStream);

template void performExclusiveScan<PairlistType::Hierarchical8x8x8>(
        size_t                                        temporaryBufferSize,
        char*                                         temporaryBuffer,
        GpuPairlist<PairlistType::Hierarchical8x8x8>* plist,
        const DeviceStream&                           deviceStream);

} // namespace gmx
