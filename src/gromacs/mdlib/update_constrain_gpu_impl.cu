#include "hip/hip_runtime.h"
/*
 * This file is part of the GROMACS molecular simulation package.
 *
 * Copyright (c) 2019,2020, by the GROMACS development team, led by
 * Mark Abraham, David van der Spoel, Berk Hess, and Erik Lindahl,
 * and including many others, as listed in the AUTHORS file in the
 * top-level source directory and at http://www.gromacs.org.
 *
 * GROMACS is free software; you can redistribute it and/or
 * modify it under the terms of the GNU Lesser General Public License
 * as published by the Free Software Foundation; either version 2.1
 * of the License, or (at your option) any later version.
 *
 * GROMACS is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
 * Lesser General Public License for more details.
 *
 * You should have received a copy of the GNU Lesser General Public
 * License along with GROMACS; if not, see
 * http://www.gnu.org/licenses, or write to the Free Software Foundation,
 * Inc., 51 Franklin Street, Fifth Floor, Boston, MA  02110-1301  USA.
 *
 * If you want to redistribute modifications to GROMACS, please
 * consider that scientific software is very special. Version
 * control is crucial - bugs must be traceable. We will be happy to
 * consider code for inclusion in the official distribution, but
 * derived work must not be called official GROMACS. Details are found
 * in the README & COPYING files - if they are missing, get the
 * official version at http://www.gromacs.org.
 *
 * To help us fund GROMACS development, we humbly ask that you cite
 * the research papers on the package. Check out http://www.gromacs.org.
 */
/*! \internal \file
 *
 * \brief Implements update and constraints class using CUDA.
 *
 * The class combines Leap-Frog integrator with LINCS and SETTLE constraints.
 *
 * \todo The computational procedures in members should be integrated to improve
 *       computational performance.
 *
 * \author Artem Zhmurov <zhmurov@gmail.com>
 *
 * \ingroup module_mdlib
 */
#include "gmxpre.h"

#include "update_constrain_gpu_impl.h"

#include <assert.h>
#include <stdio.h>

#include <cmath>

#include <algorithm>

#include "gromacs/gpu_utils/cudautils.cuh"
#include "gromacs/gpu_utils/devicebuffer.h"
#include "gromacs/gpu_utils/gputraits.cuh"
#include "gromacs/gpu_utils/vectype_ops.cuh"
#include "gromacs/mdlib/leapfrog_gpu.cuh"
#include "gromacs/mdlib/lincs_gpu.cuh"
#include "gromacs/mdlib/settle_gpu.cuh"
#include "gromacs/mdlib/update_constrain_gpu.h"

namespace gmx
{
/*!\brief Number of CUDA threads in a block
 *
 * \todo Check if using smaller block size will lead to better prformance.
 */
constexpr static int c_threadsPerBlock = 256;
//! Maximum number of threads in a block (for __launch_bounds__)
constexpr static int c_maxThreadsPerBlock = c_threadsPerBlock;

/*! \brief Scaling matrix struct.
 *
 * \todo Should be generalized.
 */
struct ScalingMatrix
{
    float xx, yy, zz, yx, zx, zy;
};

__launch_bounds__(c_maxThreadsPerBlock) __global__
        static void scaleCoordinates_kernel(const int numAtoms,
                                            float3* __restrict__ gm_x,
                                            const ScalingMatrix scalingMatrix)
{
    int threadIndex = blockIdx.x * blockDim.x + threadIdx.x;
    if (threadIndex < numAtoms)
    {
        float3 x = gm_x[threadIndex];

        x.x = scalingMatrix.xx * x.x + scalingMatrix.yx * x.y + scalingMatrix.zx * x.z;
        x.y = scalingMatrix.yy * x.y + scalingMatrix.zy * x.z;
        x.z = scalingMatrix.zz * x.z;

        gm_x[threadIndex] = x;
    }
}

void UpdateConstrainGpu::Impl::integrate(GpuEventSynchronizer*             fReadyOnDevice,
                                         const real                        dt,
                                         const bool                        updateVelocities,
                                         const bool                        computeVirial,
                                         tensor                            virial,
                                         const bool                        doTemperatureScaling,
                                         gmx::ArrayRef<const t_grp_tcstat> tcstat,
                                         const bool                        doParrinelloRahman,
                                         const float                       dtPressureCouple,
                                         const matrix                      prVelocityScalingMatrix)
{
    // Clearing virial matrix
    // TODO There is no point in having separate virial matrix for constraints
    clear_mat(virial);

    // Make sure that the forces are ready on device before proceeding with the update.
    fReadyOnDevice->enqueueWaitEvent(commandStream_);

    // The integrate should save a copy of the current coordinates in d_xp_ and write updated once
    // into d_x_. The d_xp_ is only needed by constraints.
    integrator_->integrate(d_x_, d_xp_, d_v_, d_f_, dt, doTemperatureScaling, tcstat,
                           doParrinelloRahman, dtPressureCouple, prVelocityScalingMatrix);
    // Constraints need both coordinates before (d_x_) and after (d_xp_) update. However, after constraints
    // are applied, the d_x_ can be discarded. So we intentionally swap the d_x_ and d_xp_ here to avoid the
    // d_xp_ -> d_x_ copy after constraints. Note that the integrate saves them in the wrong order as well.
    lincsGpu_->apply(d_xp_, d_x_, updateVelocities, d_v_, 1.0 / dt, computeVirial, virial, pbcAiuc_);
    settleGpu_->apply(d_xp_, d_x_, updateVelocities, d_v_, 1.0 / dt, computeVirial, virial, pbcAiuc_);

    // scaledVirial -> virial (methods above returns scaled values)
    float scaleFactor = 0.5f / (dt * dt);
    for (int i = 0; i < DIM; i++)
    {
        for (int j = 0; j < DIM; j++)
        {
            virial[i][j] = scaleFactor * virial[i][j];
        }
    }

    coordinatesReady_->markEvent(commandStream_);

    return;
}

void UpdateConstrainGpu::Impl::scaleCoordinates(const matrix scalingMatrix)
{
    ScalingMatrix mu;
    mu.xx = scalingMatrix[XX][XX];
    mu.yy = scalingMatrix[YY][YY];
    mu.zz = scalingMatrix[ZZ][ZZ];
    mu.yx = scalingMatrix[YY][XX];
    mu.zx = scalingMatrix[ZZ][XX];
    mu.zy = scalingMatrix[ZZ][YY];

    const auto kernelArgs = prepareGpuKernelArguments(
            scaleCoordinates_kernel, coordinateScalingKernelLaunchConfig_, &numAtoms_, &d_x_, &mu);
    launchGpuKernel(scaleCoordinates_kernel, coordinateScalingKernelLaunchConfig_, nullptr,
                    "scaleCoordinates_kernel", kernelArgs);
    // TODO: Although this only happens on the pressure coupling steps, this synchronization
    //       can affect the perfornamce if nstpcouple is small.
    gpuStreamSynchronize(commandStream_);
}

UpdateConstrainGpu::Impl::Impl(const t_inputrec&     ir,
                               const gmx_mtop_t&     mtop,
                               const void*           commandStream,
                               GpuEventSynchronizer* xUpdatedOnDevice) :
    coordinatesReady_(xUpdatedOnDevice)
{
    GMX_ASSERT(xUpdatedOnDevice != nullptr, "The event synchronizer can not be nullptr.");
    commandStream != nullptr ? commandStream_ = *static_cast<const CommandStream*>(commandStream)
                             : commandStream_ = nullptr;


    integrator_ = std::make_unique<LeapFrogGpu>(commandStream_);
    lincsGpu_   = std::make_unique<LincsGpu>(ir.nLincsIter, ir.nProjOrder, commandStream_);
    settleGpu_  = std::make_unique<SettleGpu>(mtop, commandStream_);

    coordinateScalingKernelLaunchConfig_.blockSize[0]     = c_threadsPerBlock;
    coordinateScalingKernelLaunchConfig_.blockSize[1]     = 1;
    coordinateScalingKernelLaunchConfig_.blockSize[2]     = 1;
    coordinateScalingKernelLaunchConfig_.sharedMemorySize = 0;
    coordinateScalingKernelLaunchConfig_.stream           = commandStream_;
}

UpdateConstrainGpu::Impl::~Impl() {}

void UpdateConstrainGpu::Impl::set(DeviceBuffer<RVec>       d_x,
                                   DeviceBuffer<RVec>       d_v,
                                   const DeviceBuffer<RVec> d_f,
                                   const t_idef&            idef,
                                   const t_mdatoms&         md,
                                   const int                numTempScaleValues)
{
    GMX_ASSERT(d_x != nullptr, "Coordinates device buffer should not be null.");
    GMX_ASSERT(d_v != nullptr, "Velocities device buffer should not be null.");
    GMX_ASSERT(d_f != nullptr, "Forces device buffer should not be null.");

    d_x_ = reinterpret_cast<float3*>(d_x);
    d_v_ = reinterpret_cast<float3*>(d_v);
    d_f_ = reinterpret_cast<float3*>(d_f);

    numAtoms_ = md.nr;

    reallocateDeviceBuffer(&d_xp_, numAtoms_, &numXp_, &numXpAlloc_, nullptr);

    reallocateDeviceBuffer(&d_inverseMasses_, numAtoms_, &numInverseMasses_,
                           &numInverseMassesAlloc_, nullptr);

    // Integrator should also update something, but it does not even have a method yet
    integrator_->set(md, numTempScaleValues, md.cTC);
    lincsGpu_->set(idef, md);
    settleGpu_->set(idef, md);

    coordinateScalingKernelLaunchConfig_.gridSize[0] =
            (numAtoms_ + c_threadsPerBlock - 1) / c_threadsPerBlock;
}

void UpdateConstrainGpu::Impl::setPbc(const PbcType pbcType, const matrix box)
{
    setPbcAiuc(numPbcDimensions(pbcType), box, &pbcAiuc_);
}

GpuEventSynchronizer* UpdateConstrainGpu::Impl::getCoordinatesReadySync()
{
    return coordinatesReady_;
}

UpdateConstrainGpu::UpdateConstrainGpu(const t_inputrec&     ir,
                                       const gmx_mtop_t&     mtop,
                                       const void*           commandStream,
                                       GpuEventSynchronizer* xUpdatedOnDevice) :
    impl_(new Impl(ir, mtop, commandStream, xUpdatedOnDevice))
{
}

UpdateConstrainGpu::~UpdateConstrainGpu() = default;

void UpdateConstrainGpu::integrate(GpuEventSynchronizer*             fReadyOnDevice,
                                   const real                        dt,
                                   const bool                        updateVelocities,
                                   const bool                        computeVirial,
                                   tensor                            virialScaled,
                                   const bool                        doTemperatureScaling,
                                   gmx::ArrayRef<const t_grp_tcstat> tcstat,
                                   const bool                        doParrinelloRahman,
                                   const float                       dtPressureCouple,
                                   const matrix                      prVelocityScalingMatrix)
{
    impl_->integrate(fReadyOnDevice, dt, updateVelocities, computeVirial, virialScaled, doTemperatureScaling,
                     tcstat, doParrinelloRahman, dtPressureCouple, prVelocityScalingMatrix);
}

void UpdateConstrainGpu::scaleCoordinates(const matrix scalingMatrix)
{
    impl_->scaleCoordinates(scalingMatrix);
}

void UpdateConstrainGpu::set(DeviceBuffer<RVec>       d_x,
                             DeviceBuffer<RVec>       d_v,
                             const DeviceBuffer<RVec> d_f,
                             const t_idef&            idef,
                             const t_mdatoms&         md,
                             const int                numTempScaleValues)
{
    impl_->set(d_x, d_v, d_f, idef, md, numTempScaleValues);
}

void UpdateConstrainGpu::setPbc(const PbcType pbcType, const matrix box)
{
    impl_->setPbc(pbcType, box);
}

GpuEventSynchronizer* UpdateConstrainGpu::getCoordinatesReadySync()
{
    return impl_->getCoordinatesReadySync();
}

bool UpdateConstrainGpu::isNumCoupledConstraintsSupported(const gmx_mtop_t& mtop)
{
    return LincsGpu::isNumCoupledConstraintsSupported(mtop);
}

} // namespace gmx
