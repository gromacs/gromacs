#include "hip/hip_runtime.h"
/*
 * This file is part of the GROMACS molecular simulation package.
 *
 * Copyright (c) 2018,2019, by the GROMACS development team, led by
 * Mark Abraham, David van der Spoel, Berk Hess, and Erik Lindahl,
 * and including many others, as listed in the AUTHORS file in the
 * top-level source directory and at http://www.gromacs.org.
 *
 * GROMACS is free software; you can redistribute it and/or
 * modify it under the terms of the GNU Lesser General Public License
 * as published by the Free Software Foundation; either version 2.1
 * of the License, or (at your option) any later version.
 *
 * GROMACS is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
 * Lesser General Public License for more details.
 *
 * You should have received a copy of the GNU Lesser General Public
 * License along with GROMACS; if not, see
 * http://www.gnu.org/licenses, or write to the Free Software Foundation,
 * Inc., 51 Franklin Street, Fifth Floor, Boston, MA  02110-1301  USA.
 *
 * If you want to redistribute modifications to GROMACS, please
 * consider that scientific software is very special. Version
 * control is crucial - bugs must be traceable. We will be happy to
 * consider code for inclusion in the official distribution, but
 * derived work must not be called official GROMACS. Details are found
 * in the README & COPYING files - if they are missing, get the
 * official version at http://www.gromacs.org.
 *
 * To help us fund GROMACS development, we humbly ask that you cite
 * the research papers on the package. Check out http://www.gromacs.org.
 */
/*! \internal \file
 * \brief Subroutines to run LINCS on GPU
 *
 * Copies data to GPU, runs LINCS and copies the results back.
 *
 * \author Artem Zhmurov <zhmurov@gmail.com>
 * \ingroup module_mdlib
 */
#include "gmxpre.h"

#include "constrtestrunners.h"

#include <assert.h>

#include <cmath>

#include <algorithm>
#include <vector>

#include "gromacs/gpu_utils/devicebuffer.cuh"
#include "gromacs/gpu_utils/gpu_utils.h"
#include "gromacs/mdlib/lincs_cuda.cuh"
#include "gromacs/pbcutil/pbc.h"
#include "gromacs/utility/unique_cptr.h"

namespace gmx
{
namespace test
{

/*! \brief
 * Initialize and apply LINCS constraints on CUDA-enabled GPU.
 *
 * \param[in] testData        Test data structure.
 * \param[in] pbc             Periodic boundary data.
 */
void applyLincsCuda(ConstraintsTestData* testData, t_pbc pbc)
{
    auto lincsCuda =
            std::make_unique<LincsCuda>(testData->ir_.nLincsIter, testData->ir_.nProjOrder, nullptr);

    bool    updateVelocities = true;
    int     numAtoms         = testData->numAtoms_;
    float3 *d_x, *d_xp, *d_v;

    lincsCuda->set(testData->idef_, testData->md_);
    lincsCuda->setPbc(&pbc);

    allocateDeviceBuffer(&d_x, numAtoms, nullptr);
    allocateDeviceBuffer(&d_xp, numAtoms, nullptr);
    allocateDeviceBuffer(&d_v, numAtoms, nullptr);

    copyToDeviceBuffer(&d_x, (float3*)(testData->x_.data()), 0, numAtoms, nullptr,
                       GpuApiCallBehavior::Sync, nullptr);
    copyToDeviceBuffer(&d_xp, (float3*)(testData->xPrime_.data()), 0, numAtoms, nullptr,
                       GpuApiCallBehavior::Sync, nullptr);
    if (updateVelocities)
    {
        copyToDeviceBuffer(&d_v, (float3*)(testData->v_.data()), 0, numAtoms, nullptr,
                           GpuApiCallBehavior::Sync, nullptr);
    }
    lincsCuda->apply(d_x, d_xp, updateVelocities, d_v, testData->invdt_, testData->computeVirial_,
                     testData->virialScaled_);

    copyFromDeviceBuffer((float3*)(testData->xPrime_.data()), &d_xp, 0, numAtoms, nullptr,
                         GpuApiCallBehavior::Sync, nullptr);
    if (updateVelocities)
    {
        copyFromDeviceBuffer((float3*)(testData->v_.data()), &d_v, 0, numAtoms, nullptr,
                             GpuApiCallBehavior::Sync, nullptr);
    }

    freeDeviceBuffer(&d_x);
    freeDeviceBuffer(&d_xp);
    freeDeviceBuffer(&d_v);
}

} // namespace test
} // namespace gmx
