/*
 * This file is part of the GROMACS molecular simulation package.
 *
 * Copyright (c) 2012,2013,2014,2015,2016,2017,2018, by the GROMACS development team, led by
 * Mark Abraham, David van der Spoel, Berk Hess, and Erik Lindahl,
 * and including many others, as listed in the AUTHORS file in the
 * top-level source directory and at http://www.gromacs.org.
 *
 * GROMACS is free software; you can redistribute it and/or
 * modify it under the terms of the GNU Lesser General Public License
 * as published by the Free Software Foundation; either version 2.1
 * of the License, or (at your option) any later version.
 *
 * GROMACS is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
 * Lesser General Public License for more details.
 *
 * You should have received a copy of the GNU Lesser General Public
 * License along with GROMACS; if not, see
 * http://www.gnu.org/licenses, or write to the Free Software Foundation,
 * Inc., 51 Franklin Street, Fifth Floor, Boston, MA  02110-1301  USA.
 *
 * If you want to redistribute modifications to GROMACS, please
 * consider that scientific software is very special. Version
 * control is crucial - bugs must be traceable. We will be happy to
 * consider code for inclusion in the official distribution, but
 * derived work must not be called official GROMACS. Details are found
 * in the README & COPYING files - if they are missing, get the
 * official version at http://www.gromacs.org.
 *
 * To help us fund GROMACS development, we humbly ask that you cite
 * the research papers on the package. Check out http://www.gromacs.org.
 */
/*! \file
 *  \brief Define CUDA implementation of nbnxn_gpu_data_mgmt.h
 *
 *  \author Szilard Pall <pall.szilard@gmail.com>
 */
#include "gmxpre.h"

#include <assert.h>
#include <stdarg.h>
#include <stdio.h>
#include <stdlib.h>

#include "gromacs/gpu_utils/cudautils.cuh"
#include "gromacs/gpu_utils/gpu_utils.h"
#include "gromacs/gpu_utils/pmalloc_cuda.h"
#include "gromacs/hardware/gpu_hw_info.h"
#include "gromacs/math/vectypes.h"
#include "gromacs/mdlib/force_flags.h"
#include "gromacs/mdlib/nb_verlet.h"
#include "gromacs/mdlib/nbnxn_consts.h"
#include "gromacs/mdlib/nbnxn_gpu_data_mgmt.h"
#include "gromacs/mdtypes/interaction_const.h"
#include "gromacs/mdtypes/md_enums.h"
#include "gromacs/pbcutil/ishift.h"
#include "gromacs/timing/gpu_timing.h"
#include "gromacs/utility/basedefinitions.h"
#include "gromacs/utility/cstringutil.h"
#include "gromacs/utility/fatalerror.h"
#include "gromacs/utility/real.h"
#include "gromacs/utility/smalloc.h"

#include "nbnxn_cuda.h"
#include "nbnxn_cuda_types.h"

/* This is a heuristically determined parameter for the Fermi, Kepler
 * and Maxwell architectures for the minimum size of ci lists by multiplying
 * this constant with the # of multiprocessors on the current device.
 * Since the maximum number of blocks per multiprocessor is 16, the ideal
 * count for small systems is 32 or 48 blocks per multiprocessor. Because
 * there is a bit of fluctuations in the generated block counts, we use
 * a target of 44 instead of the ideal value of 48.
 */
static unsigned int gpu_min_ci_balanced_factor = 44;

/* Fw. decl. */
static void nbnxn_cuda_clear_e_fshift(gmx_nbnxn_cuda_t *nb);

/* Fw. decl, */
static void nbnxn_cuda_free_nbparam_table(cu_nbparam_t            *nbparam,
                                          const gmx_device_info_t *dev_info);

/*! \brief Return whether combination rules are used.
 *
 * \param[in]   pointer to nonbonded paramter struct
 * \return      true if combination rules are used in this run, false otherwise
 */
static inline bool useLjCombRule(const cu_nbparam_t  *nbparam)
{
    return (nbparam->vdwtype == evdwCuCUTCOMBGEOM ||
            nbparam->vdwtype == evdwCuCUTCOMBLB);
}

/*! \brief Initialized the Ewald Coulomb correction GPU table.

    Tabulates the Ewald Coulomb force and initializes the size/scale
    and the table GPU array. If called with an already allocated table,
    it just re-uploads the table.
 */
static void init_ewald_coulomb_force_table(const interaction_const_t *ic,
                                           cu_nbparam_t              *nbp,
                                           const gmx_device_info_t   *dev_info)
{
    if (nbp->coulomb_tab != NULL)
    {
        nbnxn_cuda_free_nbparam_table(nbp, dev_info);
    }

    nbp->coulomb_tab_scale = ic->tabq_scale;
    initParamLookupTable(nbp->coulomb_tab, nbp->coulomb_tab_texobj,
                         &nbnxn_cuda_get_coulomb_tab_texref(),
                         ic->tabq_coul_F, ic->tabq_size, dev_info);
}


/*! Initializes the atomdata structure first time, it only gets filled at
    pair-search. */
static void init_atomdata_first(cu_atomdata_t *ad, int ntypes)
{
    hipError_t stat;

    ad->ntypes  = ntypes;
    stat        = hipMalloc((void**)&ad->shift_vec, SHIFTS*sizeof(*ad->shift_vec));
    CU_RET_ERR(stat, "hipMalloc failed on ad->shift_vec");
    ad->bShiftVecUploaded = false;

    stat = hipMalloc((void**)&ad->fshift, SHIFTS*sizeof(*ad->fshift));
    CU_RET_ERR(stat, "hipMalloc failed on ad->fshift");

    stat = hipMalloc((void**)&ad->e_lj, sizeof(*ad->e_lj));
    CU_RET_ERR(stat, "hipMalloc failed on ad->e_lj");
    stat = hipMalloc((void**)&ad->e_el, sizeof(*ad->e_el));
    CU_RET_ERR(stat, "hipMalloc failed on ad->e_el");

    /* initialize to NULL poiters to data that is not allocated here and will
       need reallocation in nbnxn_cuda_init_atomdata */
    ad->xq = NULL;
    ad->f  = NULL;

    /* size -1 indicates that the respective array hasn't been initialized yet */
    ad->natoms = -1;
    ad->nalloc = -1;
}

/*! Selects the Ewald kernel type, analytical on SM 3.0 and later, tabulated on
    earlier GPUs, single or twin cut-off. */
static int pick_ewald_kernel_type(bool                     bTwinCut,
                                  const gmx_device_info_t *dev_info)
{
    bool bUseAnalyticalEwald, bForceAnalyticalEwald, bForceTabulatedEwald;
    int  kernel_type;

    /* Benchmarking/development environment variables to force the use of
       analytical or tabulated Ewald kernel. */
    bForceAnalyticalEwald = (getenv("GMX_CUDA_NB_ANA_EWALD") != NULL);
    bForceTabulatedEwald  = (getenv("GMX_CUDA_NB_TAB_EWALD") != NULL);

    if (bForceAnalyticalEwald && bForceTabulatedEwald)
    {
        gmx_incons("Both analytical and tabulated Ewald CUDA non-bonded kernels "
                   "requested through environment variables.");
    }

    /* By default, on SM 3.0 and later use analytical Ewald, on earlier tabulated. */
    if ((dev_info->prop.major >= 3 || bForceAnalyticalEwald) && !bForceTabulatedEwald)
    {
        bUseAnalyticalEwald = true;

        if (debug)
        {
            fprintf(debug, "Using analytical Ewald CUDA kernels\n");
        }
    }
    else
    {
        bUseAnalyticalEwald = false;

        if (debug)
        {
            fprintf(debug, "Using tabulated Ewald CUDA kernels\n");
        }
    }

    /* Use twin cut-off kernels if requested by bTwinCut or the env. var.
       forces it (use it for debugging/benchmarking only). */
    if (!bTwinCut && (getenv("GMX_CUDA_NB_EWALD_TWINCUT") == NULL))
    {
        kernel_type = bUseAnalyticalEwald ? eelCuEWALD_ANA : eelCuEWALD_TAB;
    }
    else
    {
        kernel_type = bUseAnalyticalEwald ? eelCuEWALD_ANA_TWIN : eelCuEWALD_TAB_TWIN;
    }

    return kernel_type;
}

/*! Copies all parameters related to the cut-off from ic to nbp */
static void set_cutoff_parameters(cu_nbparam_t              *nbp,
                                  const interaction_const_t *ic,
                                  const NbnxnListParameters *listParams)
{
    nbp->ewald_beta        = ic->ewaldcoeff_q;
    nbp->sh_ewald          = ic->sh_ewald;
    nbp->epsfac            = ic->epsfac;
    nbp->two_k_rf          = 2.0 * ic->k_rf;
    nbp->c_rf              = ic->c_rf;
    nbp->rvdw_sq           = ic->rvdw * ic->rvdw;
    nbp->rcoulomb_sq       = ic->rcoulomb * ic->rcoulomb;
    nbp->rlistOuter_sq     = listParams->rlistOuter * listParams->rlistOuter;
    nbp->rlistInner_sq     = listParams->rlistInner * listParams->rlistInner;
    nbp->useDynamicPruning = listParams->useDynamicPruning;

    nbp->sh_lj_ewald       = ic->sh_lj_ewald;
    nbp->ewaldcoeff_lj     = ic->ewaldcoeff_lj;

    nbp->rvdw_switch       = ic->rvdw_switch;
    nbp->dispersion_shift  = ic->dispersion_shift;
    nbp->repulsion_shift   = ic->repulsion_shift;
    nbp->vdw_switch        = ic->vdw_switch;
}

/*! Initializes the nonbonded parameter data structure. */
static void init_nbparam(cu_nbparam_t              *nbp,
                         const interaction_const_t *ic,
                         const NbnxnListParameters *listParams,
                         const nbnxn_atomdata_t    *nbat,
                         const gmx_device_info_t   *dev_info)
{
    int         ntypes;

    ntypes  = nbat->ntype;

    set_cutoff_parameters(nbp, ic, listParams);

    /* The kernel code supports LJ combination rules (geometric and LB) for
     * all kernel types, but we only generate useful combination rule kernels.
     * We currently only use LJ combination rule (geometric and LB) kernels
     * for plain cut-off LJ. On Maxwell the force only kernels speed up 15%
     * with PME and 20% with RF, the other kernels speed up about half as much.
     * For LJ force-switch the geometric rule would give 7% speed-up, but this
     * combination is rarely used. LJ force-switch with LB rule is more common,
     * but gives only 1% speed-up.
     */
    if (ic->vdwtype == evdwCUT)
    {
        switch (ic->vdw_modifier)
        {
            case eintmodNONE:
            case eintmodPOTSHIFT:
                switch (nbat->comb_rule)
                {
                    case ljcrNONE:
                        nbp->vdwtype = evdwCuCUT;
                        break;
                    case ljcrGEOM:
                        nbp->vdwtype = evdwCuCUTCOMBGEOM;
                        break;
                    case ljcrLB:
                        nbp->vdwtype = evdwCuCUTCOMBLB;
                        break;
                    default:
                        gmx_incons("The requested LJ combination rule is not implemented in the CUDA GPU accelerated kernels!");
                        break;
                }
                break;
            case eintmodFORCESWITCH:
                nbp->vdwtype = evdwCuFSWITCH;
                break;
            case eintmodPOTSWITCH:
                nbp->vdwtype = evdwCuPSWITCH;
                break;
            default:
                gmx_incons("The requested VdW interaction modifier is not implemented in the CUDA GPU accelerated kernels!");
                break;
        }
    }
    else if (ic->vdwtype == evdwPME)
    {
        if (ic->ljpme_comb_rule == ljcrGEOM)
        {
            assert(nbat->comb_rule == ljcrGEOM);
            nbp->vdwtype = evdwCuEWALDGEOM;
        }
        else
        {
            assert(nbat->comb_rule == ljcrLB);
            nbp->vdwtype = evdwCuEWALDLB;
        }
    }
    else
    {
        gmx_incons("The requested VdW type is not implemented in the CUDA GPU accelerated kernels!");
    }

    if (ic->eeltype == eelCUT)
    {
        nbp->eeltype = eelCuCUT;
    }
    else if (EEL_RF(ic->eeltype))
    {
        nbp->eeltype = eelCuRF;
    }
    else if ((EEL_PME(ic->eeltype) || ic->eeltype == eelEWALD))
    {
        /* Initially rcoulomb == rvdw, so it's surely not twin cut-off. */
        nbp->eeltype = pick_ewald_kernel_type(false, dev_info);
    }
    else
    {
        /* Shouldn't happen, as this is checked when choosing Verlet-scheme */
        gmx_incons("The requested electrostatics type is not implemented in the CUDA GPU accelerated kernels!");
    }

    /* generate table for PME */
    nbp->coulomb_tab = NULL;
    if (nbp->eeltype == eelCuEWALD_TAB || nbp->eeltype == eelCuEWALD_TAB_TWIN)
    {
        init_ewald_coulomb_force_table(ic, nbp, dev_info);
    }

    /* set up LJ parameter lookup table */
    if (!useLjCombRule(nbp))
    {
        initParamLookupTable(nbp->nbfp, nbp->nbfp_texobj,
                             &nbnxn_cuda_get_nbfp_texref(),
                             nbat->nbfp, 2*ntypes*ntypes, dev_info);
    }

    /* set up LJ-PME parameter lookup table */
    if (ic->vdwtype == evdwPME)
    {
        initParamLookupTable(nbp->nbfp_comb, nbp->nbfp_comb_texobj,
                             &nbnxn_cuda_get_nbfp_comb_texref(),
                             nbat->nbfp_comb, 2*ntypes, dev_info);
    }
}

/*! Re-generate the GPU Ewald force table, resets rlist, and update the
 *  electrostatic type switching to twin cut-off (or back) if needed. */
void nbnxn_gpu_pme_loadbal_update_param(const nonbonded_verlet_t    *nbv,
                                        const interaction_const_t   *ic,
                                        const NbnxnListParameters   *listParams)
{
    if (!nbv || nbv->grp[0].kernel_type != nbnxnk8x8x8_GPU)
    {
        return;
    }
    gmx_nbnxn_cuda_t *nb    = nbv->gpu_nbv;
    cu_nbparam_t     *nbp   = nb->nbparam;

    set_cutoff_parameters(nbp, ic, listParams);

    nbp->eeltype        = pick_ewald_kernel_type(ic->rcoulomb != ic->rvdw,
                                                 nb->dev_info);

    init_ewald_coulomb_force_table(ic, nb->nbparam, nb->dev_info);
}

/*! Initializes the pair list data structure. */
static void init_plist(cu_plist_t *pl)
{
    /* initialize to NULL pointers to data that is not allocated here and will
       need reallocation in nbnxn_gpu_init_pairlist */
    pl->sci      = NULL;
    pl->cj4      = NULL;
    pl->imask    = NULL;
    pl->excl     = NULL;

    /* size -1 indicates that the respective array hasn't been initialized yet */
    pl->na_c           = -1;
    pl->nsci           = -1;
    pl->sci_nalloc     = -1;
    pl->ncj4           = -1;
    pl->cj4_nalloc     = -1;
    pl->nimask         = -1;
    pl->imask_nalloc   = -1;
    pl->nexcl          = -1;
    pl->excl_nalloc    = -1;
    pl->haveFreshList  = false;
}

/*! Initializes the timer data structure. */
static void init_timers(cu_timers_t *t, bool bUseTwoStreams)
{
    /* The non-local counters/stream (second in the array) are needed only with DD. */
    for (int i = 0; i <= (bUseTwoStreams ? 1 : 0); i++)
    {
        t->didPairlistH2D[i]  = false;
        t->didPrune[i]        = false;
        t->didRollingPrune[i] = false;
    }
}

/*! Initializes the timings data structure. */
static void init_timings(gmx_wallclock_gpu_nbnxn_t *t)
{
    int i, j;

    t->nb_h2d_t = 0.0;
    t->nb_d2h_t = 0.0;
    t->nb_c     = 0;
    t->pl_h2d_t = 0.0;
    t->pl_h2d_c = 0;
    for (i = 0; i < 2; i++)
    {
        for (j = 0; j < 2; j++)
        {
            t->ktime[i][j].t = 0.0;
            t->ktime[i][j].c = 0;
        }
    }
    t->pruneTime.c        = 0;
    t->pruneTime.t        = 0.0;
    t->dynamicPruneTime.c = 0;
    t->dynamicPruneTime.t = 0.0;
}

/*! Initializes simulation constant data. */
static void nbnxn_cuda_init_const(gmx_nbnxn_cuda_t               *nb,
                                  const interaction_const_t      *ic,
                                  const NbnxnListParameters      *listParams,
                                  const nbnxn_atomdata_t         *nbat)
{
    init_atomdata_first(nb->atdat, nbat->ntype);
    init_nbparam(nb->nbparam, ic, listParams, nbat, nb->dev_info);

    /* clear energy and shift force outputs */
    nbnxn_cuda_clear_e_fshift(nb);
}

void nbnxn_gpu_init(gmx_nbnxn_cuda_t         **p_nb,
                    const gmx_device_info_t   *deviceInfo,
                    const interaction_const_t *ic,
                    const NbnxnListParameters *listParams,
                    const nbnxn_atomdata_t    *nbat,
                    int                        /*rank*/,
                    gmx_bool                   bLocalAndNonlocal)
{
    hipError_t       stat;
    gmx_nbnxn_cuda_t *nb;

    if (p_nb == NULL)
    {
        return;
    }

    snew(nb, 1);
    snew(nb->atdat, 1);
    snew(nb->nbparam, 1);
    snew(nb->plist[eintLocal], 1);
    if (bLocalAndNonlocal)
    {
        snew(nb->plist[eintNonlocal], 1);
    }

    nb->bUseTwoStreams = bLocalAndNonlocal;

    nb->timers = new cu_timers_t();
    snew(nb->timings, 1);

    /* init nbst */
    pmalloc((void**)&nb->nbst.e_lj, sizeof(*nb->nbst.e_lj));
    pmalloc((void**)&nb->nbst.e_el, sizeof(*nb->nbst.e_el));
    pmalloc((void**)&nb->nbst.fshift, SHIFTS * sizeof(*nb->nbst.fshift));

    init_plist(nb->plist[eintLocal]);

    /* set device info, just point it to the right GPU among the detected ones */
    nb->dev_info = deviceInfo;

    /* local/non-local GPU streams */
    stat = hipStreamCreate(&nb->stream[eintLocal]);
    CU_RET_ERR(stat, "hipStreamCreate on stream[eintLocal] failed");
    if (nb->bUseTwoStreams)
    {
        init_plist(nb->plist[eintNonlocal]);

        /* Note that the device we're running on does not have to support
         * priorities, because we are querying the priority range which in this
         * case will be a single value.
         */
        int highest_priority;
        stat = hipDeviceGetStreamPriorityRange(NULL, &highest_priority);
        CU_RET_ERR(stat, "hipDeviceGetStreamPriorityRange failed");

        stat = hipStreamCreateWithPriority(&nb->stream[eintNonlocal],
                                            hipStreamDefault,
                                            highest_priority);
        CU_RET_ERR(stat, "hipStreamCreateWithPriority on stream[eintNonlocal] failed");
    }

    /* init events for sychronization (timing disabled for performance reasons!) */
    stat = hipEventCreateWithFlags(&nb->nonlocal_done, hipEventDisableTiming);
    CU_RET_ERR(stat, "hipEventCreate on nonlocal_done failed");
    stat = hipEventCreateWithFlags(&nb->misc_ops_and_local_H2D_done, hipEventDisableTiming);
    CU_RET_ERR(stat, "hipEventCreate on misc_ops_and_local_H2D_done failed");

    /* WARNING: CUDA timings are incorrect with multiple streams.
     *          This is the main reason why they are disabled by default.
     */
    // TODO: Consider turning on by default when we can detect nr of streams.
    nb->bDoTime = (getenv("GMX_ENABLE_GPU_TIMING") != NULL);

    if (nb->bDoTime)
    {
        init_timers(nb->timers, nb->bUseTwoStreams);
        init_timings(nb->timings);
    }

    /* set the kernel type for the current GPU */
    /* pick L1 cache configuration */
    nbnxn_cuda_set_cacheconfig(nb->dev_info);

    nbnxn_cuda_init_const(nb, ic, listParams, nbat);

    *p_nb = nb;

    if (debug)
    {
        fprintf(debug, "Initialized CUDA data structures.\n");
    }
}

void nbnxn_gpu_init_pairlist(gmx_nbnxn_cuda_t       *nb,
                             const nbnxn_pairlist_t *h_plist,
                             int                     iloc)
{
    char          sbuf[STRLEN];
    bool          bDoTime    =  (nb->bDoTime && h_plist->nsci > 0);
    hipStream_t  stream     = nb->stream[iloc];
    cu_plist_t   *d_plist    = nb->plist[iloc];

    if (d_plist->na_c < 0)
    {
        d_plist->na_c = h_plist->na_ci;
    }
    else
    {
        if (d_plist->na_c != h_plist->na_ci)
        {
            sprintf(sbuf, "In cu_init_plist: the #atoms per cell has changed (from %d to %d)",
                    d_plist->na_c, h_plist->na_ci);
            gmx_incons(sbuf);
        }
    }

    if (bDoTime)
    {
        nb->timers->pl_h2d[iloc].openTimingRegion(stream);
        nb->timers->didPairlistH2D[iloc] = true;
    }

    cu_realloc_buffered((void **)&d_plist->sci, h_plist->sci, sizeof(*d_plist->sci),
                        &d_plist->nsci, &d_plist->sci_nalloc,
                        h_plist->nsci,
                        stream, true);

    cu_realloc_buffered((void **)&d_plist->cj4, h_plist->cj4, sizeof(*d_plist->cj4),
                        &d_plist->ncj4, &d_plist->cj4_nalloc,
                        h_plist->ncj4,
                        stream, true);

    /* this call only allocates space on the device (no data is transferred) */
    cu_realloc_buffered((void **)&d_plist->imask, NULL, sizeof(*d_plist->imask),
                        &d_plist->nimask, &d_plist->imask_nalloc,
                        h_plist->ncj4*c_nbnxnGpuClusterpairSplit,
                        stream, true);

    cu_realloc_buffered((void **)&d_plist->excl, h_plist->excl, sizeof(*d_plist->excl),
                        &d_plist->nexcl, &d_plist->excl_nalloc,
                        h_plist->nexcl,
                        stream, true);

    if (bDoTime)
    {
        nb->timers->pl_h2d[iloc].closeTimingRegion(stream);
    }

    /* the next use of thist list we be the first one, so we need to prune */
    d_plist->haveFreshList = true;
}

void nbnxn_gpu_upload_shiftvec(gmx_nbnxn_cuda_t       *nb,
                               const nbnxn_atomdata_t *nbatom)
{
    cu_atomdata_t *adat  = nb->atdat;
    hipStream_t   ls    = nb->stream[eintLocal];

    /* only if we have a dynamic box */
    if (nbatom->bDynamicBox || !adat->bShiftVecUploaded)
    {
        cu_copy_H2D_async(adat->shift_vec, nbatom->shift_vec,
                          SHIFTS * sizeof(*adat->shift_vec), ls);
        adat->bShiftVecUploaded = true;
    }
}

/*! Clears the first natoms_clear elements of the GPU nonbonded force output array. */
static void nbnxn_cuda_clear_f(gmx_nbnxn_cuda_t *nb, int natoms_clear)
{
    hipError_t    stat;
    cu_atomdata_t *adat  = nb->atdat;
    hipStream_t   ls    = nb->stream[eintLocal];

    stat = hipMemsetAsync(adat->f, 0, natoms_clear * sizeof(*adat->f), ls);
    CU_RET_ERR(stat, "hipMemsetAsync on f falied");
}

/*! Clears nonbonded shift force output array and energy outputs on the GPU. */
static void nbnxn_cuda_clear_e_fshift(gmx_nbnxn_cuda_t *nb)
{
    hipError_t    stat;
    cu_atomdata_t *adat  = nb->atdat;
    hipStream_t   ls    = nb->stream[eintLocal];

    stat = hipMemsetAsync(adat->fshift, 0, SHIFTS * sizeof(*adat->fshift), ls);
    CU_RET_ERR(stat, "hipMemsetAsync on fshift falied");
    stat = hipMemsetAsync(adat->e_lj, 0, sizeof(*adat->e_lj), ls);
    CU_RET_ERR(stat, "hipMemsetAsync on e_lj falied");
    stat = hipMemsetAsync(adat->e_el, 0, sizeof(*adat->e_el), ls);
    CU_RET_ERR(stat, "hipMemsetAsync on e_el falied");
}

void nbnxn_gpu_clear_outputs(gmx_nbnxn_cuda_t *nb, int flags)
{
    nbnxn_cuda_clear_f(nb, nb->atdat->natoms);
    /* clear shift force array and energies if the outputs were
       used in the current step */
    if (flags & GMX_FORCE_VIRIAL)
    {
        nbnxn_cuda_clear_e_fshift(nb);
    }
}

void nbnxn_gpu_init_atomdata(gmx_nbnxn_cuda_t              *nb,
                             const nbnxn_atomdata_t        *nbat)
{
    hipError_t    stat;
    int            nalloc, natoms;
    bool           realloced;
    bool           bDoTime   = nb->bDoTime;
    cu_timers_t   *timers    = nb->timers;
    cu_atomdata_t *d_atdat   = nb->atdat;
    hipStream_t   ls        = nb->stream[eintLocal];

    natoms    = nbat->natoms;
    realloced = false;

    if (bDoTime)
    {
        /* time async copy */
        timers->atdat.openTimingRegion(ls);
    }

    /* need to reallocate if we have to copy more atoms than the amount of space
       available and only allocate if we haven't initialized yet, i.e d_atdat->natoms == -1 */
    if (natoms > d_atdat->nalloc)
    {
        nalloc = over_alloc_small(natoms);

        /* free up first if the arrays have already been initialized */
        if (d_atdat->nalloc != -1)
        {
            cu_free_buffered(d_atdat->f, &d_atdat->natoms, &d_atdat->nalloc);
            cu_free_buffered(d_atdat->xq);
            cu_free_buffered(d_atdat->atom_types);
            cu_free_buffered(d_atdat->lj_comb);
        }

        stat = hipMalloc((void **)&d_atdat->f, nalloc*sizeof(*d_atdat->f));
        CU_RET_ERR(stat, "hipMalloc failed on d_atdat->f");
        stat = hipMalloc((void **)&d_atdat->xq, nalloc*sizeof(*d_atdat->xq));
        CU_RET_ERR(stat, "hipMalloc failed on d_atdat->xq");
        if (useLjCombRule(nb->nbparam))
        {
            stat = hipMalloc((void **)&d_atdat->lj_comb, nalloc*sizeof(*d_atdat->lj_comb));
            CU_RET_ERR(stat, "hipMalloc failed on d_atdat->lj_comb");
        }
        else
        {
            stat = hipMalloc((void **)&d_atdat->atom_types, nalloc*sizeof(*d_atdat->atom_types));
            CU_RET_ERR(stat, "hipMalloc failed on d_atdat->atom_types");
        }

        d_atdat->nalloc = nalloc;
        realloced       = true;
    }

    d_atdat->natoms       = natoms;
    d_atdat->natoms_local = nbat->natoms_local;

    /* need to clear GPU f output if realloc happened */
    if (realloced)
    {
        nbnxn_cuda_clear_f(nb, nalloc);
    }

    if (useLjCombRule(nb->nbparam))
    {
        cu_copy_H2D_async(d_atdat->lj_comb, nbat->lj_comb,
                          natoms*sizeof(*d_atdat->lj_comb), ls);
    }
    else
    {
        cu_copy_H2D_async(d_atdat->atom_types, nbat->type,
                          natoms*sizeof(*d_atdat->atom_types), ls);
    }

    if (bDoTime)
    {
        timers->atdat.closeTimingRegion(ls);
    }
}

static void nbnxn_cuda_free_nbparam_table(cu_nbparam_t            *nbparam,
                                          const gmx_device_info_t *dev_info)
{
    if (nbparam->eeltype == eelCuEWALD_TAB || nbparam->eeltype == eelCuEWALD_TAB_TWIN)
    {
        destroyParamLookupTable(nbparam->coulomb_tab, nbparam->coulomb_tab_texobj,
                                &nbnxn_cuda_get_coulomb_tab_texref(), dev_info);
    }
}

void nbnxn_gpu_free(gmx_nbnxn_cuda_t *nb)
{
    hipError_t      stat;
    cu_atomdata_t   *atdat;
    cu_nbparam_t    *nbparam;
    cu_plist_t      *plist, *plist_nl;

    if (nb == NULL)
    {
        return;
    }

    atdat       = nb->atdat;
    nbparam     = nb->nbparam;
    plist       = nb->plist[eintLocal];
    plist_nl    = nb->plist[eintNonlocal];

    nbnxn_cuda_free_nbparam_table(nbparam, nb->dev_info);

    stat = hipEventDestroy(nb->nonlocal_done);
    CU_RET_ERR(stat, "hipEventDestroy failed on timers->nonlocal_done");
    stat = hipEventDestroy(nb->misc_ops_and_local_H2D_done);
    CU_RET_ERR(stat, "hipEventDestroy failed on timers->misc_ops_and_local_H2D_done");

    delete nb->timers;
    if (nb->bDoTime)
    {
        /* The non-local counters/stream (second in the array) are needed only with DD. */
        for (int i = 0; i <= (nb->bUseTwoStreams ? 1 : 0); i++)
        {
            stat = hipStreamDestroy(nb->stream[i]);
            CU_RET_ERR(stat, "hipStreamDestroy failed on stream");
        }
    }

    if (!useLjCombRule(nb->nbparam))
    {
        destroyParamLookupTable(nbparam->nbfp, nbparam->nbfp_texobj,
                                &nbnxn_cuda_get_nbfp_texref(), nb->dev_info);

    }

    if (nbparam->vdwtype == evdwCuEWALDGEOM || nbparam->vdwtype == evdwCuEWALDLB)
    {
        destroyParamLookupTable(nbparam->nbfp_comb, nbparam->nbfp_comb_texobj,
                                &nbnxn_cuda_get_nbfp_comb_texref(), nb->dev_info);
    }

    stat = hipFree(atdat->shift_vec);
    CU_RET_ERR(stat, "hipFree failed on atdat->shift_vec");
    stat = hipFree(atdat->fshift);
    CU_RET_ERR(stat, "hipFree failed on atdat->fshift");

    stat = hipFree(atdat->e_lj);
    CU_RET_ERR(stat, "hipFree failed on atdat->e_lj");
    stat = hipFree(atdat->e_el);
    CU_RET_ERR(stat, "hipFree failed on atdat->e_el");

    cu_free_buffered(atdat->f, &atdat->natoms, &atdat->nalloc);
    cu_free_buffered(atdat->xq);
    cu_free_buffered(atdat->atom_types, &atdat->ntypes);
    cu_free_buffered(atdat->lj_comb);

    cu_free_buffered(plist->sci, &plist->nsci, &plist->sci_nalloc);
    cu_free_buffered(plist->cj4, &plist->ncj4, &plist->cj4_nalloc);
    cu_free_buffered(plist->imask, &plist->nimask, &plist->imask_nalloc);
    cu_free_buffered(plist->excl, &plist->nexcl, &plist->excl_nalloc);
    if (nb->bUseTwoStreams)
    {
        cu_free_buffered(plist_nl->sci, &plist_nl->nsci, &plist_nl->sci_nalloc);
        cu_free_buffered(plist_nl->cj4, &plist_nl->ncj4, &plist_nl->cj4_nalloc);
        cu_free_buffered(plist_nl->imask, &plist_nl->nimask, &plist_nl->imask_nalloc);
        cu_free_buffered(plist_nl->excl, &plist_nl->nexcl, &plist->excl_nalloc);
    }

    sfree(atdat);
    sfree(nbparam);
    sfree(plist);
    if (nb->bUseTwoStreams)
    {
        sfree(plist_nl);
    }
    sfree(nb->timings);
    sfree(nb);

    if (debug)
    {
        fprintf(debug, "Cleaned up CUDA data structures.\n");
    }
}

//! This function is documented in the header file
gmx_wallclock_gpu_nbnxn_t *nbnxn_gpu_get_timings(gmx_nbnxn_cuda_t *nb)
{
    return (nb != nullptr && nb->bDoTime) ? nb->timings : nullptr;
}

void nbnxn_gpu_reset_timings(nonbonded_verlet_t* nbv)
{
    if (nbv->gpu_nbv && nbv->gpu_nbv->bDoTime)
    {
        init_timings(nbv->gpu_nbv->timings);
    }
}

int nbnxn_gpu_min_ci_balanced(gmx_nbnxn_cuda_t *nb)
{
    return nb != NULL ?
           gpu_min_ci_balanced_factor*nb->dev_info->prop.multiProcessorCount : 0;

}

gmx_bool nbnxn_gpu_is_kernel_ewald_analytical(const gmx_nbnxn_cuda_t *nb)
{
    return ((nb->nbparam->eeltype == eelCuEWALD_ANA) ||
            (nb->nbparam->eeltype == eelCuEWALD_ANA_TWIN));
}
