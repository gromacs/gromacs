#include "hip/hip_runtime.h"
/* -*- mode: c; tab-width: 4; indent-tabs-mode: nil; c-basic-offset: 4; c-file-style: "stroustrup"; -*-
 *
 *
 *                This source code is part of
 *
 *                 G   R   O   M   A   C   S
 *
 *          GROningen MAchine for Chemical Simulations
 *
 * Written by David van der Spoel, Erik Lindahl, Berk Hess, and others.
 * Copyright (c) 1991-2000, University of Groningen, The Netherlands.
 * Copyright (c) 2001-2012, The GROMACS development team,
 * check out http://www.gromacs.org for more information.
 *
 * This program is free software; you can redistribute it and/or
 * modify it under the terms of the GNU General Public License
 * as published by the Free Software Foundation; either version 2
 * of the License, or (at your option) any later version.
 *
 * If you want to redistribute modifications, please consider that
 * scientific software is very special. Version control is crucial -
 * bugs must be traceable. We will be happy to consider code for
 * inclusion in the official distribution, but derived work must not
 * be called official GROMACS. Details are found in the README & COPYING
 * files - if they are missing, get the official version at www.gromacs.org.
 *
 * To help us fund GROMACS development, we humbly ask that you cite
 * the papers on the package - you can find them in the top README file.
 *
 * For more info, check our website at http://www.gromacs.org
 *
 * And Hey:
 * Gallium Rubidium Oxygen Manganese Argon Carbon Silicon
 */
#ifdef HAVE_CONFIG_H
#include <config.h>
#endif

#include <stdlib.h>
#include <stdio.h>
#include <assert.h>

#include "gmx_fatal.h"
#include "smalloc.h"
#include "tables.h"
#include "typedefs.h"
#include "types/nb_verlet.h"
#include "types/interaction_const.h"
#include "types/force_flags.h"
#include "../nbnxn_consts.h"

#include "nbnxn_cuda_types.h"
#include "../../gmxlib/cuda_tools/cudautils.cuh"
#include "nbnxn_cuda_data_mgmt.h"
#include "pmalloc_cuda.h"
#include "gpu_utils.h"

static bool bUseCudaEventBlockingSync = false; /* makes the CPU thread block */

/* This is a heuristically determined parameter for the Fermi architecture for
 * the minimum size of ci lists by multiplying this constant with the # of
 * multiprocessors on the current device.
 */
static unsigned int gpu_min_ci_balanced_factor = 40;

/* Functions from nbnxn_cuda.cu */
extern void nbnxn_cuda_set_cacheconfig(cuda_dev_info_t *devinfo);
extern const struct texture<float, 1, hipReadModeElementType>& nbnxn_cuda_get_nbfp_texref();
extern const struct texture<float, 1, hipReadModeElementType>& nbnxn_cuda_get_coulomb_tab_texref();

/* Fw. decl. */
static void nbnxn_cuda_clear_e_fshift(nbnxn_cuda_ptr_t cu_nb);


/*! Tabulates the Ewald Coulomb force and initializes the size/scale
    and the table GPU array. If called with an already allocated table,
    it just re-uploads the table.
 */
static void init_ewald_coulomb_force_table(cu_nbparam_t *nbp)
{
    float       *ftmp, *coul_tab;
    int         tabsize;
    double      tabscale;
    hipError_t stat;

    tabsize     = GPU_EWALD_COULOMB_FORCE_TABLE_SIZE;
    /* Subtract 2 iso 1 to avoid access out of range due to rounding */
    tabscale    = (tabsize - 2) / sqrt(nbp->rcoulomb_sq);

    pmalloc((void**)&ftmp, tabsize*sizeof(*ftmp));

    table_spline3_fill_ewald_lr(ftmp, NULL, tabsize, tableformatF,
                                1/tabscale, nbp->ewald_beta);

    /* If the table pointer == NULL the table is generated the first time =>
       the array pointer will be saved to nbparam and the texture is bound.
     */
    coul_tab = nbp->coulomb_tab;
    if (coul_tab == NULL)
    {
        stat = hipMalloc((void **)&coul_tab, tabsize*sizeof(*coul_tab));
        CU_RET_ERR(stat, "hipMalloc failed on coul_tab");

        nbp->coulomb_tab = coul_tab;

        hipChannelFormatDesc cd   = hipCreateChannelDesc<float>();
        stat = hipBindTexture(NULL, &nbnxn_cuda_get_coulomb_tab_texref(),
                               coul_tab, &cd, tabsize*sizeof(*coul_tab));
        CU_RET_ERR(stat, "hipBindTexture on coul_tab failed");
    }

    cu_copy_H2D(coul_tab, ftmp, tabsize*sizeof(*coul_tab));

    nbp->coulomb_tab_size     = tabsize;
    nbp->coulomb_tab_scale    = tabscale;

    pfree(ftmp);
}


/*! Initializes the atomdata structure first time, it only gets filled at
    pair-search. */
static void init_atomdata_first(cu_atomdata_t *ad, int ntypes)
{
    hipError_t stat;

    ad->ntypes  = ntypes;
    stat = hipMalloc((void**)&ad->shift_vec, SHIFTS*sizeof(*ad->shift_vec));
    CU_RET_ERR(stat, "hipMalloc failed on ad->shift_vec");
    ad->bShiftVecUploaded = false;

    stat = hipMalloc((void**)&ad->fshift, SHIFTS*sizeof(*ad->fshift));
    CU_RET_ERR(stat, "hipMalloc failed on ad->fshift");

    stat = hipMalloc((void**)&ad->e_lj, sizeof(*ad->e_lj));
    CU_RET_ERR(stat, "hipMalloc failed on ad->e_lj");
    stat = hipMalloc((void**)&ad->e_el, sizeof(*ad->e_el));
    CU_RET_ERR(stat, "hipMalloc failed on ad->e_el");

    /* initialize to NULL poiters to data that is not allocated here and will
       need reallocation in nbnxn_cuda_init_atomdata */
    ad->xq = NULL;
    ad->f  = NULL;

    /* size -1 indicates that the respective array hasn't been initialized yet */
    ad->natoms = -1;
    ad->nalloc = -1;
}

/*! Initializes the nonbonded parameter data structure. */
static void init_nbparam(cu_nbparam_t *nbp,
                         const interaction_const_t *ic,
                         const nonbonded_verlet_t *nbv)
{
    hipError_t stat;
    int         ntypes, nnbfp;

    ntypes  = nbv->grp[0].nbat->ntype;

    nbp->ewald_beta = ic->ewaldcoeff;
    nbp->sh_ewald   = ic->sh_ewald;
    nbp->epsfac     = ic->epsfac;
    nbp->two_k_rf   = 2.0 * ic->k_rf;
    nbp->c_rf       = ic->c_rf;
    nbp->rvdw_sq    = ic->rvdw * ic->rvdw;
    nbp->rcoulomb_sq= ic->rcoulomb * ic->rcoulomb;
    nbp->rlist_sq   = ic->rlist * ic->rlist;
    nbp->sh_invrc6  = ic->sh_invrc6;

    if (ic->eeltype == eelCUT)
    {
        nbp->eeltype = eelCuCUT;
    }
    else if (EEL_RF(ic->eeltype))
    {
        nbp->eeltype = eelCuRF;
    }
    else if ((EEL_PME(ic->eeltype) || ic->eeltype==eelEWALD))
    {
        /* Initially rcoulomb == rvdw, so it's surely not twin cut-off, unless
           forced by the env. var. (used only for benchmarking). */
        if (getenv("GMX_CUDA_NB_EWALD_TWINCUT") == NULL)
        {
            nbp->eeltype = eelCuEWALD;
        }
        else
        {
            nbp->eeltype = eelCuEWALD_TWIN;
        }
    }
    else
    {
        /* Shouldn't happen, as this is checked when choosing Verlet-scheme */
        gmx_incons("The requested electrostatics type is not implemented in the CUDA GPU accelerated kernels!");
    }

    /* generate table for PME */
    if (nbp->eeltype == eelCuEWALD)
    {
        nbp->coulomb_tab = NULL;
        init_ewald_coulomb_force_table(nbp);
    }

    nnbfp = 2*ntypes*ntypes;
    stat = hipMalloc((void **)&nbp->nbfp, nnbfp*sizeof(*nbp->nbfp));
    CU_RET_ERR(stat, "hipMalloc failed on nbp->nbfp");
    cu_copy_H2D(nbp->nbfp, nbv->grp[0].nbat->nbfp, nnbfp*sizeof(*nbp->nbfp));

    hipChannelFormatDesc cd   = hipCreateChannelDesc<float>();
    stat = hipBindTexture(NULL, &nbnxn_cuda_get_nbfp_texref(),
                           nbp->nbfp, &cd, nnbfp*sizeof(*nbp->nbfp));
    CU_RET_ERR(stat, "hipBindTexture on nbfp failed");
}

/*! Re-generate the GPU Ewald force table, resets rlist, and update the
 *  electrostatic type switching to twin cut-off (or back) if needed. */
void nbnxn_cuda_pme_loadbal_update_param(nbnxn_cuda_ptr_t cu_nb,
                                         const interaction_const_t *ic)
{
    cu_nbparam_t *nbp = cu_nb->nbparam;

    nbp->rlist_sq       = ic->rlist * ic->rlist;
    nbp->rcoulomb_sq    = ic->rcoulomb * ic->rcoulomb;
    nbp->ewald_beta     = ic->ewaldcoeff;

    /* When switching to/from twin cut-off, the electrostatics type needs updating.
       (The env. var. that forces twin cut-off is for benchmarking only!) */
    if (ic->rcoulomb == ic->rvdw &&
        getenv("GMX_CUDA_NB_EWALD_TWINCUT") == NULL)
    {
        nbp->eeltype = eelCuEWALD;
    }
    else
    {
        nbp->eeltype = eelCuEWALD_TWIN;
    }

    init_ewald_coulomb_force_table(cu_nb->nbparam);
}

/*! Initializes the pair list data structure. */
static void init_plist(cu_plist_t *pl)
{
    /* initialize to NULL pointers to data that is not allocated here and will
       need reallocation in nbnxn_cuda_init_pairlist */
    pl->sci     = NULL;
    pl->cj4     = NULL;
    pl->excl    = NULL;

    /* size -1 indicates that the respective array hasn't been initialized yet */
    pl->na_c        = -1;
    pl->nsci        = -1;
    pl->sci_nalloc  = -1;
    pl->ncj4        = -1;
    pl->cj4_nalloc  = -1;
    pl->nexcl       = -1;
    pl->excl_nalloc = -1;
    pl->bDoPrune    = false;
}

/*! Initializes the timer data structure. */
static void init_timers(cu_timers_t *t, bool bUseTwoStreams)
{
    hipError_t stat;
    int eventflags = ( bUseCudaEventBlockingSync ? hipEventBlockingSync: hipEventDefault );

    stat = hipEventCreateWithFlags(&(t->start_atdat), eventflags);
    CU_RET_ERR(stat, "hipEventCreate on start_atdat failed");
    stat = hipEventCreateWithFlags(&(t->stop_atdat), eventflags);
    CU_RET_ERR(stat, "hipEventCreate on stop_atdat failed");

    /* The non-local counters/stream (second in the array) are needed only with DD. */
    for (int i = 0; i <= (bUseTwoStreams ? 1 : 0); i++)
    {
        stat = hipEventCreateWithFlags(&(t->start_nb_k[i]), eventflags);
        CU_RET_ERR(stat, "hipEventCreate on start_nb_k failed");
        stat = hipEventCreateWithFlags(&(t->stop_nb_k[i]), eventflags);
        CU_RET_ERR(stat, "hipEventCreate on stop_nb_k failed");


        stat = hipEventCreateWithFlags(&(t->start_pl_h2d[i]), eventflags);
        CU_RET_ERR(stat, "hipEventCreate on start_pl_h2d failed");
        stat = hipEventCreateWithFlags(&(t->stop_pl_h2d[i]), eventflags);
        CU_RET_ERR(stat, "hipEventCreate on stop_pl_h2d failed");

        stat = hipEventCreateWithFlags(&(t->start_nb_h2d[i]), eventflags);
        CU_RET_ERR(stat, "hipEventCreate on start_nb_h2d failed");
        stat = hipEventCreateWithFlags(&(t->stop_nb_h2d[i]), eventflags);
        CU_RET_ERR(stat, "hipEventCreate on stop_nb_h2d failed");

        stat = hipEventCreateWithFlags(&(t->start_nb_d2h[i]), eventflags);
        CU_RET_ERR(stat, "hipEventCreate on start_nb_d2h failed");
        stat = hipEventCreateWithFlags(&(t->stop_nb_d2h[i]), eventflags);
        CU_RET_ERR(stat, "hipEventCreate on stop_nb_d2h failed");
    }
}

/*! Initializes the timings data structure. */
static void init_timings(wallclock_gpu_t *t)
{
    int i, j;

    t->nb_h2d_t = 0.0;
    t->nb_d2h_t = 0.0;
    t->nb_c    = 0;
    t->pl_h2d_t = 0.0;
    t->pl_h2d_c = 0;
    for (i = 0; i < 2; i++)
    {
        for(j = 0; j < 2; j++)
        {
            t->ktime[i][j].t = 0.0;
            t->ktime[i][j].c = 0;
        }
    }
}

/* Decide which kernel version to use (default or legacy) based on:
 *  - CUDA version
 *  - non-bonded kernel selector environment variables
 *  - GPU SM version TODO ???
 */
static int pick_nbnxn_kernel_version()
{
    bool bLegacyKernel, bDefaultKernel, bCUDA40, bCUDA32;
    char sbuf[STRLEN];
    int  kver;

    /* legacy kernel (former k2), kept for now for backward compatibility,
       faster than the default with  CUDA 3.2/4.0 (TODO: on Kepler?). */
    bLegacyKernel  = (getenv("GMX_CUDA_NB_LEGACY") != NULL);
    /* default kernel (former k3). */
    bDefaultKernel = (getenv("GMX_CUDA_NB_DEFAULT") != NULL);

    if ((unsigned)(bLegacyKernel + bDefaultKernel) > 1)
    {
        gmx_fatal(FARGS, "Multiple CUDA non-bonded kernels requested; to manually pick a kernel set only one \n"
                  "of the following environment variables: \n"
                  "GMX_CUDA_NB_DEFAULT, GMX_CUDA_NB_LEGACY");
    }

    bCUDA32 = bCUDA40 = false;
#if CUDA_VERSION == 3200
    bCUDA32 = true;
    sprintf(sbuf, "3.2");
#elif CUDA_VERSION == 4000
    bCUDA40 = true;
    sprintf(sbuf, "4.0");
#endif

    /* default is default ;) */
    kver = eNbnxnCuKDefault;

    if (bCUDA32 || bCUDA40)
    {
        /* use legacy kernel unless something else is forced by an env. var */
        if (bDefaultKernel)
        {
            fprintf(stderr,
                    "\nNOTE: CUDA %s compilation detected; with this compiler version the legacy\n"
                    "      non-bonded kernels perform best. However, the default kernels were\n"
                    "      selected by the GMX_CUDA_NB_DEFAULT environment variable.\n"
                    "      For best performance upgrade your CUDA toolkit.",
                    sbuf);
        }
        else
        {
            kver = eNbnxnCuKLegacy;
        }
    }
    else
    {
        /* issue not if the non-default kernel is forced by an env. var */
        if (bLegacyKernel)
        {
            fprintf(stderr,
                    "\nNOTE: Legacy non-bonded CUDA kernels were selected by the GMX_CUDA_NB_LEGACY\n"
                    "      env. var. Consider using using the default kernels which should be faster!\n");

            kver = eNbnxnCuKLegacy;
        }
    }

    return kver;
}

void nbnxn_cuda_init(FILE *fplog,
                     nbnxn_cuda_ptr_t *p_cu_nb,
                     gmx_gpu_info_t *gpu_info, int my_gpu_index,
                     gmx_bool bLocalAndNonlocal)
{
    hipError_t stat;
    nbnxn_cuda_ptr_t  nb;
    char sbuf[STRLEN];
    bool bStreamSync, bNoStreamSync, bTMPIAtomics, bX86;

    assert(gpu_info);

    if (p_cu_nb == NULL) return;

    snew(nb, 1);
    snew(nb->atdat, 1);
    snew(nb->nbparam, 1);
    snew(nb->plist[eintLocal], 1);
    if (bLocalAndNonlocal)
    {
        snew(nb->plist[eintNonlocal], 1);
    }

    nb->bUseTwoStreams = bLocalAndNonlocal;

    snew(nb->timers, 1);
    snew(nb->timings, 1);

    /* init nbst */
    pmalloc((void**)&nb->nbst.e_lj, sizeof(*nb->nbst.e_lj));
    pmalloc((void**)&nb->nbst.e_el, sizeof(*nb->nbst.e_el));
    pmalloc((void**)&nb->nbst.fshift, SHIFTS * sizeof(*nb->nbst.fshift));

    init_plist(nb->plist[eintLocal]);

    /* local/non-local GPU streams */
    stat = hipStreamCreate(&nb->stream[eintLocal]);
    CU_RET_ERR(stat, "hipStreamCreate on stream[eintLocal] failed");
    if (nb->bUseTwoStreams)
    {
        init_plist(nb->plist[eintNonlocal]);
        stat = hipStreamCreate(&nb->stream[eintNonlocal]);
        CU_RET_ERR(stat, "hipStreamCreate on stream[eintNonlocal] failed");
    }

    /* init events for sychronization (timing disabled for performance reasons!) */
    stat = hipEventCreateWithFlags(&nb->nonlocal_done, hipEventDisableTiming);
    CU_RET_ERR(stat, "hipEventCreate on nonlocal_done failed");
    stat = hipEventCreateWithFlags(&nb->misc_ops_done, hipEventDisableTiming);
    CU_RET_ERR(stat, "hipEventCreate on misc_ops_one failed");

    /* set device info, just point it to the right GPU among the detected ones */
    nb->dev_info = &gpu_info->cuda_dev[get_gpu_device_id(gpu_info, my_gpu_index)];

    /* On GPUs with ECC enabled, hipStreamSynchronize shows a large overhead
     * (which increases with shorter time/step) caused by a known CUDA driver bug.
     * To work around the issue we'll use an (admittedly fragile) memory polling
     * waiting to preserve performance. This requires support for atomic
     * operations and only works on x86/x86_64.
     * With polling wait event-timing also needs to be disabled.
     */

    bStreamSync    = getenv("GMX_CUDA_STREAMSYNC") != NULL;
    bNoStreamSync  = getenv("GMX_NO_CUDA_STREAMSYNC") != NULL;

#ifdef TMPI_ATOMICS
    bTMPIAtomics = true;
#else
    bTMPIAtomics = false;
#endif

#if defined(i386) || defined(__x86_64__)
    bX86 = true;
#else
    bX86 = false;
#endif

    if (bStreamSync && bNoStreamSync)
    {
        gmx_fatal(FARGS, "Conflicting environment variables: both GMX_CUDA_STREAMSYNC and GMX_NO_CUDA_STREAMSYNC defined");
    }

    if (nb->dev_info->prop.ECCEnabled == 1)
    {
        if (bStreamSync)
        {
            nb->bUseStreamSync = true;

            sprintf(sbuf,
                    "NOTE: Using a GPU with ECC enabled, but hipStreamSynchronize-based waiting is\n"
                    "      forced by the GMX_CUDA_STREAMSYNC env. var. Due to a CUDA bug, this \n"
                    "      combination causes performance loss.");
            fprintf(stderr, "\n%s\n", sbuf);
            if (fplog)
            {
                fprintf(fplog, "\n%s\n", sbuf);
            }
        }
        else
        {
            /* can use polling wait only on x86/x86_64 *if* atomics are available */
            nb->bUseStreamSync = ((bX86 && bTMPIAtomics) == false);

            if (!bX86)
            {
                sprintf(sbuf,
                        "Using a GPU with ECC on; the standard hipStreamSynchronize waiting, due to a\n"
                        "      CUDA bug, causes performance loss when used in combination with ECC.\n"
                        "      However, the polling waiting workaround can not be used as it is only\n"
                        "      supported on x86/x86_64, but not on the current architecture.");
                gmx_warning("%s\n", sbuf);
                if (fplog)
                {
                    fprintf(fplog, "\n%s\n", sbuf);
                }

            }
            else if (bTMPIAtomics)
            {
                if (fplog)
                {
                    fprintf(fplog,
                            "NOTE: Using a GPU with ECC enabled; will use polling waiting.\n");
                }
            }
            else
            {
                sprintf(sbuf,
                        "Using a GPU with ECC on; the standard hipStreamSynchronize waiting, due to a\n"
                        "      CUDA bug, causes performance loss when used in combination with ECC.\n"
                        "      However, the polling waiting workaround can not be used as atomic\n"
                        "      operations are not supported by the current CPU+compiler combination.");
                gmx_warning("%s\n", sbuf);
                if (fplog)
                {
                    fprintf(fplog, "\n%s\n", sbuf);
                }
            }
        }
    }
    else
    {
        if (bNoStreamSync)
        {
            nb->bUseStreamSync = false;

            sprintf(sbuf,
                    "NOTE: Using a GPU with no/disabled ECC, but hipStreamSynchronize-based waiting\n"
                    "      is turned off and polling turned on by the GMX_NO_CUDA_STREAMSYNC env. var.");
            fprintf(stderr, "\n%s\n", sbuf);
            if (fplog)
            {
                fprintf(fplog, "\n%s\n", sbuf);
            }
        }
        else
        {
            /* no/off ECC, hipStreamSynchronize not turned off by env. var. */
            nb->bUseStreamSync = true;
        }
    }

    /* CUDA timing disabled as event timers don't work:
       - with multiple streams = domain-decomposition;
       - with the polling waiting hack (without hipStreamSynchronize);
       - when turned off by GMX_DISABLE_CUDA_TIMING.
     */
    nb->bDoTime = (!nb->bUseTwoStreams && nb->bUseStreamSync &&
                   (getenv("GMX_DISABLE_CUDA_TIMING") == NULL));

    if (nb->bDoTime)
    {
        init_timers(nb->timers, nb->bUseTwoStreams);
        init_timings(nb->timings);
    }

    /* set the kernel type for the current GPU */
    nb->kernel_ver = pick_nbnxn_kernel_version();
    /* pick L1 cache configuration */
    nbnxn_cuda_set_cacheconfig(nb->dev_info);

    *p_cu_nb = nb;

    if (debug)
    {
        fprintf(debug, "Initialized CUDA data structures.\n");
    }
}

void nbnxn_cuda_init_const(nbnxn_cuda_ptr_t cu_nb,
                           const interaction_const_t *ic,
                           const nonbonded_verlet_t *nbv)
{
    init_atomdata_first(cu_nb->atdat, nbv->grp[0].nbat->ntype);
    init_nbparam(cu_nb->nbparam, ic, nbv);

    /* clear energy and shift force outputs */
    nbnxn_cuda_clear_e_fshift(cu_nb);
}

void nbnxn_cuda_init_pairlist(nbnxn_cuda_ptr_t cu_nb,
                              const nbnxn_pairlist_t *h_plist,
                              int iloc)
{
    char         sbuf[STRLEN];
    hipError_t  stat;
    bool         bDoTime    = cu_nb->bDoTime;
    hipStream_t stream     = cu_nb->stream[iloc];
    cu_plist_t   *d_plist   = cu_nb->plist[iloc];

    if (d_plist->na_c < 0)
    {
        d_plist->na_c = h_plist->na_ci;
    }
    else
    {
        if (d_plist->na_c != h_plist->na_ci)
        {
            sprintf(sbuf, "In cu_init_plist: the #atoms per cell has changed (from %d to %d)",
                    d_plist->na_c, h_plist->na_ci);
            gmx_incons(sbuf);
        }
    }

    if (bDoTime)
    {
        stat = hipEventRecord(cu_nb->timers->start_pl_h2d[iloc], stream);
        CU_RET_ERR(stat, "hipEventRecord failed");
    }

    cu_realloc_buffered((void **)&d_plist->sci, h_plist->sci, sizeof(*d_plist->sci),
                         &d_plist->nsci, &d_plist->sci_nalloc,
                         h_plist->nsci,
                         stream, true);

    cu_realloc_buffered((void **)&d_plist->cj4, h_plist->cj4, sizeof(*d_plist->cj4),
                         &d_plist->ncj4, &d_plist->cj4_nalloc,
                         h_plist->ncj4,
                         stream, true);

    cu_realloc_buffered((void **)&d_plist->excl, h_plist->excl, sizeof(*d_plist->excl),
                         &d_plist->nexcl, &d_plist->excl_nalloc,
                         h_plist->nexcl,
                         stream, true);

    if (bDoTime)
    {
        stat = hipEventRecord(cu_nb->timers->stop_pl_h2d[iloc], stream);
        CU_RET_ERR(stat, "hipEventRecord failed");
    }

    /* need to prune the pair list during the next step */
    d_plist->bDoPrune = true;
}

void nbnxn_cuda_upload_shiftvec(nbnxn_cuda_ptr_t cu_nb,
                                const nbnxn_atomdata_t *nbatom)
{
    cu_atomdata_t *adat = cu_nb->atdat;
    hipStream_t  ls    = cu_nb->stream[eintLocal];

    /* only if we have a dynamic box */
    if (nbatom->bDynamicBox || !adat->bShiftVecUploaded)
    {
        cu_copy_H2D_async(adat->shift_vec, nbatom->shift_vec, 
                          SHIFTS * sizeof(*adat->shift_vec), ls);
        adat->bShiftVecUploaded = true;
    }
}

/*! Clears the first natoms_clear elements of the GPU nonbonded force output array. */
static void nbnxn_cuda_clear_f(nbnxn_cuda_ptr_t cu_nb, int natoms_clear)
{
    hipError_t   stat;
    cu_atomdata_t *adat = cu_nb->atdat;
    hipStream_t  ls    = cu_nb->stream[eintLocal];

    stat = hipMemsetAsync(adat->f, 0, natoms_clear * sizeof(*adat->f), ls);
    CU_RET_ERR(stat, "hipMemsetAsync on f falied");
}

/*! Clears nonbonded shift force output array and energy outputs on the GPU. */
static void nbnxn_cuda_clear_e_fshift(nbnxn_cuda_ptr_t cu_nb)
{
    hipError_t   stat;
    cu_atomdata_t *adat = cu_nb->atdat;
    hipStream_t  ls    = cu_nb->stream[eintLocal];

    stat = hipMemsetAsync(adat->fshift, 0, SHIFTS * sizeof(*adat->fshift), ls);
    CU_RET_ERR(stat, "hipMemsetAsync on fshift falied");
    stat = hipMemsetAsync(adat->e_lj, 0, sizeof(*adat->e_lj), ls);
    CU_RET_ERR(stat, "hipMemsetAsync on e_lj falied");
    stat = hipMemsetAsync(adat->e_el, 0, sizeof(*adat->e_el), ls);
    CU_RET_ERR(stat, "hipMemsetAsync on e_el falied");
}

void nbnxn_cuda_clear_outputs(nbnxn_cuda_ptr_t cu_nb, int flags)
{
    nbnxn_cuda_clear_f(cu_nb, cu_nb->atdat->natoms);
    /* clear shift force array and energies if the outputs were 
       used in the current step */
    if (flags & GMX_FORCE_VIRIAL)
    {
        nbnxn_cuda_clear_e_fshift(cu_nb);
    }
}

void nbnxn_cuda_init_atomdata(nbnxn_cuda_ptr_t cu_nb,
                              const nbnxn_atomdata_t *nbat)
{
    hipError_t   stat;
    int           nalloc, natoms;
    bool          realloced;
    bool          bDoTime   = cu_nb->bDoTime;
    cu_timers_t   *timers   = cu_nb->timers;
    cu_atomdata_t *d_atdat  = cu_nb->atdat;
    hipStream_t  ls        = cu_nb->stream[eintLocal];

    natoms = nbat->natoms;
    realloced = false;

    if (bDoTime)
    {
        /* time async copy */
        stat = hipEventRecord(timers->start_atdat, ls);
        CU_RET_ERR(stat, "hipEventRecord failed");
    }

    /* need to reallocate if we have to copy more atoms than the amount of space
       available and only allocate if we haven't initialized yet, i.e d_atdat->natoms == -1 */
    if (natoms > d_atdat->nalloc)
    {
        nalloc = over_alloc_small(natoms);

        /* free up first if the arrays have already been initialized */
        if (d_atdat->nalloc != -1)
        {
            cu_free_buffered(d_atdat->f, &d_atdat->natoms, &d_atdat->nalloc);
            cu_free_buffered(d_atdat->xq);
            cu_free_buffered(d_atdat->atom_types);
        }

        stat = hipMalloc((void **)&d_atdat->f, nalloc*sizeof(*d_atdat->f));
        CU_RET_ERR(stat, "hipMalloc failed on d_atdat->f");
        stat = hipMalloc((void **)&d_atdat->xq, nalloc*sizeof(*d_atdat->xq));
        CU_RET_ERR(stat, "hipMalloc failed on d_atdat->xq");

        stat = hipMalloc((void **)&d_atdat->atom_types, nalloc*sizeof(*d_atdat->atom_types));
        CU_RET_ERR(stat, "hipMalloc failed on d_atdat->atom_types");

        d_atdat->nalloc = nalloc;
        realloced = true;
    }

    d_atdat->natoms = natoms;
    d_atdat->natoms_local = nbat->natoms_local;

    /* need to clear GPU f output if realloc happened */
    if (realloced)
    {
        nbnxn_cuda_clear_f(cu_nb, nalloc);
    }

    cu_copy_H2D_async(d_atdat->atom_types, nbat->type,
                      natoms*sizeof(*d_atdat->atom_types), ls);

    if (bDoTime)
    {
        stat = hipEventRecord(timers->stop_atdat, ls);
        CU_RET_ERR(stat, "hipEventRecord failed");
    }
}

void nbnxn_cuda_free(FILE *fplog, nbnxn_cuda_ptr_t cu_nb)
{
    hipError_t     stat;
    cu_atomdata_t   *atdat;
    cu_nbparam_t    *nbparam;
    cu_plist_t      *plist, *plist_nl;
    cu_timers_t     *timers;

    if (cu_nb == NULL) return;

    atdat       = cu_nb->atdat;
    nbparam     = cu_nb->nbparam;
    plist       = cu_nb->plist[eintLocal];
    plist_nl    = cu_nb->plist[eintNonlocal];
    timers      = cu_nb->timers;

    if (nbparam->eeltype == eelCuEWALD || nbparam->eeltype == eelCuEWALD_TWIN)
    {
      stat = hipUnbindTexture(nbnxn_cuda_get_coulomb_tab_texref());
      CU_RET_ERR(stat, "hipUnbindTexture on coulomb_tab failed");
      cu_free_buffered(nbparam->coulomb_tab, &nbparam->coulomb_tab_size);
    }

    stat = hipEventDestroy(cu_nb->nonlocal_done);
    CU_RET_ERR(stat, "hipEventDestroy failed on timers->nonlocal_done");
    stat = hipEventDestroy(cu_nb->misc_ops_done);
    CU_RET_ERR(stat, "hipEventDestroy failed on timers->misc_ops_done");

    if (cu_nb->bDoTime)
    {
        stat = hipEventDestroy(timers->start_atdat);
        CU_RET_ERR(stat, "hipEventDestroy failed on timers->start_atdat");
        stat = hipEventDestroy(timers->stop_atdat);
        CU_RET_ERR(stat, "hipEventDestroy failed on timers->stop_atdat");

        /* The non-local counters/stream (second in the array) are needed only with DD. */
        for (int i = 0; i <= (cu_nb->bUseTwoStreams ? 1 : 0); i++)
        {
            stat = hipEventDestroy(timers->start_nb_k[i]);
            CU_RET_ERR(stat, "hipEventDestroy failed on timers->start_nb_k");
            stat = hipEventDestroy(timers->stop_nb_k[i]);
            CU_RET_ERR(stat, "hipEventDestroy failed on timers->stop_nb_k");

            stat = hipEventDestroy(timers->start_pl_h2d[i]);
            CU_RET_ERR(stat, "hipEventDestroy failed on timers->start_pl_h2d");
            stat = hipEventDestroy(timers->stop_pl_h2d[i]);
            CU_RET_ERR(stat, "hipEventDestroy failed on timers->stop_pl_h2d");

            stat = hipStreamDestroy(cu_nb->stream[i]);
            CU_RET_ERR(stat, "hipStreamDestroy failed on stream");

            stat = hipEventDestroy(timers->start_nb_h2d[i]);
            CU_RET_ERR(stat, "hipEventDestroy failed on timers->start_nb_h2d");
            stat = hipEventDestroy(timers->stop_nb_h2d[i]);
            CU_RET_ERR(stat, "hipEventDestroy failed on timers->stop_nb_h2d");

            stat = hipEventDestroy(timers->start_nb_d2h[i]);
            CU_RET_ERR(stat, "hipEventDestroy failed on timers->start_nb_d2h");
            stat = hipEventDestroy(timers->stop_nb_d2h[i]);
            CU_RET_ERR(stat, "hipEventDestroy failed on timers->stop_nb_d2h");
        }
    }

    stat = hipUnbindTexture(nbnxn_cuda_get_nbfp_texref());
    CU_RET_ERR(stat, "hipUnbindTexture on coulomb_tab failed");
    cu_free_buffered(nbparam->nbfp);

    stat = hipFree(atdat->shift_vec);
    CU_RET_ERR(stat, "hipFree failed on atdat->shift_vec");
    stat = hipFree(atdat->fshift);
    CU_RET_ERR(stat, "hipFree failed on atdat->fshift");

    stat = hipFree(atdat->e_lj);
    CU_RET_ERR(stat, "hipFree failed on atdat->e_lj");
    stat = hipFree(atdat->e_el);
    CU_RET_ERR(stat, "hipFree failed on atdat->e_el");

    cu_free_buffered(atdat->f, &atdat->natoms, &atdat->nalloc);
    cu_free_buffered(atdat->xq);
    cu_free_buffered(atdat->atom_types, &atdat->ntypes);

    cu_free_buffered(plist->sci, &plist->nsci, &plist->sci_nalloc);
    cu_free_buffered(plist->cj4, &plist->ncj4, &plist->cj4_nalloc);
    cu_free_buffered(plist->excl, &plist->nexcl, &plist->excl_nalloc);
    if (cu_nb->bUseTwoStreams)
    {
        cu_free_buffered(plist_nl->sci, &plist_nl->nsci, &plist_nl->sci_nalloc);
        cu_free_buffered(plist_nl->cj4, &plist_nl->ncj4, &plist_nl->cj4_nalloc);
        cu_free_buffered(plist_nl->excl, &plist_nl->nexcl, &plist->excl_nalloc);
    }

    if (debug)
    {
        fprintf(debug, "Cleaned up CUDA data structures.\n");
    }
}

void cu_synchstream_atdat(nbnxn_cuda_ptr_t cu_nb, int iloc)
{
    hipError_t stat;
    hipStream_t stream = cu_nb->stream[iloc];

    stat = hipStreamWaitEvent(stream, cu_nb->timers->stop_atdat, 0);
    CU_RET_ERR(stat, "hipStreamWaitEvent failed");
}

wallclock_gpu_t * nbnxn_cuda_get_timings(nbnxn_cuda_ptr_t cu_nb)
{
    return (cu_nb != NULL && cu_nb->bDoTime) ? cu_nb->timings : NULL;
}

void nbnxn_cuda_reset_timings(nbnxn_cuda_ptr_t cu_nb)
{
    if (cu_nb->bDoTime)
    {
        init_timings(cu_nb->timings);
    }
}

int nbnxn_cuda_min_ci_balanced(nbnxn_cuda_ptr_t cu_nb)
{
    return cu_nb != NULL ?
        gpu_min_ci_balanced_factor*cu_nb->dev_info->prop.multiProcessorCount : 0;

}
