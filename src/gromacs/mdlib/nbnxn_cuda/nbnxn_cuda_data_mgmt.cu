/*
 * This file is part of the GROMACS molecular simulation package.
 *
 * Copyright (c) 2012,2013,2014,2015, by the GROMACS development team, led by
 * Mark Abraham, David van der Spoel, Berk Hess, and Erik Lindahl,
 * and including many others, as listed in the AUTHORS file in the
 * top-level source directory and at http://www.gromacs.org.
 *
 * GROMACS is free software; you can redistribute it and/or
 * modify it under the terms of the GNU Lesser General Public License
 * as published by the Free Software Foundation; either version 2.1
 * of the License, or (at your option) any later version.
 *
 * GROMACS is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
 * Lesser General Public License for more details.
 *
 * You should have received a copy of the GNU Lesser General Public
 * License along with GROMACS; if not, see
 * http://www.gnu.org/licenses, or write to the Free Software Foundation,
 * Inc., 51 Franklin Street, Fifth Floor, Boston, MA  02110-1301  USA.
 *
 * If you want to redistribute modifications to GROMACS, please
 * consider that scientific software is very special. Version
 * control is crucial - bugs must be traceable. We will be happy to
 * consider code for inclusion in the official distribution, but
 * derived work must not be called official GROMACS. Details are found
 * in the README & COPYING files - if they are missing, get the
 * official version at http://www.gromacs.org.
 *
 * To help us fund GROMACS development, we humbly ask that you cite
 * the research papers on the package. Check out http://www.gromacs.org.
 */
/*! \file
 *  \brief Define CUDA implementation of nbnxn_gpu_data_mgmt.h
 *
 *  \author Szilard Pall <pall.szilard@gmail.com>
 *  \author Alfredo Metere <alfredometere2@gmail.com>
 */
#include "gmxpre.h"

#include "config.h"

#include <assert.h>
#include <stdarg.h>
#include <stdio.h>
#include <stdlib.h>

#include <hip/hip_runtime_api.h>

#include "gromacs/gmxlib/cuda_tools/cudautils.cuh"
#include "gromacs/gmxlib/cuda_tools/pmalloc_cuda.h"
#include "gromacs/gmxlib/gpu_utils/gpu_utils.h"
#include "gromacs/legacyheaders/gmx_detect_hardware.h"
#include "gromacs/legacyheaders/typedefs.h"
#include "gromacs/legacyheaders/types/enums.h"
#include "gromacs/legacyheaders/types/force_flags.h"
#include "gromacs/legacyheaders/types/interaction_const.h"
#include "gromacs/mdlib/nb_verlet.h"
#include "gromacs/mdlib/nbnxn_consts.h"
#include "gromacs/mdlib/nbnxn_gpu_data_mgmt.h"
#include "gromacs/pbcutil/ishift.h"
#include "gromacs/timing/gpu_timing.h"
#include "gromacs/utility/basedefinitions.h"
#include "gromacs/utility/cstringutil.h"
#include "gromacs/utility/fatalerror.h"
#include "gromacs/utility/smalloc.h"

#include "nbnxn_cuda_types.h"

static bool bUseCudaEventBlockingSync = false; /* makes the CPU thread block */

/* This is a heuristically determined parameter for the Fermi, Kepler
 * and Maxwell architectures for the minimum size of ci lists by multiplying
 * this constant with the # of multiprocessors on the current device.
 * Since the maximum number of blocks per multiprocessor is 16, the ideal
 * count for small systems is 32 or 48 blocks per multiprocessor. Because
 * there is a bit of fluctuations in the generated block counts, we use
 * a target of 44 instead of the ideal value of 48.
 */
static unsigned int gpu_min_ci_balanced_factor = 44;

/* Functions from nbnxn_cuda.cu */
extern void nbnxn_cuda_set_cacheconfig(gmx_device_info_t *devinfo);
extern const struct texture<float, 1, hipReadModeElementType> &nbnxn_cuda_get_nbfp_texref();
extern const struct texture<float, 1, hipReadModeElementType> &nbnxn_cuda_get_nbfp_comb_texref();
extern const struct texture<float, 1, hipReadModeElementType> &nbnxn_cuda_get_coulomb_tab_texref();

/* User tables */
extern const struct texture<float, 1, hipReadModeElementType> &nbnxn_cuda_get_nb_generic_Ftab_texref();
extern const struct texture<float, 1, hipReadModeElementType> &nbnxn_cuda_get_nb_generic_Vtab_texref();

extern const struct texture<float, 1, hipReadModeElementType> &nbnxn_cuda_get_nb_vdw_LJ6_Ftab_texref();
extern const struct texture<float, 1, hipReadModeElementType> &nbnxn_cuda_get_nb_vdw_LJ6_Vtab_texref();
extern const struct texture<float, 1, hipReadModeElementType> &nbnxn_cuda_get_nb_vdw_LJ12_Ftab_texref();
extern const struct texture<float, 1, hipReadModeElementType> &nbnxn_cuda_get_nb_vdw_LJ12_Vtab_texref();

extern const struct texture<float, 1, hipReadModeElementType> &nbnxn_cuda_get_nb_coul_Ftab_texref();
extern const struct texture<float, 1, hipReadModeElementType> &nbnxn_cuda_get_nb_coul_Vtab_texref();

/* Fw. decl. */
static void nbnxn_cuda_clear_e_fshift(gmx_nbnxn_cuda_t *nb);

/* Fw. decl, */
static void nbnxn_cuda_free_nbparam_table(cu_nbparam_t            *nbparam,
                                          const gmx_device_info_t *dev_info);

static bool use_texobj(const gmx_device_info_t *dev_info)
{
    /* Only device CC >= 3.0 (Kepler and later) support texture objects */
    return (dev_info->prop.major >= 3);
}

/*! Tabulates the Ewald Coulomb force and initializes the size/scale
    and the table GPU array. If called with an already allocated table,
    it just re-uploads the table.
 */
static void init_ewald_coulomb_force_table(const interaction_const_t *ic,
                                           cu_nbparam_t              *nbp,
                                           const gmx_device_info_t   *dev_info)
{
    float       *coul_tab;
    hipError_t  stat;

    if (nbp->coulomb_tab != NULL)
    {
        nbnxn_cuda_free_nbparam_table(nbp, dev_info);
    }

    stat = hipMalloc((void **)&coul_tab, ic->tabq_size*sizeof(*coul_tab));
    CU_RET_ERR(stat, "hipMalloc failed on coul_tab");

    nbp->coulomb_tab = coul_tab;

    /* Only device CC >= 3.0 (Kepler and later) support texture objects */
    if (use_texobj(dev_info))
    {
        hipResourceDesc rd;
        memset(&rd, 0, sizeof(rd));
        rd.resType                  = hipResourceTypeLinear;
        rd.res.linear.devPtr        = nbp->coulomb_tab;
        rd.res.linear.desc.f        = hipChannelFormatKindFloat;
        rd.res.linear.desc.x        = 32;
        rd.res.linear.sizeInBytes   = ic->tabq_size*sizeof(*coul_tab);

        hipTextureDesc td;
        memset(&td, 0, sizeof(td));
        td.readMode                 = hipReadModeElementType;
        stat = hipCreateTextureObject(&nbp->coulomb_tab_texobj, &rd, &td, NULL);
        CU_RET_ERR(stat, "hipCreateTextureObject on coulomb_tab_texobj failed");
    }
    else
    {
        GMX_UNUSED_VALUE(dev_info);
        hipChannelFormatDesc cd   = hipCreateChannelDesc<float>();
        stat = hipBindTexture(NULL, &nbnxn_cuda_get_coulomb_tab_texref(),
                               coul_tab, &cd,
                               ic->tabq_size*sizeof(*coul_tab));
        CU_RET_ERR(stat, "hipBindTexture on coulomb_tab_texref failed");
    }

    cu_copy_H2D(coul_tab, ic->tabq_coul_F, ic->tabq_size*sizeof(*coul_tab));

    nbp->coulomb_tab_size     = ic->tabq_size;
    nbp->coulomb_tab_scale    = ic->tabq_scale;
}

/*! Initializes the Non-bonded force table with the size/scale
    and the table GPU array. If called with an already allocated table,
    it just re-uploads the table.
 */
static void init_nb_generic_Ftables(const interaction_const_t  *ic,
                                    cu_nbparam_t               *nbp,
                                    const gmx_device_info_t    *dev_info)
{
    float       *nbgFTab;
    int          tabq_size;
    hipError_t  gFstat;

    if (debug)
    {
        FILE        *fFgeneric, *fVgeneric;
        fFgeneric = fopen("generic_Ftable.csv", "w");
        fVgeneric = fopen("generic_Vtable.csv", "w");

        for (int i = 0; i < ic->tabq_size; i++)
        {
            fprintf(fFgeneric, "%d,%12.10f\n", i, ic->tabVerlet_nbtab_F[i]);
            fprintf(fVgeneric, "%d,%12.10f\n", i, ic->tabVerlet_nbtab_V[i]);
        }
        fprintf (debug, "Generic table CSV files created\n");
        fclose(fFgeneric);
        fclose(fVgeneric);
    }

    tabq_size = ic->tabq_size;

    if (nbp->nb_generic_Ftab != NULL)
    {
        nbnxn_cuda_free_nbparam_table(nbp, dev_info);
    }

    gFstat = hipMalloc((float **)&nbgFTab, tabq_size*sizeof(*nbgFTab));
    CU_RET_ERR(gFstat, "hipMalloc failed on nbFTab");

    nbp->nb_generic_Ftab = nbgFTab;
    /* Only device CC >= 3.0 (Kepler and later) support texture objects */
    if (use_texobj(dev_info))
    {
        hipResourceDesc gFrd;

        memset(&gFrd, 0, sizeof(gFrd));

        gFrd.resType                  = hipResourceTypeLinear;
        gFrd.res.linear.devPtr        = nbp->nb_generic_Ftab;
        gFrd.res.linear.desc.f        = hipChannelFormatKindFloat;
        gFrd.res.linear.desc.x        = 32;
        gFrd.res.linear.sizeInBytes   = ic->tabq_size*sizeof(*nbgFTab);

        hipTextureDesc gFtd;

        memset(&gFtd, 0, sizeof(gFtd));

        gFtd.readMode                 = hipReadModeElementType;

        gFstat = hipCreateTextureObject(&nbp->nb_generic_Ftab_texobj, &gFrd, &gFtd, NULL);
        CU_RET_ERR(gFstat, "hipCreateTextureObject on nb_generic_Ftab_texobj failed");
    }
    else
    {
        GMX_UNUSED_VALUE(dev_info);
        hipChannelFormatDesc gFcd   = hipCreateChannelDesc<float>();

        gFstat = hipBindTexture(NULL, &nbnxn_cuda_get_nb_generic_Ftab_texref(),
                                 nbgFTab, &gFcd,
                                 ic->tabq_size*sizeof(*nbgFTab));
        CU_RET_ERR(gFstat, "hipBindTexture on nb_generic_Ftab_texref failed");
    }

    cu_copy_H2D(nbgFTab, ic->tabVerlet_nbtab_F, ic->tabq_size*sizeof(*nbgFTab));

    nbp->nb_generic_tab_size          = ic->tabq_size;
    nbp->nb_generic_tab_scale         = ic->tabq_scale;
}

static void init_nb_generic_Vtables(const interaction_const_t  *ic,
                                    cu_nbparam_t               *nbp,
                                    const gmx_device_info_t    *dev_info)
{
    float       *nbgVTab;

    hipError_t  gVstat;

    if (nbp->nb_generic_Vtab != NULL)
    {
        nbnxn_cuda_free_nbparam_table(nbp, dev_info);
    }

    gVstat = hipMalloc((float **)&nbgVTab, ic->tabq_size*sizeof(*nbgVTab));
    CU_RET_ERR(gVstat, "hipMalloc failed on nb_generic_VTab");

    nbp->nb_generic_Vtab = nbgVTab;
    /* Only device CC >= 3.0 (Kepler and later) support texture objects */
    if (use_texobj(dev_info))
    {
        hipResourceDesc gVrd;

        memset(&gVrd, 0, sizeof(gVrd));
        gVrd.resType                  = hipResourceTypeLinear;
        gVrd.res.linear.devPtr        = nbp->nb_generic_Vtab;
        gVrd.res.linear.desc.f        = hipChannelFormatKindFloat;
        gVrd.res.linear.desc.x        = 32;
        gVrd.res.linear.sizeInBytes   = ic->tabq_size*sizeof(*nbgVTab);

        hipTextureDesc gVtd;
        memset(&gVtd, 0, sizeof(gVtd));
        gVtd.readMode                 = hipReadModeElementType;

        gVstat = hipCreateTextureObject(&nbp->nb_generic_Vtab_texobj, &gVrd, &gVtd, NULL);
        CU_RET_ERR(gVstat, "hipCreateTextureObject on nb_generic_Vtab_texobj failed");

    }
    else
    {
        GMX_UNUSED_VALUE(dev_info);
        hipChannelFormatDesc gVcd   = hipCreateChannelDesc<float>();

        gVstat = hipBindTexture(NULL, &nbnxn_cuda_get_nb_generic_Vtab_texref(),
                                 nbgVTab, &gVcd,
                                 ic->tabq_size*sizeof(*nbgVTab));
        CU_RET_ERR(gVstat, "hipBindTexture on nb_generic_Vtab_texref failed");
    }

    cu_copy_H2D(nbgVTab, ic->tabVerlet_nbtab_V, ic->tabq_size*sizeof(*nbgVTab));

    nbp->nb_generic_tab_size          = ic->tabq_size;
    nbp->nb_generic_tab_scale         = ic->tabq_scale;
}

static void init_nb_coul_Ftables(const interaction_const_t  *ic,
                                 cu_nbparam_t               *nbp,
                                 const gmx_device_info_t    *dev_info)
{
    float       *nbcFTab;
    int          tabq_size;
    hipError_t  cFstat;

    if (debug)
    {
        FILE *fFcoul, *fVcoul;
        fFcoul = fopen("coulFtable.csv", "w");
        fVcoul = fopen("coulVtable.csv", "w");

        for (int i = 0; i < ic->tabq_size; i++)
        {
            fprintf(fFcoul, "%d,%12.10f\n", i, ic->tabVerlet_coul_F[i]);
            fprintf(fVcoul, "%d,%12.10f\n", i, ic->tabVerlet_coul_V[i]);
        }
        printf ("CSV files created\n");
        fclose(fFcoul);
        fclose(fVcoul);
    }

    tabq_size = ic->tabq_size;

    if (nbp->nb_coul_Ftab != NULL)
    {
        nbnxn_cuda_free_nbparam_table(nbp, dev_info);
    }

    cFstat = hipMalloc((float **)&nbcFTab, tabq_size*sizeof(*nbcFTab));
    CU_RET_ERR(cFstat, "hipMalloc failed on coul_nbFTab");

    nbp->nb_coul_Ftab = nbcFTab;
    /* Only device CC >= 3.0 (Kepler and later) support texture objects */
    if (use_texobj(dev_info))
    {
        hipResourceDesc cFrd;

        memset(&cFrd, 0, sizeof(cFrd));

        cFrd.resType                  = hipResourceTypeLinear;
        cFrd.res.linear.devPtr        = nbp->nb_coul_Ftab;
        cFrd.res.linear.desc.f        = hipChannelFormatKindFloat;
        cFrd.res.linear.desc.x        = 32;
        cFrd.res.linear.sizeInBytes   = ic->tabq_size*sizeof(*nbcFTab);

        hipTextureDesc cFtd;

        memset(&cFtd, 0, sizeof(cFtd));

        cFtd.readMode                 = hipReadModeElementType;

        cFstat = hipCreateTextureObject(&nbp->nb_coul_Ftab_texobj, &cFrd, &cFtd, NULL);
        CU_RET_ERR(cFstat, "hipCreateTextureObject on nb_coul_Ftab_texobj failed");
    }
    else
    {
        GMX_UNUSED_VALUE(dev_info);
        hipChannelFormatDesc cFcd   = hipCreateChannelDesc<float>();

        cFstat = hipBindTexture(NULL, &nbnxn_cuda_get_nb_coul_Ftab_texref(),
                                 nbcFTab, &cFcd,
                                 ic->tabq_size*sizeof(*nbcFTab));
        CU_RET_ERR(cFstat, "hipBindTexture on nb_coul_Ftab_texref failed");
    }

    cu_copy_H2D(nbcFTab, ic->tabVerlet_coul_F, ic->tabq_size*sizeof(*nbcFTab));

    nbp->nb_coul_tab_size          = ic->tabq_size;
    nbp->nb_coul_tab_scale         = ic->tabq_scale;
}

static void init_nb_coul_Vtables(const interaction_const_t  *ic,
                                 cu_nbparam_t               *nbp,
                                 const gmx_device_info_t    *dev_info)
{
    float       *nbcVTab;
    hipError_t  cVstat;

    if (nbp->nb_coul_Vtab != NULL)
    {
        nbnxn_cuda_free_nbparam_table(nbp, dev_info);
    }

    cVstat = hipMalloc((float **)&nbcVTab, ic->tabq_size*sizeof(*nbcVTab));
    CU_RET_ERR(cVstat, "hipMalloc failed on coul_nbVTab");

    nbp->nb_coul_Vtab = nbcVTab;

    if (use_texobj(dev_info))
    {
        hipResourceDesc cVrd;

        memset(&cVrd, 0, sizeof(cVrd));
        cVrd.resType                  = hipResourceTypeLinear;
        cVrd.res.linear.devPtr        = nbp->nb_coul_Vtab;
        cVrd.res.linear.desc.f        = hipChannelFormatKindFloat;
        cVrd.res.linear.desc.x        = 32;
        cVrd.res.linear.sizeInBytes   = ic->tabq_size*sizeof(*nbcVTab);

        hipTextureDesc cVtd;
        memset(&cVtd, 0, sizeof(cVtd));
        cVtd.readMode                 = hipReadModeElementType;

        cVstat = hipCreateTextureObject(&nbp->nb_coul_Vtab_texobj, &cVrd, &cVtd, NULL);
        CU_RET_ERR(cVstat, "hipCreateTextureObject on nb_coul_Vtab_texobj failed");


    }
    else
    {
        GMX_UNUSED_VALUE(dev_info);
        hipChannelFormatDesc cVcd   = hipCreateChannelDesc<float>();

        cVstat = hipBindTexture(NULL, &nbnxn_cuda_get_nb_coul_Vtab_texref(),
                                 nbcVTab, &cVcd,
                                 ic->tabq_size*sizeof(*nbcVTab));
        CU_RET_ERR(cVstat, "hipBindTexture on nb_coul_Vtab_texref failed");
    }

    cu_copy_H2D(nbcVTab, ic->tabVerlet_coul_V, ic->tabq_size*sizeof(*nbcVTab));

    nbp->nb_coul_tab_size          = ic->tabq_size;
    nbp->nb_coul_tab_scale         = ic->tabq_scale;
}

static void init_nb_vdw_LJ6_Ftables(const interaction_const_t  *ic,
                                    cu_nbparam_t               *nbp,
                                    const gmx_device_info_t    *dev_info)
{
    float       *nbv6FTab;
    int          tabq_size;
    hipError_t  v6Fstat;

    if (debug)
    {
        FILE *fFvdw6, *fVvdw6;
        fFvdw6 = fopen("vdw_LJ6_Ftable.csv", "w");
        fVvdw6 = fopen("vdw_LJ6_Vtable.csv", "w");

        for (int i = 0; i < ic->tabq_size; i++)
        {
            fprintf(fFvdw6, "%d,%12.10f\n", i, ic->tabVerlet_vdw_LJ6_F[i]);
            fprintf(fVvdw6, "%d,%12.10f\n", i, ic->tabVerlet_vdw_LJ6_V[i]);
        }
        fprintf (debug, "vdw CSV files created\n");
        fclose(fFvdw6);
        fclose(fVvdw6);
    }

    tabq_size = ic->tabq_size;

    if (nbp->nb_vdw_LJ6_Ftab != NULL)
    {
        nbnxn_cuda_free_nbparam_table(nbp, dev_info);
    }

    v6Fstat = hipMalloc((float **)&nbv6FTab, tabq_size*sizeof(*nbv6FTab));
    CU_RET_ERR(v6Fstat, "hipMalloc failed on vdw LJ6 nbFTab");

    nbp->nb_vdw_LJ6_Ftab = nbv6FTab;

    if (use_texobj(dev_info))
    {
        hipResourceDesc v6Frd;

        memset(&v6Frd, 0, sizeof(v6Frd));

        v6Frd.resType                  = hipResourceTypeLinear;
        v6Frd.res.linear.devPtr        = nbp->nb_vdw_LJ6_Ftab;
        v6Frd.res.linear.desc.f        = hipChannelFormatKindFloat;
        v6Frd.res.linear.desc.x        = 32;
        v6Frd.res.linear.sizeInBytes   = ic->tabq_size*sizeof(*nbv6FTab);

        hipTextureDesc v6Ftd;

        memset(&v6Ftd, 0, sizeof(v6Ftd));

        v6Ftd.readMode                 = hipReadModeElementType;

        v6Fstat = hipCreateTextureObject(&nbp->nb_vdw_LJ6_Ftab_texobj, &v6Frd, &v6Ftd, NULL);
        CU_RET_ERR(v6Fstat, "hipCreateTextureObject on nb_vdw_LJ6_Ftab_texobj failed");
    }
    else
    {
        GMX_UNUSED_VALUE(dev_info);
        hipChannelFormatDesc v6Fcd   = hipCreateChannelDesc<float>();

        v6Fstat = hipBindTexture(NULL, &nbnxn_cuda_get_nb_vdw_LJ6_Ftab_texref(),
                                  nbv6FTab, &v6Fcd,
                                  ic->tabq_size*sizeof(*nbv6FTab));
        CU_RET_ERR(v6Fstat, "hipBindTexture on nb_vdw_Ftab_texref failed");
    }

    cu_copy_H2D(nbv6FTab, ic->tabVerlet_vdw_LJ6_F, ic->tabq_size*sizeof(*nbv6FTab));

    nbp->nb_vdw_tab_size          = ic->tabq_size;
    nbp->nb_vdw_tab_scale         = ic->tabq_scale;
}
static void init_nb_vdw_LJ12_Ftables(const interaction_const_t  *ic,
                                     cu_nbparam_t               *nbp,
                                     const gmx_device_info_t    *dev_info)
{
    float       *nbv12FTab;
    int          tabq_size;
    hipError_t  v12Fstat;

    if (debug)
    {
        FILE *fFvdw12, *fVvdw12;
        fFvdw12 = fopen("vdw_LJ12_Ftable.csv", "w");
        fVvdw12 = fopen("vdw_LJ12_Vtable.csv", "w");

        for (int i = 0; i < ic->tabq_size; i++)
        {
            fprintf(fFvdw12, "%d,%12.10f\n", i, ic->tabVerlet_vdw_LJ12_F[i]);
            fprintf(fVvdw12, "%d,%12.10f\n", i, ic->tabVerlet_vdw_LJ12_V[i]);
        }
        fprintf (debug, "vdw LJ12 CSV files created\n");
        fclose(fFvdw12);
        fclose(fVvdw12);
    }

    tabq_size = ic->tabq_size;

    if (nbp->nb_vdw_LJ12_Ftab != NULL)
    {
        nbnxn_cuda_free_nbparam_table(nbp, dev_info);
    }

    v12Fstat = hipMalloc((float **)&nbv12FTab, tabq_size*sizeof(*nbv12FTab));
    CU_RET_ERR(v12Fstat, "hipMalloc failed on vdw LJ12 nbFTab");

    nbp->nb_vdw_LJ12_Ftab = nbv12FTab;

    if (use_texobj(dev_info))
    {
        hipResourceDesc v12Frd;

        memset(&v12Frd, 0, sizeof(v12Frd));

        v12Frd.resType                  = hipResourceTypeLinear;
        v12Frd.res.linear.devPtr        = nbp->nb_vdw_LJ12_Ftab;
        v12Frd.res.linear.desc.f        = hipChannelFormatKindFloat;
        v12Frd.res.linear.desc.x        = 32;
        v12Frd.res.linear.sizeInBytes   = ic->tabq_size*sizeof(*nbv12FTab);

        hipTextureDesc v12Ftd;

        memset(&v12Ftd, 0, sizeof(v12Ftd));

        v12Ftd.readMode                 = hipReadModeElementType;

        v12Fstat = hipCreateTextureObject(&nbp->nb_vdw_LJ12_Ftab_texobj, &v12Frd, &v12Ftd, NULL);
        CU_RET_ERR(v12Fstat, "hipCreateTextureObject on nb_vdw_LJ12_Ftab_texobj failed");
    }
    else
    {
        GMX_UNUSED_VALUE(dev_info);
        hipChannelFormatDesc v12Fcd   = hipCreateChannelDesc<float>();

        v12Fstat = hipBindTexture(NULL, &nbnxn_cuda_get_nb_vdw_LJ12_Ftab_texref(),
                                   nbv12FTab, &v12Fcd,
                                   ic->tabq_size*sizeof(*nbv12FTab));
        CU_RET_ERR(v12Fstat, "hipBindTexture on nb_vdw_LJ12_Ftab_texref failed");
    }

    cu_copy_H2D(nbv12FTab, ic->tabVerlet_vdw_LJ12_F, ic->tabq_size*sizeof(*nbv12FTab));

    nbp->nb_vdw_tab_size          = ic->tabq_size;
    nbp->nb_vdw_tab_scale         = ic->tabq_scale;
}

static void init_nb_vdw_LJ6_Vtables(const interaction_const_t  *ic,
                                    cu_nbparam_t               *nbp,
                                    const gmx_device_info_t    *dev_info)
{
    float       *nbv6VTab;

    hipError_t  v6Vstat;

    if (nbp->nb_vdw_LJ6_Vtab != NULL)
    {
        nbnxn_cuda_free_nbparam_table(nbp, dev_info);
    }

    v6Vstat = hipMalloc((float **)&nbv6VTab, ic->tabq_size*sizeof(*nbv6VTab));
    CU_RET_ERR(v6Vstat, "hipMalloc failed on vdw LJ6 nbvVTab");
    nbp->nb_vdw_LJ6_Vtab = nbv6VTab;

    if (use_texobj(dev_info))
    {
        hipResourceDesc v6Vrd;

        memset(&v6Vrd, 0, sizeof(v6Vrd));
        v6Vrd.resType                  = hipResourceTypeLinear;
        v6Vrd.res.linear.devPtr        = nbp->nb_vdw_LJ6_Vtab;
        v6Vrd.res.linear.desc.f        = hipChannelFormatKindFloat;
        v6Vrd.res.linear.desc.x        = 32;
        v6Vrd.res.linear.sizeInBytes   = ic->tabq_size*sizeof(*nbv6VTab);

        hipTextureDesc v6Vtd;
        memset(&v6Vtd, 0, sizeof(v6Vtd));
        v6Vtd.readMode                 = hipReadModeElementType;

        v6Vstat = hipCreateTextureObject(&nbp->nb_vdw_LJ6_Vtab_texobj, &v6Vrd, &v6Vtd, NULL);
        CU_RET_ERR(v6Vstat, "hipCreateTextureObject on nb_vdw_Vtab_texobj failed");


    }
    else
    {
        GMX_UNUSED_VALUE(dev_info);
        hipChannelFormatDesc v6Vcd   = hipCreateChannelDesc<float>();

        v6Vstat = hipBindTexture(NULL, &nbnxn_cuda_get_nb_vdw_LJ6_Vtab_texref(),
                                  nbv6VTab, &v6Vcd,
                                  ic->tabq_size*sizeof(*nbv6VTab));
        CU_RET_ERR(v6Vstat, "hipBindTexture on nb_Vtab_texref failed");
    }

    cu_copy_H2D(nbv6VTab, ic->tabVerlet_vdw_LJ6_V, ic->tabq_size*sizeof(*nbv6VTab));

    nbp->nb_vdw_tab_size          = ic->tabq_size;
    nbp->nb_vdw_tab_scale         = ic->tabq_scale;
}

static void init_nb_vdw_LJ12_Vtables(const interaction_const_t  *ic,
                                     cu_nbparam_t               *nbp,
                                     const gmx_device_info_t    *dev_info)
{
    float       *nbv12VTab;

    hipError_t  v12Vstat;

    if (nbp->nb_vdw_LJ12_Vtab != NULL)
    {
        nbnxn_cuda_free_nbparam_table(nbp, dev_info);
    }

    v12Vstat = hipMalloc((float **)&nbv12VTab, ic->tabq_size*sizeof(*nbv12VTab));
    CU_RET_ERR(v12Vstat, "hipMalloc failed on vdw LJ12 nbvVTab");
    nbp->nb_vdw_LJ12_Vtab = nbv12VTab;

    if (use_texobj(dev_info))
    {
        hipResourceDesc v12Vrd;

        memset(&v12Vrd, 0, sizeof(v12Vrd));
        v12Vrd.resType                  = hipResourceTypeLinear;
        v12Vrd.res.linear.devPtr        = nbp->nb_vdw_LJ12_Vtab;
        v12Vrd.res.linear.desc.f        = hipChannelFormatKindFloat;
        v12Vrd.res.linear.desc.x        = 32;
        v12Vrd.res.linear.sizeInBytes   = ic->tabq_size*sizeof(*nbv12VTab);

        hipTextureDesc v12Vtd;
        memset(&v12Vtd, 0, sizeof(v12Vtd));
        v12Vtd.readMode                 = hipReadModeElementType;

        v12Vstat = hipCreateTextureObject(&nbp->nb_vdw_LJ12_Vtab_texobj, &v12Vrd, &v12Vtd, NULL);
        CU_RET_ERR(v12Vstat, "hipCreateTextureObject on nb_vdw_Vtab_texobj failed");


    }
    else
    {
        GMX_UNUSED_VALUE(dev_info);
        hipChannelFormatDesc v12Vcd   = hipCreateChannelDesc<float>();

        v12Vstat = hipBindTexture(NULL, &nbnxn_cuda_get_nb_vdw_LJ12_Vtab_texref(),
                                   nbv12VTab, &v12Vcd,
                                   ic->tabq_size*sizeof(*nbv12VTab));
        CU_RET_ERR(v12Vstat, "hipBindTexture on nb_Vtab_texref failed");
    }

    cu_copy_H2D(nbv12VTab, ic->tabVerlet_vdw_LJ12_V, ic->tabq_size*sizeof(*nbv12VTab));

    nbp->nb_vdw_tab_size          = ic->tabq_size;
    nbp->nb_vdw_tab_scale         = ic->tabq_scale;
}

/*! Initializes the atomdata structure first time, it only gets filled at
    pair-search. */
static void init_atomdata_first(cu_atomdata_t *ad, int ntypes)
{
    hipError_t stat;

    ad->ntypes  = ntypes;
    stat        = hipMalloc((void**)&ad->shift_vec, SHIFTS*sizeof(*ad->shift_vec));
    CU_RET_ERR(stat, "hipMalloc failed on ad->shift_vec");
    ad->bShiftVecUploaded = false;

    stat = hipMalloc((void**)&ad->fshift, SHIFTS*sizeof(*ad->fshift));
    CU_RET_ERR(stat, "hipMalloc failed on ad->fshift");

    stat = hipMalloc((void**)&ad->e_lj, sizeof(*ad->e_lj));
    CU_RET_ERR(stat, "hipMalloc failed on ad->e_lj");
    stat = hipMalloc((void**)&ad->e_el, sizeof(*ad->e_el));
    CU_RET_ERR(stat, "hipMalloc failed on ad->e_el");

    /* initialize to NULL poiters to data that is not allocated here and will
       need reallocation in nbnxn_cuda_init_atomdata */
    ad->xq = NULL;
    ad->f  = NULL;

    /* size -1 indicates that the respective array hasn't been initialized yet */
    ad->natoms = -1;
    ad->nalloc = -1;
}

/*! Selects the Ewald kernel type, analytical on SM 3.0 and later, tabulated on
    earlier GPUs, single or twin cut-off. */
static int pick_ewald_kernel_type(bool                     bTwinCut,
                                  const gmx_device_info_t *dev_info)
{
    bool bUseAnalyticalEwald, bForceAnalyticalEwald, bForceTabulatedEwald;
    int  kernel_type;

    /* Benchmarking/development environment variables to force the use of
       analytical or tabulated Ewald kernel. */
    bForceAnalyticalEwald = (getenv("GMX_CUDA_NB_ANA_EWALD") != NULL);
    bForceTabulatedEwald  = (getenv("GMX_CUDA_NB_TAB_EWALD") != NULL);

    if (bForceAnalyticalEwald && bForceTabulatedEwald)
    {
        gmx_incons("Both analytical and tabulated Ewald CUDA non-bonded kernels "
                   "requested through environment variables.");
    }

    /* By default, on SM 3.0 and later use analytical Ewald, on earlier tabulated. */
    if ((dev_info->prop.major >= 3 || bForceAnalyticalEwald) && !bForceTabulatedEwald)
    {
        bUseAnalyticalEwald = true;

        if (debug)
        {
            fprintf(debug, "Using analytical Ewald CUDA kernels\n");
        }
    }
    else
    {
        bUseAnalyticalEwald = false;

        if (debug)
        {
            fprintf(debug, "Using tabulated Ewald CUDA kernels\n");
        }
    }

    /* Use twin cut-off kernels if requested by bTwinCut or the env. var.
       forces it (use it for debugging/benchmarking only). */
    if (!bTwinCut && (getenv("GMX_CUDA_NB_EWALD_TWINCUT") == NULL))
    {
        kernel_type = bUseAnalyticalEwald ? eelCuEWALD_ANA : eelCuEWALD_TAB;
    }
    else
    {
        kernel_type = bUseAnalyticalEwald ? eelCuEWALD_ANA_TWIN : eelCuEWALD_TAB_TWIN;
    }

    return kernel_type;
}

/*! Copies all parameters related to the cut-off from ic to nbp */
static void set_cutoff_parameters(cu_nbparam_t              *nbp,
                                  const interaction_const_t *ic)
{
    nbp->ewald_beta       = ic->ewaldcoeff_q;
    nbp->sh_ewald         = ic->sh_ewald;
    nbp->epsfac           = ic->epsfac;
    nbp->two_k_rf         = 2.0 * ic->k_rf;
    nbp->c_rf             = ic->c_rf;
    nbp->rvdw_sq          = ic->rvdw * ic->rvdw;
    nbp->rcoulomb_sq      = ic->rcoulomb * ic->rcoulomb;
    nbp->rlist_sq         = ic->rlist * ic->rlist;

    nbp->sh_lj_ewald      = ic->sh_lj_ewald;
    nbp->ewaldcoeff_lj    = ic->ewaldcoeff_lj;

    nbp->rvdw_switch      = ic->rvdw_switch;
    nbp->dispersion_shift = ic->dispersion_shift;
    nbp->repulsion_shift  = ic->repulsion_shift;
    nbp->vdw_switch       = ic->vdw_switch;
}

/*! Initializes the nonbonded parameter data structure. */
static void init_nbparam(cu_nbparam_t              *nbp,
                         const interaction_const_t *ic,
                         const nbnxn_atomdata_t    *nbat,
                         const gmx_device_info_t   *dev_info)
{
    hipError_t stat;
    int         ntypes, nnbfp, nnbfp_comb;

    ntypes  = nbat->ntype;
    set_cutoff_parameters(nbp, ic);

    if (ic->vdwtype == evdwCUT)
    {
        switch (ic->vdw_modifier)
        {
            case eintmodNONE:
            case eintmodPOTSHIFT:
                nbp->vdwtype = evdwCuCUT;
                break;
            case eintmodFORCESWITCH:
                nbp->vdwtype = evdwCuFSWITCH;
                break;
            case eintmodPOTSWITCH:
                nbp->vdwtype = evdwCuPSWITCH;
                break;
            default:
                gmx_incons("The requested VdW interaction modifier is not implemented in the CUDA GPU accelerated kernels!");
                break;
        }
    }
    if (ic->vdwtype == evdwPME)
    {
        if (ic->ljpme_comb_rule == ljcrGEOM)
        {
            assert(nbat->comb_rule == ljcrGEOM);
            nbp->vdwtype = evdwCuEWALDGEOM;
        }
        else
        {
            assert(nbat->comb_rule == ljcrLB);
            nbp->vdwtype = evdwCuEWALDLB;
        }
    }
    if (ic->vdwtype == evdwUSER)
    {
        nbp->vdwtype = evdwCuUSER;
        if (debug)
        {
            fprintf(debug, "evdwCuUSER chosen\n");
        }
    }
    if (ic->vdwtype == evdwGENERIC)
    {
        nbp->vdwtype = evdwCuGENERIC;
        if (debug)
        {
            fprintf(debug, "evdwCuGENERIC chosen\n");
        }
    }

    if (ic->vdwtype > 6 || ic->vdwtype < 0)
    {
        printf ("ic->vdwtype = %d\n", ic->vdwtype);
        gmx_incons("The requested VdW type is not implemented in the CUDA GPU accelerated kernels!");
    }

    switch (ic->eeltype)
    {
        case eelCUT:
            nbp->eeltype = eelCuCUT;
            break;

        case eelRF:
        case eelGRF:
        case eelRF_NEC:
        case eelRF_ZERO:
            nbp->eeltype = eelCuRF;
            break;

        case eelPME:
        case eelPMESWITCH:
        case eelPMEUSER:
        case eelPMEUSERSWITCH:
        case eelP3M_AD:
        case eelEWALD:
            nbp->eeltype = pick_ewald_kernel_type(false, dev_info);
            break;

        case eelNONE:
            if (debug)
            {
                fprintf(debug, "eeltype = eelCuNONE selected!\n");
            }
            nbp->eeltype = eelCuNONE;
            break;

        case eelUSER:
            if (debug)
            {
                fprintf(debug, "eeltype = eelCuUSER selected!\n");
            }
            nbp->eeltype = eelCuUSER;
            break;

        default:
            /* Shouldn't happen, as this is checked when choosing Verlet-scheme */
            gmx_incons("The requested electrostatics type is not implemented in the CUDA GPU accelerated kernels!");
    }



    if (nbp->eeltype == eelCuUSER)
    {
        if (debug)
        {
            fprintf (debug, "nbnxn_cuda_data_mgmt.cu: Selected Coulomb user tables\n");
        }
        nbp->nb_coul_Ftab = NULL;
        nbp->nb_coul_Vtab = NULL;
        init_nb_coul_Ftables(ic, nbp, dev_info);
        init_nb_coul_Vtables(ic, nbp, dev_info);
    }

    if (nbp->vdwtype == evdwCuUSER)
    {
        if (debug)
        {
            fprintf (debug, "nbnxn_cuda_data_mgmt.cu: Selected VDW user tables\n");
        }

        nbp->nb_vdw_LJ6_Ftab  = NULL;
        nbp->nb_vdw_LJ6_Vtab  = NULL;
        nbp->nb_vdw_LJ12_Ftab = NULL;
        nbp->nb_vdw_LJ12_Vtab = NULL;

        init_nb_vdw_LJ6_Ftables(ic, nbp, dev_info);
        init_nb_vdw_LJ6_Vtables(ic, nbp, dev_info);
        init_nb_vdw_LJ12_Ftables(ic, nbp, dev_info);
        init_nb_vdw_LJ12_Vtables(ic, nbp, dev_info);
    }

    if (nbp->vdwtype == evdwCuGENERIC)
    {
        if (debug)
        {
            fprintf (debug, "nbnxn_cuda_data_mgmt.cu: Selected vdw USER tables\n");
        }
        /* generate table for VdwLJ */
        nbp->nb_generic_Ftab = NULL;
        nbp->nb_generic_Vtab = NULL;

        init_nb_generic_Ftables(ic, nbp, dev_info);
        init_nb_generic_Vtables(ic, nbp, dev_info);
    }

    /* generate table for PME */
    nbp->coulomb_tab = NULL;
    if (nbp->eeltype == eelCuEWALD_TAB || nbp->eeltype == eelCuEWALD_TAB_TWIN)
    {
        init_ewald_coulomb_force_table(ic, nbp, dev_info);
    }

    nnbfp      = 2*ntypes*ntypes;
    nnbfp_comb = 2*ntypes;

    stat  = hipMalloc((void **)&nbp->nbfp, nnbfp*sizeof(*nbp->nbfp));
    CU_RET_ERR(stat, "hipMalloc failed on nbp->nbfp");
    cu_copy_H2D(nbp->nbfp, nbat->nbfp, nnbfp*sizeof(*nbp->nbfp));


    if (ic->vdwtype == evdwPME)
    {
        stat  = hipMalloc((void **)&nbp->nbfp_comb, nnbfp_comb*sizeof(*nbp->nbfp_comb));
        CU_RET_ERR(stat, "hipMalloc failed on nbp->nbfp_comb");
        cu_copy_H2D(nbp->nbfp_comb, nbat->nbfp_comb, nnbfp_comb*sizeof(*nbp->nbfp_comb));
    }

    /* Only device CC >= 3.0 (Kepler and later) support texture objects */
    if (use_texobj(dev_info))
    {
        hipResourceDesc rd;
        hipTextureDesc  td;

        memset(&rd, 0, sizeof(rd));
        rd.resType                  = hipResourceTypeLinear;
        rd.res.linear.devPtr        = nbp->nbfp;
        rd.res.linear.desc.f        = hipChannelFormatKindFloat;
        rd.res.linear.desc.x        = 32;
        rd.res.linear.sizeInBytes   = nnbfp*sizeof(*nbp->nbfp);

        memset(&td, 0, sizeof(td));
        td.readMode                 = hipReadModeElementType;
        stat = hipCreateTextureObject(&nbp->nbfp_texobj, &rd, &td, NULL);
        CU_RET_ERR(stat, "hipCreateTextureObject on nbfp_texobj failed");

        if (ic->vdwtype == evdwPME)
        {
            memset(&rd, 0, sizeof(rd));
            rd.resType                  = hipResourceTypeLinear;
            rd.res.linear.devPtr        = nbp->nbfp_comb;
            rd.res.linear.desc.f        = hipChannelFormatKindFloat;
            rd.res.linear.desc.x        = 32;
            rd.res.linear.sizeInBytes   = nnbfp_comb*sizeof(*nbp->nbfp_comb);

            memset(&td, 0, sizeof(td));
            td.readMode = hipReadModeElementType;
            stat        = hipCreateTextureObject(&nbp->nbfp_comb_texobj, &rd, &td, NULL);
            CU_RET_ERR(stat, "hipCreateTextureObject on nbfp_comb_texobj failed");
        }
    }
    else
    {
        hipChannelFormatDesc cd = hipCreateChannelDesc<float>();
        stat = hipBindTexture(NULL, &nbnxn_cuda_get_nbfp_texref(),
                               nbp->nbfp, &cd, nnbfp*sizeof(*nbp->nbfp));
        CU_RET_ERR(stat, "hipBindTexture on nbfp_texref failed");

        if (ic->vdwtype == evdwPME)
        {
            stat = hipBindTexture(NULL, &nbnxn_cuda_get_nbfp_comb_texref(),
                                   nbp->nbfp_comb, &cd, nnbfp_comb*sizeof(*nbp->nbfp_comb));
            CU_RET_ERR(stat, "hipBindTexture on nbfp_comb_texref failed");
        }
    }
}

/*! Re-generate the GPU Ewald force table, resets rlist, and update the
 *  electrostatic type switching to twin cut-off (or back) if needed. */
void nbnxn_gpu_pme_loadbal_update_param(const nonbonded_verlet_t    *nbv,
                                        const interaction_const_t   *ic)
{
    if (!nbv || nbv->grp[0].kernel_type != nbnxnk8x8x8_GPU)
    {
        return;
    }
    gmx_nbnxn_cuda_t *nb    = nbv->gpu_nbv;
    cu_nbparam_t     *nbp   = nb->nbparam;

    set_cutoff_parameters(nbp, ic);

    nbp->eeltype        = pick_ewald_kernel_type(ic->rcoulomb != ic->rvdw,
                                                 nb->dev_info);

    init_ewald_coulomb_force_table(ic, nb->nbparam, nb->dev_info);
}

/*! Initializes the pair list data structure. */
static void init_plist(cu_plist_t *pl)
{
    /* initialize to NULL pointers to data that is not allocated here and will
       need reallocation in nbnxn_gpu_init_pairlist */
    pl->sci     = NULL;
    pl->cj4     = NULL;
    pl->excl    = NULL;

    /* size -1 indicates that the respective array hasn't been initialized yet */
    pl->na_c        = -1;
    pl->nsci        = -1;
    pl->sci_nalloc  = -1;
    pl->ncj4        = -1;
    pl->cj4_nalloc  = -1;
    pl->nexcl       = -1;
    pl->excl_nalloc = -1;
    pl->bDoPrune    = false;
}

/*! Initializes the timer data structure. */
static void init_timers(cu_timers_t *t, bool bUseTwoStreams)
{
    hipError_t stat;
    int         eventflags = ( bUseCudaEventBlockingSync ? hipEventBlockingSync : hipEventDefault );

    stat = hipEventCreateWithFlags(&(t->start_atdat), eventflags);
    CU_RET_ERR(stat, "hipEventCreate on start_atdat failed");
    stat = hipEventCreateWithFlags(&(t->stop_atdat), eventflags);
    CU_RET_ERR(stat, "hipEventCreate on stop_atdat failed");

    /* The non-local counters/stream (second in the array) are needed only with DD. */
    for (int i = 0; i <= (bUseTwoStreams ? 1 : 0); i++)
    {
        stat = hipEventCreateWithFlags(&(t->start_nb_k[i]), eventflags);
        CU_RET_ERR(stat, "hipEventCreate on start_nb_k failed");
        stat = hipEventCreateWithFlags(&(t->stop_nb_k[i]), eventflags);
        CU_RET_ERR(stat, "hipEventCreate on stop_nb_k failed");


        stat = hipEventCreateWithFlags(&(t->start_pl_h2d[i]), eventflags);
        CU_RET_ERR(stat, "hipEventCreate on start_pl_h2d failed");
        stat = hipEventCreateWithFlags(&(t->stop_pl_h2d[i]), eventflags);
        CU_RET_ERR(stat, "hipEventCreate on stop_pl_h2d failed");

        stat = hipEventCreateWithFlags(&(t->start_nb_h2d[i]), eventflags);
        CU_RET_ERR(stat, "hipEventCreate on start_nb_h2d failed");
        stat = hipEventCreateWithFlags(&(t->stop_nb_h2d[i]), eventflags);
        CU_RET_ERR(stat, "hipEventCreate on stop_nb_h2d failed");

        stat = hipEventCreateWithFlags(&(t->start_nb_d2h[i]), eventflags);
        CU_RET_ERR(stat, "hipEventCreate on start_nb_d2h failed");
        stat = hipEventCreateWithFlags(&(t->stop_nb_d2h[i]), eventflags);
        CU_RET_ERR(stat, "hipEventCreate on stop_nb_d2h failed");
    }
}

/*! Initializes the timings data structure. */
static void init_timings(gmx_wallclock_gpu_t *t)
{
    int i, j;

    t->nb_h2d_t = 0.0;
    t->nb_d2h_t = 0.0;
    t->nb_c     = 0;
    t->pl_h2d_t = 0.0;
    t->pl_h2d_c = 0;
    for (i = 0; i < 2; i++)
    {
        for (j = 0; j < 2; j++)
        {
            t->ktime[i][j].t = 0.0;
            t->ktime[i][j].c = 0;
        }
    }
}

/*! Initializes simulation constant data. */
static void nbnxn_cuda_init_const(gmx_nbnxn_cuda_t               *nb,
                                  const interaction_const_t      *ic,
                                  const nonbonded_verlet_group_t *nbv_group)
{
    init_atomdata_first(nb->atdat, nbv_group[0].nbat->ntype);
    init_nbparam(nb->nbparam, ic, nbv_group[0].nbat, nb->dev_info);

    /* clear energy and shift force outputs */
    nbnxn_cuda_clear_e_fshift(nb);
}

void nbnxn_gpu_init(gmx_nbnxn_cuda_t         **p_nb,
                    const gmx_gpu_info_t      *gpu_info,
                    const gmx_gpu_opt_t       *gpu_opt,
                    const interaction_const_t *ic,
                    nonbonded_verlet_group_t  *nbv_grp,
                    int                        my_gpu_index,
                    int /*rank*/,
                    gmx_bool                   bLocalAndNonlocal)
{
    hipError_t       stat;
    gmx_nbnxn_cuda_t *nb;

    assert(gpu_info);

    if (p_nb == NULL)
    {
        return;
    }

    snew(nb, 1);
    snew(nb->atdat, 1);
    snew(nb->nbparam, 1);
    snew(nb->plist[eintLocal], 1);
    if (bLocalAndNonlocal)
    {
        snew(nb->plist[eintNonlocal], 1);
    }

    nb->bUseTwoStreams = bLocalAndNonlocal;

    snew(nb->timers, 1);
    snew(nb->timings, 1);

    /* init nbst */
    pmalloc((void**)&nb->nbst.e_lj, sizeof(*nb->nbst.e_lj));
    pmalloc((void**)&nb->nbst.e_el, sizeof(*nb->nbst.e_el));
    pmalloc((void**)&nb->nbst.fshift, SHIFTS * sizeof(*nb->nbst.fshift));

    init_plist(nb->plist[eintLocal]);

    /* set device info, just point it to the right GPU among the detected ones */
    nb->dev_info = &gpu_info->gpu_dev[get_gpu_device_id(gpu_info, gpu_opt, my_gpu_index)];

    /* local/non-local GPU streams */
    stat = hipStreamCreate(&nb->stream[eintLocal]);
    CU_RET_ERR(stat, "hipStreamCreate on stream[eintLocal] failed");
    if (nb->bUseTwoStreams)
    {
        init_plist(nb->plist[eintNonlocal]);

        /* CUDA stream priority available in the CUDA RT 5.5 API.
         * Note that the device we're running on does not have to support
         * priorities, because we are querying the priority range which in this
         * case will be a single value.
         */
#if GMX_CUDA_VERSION >= 5050
        {
            int highest_priority;
            stat = hipDeviceGetStreamPriorityRange(NULL, &highest_priority);
            CU_RET_ERR(stat, "hipDeviceGetStreamPriorityRange failed");

            stat = hipStreamCreateWithPriority(&nb->stream[eintNonlocal],
                                                hipStreamDefault,
                                                highest_priority);
            CU_RET_ERR(stat, "hipStreamCreateWithPriority on stream[eintNonlocal] failed");
        }
#else
        stat = hipStreamCreate(&nb->stream[eintNonlocal]);
        CU_RET_ERR(stat, "hipStreamCreate on stream[eintNonlocal] failed");
#endif
    }

    /* init events for sychronization (timing disabled for performance reasons!) */
    stat = hipEventCreateWithFlags(&nb->nonlocal_done, hipEventDisableTiming);
    CU_RET_ERR(stat, "hipEventCreate on nonlocal_done failed");
    stat = hipEventCreateWithFlags(&nb->misc_ops_and_local_H2D_done, hipEventDisableTiming);
    CU_RET_ERR(stat, "hipEventCreate on misc_ops_and_local_H2D_done failed");

    /* CUDA timing disabled as event timers don't work:
       - with multiple streams = domain-decomposition;
       - when turned off by GMX_DISABLE_CUDA_TIMING.
     */
    nb->bDoTime = (!nb->bUseTwoStreams &&
                   (getenv("GMX_DISABLE_CUDA_TIMING") == NULL));

    if (nb->bDoTime)
    {
        init_timers(nb->timers, nb->bUseTwoStreams);
        init_timings(nb->timings);
    }

    /* set the kernel type for the current GPU */
    /* pick L1 cache configuration */
    nbnxn_cuda_set_cacheconfig(nb->dev_info);

    nbnxn_cuda_init_const(nb, ic, nbv_grp);

    *p_nb = nb;

    if (debug)
    {
        fprintf(debug, "Initialized CUDA data structures.\n");
    }
}

void nbnxn_gpu_init_pairlist(gmx_nbnxn_cuda_t       *nb,
                             const nbnxn_pairlist_t *h_plist,
                             int                     iloc)
{
    char          sbuf[STRLEN];
    hipError_t   stat;
    bool          bDoTime    = nb->bDoTime;
    hipStream_t  stream     = nb->stream[iloc];
    cu_plist_t   *d_plist    = nb->plist[iloc];

    if (d_plist->na_c < 0)
    {
        d_plist->na_c = h_plist->na_ci;
    }
    else
    {
        if (d_plist->na_c != h_plist->na_ci)
        {
            sprintf(sbuf, "In cu_init_plist: the #atoms per cell has changed (from %d to %d)",
                    d_plist->na_c, h_plist->na_ci);
            gmx_incons(sbuf);
        }
    }

    if (bDoTime)
    {
        stat = hipEventRecord(nb->timers->start_pl_h2d[iloc], stream);
        CU_RET_ERR(stat, "hipEventRecord failed");
    }

    cu_realloc_buffered((void **)&d_plist->sci, h_plist->sci, sizeof(*d_plist->sci),
                        &d_plist->nsci, &d_plist->sci_nalloc,
                        h_plist->nsci,
                        stream, true);

    cu_realloc_buffered((void **)&d_plist->cj4, h_plist->cj4, sizeof(*d_plist->cj4),
                        &d_plist->ncj4, &d_plist->cj4_nalloc,
                        h_plist->ncj4,
                        stream, true);

    cu_realloc_buffered((void **)&d_plist->excl, h_plist->excl, sizeof(*d_plist->excl),
                        &d_plist->nexcl, &d_plist->excl_nalloc,
                        h_plist->nexcl,
                        stream, true);

    if (bDoTime)
    {
        stat = hipEventRecord(nb->timers->stop_pl_h2d[iloc], stream);
        CU_RET_ERR(stat, "hipEventRecord failed");
    }

    /* need to prune the pair list during the next step */
    d_plist->bDoPrune = true;
}

void nbnxn_gpu_upload_shiftvec(gmx_nbnxn_cuda_t       *nb,
                               const nbnxn_atomdata_t *nbatom)
{
    cu_atomdata_t *adat  = nb->atdat;
    hipStream_t   ls    = nb->stream[eintLocal];

    /* only if we have a dynamic box */
    if (nbatom->bDynamicBox || !adat->bShiftVecUploaded)
    {
        cu_copy_H2D_async(adat->shift_vec, nbatom->shift_vec,
                          SHIFTS * sizeof(*adat->shift_vec), ls);
        adat->bShiftVecUploaded = true;
    }
}

/*! Clears the first natoms_clear elements of the GPU nonbonded force output array. */
static void nbnxn_cuda_clear_f(gmx_nbnxn_cuda_t *nb, int natoms_clear)
{
    hipError_t    stat;
    cu_atomdata_t *adat  = nb->atdat;
    hipStream_t   ls    = nb->stream[eintLocal];

    stat = hipMemsetAsync(adat->f, 0, natoms_clear * sizeof(*adat->f), ls);
    CU_RET_ERR(stat, "hipMemsetAsync on f falied");
}

/*! Clears nonbonded shift force output array and energy outputs on the GPU. */
static void nbnxn_cuda_clear_e_fshift(gmx_nbnxn_cuda_t *nb)
{
    hipError_t    stat;
    cu_atomdata_t *adat  = nb->atdat;
    hipStream_t   ls    = nb->stream[eintLocal];

    stat = hipMemsetAsync(adat->fshift, 0, SHIFTS * sizeof(*adat->fshift), ls);
    CU_RET_ERR(stat, "hipMemsetAsync on fshift falied");
    stat = hipMemsetAsync(adat->e_lj, 0, sizeof(*adat->e_lj), ls);
    CU_RET_ERR(stat, "hipMemsetAsync on e_lj falied");
    stat = hipMemsetAsync(adat->e_el, 0, sizeof(*adat->e_el), ls);
    CU_RET_ERR(stat, "hipMemsetAsync on e_el falied");
}

void nbnxn_gpu_clear_outputs(gmx_nbnxn_cuda_t *nb, int flags)
{
    nbnxn_cuda_clear_f(nb, nb->atdat->natoms);
    /* clear shift force array and energies if the outputs were
       used in the current step */
    if (flags & GMX_FORCE_VIRIAL)
    {
        nbnxn_cuda_clear_e_fshift(nb);
    }
}

void nbnxn_gpu_init_atomdata(gmx_nbnxn_cuda_t              *nb,
                             const struct nbnxn_atomdata_t *nbat)
{
    hipError_t    stat;
    int            nalloc, natoms;
    bool           realloced;
    bool           bDoTime   = nb->bDoTime;
    cu_timers_t   *timers    = nb->timers;
    cu_atomdata_t *d_atdat   = nb->atdat;
    hipStream_t   ls        = nb->stream[eintLocal];

    natoms    = nbat->natoms;
    realloced = false;

    if (bDoTime)
    {
        /* time async copy */
        stat = hipEventRecord(timers->start_atdat, ls);
        CU_RET_ERR(stat, "hipEventRecord failed");
    }

    /* need to reallocate if we have to copy more atoms than the amount of space
       available and only allocate if we haven't initialized yet, i.e d_atdat->natoms == -1 */
    if (natoms > d_atdat->nalloc)
    {
        nalloc = over_alloc_small(natoms);

        /* free up first if the arrays have already been initialized */
        if (d_atdat->nalloc != -1)
        {
            cu_free_buffered(d_atdat->f, &d_atdat->natoms, &d_atdat->nalloc);
            cu_free_buffered(d_atdat->xq);
            cu_free_buffered(d_atdat->atom_types);
        }

        stat = hipMalloc((void **)&d_atdat->f, nalloc*sizeof(*d_atdat->f));
        CU_RET_ERR(stat, "hipMalloc failed on d_atdat->f");
        stat = hipMalloc((void **)&d_atdat->xq, nalloc*sizeof(*d_atdat->xq));
        CU_RET_ERR(stat, "hipMalloc failed on d_atdat->xq");

        stat = hipMalloc((void **)&d_atdat->atom_types, nalloc*sizeof(*d_atdat->atom_types));
        CU_RET_ERR(stat, "hipMalloc failed on d_atdat->atom_types");

        d_atdat->nalloc = nalloc;
        realloced       = true;
    }

    d_atdat->natoms       = natoms;
    d_atdat->natoms_local = nbat->natoms_local;

    /* need to clear GPU f output if realloc happened */
    if (realloced)
    {
        nbnxn_cuda_clear_f(nb, nalloc);
    }

    cu_copy_H2D_async(d_atdat->atom_types, nbat->type,
                      natoms*sizeof(*d_atdat->atom_types), ls);

    if (bDoTime)
    {
        stat = hipEventRecord(timers->stop_atdat, ls);
        CU_RET_ERR(stat, "hipEventRecord failed");
    }
}

static void nbnxn_cuda_free_nbparam_table(cu_nbparam_t            *nbparam,
                                          const gmx_device_info_t *dev_info)
{
    if (debug)
    {
        fprintf(debug, "Called nbnxn_cuda_free_nbparam_table\n");
    }
    hipError_t stat;

    if (nbparam->eeltype == eelCuEWALD_TAB ||
        nbparam->eeltype == eelCuEWALD_TAB_TWIN
        )
    {
        /* Only device CC >= 3.0 (Kepler and later) support texture objects */
        if (use_texobj(dev_info))
        {
            stat = hipDestroyTextureObject(nbparam->coulomb_tab_texobj);
            CU_RET_ERR(stat, "hipDestroyTextureObject on coulomb_tab_texobj failed");
        }
        else
        {
            GMX_UNUSED_VALUE(dev_info);
            stat = hipUnbindTexture(nbnxn_cuda_get_coulomb_tab_texref());
            CU_RET_ERR(stat, "hipUnbindTexture on coulomb_tab_texref failed");
        }
        cu_free_buffered(nbparam->coulomb_tab, &nbparam->coulomb_tab_size);
    }

    if (nbparam->eeltype == eelCuUSER)
    {
        /* Only device CC >= 3.0 (Kepler and later) support texture objects */
        if (use_texobj(dev_info))
        {
            stat = hipDestroyTextureObject(nbparam->nb_coul_Ftab_texobj);
            CU_RET_ERR(stat, "hipDestroyTextureObject on nb_coul_Ftab_texobj failed");

            stat = hipDestroyTextureObject(nbparam->nb_coul_Vtab_texobj);
            CU_RET_ERR(stat, "hipDestroyTextureObject on nb_coul_Vtab_texobj failed");
        }
        else
        {
            GMX_UNUSED_VALUE(dev_info);
            stat = hipUnbindTexture(nbnxn_cuda_get_nb_coul_Ftab_texref());
            CU_RET_ERR(stat, "hipUnbindTexture on nb_coul_Ftab_texref failed");

            stat = hipUnbindTexture(nbnxn_cuda_get_nb_coul_Vtab_texref());
            CU_RET_ERR(stat, "hipUnbindTexture on nb_coul_Vtab_texref failed");
        }
        cu_free_buffered(nbparam->nb_coul_Ftab, &nbparam->nb_coul_tab_size);
        cu_free_buffered(nbparam->nb_coul_Vtab, &nbparam->nb_coul_tab_size);
    }


    if (nbparam->vdwtype == evdwCuUSER)
    {
        /* Only device CC >= 3.0 (Kepler and later) support texture objects */
        if (use_texobj(dev_info))
        {
            stat = hipDestroyTextureObject(nbparam->nb_vdw_LJ6_Ftab_texobj);
            CU_RET_ERR(stat, "hipDestroyTextureObject on nb_vdw_LJ6_Ftab_texobj failed");

            stat = hipDestroyTextureObject(nbparam->nb_vdw_LJ6_Vtab_texobj);
            CU_RET_ERR(stat, "hipDestroyTextureObject on nb_vdw_LJ6_Vtab_texobj failed");

            stat = hipDestroyTextureObject(nbparam->nb_vdw_LJ12_Ftab_texobj);
            CU_RET_ERR(stat, "hipDestroyTextureObject on nb_vdw_LJ12_Ftab_texobj failed");

            stat = hipDestroyTextureObject(nbparam->nb_vdw_LJ12_Vtab_texobj);
            CU_RET_ERR(stat, "hipDestroyTextureObject on nb_vdw_LJ12_Vtab_texobj failed");
        }
        else
        {
            GMX_UNUSED_VALUE(dev_info);
            stat = hipUnbindTexture(nbnxn_cuda_get_nb_vdw_LJ6_Ftab_texref());
            CU_RET_ERR(stat, "hipUnbindTexture on nb_vdw_LJ6_Ftab_texref failed");

            stat = hipUnbindTexture(nbnxn_cuda_get_nb_vdw_LJ6_Vtab_texref());
            CU_RET_ERR(stat, "hipUnbindTexture on nb_vdw_LJ6_Vtab_texref failed");

            stat = hipUnbindTexture(nbnxn_cuda_get_nb_vdw_LJ12_Ftab_texref());
            CU_RET_ERR(stat, "hipUnbindTexture on nb_vdw_LJ12_Ftab_texref failed");

            stat = hipUnbindTexture(nbnxn_cuda_get_nb_vdw_LJ12_Vtab_texref());
            CU_RET_ERR(stat, "hipUnbindTexture on nb_vdw_LJ12_Vtab_texref failed");
        }
        cu_free_buffered(nbparam->nb_vdw_LJ6_Ftab, &nbparam->nb_vdw_tab_size);
        cu_free_buffered(nbparam->nb_vdw_LJ6_Vtab, &nbparam->nb_vdw_tab_size);
        cu_free_buffered(nbparam->nb_vdw_LJ12_Ftab, &nbparam->nb_vdw_tab_size);
        cu_free_buffered(nbparam->nb_vdw_LJ12_Vtab, &nbparam->nb_vdw_tab_size);
    }

    if (nbparam->vdwtype == evdwCuGENERIC)
    {
        /* Only device CC >= 3.0 (Kepler and later) support texture objects */
        if (use_texobj(dev_info))
        {
            stat = hipDestroyTextureObject(nbparam->nb_generic_Ftab_texobj);
            CU_RET_ERR(stat, "hipDestroyTextureObject on nb_generic_Ftab_texobj failed");

            stat = hipDestroyTextureObject(nbparam->nb_generic_Vtab_texobj);
            CU_RET_ERR(stat, "hipDestroyTextureObject on nb_generic_Vtab_texobj failed");
        }
        else
        {
            GMX_UNUSED_VALUE(dev_info);
            stat = hipUnbindTexture(nbnxn_cuda_get_nb_generic_Ftab_texref());
            CU_RET_ERR(stat, "hipUnbindTexture on nb_generic_Ftab_texref failed");

            stat = hipUnbindTexture(nbnxn_cuda_get_nb_generic_Vtab_texref());
            CU_RET_ERR(stat, "hipUnbindTexture on nb_generic_Vtab_texref failed");
        }
        cu_free_buffered(nbparam->nb_generic_Ftab, &nbparam->nb_generic_tab_size);
        cu_free_buffered(nbparam->nb_generic_Vtab, &nbparam->nb_generic_tab_size);
    }

}

void nbnxn_gpu_free(gmx_nbnxn_cuda_t *nb)
{
    hipError_t      stat;
    cu_atomdata_t   *atdat;
    cu_nbparam_t    *nbparam;
    cu_plist_t      *plist, *plist_nl;
    cu_timers_t     *timers;

    /* Stopping the nvidia profiler here allows us to eliminate the subsequent
       uninitialization API calls from the trace. */
    if (getenv("NVPROF_ID") != NULL)
    {
        stat = hipProfilerStop();
        CU_RET_ERR(stat, "hipProfilerStop failed");
    }

    if (nb == NULL)
    {
        return;
    }

    atdat       = nb->atdat;
    nbparam     = nb->nbparam;
    plist       = nb->plist[eintLocal];
    plist_nl    = nb->plist[eintNonlocal];
    timers      = nb->timers;

    nbnxn_cuda_free_nbparam_table(nbparam, nb->dev_info);

    stat = hipEventDestroy(nb->nonlocal_done);
    CU_RET_ERR(stat, "hipEventDestroy failed on timers->nonlocal_done");
    stat = hipEventDestroy(nb->misc_ops_and_local_H2D_done);
    CU_RET_ERR(stat, "hipEventDestroy failed on timers->misc_ops_and_local_H2D_done");

    if (nb->bDoTime)
    {
        stat = hipEventDestroy(timers->start_atdat);
        CU_RET_ERR(stat, "hipEventDestroy failed on timers->start_atdat");
        stat = hipEventDestroy(timers->stop_atdat);
        CU_RET_ERR(stat, "hipEventDestroy failed on timers->stop_atdat");

        /* The non-local counters/stream (second in the array) are needed only with DD. */
        for (int i = 0; i <= (nb->bUseTwoStreams ? 1 : 0); i++)
        {
            stat = hipEventDestroy(timers->start_nb_k[i]);
            CU_RET_ERR(stat, "hipEventDestroy failed on timers->start_nb_k");
            stat = hipEventDestroy(timers->stop_nb_k[i]);
            CU_RET_ERR(stat, "hipEventDestroy failed on timers->stop_nb_k");

            stat = hipEventDestroy(timers->start_pl_h2d[i]);
            CU_RET_ERR(stat, "hipEventDestroy failed on timers->start_pl_h2d");
            stat = hipEventDestroy(timers->stop_pl_h2d[i]);
            CU_RET_ERR(stat, "hipEventDestroy failed on timers->stop_pl_h2d");

            stat = hipStreamDestroy(nb->stream[i]);
            CU_RET_ERR(stat, "hipStreamDestroy failed on stream");

            stat = hipEventDestroy(timers->start_nb_h2d[i]);
            CU_RET_ERR(stat, "hipEventDestroy failed on timers->start_nb_h2d");
            stat = hipEventDestroy(timers->stop_nb_h2d[i]);
            CU_RET_ERR(stat, "hipEventDestroy failed on timers->stop_nb_h2d");

            stat = hipEventDestroy(timers->start_nb_d2h[i]);
            CU_RET_ERR(stat, "hipEventDestroy failed on timers->start_nb_d2h");
            stat = hipEventDestroy(timers->stop_nb_d2h[i]);
            CU_RET_ERR(stat, "hipEventDestroy failed on timers->stop_nb_d2h");
        }
    }

    /* Only device CC >= 3.0 (Kepler and later) support texture objects */
    if (use_texobj(nb->dev_info))
    {
        stat = hipDestroyTextureObject(nbparam->nbfp_texobj);
        CU_RET_ERR(stat, "hipDestroyTextureObject on nbfp_texobj failed");
    }
    else
    {
        stat = hipUnbindTexture(nbnxn_cuda_get_nbfp_texref());
        CU_RET_ERR(stat, "hipUnbindTexture on nbfp_texref failed");
    }
    cu_free_buffered(nbparam->nbfp);

    if (nbparam->vdwtype == evdwCuEWALDGEOM || nbparam->vdwtype == evdwCuEWALDLB)
    {
        /* Only device CC >= 3.0 (Kepler and later) support texture objects */
        if (use_texobj(nb->dev_info))
        {
            stat = hipDestroyTextureObject(nbparam->nbfp_comb_texobj);
            CU_RET_ERR(stat, "hipDestroyTextureObject on nbfp_comb_texobj failed");
        }
        else
        {
            stat = hipUnbindTexture(nbnxn_cuda_get_nbfp_comb_texref());
            CU_RET_ERR(stat, "hipUnbindTexture on nbfp_comb_texref failed");
        }
        cu_free_buffered(nbparam->nbfp_comb);
    }

    stat = hipFree(atdat->shift_vec);
    CU_RET_ERR(stat, "hipFree failed on atdat->shift_vec");
    stat = hipFree(atdat->fshift);
    CU_RET_ERR(stat, "hipFree failed on atdat->fshift");

    stat = hipFree(atdat->e_lj);
    CU_RET_ERR(stat, "hipFree failed on atdat->e_lj");
    stat = hipFree(atdat->e_el);
    CU_RET_ERR(stat, "hipFree failed on atdat->e_el");

    cu_free_buffered(atdat->f, &atdat->natoms, &atdat->nalloc);
    cu_free_buffered(atdat->xq);
    cu_free_buffered(atdat->atom_types, &atdat->ntypes);

    cu_free_buffered(plist->sci, &plist->nsci, &plist->sci_nalloc);
    cu_free_buffered(plist->cj4, &plist->ncj4, &plist->cj4_nalloc);
    cu_free_buffered(plist->excl, &plist->nexcl, &plist->excl_nalloc);
    if (nb->bUseTwoStreams)
    {
        cu_free_buffered(plist_nl->sci, &plist_nl->nsci, &plist_nl->sci_nalloc);
        cu_free_buffered(plist_nl->cj4, &plist_nl->ncj4, &plist_nl->cj4_nalloc);
        cu_free_buffered(plist_nl->excl, &plist_nl->nexcl, &plist->excl_nalloc);
    }

    sfree(atdat);
    sfree(nbparam);
    sfree(plist);
    if (nb->bUseTwoStreams)
    {
        sfree(plist_nl);
    }
    sfree(timers);
    sfree(nb->timings);
    sfree(nb);

    if (debug)
    {
        fprintf(debug, "Cleaned up CUDA data structures.\n");
    }
}

void cu_synchstream_atdat(gmx_nbnxn_cuda_t *nb, int iloc)
{
    hipError_t  stat;
    hipStream_t stream = nb->stream[iloc];

    stat = hipStreamWaitEvent(stream, nb->timers->stop_atdat, 0);
    CU_RET_ERR(stat, "hipStreamWaitEvent failed");
}

gmx_wallclock_gpu_t * nbnxn_gpu_get_timings(gmx_nbnxn_cuda_t *nb)
{
    return (nb != NULL && nb->bDoTime) ? nb->timings : NULL;
}

void nbnxn_gpu_reset_timings(nonbonded_verlet_t* nbv)
{
    /* The NVPROF_ID environment variable is set by nvprof and indicates that
       mdrun is executed in the CUDA profiler.
       If nvprof was run is with "--profile-from-start off", the profiler will
       be started here. This way we can avoid tracing the CUDA events from the
       first part of the run. Starting the profiler again does nothing.
     */
    if (getenv("NVPROF_ID") != NULL)
    {
        hipError_t stat;
        stat = hipProfilerStart();
        CU_RET_ERR(stat, "hipProfilerStart failed");
    }

    if (nbv->gpu_nbv && nbv->gpu_nbv->bDoTime)
    {
        init_timings(nbv->gpu_nbv->timings);
    }
}

int nbnxn_gpu_min_ci_balanced(gmx_nbnxn_cuda_t *nb)
{
    return nb != NULL ?
           gpu_min_ci_balanced_factor*nb->dev_info->prop.multiProcessorCount : 0;

}

gmx_bool nbnxn_gpu_is_kernel_ewald_analytical(const gmx_nbnxn_cuda_t *nb)
{
    return ((nb->nbparam->eeltype == eelCuEWALD_ANA) ||
            (nb->nbparam->eeltype == eelCuEWALD_ANA_TWIN));
}
