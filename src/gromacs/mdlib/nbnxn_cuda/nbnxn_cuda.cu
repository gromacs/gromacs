#include "hip/hip_runtime.h"
/*
 * This file is part of the GROMACS molecular simulation package.
 *
 * Copyright (c) 2012,2013,2014,2015,2016,2017,2018, by the GROMACS development team, led by
 * Mark Abraham, David van der Spoel, Berk Hess, and Erik Lindahl,
 * and including many others, as listed in the AUTHORS file in the
 * top-level source directory and at http://www.gromacs.org.
 *
 * GROMACS is free software; you can redistribute it and/or
 * modify it under the terms of the GNU Lesser General Public License
 * as published by the Free Software Foundation; either version 2.1
 * of the License, or (at your option) any later version.
 *
 * GROMACS is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
 * Lesser General Public License for more details.
 *
 * You should have received a copy of the GNU Lesser General Public
 * License along with GROMACS; if not, see
 * http://www.gnu.org/licenses, or write to the Free Software Foundation,
 * Inc., 51 Franklin Street, Fifth Floor, Boston, MA  02110-1301  USA.
 *
 * If you want to redistribute modifications to GROMACS, please
 * consider that scientific software is very special. Version
 * control is crucial - bugs must be traceable. We will be happy to
 * consider code for inclusion in the official distribution, but
 * derived work must not be called official GROMACS. Details are found
 * in the README & COPYING files - if they are missing, get the
 * official version at http://www.gromacs.org.
 *
 * To help us fund GROMACS development, we humbly ask that you cite
 * the research papers on the package. Check out http://www.gromacs.org.
 */
/*! \file
 *  \brief Define CUDA implementation of nbnxn_gpu.h
 *
 *  \author Szilard Pall <pall.szilard@gmail.com>
 */
#include "gmxpre.h"

#include "config.h"

#include <assert.h>
#include <stdlib.h>

#include "gromacs/mdlib/nbnxn_gpu.h"

#if defined(_MSVC)
#include <limits>
#endif


#include "nbnxn_cuda.h"

#include "gromacs/gpu_utils/cudautils.cuh"
#include "gromacs/mdlib/force_flags.h"
#include "gromacs/mdlib/nb_verlet.h"
#include "gromacs/mdlib/nbnxn_gpu_common.h"
#include "gromacs/mdlib/nbnxn_gpu_common_utils.h"
#include "gromacs/mdlib/nbnxn_gpu_data_mgmt.h"
#include "gromacs/mdlib/nbnxn_pairlist.h"
#include "gromacs/timing/gpu_timing.h"
#include "gromacs/utility/cstringutil.h"
#include "gromacs/utility/gmxassert.h"

#include "nbnxn_cuda_types.h"


/***** The kernel declarations/definitions come here *****/

/* Top-level kernel declaration generation: will generate through multiple
 * inclusion the following flavors for all kernel declarations:
 * - force-only output;
 * - force and energy output;
 * - force-only with pair list pruning;
 * - force and energy output with pair list pruning.
 */
#define FUNCTION_DECLARATION_ONLY
/** Force only **/
#include "gromacs/mdlib/nbnxn_cuda/nbnxn_cuda_kernels.cuh"
/** Force & energy **/
#define CALC_ENERGIES
#include "gromacs/mdlib/nbnxn_cuda/nbnxn_cuda_kernels.cuh"
#undef CALC_ENERGIES

/*** Pair-list pruning kernels ***/
/** Force only **/
#define PRUNE_NBL
#include "gromacs/mdlib/nbnxn_cuda/nbnxn_cuda_kernels.cuh"
/** Force & energy **/
#define CALC_ENERGIES
#include "gromacs/mdlib/nbnxn_cuda/nbnxn_cuda_kernels.cuh"
#undef CALC_ENERGIES
#undef PRUNE_NBL

/* Prune-only kernels */
#include "gromacs/mdlib/nbnxn_cuda/nbnxn_cuda_kernel_pruneonly.cuh"
#undef FUNCTION_DECLARATION_ONLY

/* Now generate the function definitions if we are using a single compilation unit. */
#if GMX_CUDA_NB_SINGLE_COMPILATION_UNIT
#include "gromacs/mdlib/nbnxn_cuda/nbnxn_cuda_kernel_F_noprune.cu"
#include "gromacs/mdlib/nbnxn_cuda/nbnxn_cuda_kernel_F_prune.cu"
#include "gromacs/mdlib/nbnxn_cuda/nbnxn_cuda_kernel_VF_noprune.cu"
#include "gromacs/mdlib/nbnxn_cuda/nbnxn_cuda_kernel_VF_prune.cu"
#include "gromacs/mdlib/nbnxn_cuda/nbnxn_cuda_kernel_pruneonly.cu"
#endif /* GMX_CUDA_NB_SINGLE_COMPILATION_UNIT */


/*! Nonbonded kernel function pointer type */
typedef void (*nbnxn_cu_kfunc_ptr_t)(const cu_atomdata_t,
                                     const cu_nbparam_t,
                                     const cu_plist_t,
                                     bool);

/*********************************/

/*! Returns the number of blocks to be used for the nonbonded GPU kernel. */
static inline int calc_nb_kernel_nblock(int nwork_units, const gmx_device_info_t *dinfo)
{
    int max_grid_x_size;

    assert(dinfo);
    /* CUDA does not accept grid dimension of 0 (which can happen e.g. with an
       empty domain) and that case should be handled before this point. */
    assert(nwork_units > 0);

    max_grid_x_size = dinfo->prop.maxGridSize[0];

    /* do we exceed the grid x dimension limit? */
    if (nwork_units > max_grid_x_size)
    {
        gmx_fatal(FARGS, "Watch out, the input system is too large to simulate!\n"
                  "The number of nonbonded work units (=number of super-clusters) exceeds the"
                  "maximum grid size in x dimension (%d > %d)!", nwork_units, max_grid_x_size);
    }

    return nwork_units;
}


/* Constant arrays listing all kernel function pointers and enabling selection
   of a kernel in an elegant manner. */

/*! Pointers to the non-bonded kernels organized in 2-dim arrays by:
 *  electrostatics and VDW type.
 *
 *  Note that the row- and column-order of function pointers has to match the
 *  order of corresponding enumerated electrostatics and vdw types, resp.,
 *  defined in nbnxn_cuda_types.h.
 */

/*! Force-only kernel function pointers. */
static const nbnxn_cu_kfunc_ptr_t nb_kfunc_noener_noprune_ptr[eelCuNR][evdwCuNR] =
{
    { nbnxn_kernel_ElecCut_VdwLJ_F_cuda,            nbnxn_kernel_ElecCut_VdwLJCombGeom_F_cuda,            nbnxn_kernel_ElecCut_VdwLJCombLB_F_cuda,            nbnxn_kernel_ElecCut_VdwLJFsw_F_cuda,            nbnxn_kernel_ElecCut_VdwLJPsw_F_cuda,            nbnxn_kernel_ElecCut_VdwLJEwCombGeom_F_cuda,            nbnxn_kernel_ElecCut_VdwLJEwCombLB_F_cuda            },
    { nbnxn_kernel_ElecRF_VdwLJ_F_cuda,             nbnxn_kernel_ElecRF_VdwLJCombGeom_F_cuda,             nbnxn_kernel_ElecRF_VdwLJCombLB_F_cuda,             nbnxn_kernel_ElecRF_VdwLJFsw_F_cuda,             nbnxn_kernel_ElecRF_VdwLJPsw_F_cuda,             nbnxn_kernel_ElecRF_VdwLJEwCombGeom_F_cuda,             nbnxn_kernel_ElecRF_VdwLJEwCombLB_F_cuda             },
    { nbnxn_kernel_ElecEwQSTab_VdwLJ_F_cuda,        nbnxn_kernel_ElecEwQSTab_VdwLJCombGeom_F_cuda,        nbnxn_kernel_ElecEwQSTab_VdwLJCombLB_F_cuda,        nbnxn_kernel_ElecEwQSTab_VdwLJFsw_F_cuda,        nbnxn_kernel_ElecEwQSTab_VdwLJPsw_F_cuda,        nbnxn_kernel_ElecEwQSTab_VdwLJEwCombGeom_F_cuda,        nbnxn_kernel_ElecEwQSTab_VdwLJEwCombLB_F_cuda        },
    { nbnxn_kernel_ElecEwQSTabTwinCut_VdwLJ_F_cuda, nbnxn_kernel_ElecEwQSTabTwinCut_VdwLJCombGeom_F_cuda, nbnxn_kernel_ElecEwQSTabTwinCut_VdwLJCombLB_F_cuda, nbnxn_kernel_ElecEwQSTabTwinCut_VdwLJFsw_F_cuda, nbnxn_kernel_ElecEwQSTabTwinCut_VdwLJPsw_F_cuda, nbnxn_kernel_ElecEwQSTabTwinCut_VdwLJEwCombGeom_F_cuda, nbnxn_kernel_ElecEwQSTabTwinCut_VdwLJEwCombLB_F_cuda },
    { nbnxn_kernel_ElecEw_VdwLJ_F_cuda,             nbnxn_kernel_ElecEw_VdwLJCombGeom_F_cuda,             nbnxn_kernel_ElecEw_VdwLJCombLB_F_cuda,             nbnxn_kernel_ElecEw_VdwLJFsw_F_cuda,             nbnxn_kernel_ElecEw_VdwLJPsw_F_cuda,             nbnxn_kernel_ElecEw_VdwLJEwCombGeom_F_cuda,             nbnxn_kernel_ElecEw_VdwLJEwCombLB_F_cuda             },
    { nbnxn_kernel_ElecEwTwinCut_VdwLJ_F_cuda,      nbnxn_kernel_ElecEwTwinCut_VdwLJCombGeom_F_cuda,      nbnxn_kernel_ElecEwTwinCut_VdwLJCombLB_F_cuda,      nbnxn_kernel_ElecEwTwinCut_VdwLJFsw_F_cuda,      nbnxn_kernel_ElecEwTwinCut_VdwLJPsw_F_cuda,      nbnxn_kernel_ElecEwTwinCut_VdwLJEwCombGeom_F_cuda,      nbnxn_kernel_ElecEwTwinCut_VdwLJEwCombLB_F_cuda      }
};

/*! Force + energy kernel function pointers. */
static const nbnxn_cu_kfunc_ptr_t nb_kfunc_ener_noprune_ptr[eelCuNR][evdwCuNR] =
{
    { nbnxn_kernel_ElecCut_VdwLJ_VF_cuda,            nbnxn_kernel_ElecCut_VdwLJCombGeom_VF_cuda,            nbnxn_kernel_ElecCut_VdwLJCombLB_VF_cuda,            nbnxn_kernel_ElecCut_VdwLJFsw_VF_cuda,            nbnxn_kernel_ElecCut_VdwLJPsw_VF_cuda,            nbnxn_kernel_ElecCut_VdwLJEwCombGeom_VF_cuda,            nbnxn_kernel_ElecCut_VdwLJEwCombLB_VF_cuda            },
    { nbnxn_kernel_ElecRF_VdwLJ_VF_cuda,             nbnxn_kernel_ElecRF_VdwLJCombGeom_VF_cuda,             nbnxn_kernel_ElecRF_VdwLJCombLB_VF_cuda,             nbnxn_kernel_ElecRF_VdwLJFsw_VF_cuda,             nbnxn_kernel_ElecRF_VdwLJPsw_VF_cuda,             nbnxn_kernel_ElecRF_VdwLJEwCombGeom_VF_cuda,             nbnxn_kernel_ElecRF_VdwLJEwCombLB_VF_cuda             },
    { nbnxn_kernel_ElecEwQSTab_VdwLJ_VF_cuda,        nbnxn_kernel_ElecEwQSTab_VdwLJCombGeom_VF_cuda,        nbnxn_kernel_ElecEwQSTab_VdwLJCombLB_VF_cuda,        nbnxn_kernel_ElecEwQSTab_VdwLJFsw_VF_cuda,        nbnxn_kernel_ElecEwQSTab_VdwLJPsw_VF_cuda,        nbnxn_kernel_ElecEwQSTab_VdwLJEwCombGeom_VF_cuda,        nbnxn_kernel_ElecEwQSTab_VdwLJEwCombLB_VF_cuda        },
    { nbnxn_kernel_ElecEwQSTabTwinCut_VdwLJ_VF_cuda, nbnxn_kernel_ElecEwQSTabTwinCut_VdwLJCombGeom_VF_cuda, nbnxn_kernel_ElecEwQSTabTwinCut_VdwLJCombLB_VF_cuda, nbnxn_kernel_ElecEwQSTabTwinCut_VdwLJFsw_VF_cuda, nbnxn_kernel_ElecEwQSTabTwinCut_VdwLJPsw_VF_cuda, nbnxn_kernel_ElecEwQSTabTwinCut_VdwLJEwCombGeom_VF_cuda, nbnxn_kernel_ElecEwQSTabTwinCut_VdwLJEwCombLB_VF_cuda },
    { nbnxn_kernel_ElecEw_VdwLJ_VF_cuda,             nbnxn_kernel_ElecEw_VdwLJCombGeom_VF_cuda,             nbnxn_kernel_ElecEw_VdwLJCombLB_VF_cuda,             nbnxn_kernel_ElecEw_VdwLJFsw_VF_cuda,             nbnxn_kernel_ElecEw_VdwLJPsw_VF_cuda,             nbnxn_kernel_ElecEw_VdwLJEwCombGeom_VF_cuda,             nbnxn_kernel_ElecEw_VdwLJEwCombLB_VF_cuda             },
    { nbnxn_kernel_ElecEwTwinCut_VdwLJ_VF_cuda,      nbnxn_kernel_ElecEwTwinCut_VdwLJCombGeom_VF_cuda,      nbnxn_kernel_ElecEwTwinCut_VdwLJCombLB_VF_cuda,      nbnxn_kernel_ElecEwTwinCut_VdwLJFsw_VF_cuda,      nbnxn_kernel_ElecEwTwinCut_VdwLJPsw_VF_cuda,      nbnxn_kernel_ElecEwTwinCut_VdwLJEwCombGeom_VF_cuda,      nbnxn_kernel_ElecEwTwinCut_VdwLJEwCombLB_VF_cuda      }
};

/*! Force + pruning kernel function pointers. */
static const nbnxn_cu_kfunc_ptr_t nb_kfunc_noener_prune_ptr[eelCuNR][evdwCuNR] =
{
    { nbnxn_kernel_ElecCut_VdwLJ_F_prune_cuda,            nbnxn_kernel_ElecCut_VdwLJCombGeom_F_prune_cuda,            nbnxn_kernel_ElecCut_VdwLJCombLB_F_prune_cuda,            nbnxn_kernel_ElecCut_VdwLJFsw_F_prune_cuda,            nbnxn_kernel_ElecCut_VdwLJPsw_F_prune_cuda,            nbnxn_kernel_ElecCut_VdwLJEwCombGeom_F_prune_cuda,            nbnxn_kernel_ElecCut_VdwLJEwCombLB_F_prune_cuda             },
    { nbnxn_kernel_ElecRF_VdwLJ_F_prune_cuda,             nbnxn_kernel_ElecRF_VdwLJCombGeom_F_prune_cuda,             nbnxn_kernel_ElecRF_VdwLJCombLB_F_prune_cuda,             nbnxn_kernel_ElecRF_VdwLJFsw_F_prune_cuda,             nbnxn_kernel_ElecRF_VdwLJPsw_F_prune_cuda,             nbnxn_kernel_ElecRF_VdwLJEwCombGeom_F_prune_cuda,             nbnxn_kernel_ElecRF_VdwLJEwCombLB_F_prune_cuda              },
    { nbnxn_kernel_ElecEwQSTab_VdwLJ_F_prune_cuda,        nbnxn_kernel_ElecEwQSTab_VdwLJCombGeom_F_prune_cuda,        nbnxn_kernel_ElecEwQSTab_VdwLJCombLB_F_prune_cuda,        nbnxn_kernel_ElecEwQSTab_VdwLJFsw_F_prune_cuda,        nbnxn_kernel_ElecEwQSTab_VdwLJPsw_F_prune_cuda,        nbnxn_kernel_ElecEwQSTab_VdwLJEwCombGeom_F_prune_cuda,        nbnxn_kernel_ElecEwQSTab_VdwLJEwCombLB_F_prune_cuda         },
    { nbnxn_kernel_ElecEwQSTabTwinCut_VdwLJ_F_prune_cuda, nbnxn_kernel_ElecEwQSTabTwinCut_VdwLJCombGeom_F_prune_cuda, nbnxn_kernel_ElecEwQSTabTwinCut_VdwLJCombLB_F_prune_cuda, nbnxn_kernel_ElecEwQSTabTwinCut_VdwLJFsw_F_prune_cuda, nbnxn_kernel_ElecEwQSTabTwinCut_VdwLJPsw_F_prune_cuda, nbnxn_kernel_ElecEwQSTabTwinCut_VdwLJEwCombGeom_F_prune_cuda, nbnxn_kernel_ElecEwQSTabTwinCut_VdwLJEwCombLB_F_prune_cuda  },
    { nbnxn_kernel_ElecEw_VdwLJ_F_prune_cuda,             nbnxn_kernel_ElecEw_VdwLJCombGeom_F_prune_cuda,             nbnxn_kernel_ElecEw_VdwLJCombLB_F_prune_cuda,             nbnxn_kernel_ElecEw_VdwLJFsw_F_prune_cuda,             nbnxn_kernel_ElecEw_VdwLJPsw_F_prune_cuda,             nbnxn_kernel_ElecEw_VdwLJEwCombGeom_F_prune_cuda,             nbnxn_kernel_ElecEw_VdwLJEwCombLB_F_prune_cuda              },
    { nbnxn_kernel_ElecEwTwinCut_VdwLJ_F_prune_cuda,      nbnxn_kernel_ElecEwTwinCut_VdwLJCombGeom_F_prune_cuda,      nbnxn_kernel_ElecEwTwinCut_VdwLJCombLB_F_prune_cuda,      nbnxn_kernel_ElecEwTwinCut_VdwLJFsw_F_prune_cuda,      nbnxn_kernel_ElecEwTwinCut_VdwLJPsw_F_prune_cuda,      nbnxn_kernel_ElecEwTwinCut_VdwLJEwCombGeom_F_prune_cuda,      nbnxn_kernel_ElecEwTwinCut_VdwLJEwCombLB_F_prune_cuda       }
};

/*! Force + energy + pruning kernel function pointers. */
static const nbnxn_cu_kfunc_ptr_t nb_kfunc_ener_prune_ptr[eelCuNR][evdwCuNR] =
{
    { nbnxn_kernel_ElecCut_VdwLJ_VF_prune_cuda,            nbnxn_kernel_ElecCut_VdwLJCombGeom_VF_prune_cuda,            nbnxn_kernel_ElecCut_VdwLJCombLB_VF_prune_cuda,            nbnxn_kernel_ElecCut_VdwLJFsw_VF_prune_cuda,            nbnxn_kernel_ElecCut_VdwLJPsw_VF_prune_cuda,            nbnxn_kernel_ElecCut_VdwLJEwCombGeom_VF_prune_cuda,            nbnxn_kernel_ElecCut_VdwLJEwCombLB_VF_prune_cuda            },
    { nbnxn_kernel_ElecRF_VdwLJ_VF_prune_cuda,             nbnxn_kernel_ElecRF_VdwLJCombGeom_VF_prune_cuda,             nbnxn_kernel_ElecRF_VdwLJCombLB_VF_prune_cuda,             nbnxn_kernel_ElecRF_VdwLJFsw_VF_prune_cuda,             nbnxn_kernel_ElecRF_VdwLJPsw_VF_prune_cuda,             nbnxn_kernel_ElecRF_VdwLJEwCombGeom_VF_prune_cuda,             nbnxn_kernel_ElecRF_VdwLJEwCombLB_VF_prune_cuda             },
    { nbnxn_kernel_ElecEwQSTab_VdwLJ_VF_prune_cuda,        nbnxn_kernel_ElecEwQSTab_VdwLJCombGeom_VF_prune_cuda,        nbnxn_kernel_ElecEwQSTab_VdwLJCombLB_VF_prune_cuda,        nbnxn_kernel_ElecEwQSTab_VdwLJFsw_VF_prune_cuda,        nbnxn_kernel_ElecEwQSTab_VdwLJPsw_VF_prune_cuda,        nbnxn_kernel_ElecEwQSTab_VdwLJEwCombGeom_VF_prune_cuda,        nbnxn_kernel_ElecEwQSTab_VdwLJEwCombLB_VF_prune_cuda        },
    { nbnxn_kernel_ElecEwQSTabTwinCut_VdwLJ_VF_prune_cuda, nbnxn_kernel_ElecEwQSTabTwinCut_VdwLJCombGeom_VF_prune_cuda, nbnxn_kernel_ElecEwQSTabTwinCut_VdwLJCombLB_VF_prune_cuda, nbnxn_kernel_ElecEwQSTabTwinCut_VdwLJFsw_VF_prune_cuda, nbnxn_kernel_ElecEwQSTabTwinCut_VdwLJPsw_VF_prune_cuda, nbnxn_kernel_ElecEwQSTabTwinCut_VdwLJEwCombGeom_VF_prune_cuda, nbnxn_kernel_ElecEwQSTabTwinCut_VdwLJEwCombLB_VF_prune_cuda },
    { nbnxn_kernel_ElecEw_VdwLJ_VF_prune_cuda,             nbnxn_kernel_ElecEw_VdwLJCombGeom_VF_prune_cuda,             nbnxn_kernel_ElecEw_VdwLJCombLB_VF_prune_cuda,             nbnxn_kernel_ElecEw_VdwLJFsw_VF_prune_cuda,             nbnxn_kernel_ElecEw_VdwLJPsw_VF_prune_cuda,             nbnxn_kernel_ElecEw_VdwLJEwCombGeom_VF_prune_cuda,             nbnxn_kernel_ElecEw_VdwLJEwCombLB_VF_prune_cuda             },
    { nbnxn_kernel_ElecEwTwinCut_VdwLJ_VF_prune_cuda,      nbnxn_kernel_ElecEwTwinCut_VdwLJCombGeom_VF_prune_cuda,      nbnxn_kernel_ElecEwTwinCut_VdwLJCombLB_VF_prune_cuda,      nbnxn_kernel_ElecEwTwinCut_VdwLJFsw_VF_prune_cuda,      nbnxn_kernel_ElecEwTwinCut_VdwLJPsw_VF_prune_cuda,      nbnxn_kernel_ElecEwTwinCut_VdwLJEwCombGeom_VF_prune_cuda,      nbnxn_kernel_ElecEwTwinCut_VdwLJEwCombLB_VF_prune_cuda      }
};

/*! Return a pointer to the kernel version to be executed at the current step. */
static inline nbnxn_cu_kfunc_ptr_t select_nbnxn_kernel(int                                  eeltype,
                                                       int                                  evdwtype,
                                                       bool                                 bDoEne,
                                                       bool                                 bDoPrune,
                                                       const gmx_device_info_t gmx_unused  *devInfo)
{
    nbnxn_cu_kfunc_ptr_t res;

    GMX_ASSERT(eeltype < eelCuNR,
               "The electrostatics type requested is not implemented in the CUDA kernels.");
    GMX_ASSERT(evdwtype < evdwCuNR,
               "The VdW type requested is not implemented in the CUDA kernels.");

    /* assert assumptions made by the kernels */
    GMX_ASSERT(c_nbnxnGpuClusterSize*c_nbnxnGpuClusterSize/c_nbnxnGpuClusterpairSplit == devInfo->prop.warpSize,
               "The CUDA kernels require the cluster_size_i*cluster_size_j/nbnxn_gpu_clusterpair_split to match the warp size of the architecture targeted.");

    if (bDoEne)
    {
        if (bDoPrune)
        {
            res = nb_kfunc_ener_prune_ptr[eeltype][evdwtype];
        }
        else
        {
            res = nb_kfunc_ener_noprune_ptr[eeltype][evdwtype];
        }
    }
    else
    {
        if (bDoPrune)
        {
            res = nb_kfunc_noener_prune_ptr[eeltype][evdwtype];
        }
        else
        {
            res = nb_kfunc_noener_noprune_ptr[eeltype][evdwtype];
        }
    }

    return res;
}

/*! \brief Calculates the amount of shared memory required by the nonbonded kernel in use. */
static inline int calc_shmem_required_nonbonded(const int num_threads_z, const gmx_device_info_t gmx_unused *dinfo, const cu_nbparam_t *nbp)
{
    int shmem;

    assert(dinfo);

    /* size of shmem (force-buffers/xq/atom type preloading) */
    /* NOTE: with the default kernel on sm3.0 we need shmem only for pre-loading */
    /* i-atom x+q in shared memory */
    shmem  = c_numClPerSupercl * c_clSize * sizeof(float4);
    /* cj in shared memory, for each warp separately */
    shmem += num_threads_z * c_nbnxnGpuClusterpairSplit * c_nbnxnGpuJgroupSize * sizeof(int);
    if (dinfo->prop.major >= 3)
    {
        if (nbp->vdwtype == evdwCuCUTCOMBGEOM ||
            nbp->vdwtype == evdwCuCUTCOMBLB)
        {
            /* i-atom LJ combination parameters in shared memory */
            shmem += c_numClPerSupercl * c_clSize * sizeof(float2);
        }
        else
        {
            /* i-atom types in shared memory */
            shmem += c_numClPerSupercl * c_clSize * sizeof(int);
        }
    }
    if (dinfo->prop.major < 3)
    {
        /* force reduction buffers in shared memory */
        shmem += c_clSize * c_clSize * 3 * sizeof(float);
    }
    return shmem;
}

/*! As we execute nonbonded workload in separate streams, before launching
   the kernel we need to make sure that he following operations have completed:
   - atomdata allocation and related H2D transfers (every nstlist step);
   - pair list H2D transfer (every nstlist step);
   - shift vector H2D transfer (every nstlist step);
   - force (+shift force and energy) output clearing (every step).

   These operations are issued in the local stream at the beginning of the step
   and therefore always complete before the local kernel launch. The non-local
   kernel is launched after the local on the same device/context hence it is
   inherently scheduled after the operations in the local stream (including the
   above "misc_ops") on pre-GK110 devices with single hardware queue, but on later
   devices with multiple hardware queues the dependency needs to be enforced.
   We use the misc_ops_and_local_H2D_done event to record the point where
   the local x+q H2D (and all preceding) tasks are complete and synchronize
   with this event in the non-local stream before launching the non-bonded kernel.
 */
void nbnxn_gpu_launch_kernel(gmx_nbnxn_cuda_t       *nb,
                             const nbnxn_atomdata_t *nbatom,
                             int                     flags,
                             int                     iloc)
{
    hipError_t          stat;
    int                  adat_begin, adat_len; /* local/nonlocal offset and length used for xq and f */
    /* CUDA kernel launch-related stuff */
    int                  nblock;
    dim3                 dim_block, dim_grid;
    nbnxn_cu_kfunc_ptr_t nb_kernel = nullptr; /* fn pointer to the nonbonded kernel */

    cu_atomdata_t       *adat    = nb->atdat;
    cu_nbparam_t        *nbp     = nb->nbparam;
    cu_plist_t          *plist   = nb->plist[iloc];
    cu_timers_t         *t       = nb->timers;
    hipStream_t         stream  = nb->stream[iloc];

    bool                 bCalcEner   = flags & GMX_FORCE_ENERGY;
    bool                 bCalcFshift = flags & GMX_FORCE_VIRIAL;
    bool                 bDoTime     = nb->bDoTime;

    /* Don't launch the non-local kernel if there is no work to do.
       Doing the same for the local kernel is more complicated, since the
       local part of the force array also depends on the non-local kernel.
       So to avoid complicating the code and to reduce the risk of bugs,
       we always call the local kernel, the local x+q copy and later (not in
       this function) the stream wait, local f copyback and the f buffer
       clearing. All these operations, except for the local interaction kernel,
       are needed for the non-local interactions. The skip of the local kernel
       call is taken care of later in this function. */
    if (canSkipWork(nb, iloc))
    {
        plist->haveFreshList = false;

        return;
    }

    /* calculate the atom data index range based on locality */
    if (LOCAL_I(iloc))
    {
        adat_begin  = 0;
        adat_len    = adat->natoms_local;
    }
    else
    {
        adat_begin  = adat->natoms_local;
        adat_len    = adat->natoms - adat->natoms_local;
    }

    /* beginning of timed HtoD section */
    if (bDoTime)
    {
        t->nb_h2d[iloc].openTimingRegion(stream);
    }

    /* HtoD x, q */
    cu_copy_H2D_async(adat->xq + adat_begin, nbatom->x + adat_begin * 4,
                      adat_len * sizeof(*adat->xq), stream);

    if (bDoTime)
    {
        t->nb_h2d[iloc].closeTimingRegion(stream);
    }

    /* When we get here all misc operations issues in the local stream as well as
       the local xq H2D are done,
       so we record that in the local stream and wait for it in the nonlocal one. */
    if (nb->bUseTwoStreams)
    {
        if (iloc == eintLocal)
        {
            stat = hipEventRecord(nb->misc_ops_and_local_H2D_done, stream);
            CU_RET_ERR(stat, "hipEventRecord on misc_ops_and_local_H2D_done failed");
        }
        else
        {
            stat = hipStreamWaitEvent(stream, nb->misc_ops_and_local_H2D_done, 0);
            CU_RET_ERR(stat, "hipStreamWaitEvent on misc_ops_and_local_H2D_done failed");
        }
    }

    if (nbp->useDynamicPruning && plist->haveFreshList)
    {
        /* Prunes for rlistOuter and rlistInner, sets plist->haveFreshList=false
           (TODO: ATM that's the way the timing accounting can distinguish between
           separate prune kernel and combined force+prune, maybe we need a better way?).
         */
        nbnxn_gpu_launch_kernel_pruneonly(nb, iloc, 1);
    }

    if (plist->nsci == 0)
    {
        /* Don't launch an empty local kernel (not allowed with CUDA) */
        return;
    }

    /* beginning of timed nonbonded calculation section */
    if (bDoTime)
    {
        t->nb_k[iloc].openTimingRegion(stream);
    }

    /* get the pointer to the kernel flavor we need to use */
    nb_kernel = select_nbnxn_kernel(nbp->eeltype,
                                    nbp->vdwtype,
                                    bCalcEner,
                                    (plist->haveFreshList && !nb->timers->didPrune[iloc]),
                                    nb->dev_info);

    /* Kernel launch config:
     * - The thread block dimensions match the size of i-clusters, j-clusters,
     *   and j-cluster concurrency, in x, y, and z, respectively.
     * - The 1D block-grid contains as many blocks as super-clusters.
     */
    int num_threads_z = 1;
    if (nb->dev_info->prop.major == 3 && nb->dev_info->prop.minor == 7)
    {
        num_threads_z = 2;
    }
    nblock    = calc_nb_kernel_nblock(plist->nsci, nb->dev_info);

    KernelLaunchConfig config;
    config.blockSize[0]     = c_clSize;
    config.blockSize[1]     = c_clSize;
    config.blockSize[2]     = num_threads_z;
    config.gridSize[0]      = nblock;
    config.sharedMemorySize = calc_shmem_required_nonbonded(num_threads_z, nb->dev_info, nbp);
    config.stream           = stream;

    if (debug)
    {
        fprintf(debug, "Non-bonded GPU launch configuration:\n\tThread block: %zux%zux%zu\n\t"
                "\tGrid: %zux%zu\n\t#Super-clusters/clusters: %d/%d (%d)\n"
                "\tShMem: %zu\n",
                config.blockSize[0], config.blockSize[1], config.blockSize[2],
                config.gridSize[0], config.gridSize[1], plist->nsci*c_numClPerSupercl,
                c_numClPerSupercl, plist->na_c,
                config.sharedMemorySize);
    }

    auto      *timingEvent = bDoTime ? t->nb_k[iloc].fetchNextEvent() : nullptr;
    const auto kernelArgs  = prepareGpuKernelArguments(nb_kernel, config, adat, nbp, plist, &bCalcFshift);
    launchGpuKernel(nb_kernel, config, timingEvent, "k_calc_nb", kernelArgs);

    if (bDoTime)
    {
        t->nb_k[iloc].closeTimingRegion(stream);
    }

    if (GMX_NATIVE_WINDOWS)
    {
        /* Windows: force flushing WDDM queue */
        hipStreamQuery(stream);
    }
}

/*! Calculates the amount of shared memory required by the CUDA kernel in use. */
static inline int calc_shmem_required_prune(const int num_threads_z)
{
    int shmem;

    /* i-atom x in shared memory */
    shmem  = c_numClPerSupercl * c_clSize * sizeof(float4);
    /* cj in shared memory, for each warp separately */
    shmem += num_threads_z * c_nbnxnGpuClusterpairSplit * c_nbnxnGpuJgroupSize * sizeof(int);

    return shmem;
}

void nbnxn_gpu_launch_kernel_pruneonly(gmx_nbnxn_cuda_t       *nb,
                                       int                     iloc,
                                       int                     numParts)
{
    cu_atomdata_t       *adat    = nb->atdat;
    cu_nbparam_t        *nbp     = nb->nbparam;
    cu_plist_t          *plist   = nb->plist[iloc];
    cu_timers_t         *t       = nb->timers;
    hipStream_t         stream  = nb->stream[iloc];

    bool                 bDoTime = nb->bDoTime;

    if (plist->haveFreshList)
    {
        GMX_ASSERT(numParts == 1, "With first pruning we expect 1 part");

        /* Set rollingPruningNumParts to signal that it is not set */
        plist->rollingPruningNumParts = 0;
        plist->rollingPruningPart     = 0;
    }
    else
    {
        if (plist->rollingPruningNumParts == 0)
        {
            plist->rollingPruningNumParts = numParts;
        }
        else
        {
            GMX_ASSERT(numParts == plist->rollingPruningNumParts, "It is not allowed to change numParts in between list generation steps");
        }
    }

    /* Use a local variable for part and update in plist, so we can return here
     * without duplicating the part increment code.
     */
    int part = plist->rollingPruningPart;

    plist->rollingPruningPart++;
    if (plist->rollingPruningPart >= plist->rollingPruningNumParts)
    {
        plist->rollingPruningPart = 0;
    }

    /* Compute the number of list entries to prune in this pass */
    int numSciInPart = (plist->nsci - part)/numParts;

    /* Don't launch the kernel if there is no work to do (not allowed with CUDA) */
    if (numSciInPart <= 0)
    {
        plist->haveFreshList = false;

        return;
    }

    GpuRegionTimer *timer = nullptr;
    if (bDoTime)
    {
        timer = &(plist->haveFreshList ? t->prune_k[iloc] : t->rollingPrune_k[iloc]);
    }

    /* beginning of timed prune calculation section */
    if (bDoTime)
    {
        timer->openTimingRegion(stream);
    }

    /* Kernel launch config:
     * - The thread block dimensions match the size of i-clusters, j-clusters,
     *   and j-cluster concurrency, in x, y, and z, respectively.
     * - The 1D block-grid contains as many blocks as super-clusters.
     */
    int                num_threads_z  = c_cudaPruneKernelJ4Concurrency;
    int                nblock         = calc_nb_kernel_nblock(numSciInPart, nb->dev_info);
    KernelLaunchConfig config;
    config.blockSize[0]     = c_clSize;
    config.blockSize[1]     = c_clSize;
    config.blockSize[2]     = num_threads_z;
    config.gridSize[0]      = nblock;
    config.sharedMemorySize = calc_shmem_required_prune(num_threads_z);
    config.stream           = stream;

    if (debug)
    {
        fprintf(debug, "Pruning GPU kernel launch configuration:\n\tThread block: %zux%zux%zu\n\t"
                "\tGrid: %zux%zu\n\t#Super-clusters/clusters: %d/%d (%d)\n"
                "\tShMem: %zu\n",
                config.blockSize[0], config.blockSize[1], config.blockSize[2],
                config.gridSize[0], config.gridSize[1], numSciInPart*c_numClPerSupercl,
                c_numClPerSupercl, plist->na_c,
                config.sharedMemorySize);
    }

    auto          *timingEvent  = bDoTime ? timer->fetchNextEvent() : nullptr;
    constexpr char kernelName[] = "k_pruneonly";
    const auto    &kernel       = plist->haveFreshList ? nbnxn_kernel_prune_cuda<true> : nbnxn_kernel_prune_cuda<false>;
    const auto     kernelArgs   = prepareGpuKernelArguments(kernel, config, adat, nbp, plist, &numParts, &part);
    launchGpuKernel(kernel, config, timingEvent, kernelName, kernelArgs);

    /* TODO: consider a more elegant way to track which kernel has been called
       (combined or separate 1st pass prune, rolling prune). */
    if (plist->haveFreshList)
    {
        plist->haveFreshList         = false;
        /* Mark that pruning has been done */
        nb->timers->didPrune[iloc] = true;
    }
    else
    {
        /* Mark that rolling pruning has been done */
        nb->timers->didRollingPrune[iloc] = true;
    }

    if (bDoTime)
    {
        timer->closeTimingRegion(stream);
    }

    if (GMX_NATIVE_WINDOWS)
    {
        /* Windows: force flushing WDDM queue */
        hipStreamQuery(stream);
    }
}

void nbnxn_gpu_launch_cpyback(gmx_nbnxn_cuda_t       *nb,
                              const nbnxn_atomdata_t *nbatom,
                              int                     flags,
                              int                     aloc)
{
    hipError_t stat;
    int         adat_begin, adat_len; /* local/nonlocal offset and length used for xq and f */

    /* determine interaction locality from atom locality */
    int              iloc = gpuAtomToInteractionLocality(aloc);

    cu_atomdata_t   *adat    = nb->atdat;
    cu_timers_t     *t       = nb->timers;
    bool             bDoTime = nb->bDoTime;
    hipStream_t     stream  = nb->stream[iloc];

    bool             bCalcEner   = flags & GMX_FORCE_ENERGY;
    bool             bCalcFshift = flags & GMX_FORCE_VIRIAL;

    /* don't launch non-local copy-back if there was no non-local work to do */
    if (canSkipWork(nb, iloc))
    {
        return;
    }

    getGpuAtomRange(adat, aloc, &adat_begin, &adat_len);

    /* beginning of timed D2H section */
    if (bDoTime)
    {
        t->nb_d2h[iloc].openTimingRegion(stream);
    }

    /* With DD the local D2H transfer can only start after the non-local
       kernel has finished. */
    if (iloc == eintLocal && nb->bUseTwoStreams)
    {
        stat = hipStreamWaitEvent(stream, nb->nonlocal_done, 0);
        CU_RET_ERR(stat, "hipStreamWaitEvent on nonlocal_done failed");
    }

    /* DtoH f */
    cu_copy_D2H_async(nbatom->out[0].f + adat_begin * 3, adat->f + adat_begin,
                      (adat_len)*sizeof(*adat->f), stream);

    /* After the non-local D2H is launched the nonlocal_done event can be
       recorded which signals that the local D2H can proceed. This event is not
       placed after the non-local kernel because we want the non-local data
       back first. */
    if (iloc == eintNonlocal)
    {
        stat = hipEventRecord(nb->nonlocal_done, stream);
        CU_RET_ERR(stat, "hipEventRecord on nonlocal_done failed");
    }

    /* only transfer energies in the local stream */
    if (LOCAL_I(iloc))
    {
        /* DtoH fshift */
        if (bCalcFshift)
        {
            cu_copy_D2H_async(nb->nbst.fshift, adat->fshift,
                              SHIFTS * sizeof(*nb->nbst.fshift), stream);
        }

        /* DtoH energies */
        if (bCalcEner)
        {
            cu_copy_D2H_async(nb->nbst.e_lj, adat->e_lj,
                              sizeof(*nb->nbst.e_lj), stream);
            cu_copy_D2H_async(nb->nbst.e_el, adat->e_el,
                              sizeof(*nb->nbst.e_el), stream);
        }
    }

    if (bDoTime)
    {
        t->nb_d2h[iloc].closeTimingRegion(stream);
    }
}

void nbnxn_cuda_set_cacheconfig(const gmx_device_info_t *devinfo)
{
    hipError_t stat;

    for (int i = 0; i < eelCuNR; i++)
    {
        for (int j = 0; j < evdwCuNR; j++)
        {
            if (devinfo->prop.major >= 3)
            {
                /* Default kernel on sm 3.x and later 32/32 kB Shared/L1 */
                hipFuncSetCacheConfig(reinterpret_cast<const void*>(nb_kfunc_ener_prune_ptr[i][j]), hipFuncCachePreferEqual);
                hipFuncSetCacheConfig(reinterpret_cast<const void*>(nb_kfunc_ener_noprune_ptr[i][j]), hipFuncCachePreferEqual);
                hipFuncSetCacheConfig(reinterpret_cast<const void*>(nb_kfunc_noener_prune_ptr[i][j]), hipFuncCachePreferEqual);
                stat = hipFuncSetCacheConfig(reinterpret_cast<const void*>(nb_kfunc_noener_noprune_ptr[i][j]), hipFuncCachePreferEqual);
            }
            else
            {
                /* On Fermi prefer L1 gives 2% higher performance */
                /* Default kernel on sm_2.x 16/48 kB Shared/L1 */
                hipFuncSetCacheConfig(reinterpret_cast<const void*>(nb_kfunc_ener_prune_ptr[i][j]), hipFuncCachePreferL1);
                hipFuncSetCacheConfig(reinterpret_cast<const void*>(nb_kfunc_ener_noprune_ptr[i][j]), hipFuncCachePreferL1);
                hipFuncSetCacheConfig(reinterpret_cast<const void*>(nb_kfunc_noener_prune_ptr[i][j]), hipFuncCachePreferL1);
                stat = hipFuncSetCacheConfig(reinterpret_cast<const void*>(nb_kfunc_noener_noprune_ptr[i][j]), hipFuncCachePreferL1);
            }
            CU_RET_ERR(stat, "hipFuncSetCacheConfig failed");
        }
    }
}
