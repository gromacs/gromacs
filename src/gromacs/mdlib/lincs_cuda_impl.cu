#include "hip/hip_runtime.h"
/*
 * This file is part of the GROMACS molecular simulation package.
 *
 * Copyright (c) 2019, by the GROMACS development team, led by
 * Mark Abraham, David van der Spoel, Berk Hess, and Erik Lindahl,
 * and including many others, as listed in the AUTHORS file in the
 * top-level source directory and at http://www.gromacs.org.
 *
 * GROMACS is free software; you can redistribute it and/or
 * modify it under the terms of the GNU Lesser General Public License
 * as published by the Free Software Foundation; either version 2.1
 * of the License, or (at your option) any later version.
 *
 * GROMACS is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
 * Lesser General Public License for more details.
 *
 * You should have received a copy of the GNU Lesser General Public
 * License along with GROMACS; if not, see
 * http://www.gnu.org/licenses, or write to the Free Software Foundation,
 * Inc., 51 Franklin Street, Fifth Floor, Boston, MA  02110-1301  USA.
 *
 * If you want to redistribute modifications to GROMACS, please
 * consider that scientific software is very special. Version
 * control is crucial - bugs must be traceable. We will be happy to
 * consider code for inclusion in the official distribution, but
 * derived work must not be called official GROMACS. Details are found
 * in the README & COPYING files - if they are missing, get the
 * official version at http://www.gromacs.org.
 *
 * To help us fund GROMACS development, we humbly ask that you cite
 * the research papers on the package. Check out http://www.gromacs.org.
 */
#include "gmxpre.h"

#include "lincs_cuda_impl.h"

#include <assert.h>
#include <stdio.h>

#include <cmath>

#include <algorithm>

#include "gromacs/gpu_utils/cudautils.cuh"
#include "gromacs/gpu_utils/gputraits.cuh"
#include "gromacs/gpu_utils/vectype_ops.cuh"
#include "gromacs/math/vec.h"
#include "gromacs/mdlib/constr.h"
#include "gromacs/mdlib/lincs_cuda.h"
#include "gromacs/mdtypes/commrec.h"
#include "gromacs/mdtypes/group.h"
#include "gromacs/pbcutil/gpu_pbc.cuh"
#include "gromacs/pbcutil/pbc.h"

#if defined(_MSVC)
#include <limits>
#endif

#define GMX_LINCS_CUDA_TPB 256


/*! \brief Main kernel for LINCS constraints.
 *
 * \todo Combine arguments
 * \todo Move everything to local/shared memory, try to get rid of atomics.
 * \todo Template updateVelocities and virial.
 *
 * \param[in]     ncons                     Total number of the constrain threads (empty spaces at the end of thread blocks included)
 * \param[in]     x                         Coordinates before the timestep
 * \param[in,out] xp                        Coordinates after the timestep. Will be updated to satisfy the constraints by this kernel.
 * \param[in]     nIter                     Number of iterations used to compute inverse matrix.
 * \param[in]     nOrder                    LINCS projection order for correcting the direction of constraint.
 * \param[in]     constraints               List of constraints.
 * \param[in]     constraintsR0             List of target distances for constraints.
 * \param[in]     coupledConstraintsCounts  Numbers of coupled constraints for each constraint
 * \param[in]     coupledConstraintsIdxes   Indexes of coupled constraints
 * \param[in]     massFactors               Mass factors: ( (+/-) * (1/sqrt(1/m1 + 1/m2)) * (1/m2) * 1/sqrt(1/m2 + 1/m3)),
 *                                          where m1 and m3 are coupled through m2 and sign + or - indicates the order,
 *                                          in which they are arranged in atoms array.)
 * \param[in]     matrixA                   Place to store constraints matrix.
 * \param[in]     pbcAiuc                   Periodic boundary information
 * \param[in]     mlambda                   Lagrange multiplier times mass, saved by this kernel for virial evaluation.
 * \param[in]     updateVelocities          If the velocities should be computed.
 * \param[in,out] v                         Velocities to update.
 * \param[in]     invdt                     Inverse timestep (needed to update velocities).
 */
__global__ void lincs_kernel(const int            ncons,
                             const float3        *x,
                             float3              *xp,
                             const int            nIter,
                             const int            nOrder,
                             const int2          *constraints,
                             const real          *constraintsR0,
                             const int           *coupledConstraintsCounts,
                             const int           *coupledConstraintsIdxes,
                             const real          *massFactors,
                             real                *matrixA,
                             const PbcAiuc        pbcAiuc,
                             real                *mlambda,
                             const real          *invmass,
                             const bool           updateVelocities,
                             float3              *v,
                             const real           invdt)
{

    int c           = blockIdx.x*blockDim.x+threadIdx.x;
    int cs          = threadIdx.x;
    int blockStarts = blockIdx.x*blockDim.x;

    extern __shared__ float3 r[];
    extern __shared__ float  rhs[];

    if (c < ncons)
    {

        float mvb;

        int2  pair = constraints[c];
        int   i    = pair.x;
        int   j    = pair.y;

        real  ml = 0.0;

        if (i != -1)
        {
            float rlen0 = constraintsR0[c];

            float im1      = invmass[i];
            float im2      = invmass[j];

            float sqrtmu = rsqrt(im1 + im2);


            float3 xi = x[i];
            float3 xj = x[j];

            float3 dx   = pbcDxAiucFloat3(pbcAiuc, xi, xj);
            float  rlen = rsqrtf(dx.x*dx.x + dx.y*dx.y + dx.z*dx.z);

            float3 rc = rlen*dx;
            r[cs] = rc;

            xi = xp[i];
            xj = xp[j];
            dx = pbcDxAiucFloat3(pbcAiuc, xi, xj);

            mvb = sqrtmu*((rc.x*dx.x + rc.y*dx.y + rc.z*dx.z) - rlen0);

            float sol  = mvb;

            __syncthreads();

            int coupledConstraintsCount = coupledConstraintsCounts[c];

            for (int n = 0; n < coupledConstraintsCount; n++)
            {
                int    index = n*ncons + c;
                int    c1    = coupledConstraintsIdxes[index]; //Can be moved to local/shared memory

                float3 rc1 = r[c1-blockStarts];
                matrixA[index] = massFactors[index]*(rc.x*rc1.x + rc.y*rc1.y + rc.z*rc1.z);

            }

            rhs[cs] = mvb;
            __syncthreads();

            for (int rec = 0; rec < nOrder; rec++)
            {
                mvb = 0;

                for (int n = 0; n < coupledConstraintsCount; n++)
                {
                    int index = n*ncons + c;
                    int c1    = coupledConstraintsIdxes[index];

                    mvb = mvb + matrixA[index]*rhs[c1-blockStarts + blockDim.x*(rec % 2)];

                }
                rhs[cs + blockDim.x*((rec + 1) % 2)] = mvb;
                sol  = sol + mvb;
                __syncthreads();

            }

            ml = sqrtmu*sol;

            mvb      = ml;

            float3 tmp     = rc*mvb;

            atomicAdd(&xp[i], -tmp*im1);
            atomicAdd(&xp[j], tmp*im2);

            __syncthreads();

            for (int iter = 0; iter < nIter; iter++)
            {
                float len2, dlen2;

                xi = xp[i];
                xj = xp[j];


                dx = pbcDxAiucFloat3(pbcAiuc, xi, xj);

                len2  = rlen0*rlen0;
                dlen2 = 2.0f*len2 - norm2(dx);

                if (dlen2 > 0)
                {
                    mvb = sqrtmu*(rlen0 - dlen2*rsqrt(dlen2));
                }
                else
                {
                    mvb = sqrtmu*rlen0;
                }

                rhs[cs]  = mvb;
                sol      = mvb;
                __syncthreads();

                for (int rec = 0; rec < nOrder; rec++)
                {
                    mvb = 0;

                    for (int n = 0; n < coupledConstraintsCount; n++)
                    {
                        int index = n*ncons + c;
                        int c1    = coupledConstraintsIdxes[index];

                        mvb = mvb + matrixA[index]*rhs[c1-blockStarts + blockDim.x*(rec % 2)];

                    }
                    rhs[cs + blockDim.x*((rec + 1) % 2)] = mvb;
                    sol  = sol + mvb;
                    __syncthreads();

                }

                mvb         = sqrtmu*sol;
                float sqrtmu_sol  = mvb;
                ml += mvb;

                mvb      = sqrtmu_sol;

                float3 tmp = rc*mvb;

                atomicAdd(&xp[i], -tmp*im1);
                atomicAdd(&xp[j], tmp*im2);
                __syncthreads();
            }

            if (updateVelocities)
            {
                mvb      = invdt*ml;

                float3 tmp     = rc*mvb;

                atomicAdd(&v[i], -tmp*im1);
                atomicAdd(&v[j], tmp*im2);
            }

        }
        mlambda[c] = ml; // Needed for virial
    }

    return;
}

/*! \brief Kernel to compute virial.
 *
 * \todo Make it into a templated version of the main kernel?
 *
 * \param[in]     ncons                     Total number of the constrain threads (empty spaces at the end of thread blocks included)
 * \param[in]     x                         Coordinates before the timestep
 * \param[in,out] virialScaled              Scaled virial tensor to be updated.
 * \param[in]     mlambda                   Lagrange multiplier times mass, saved by this kernel for virial evaluation.
 * \param[in]     constraints               List of constraints.
 * \param[in]     constraintsR0             List of target distances for constraints.
 * \param[in]     pbcAiuc                   Periodic boundary information
 */
__global__ void lincs_virial_kernel(const int             ncons,
                                    const float3         *x,
                                    real                 *virialScaled,
                                    const real           *mlambda,
                                    const int2           *constraints,
                                    const real           *constraintsR0,
                                    const PbcAiuc         pbcAiuc)
{


    int c = blockIdx.x*blockDim.x+threadIdx.x;

    if (c < ncons)
    {
        float tmp0, tmp1;

        int2  pair = constraints[c];
        int   i    = pair.x;
        int   j    = pair.y;

        if (i != -1)
        {

            float3 xi = x[i];
            float3 xj = x[j];

            float3 dx   = pbcDxAiucFloat3(pbcAiuc, xi, xj);
            float  rlen = rsqrtf(dx.x*dx.x + dx.y*dx.y + dx.z*dx.z);

            rvec   rc;

            rc[0] = rlen*dx.x;
            rc[1] = rlen*dx.y;
            rc[2] = rlen*dx.z;

            tmp0 = -constraintsR0[c]*mlambda[c];
            for (int d1 = 0; d1 < DIM; d1++)
            {
                tmp1 = tmp0*rc[d1];
                for (int d2 = 0; d2 < DIM; d2++)
                {
                    atomicAdd(&(virialScaled[d1*DIM+d2]), -tmp1*rc[d2]);
                }
            }
        }
    }
    return;
}

/*
 * Temporary solution.
 */
#define cudaCheckError() {                                          \
        hipError_t e = hipGetLastError();                                 \
        if (e != hipSuccess) {                                              \
            printf("Cuda failure %s:%d: '%s'\n", __FILE__, __LINE__, hipGetErrorString(e));           \
            exit(0); \
        }                                                                 \
}

/*! \brief Create LINCS object
 *
 * \param [in] nAtom  Number of atoms that will be handles by LINCS.
 *                    Used to compute the memory size in allocations and copy.
 * \param [in] nIter  Number of iterations used to compute inverse matrix.
 * \param [in] nOrder LINCS projection order for correcting the direction of constraint.
 */
LincsCuda::Impl::Impl(int nAtom,
                      int nIter,
                      int nOrder)
    : nAtom(nAtom), nIter(nIter), nOrder(nOrder)

{
    GMX_ASSERT(sizeof(real) == sizeof(float), "Real numbers should be in single precision in GPU code.");
    hipMalloc(&xDevice, nAtom*DIM*sizeof(float));
    hipMalloc(&xpDevice, nAtom*DIM*sizeof(float));
    hipMalloc(&vDevice, nAtom*DIM*sizeof(float));

    hipMalloc(&virialScaledDevice, DIM*DIM*sizeof(float));
    maxConstraintsNumberSoFar = 0;
    hipStreamCreate(&stream);
    cudaCheckError();
}

LincsCuda::Impl::~Impl()
{
}


/*! \brief Apply LINCS.
 *
 * Applies LINCS to coordinates and velocities, stored on GPU.
 * Data at pointers xPrime and v (class fields) change in the GPU
 * memory. The results are not automatically copied back to the CPU
 * memory. Method uses this class data structures which should be
 * updated when needed using update method.
 *
 * \param[in] updateVelocities  If the velocities should be constrained.
 * \param[in] invdt             Inversed timestep (to scale Lagrange
 *                              multipliers when velocities are updated)
 * \param[in] bCalcVir          If virial should be updated.
 * \param[in] scaleLambda       If the Lagrange multipliers should be scaled
 *                              before virial is computed.
 * \param[in,out] virialScaled  Scaled virial tensor to be updated.
 */
void LincsCuda::Impl::apply(const bool       updateVelocities,
                            const real       invdt,
                            const gmx_bool   bCalcVir,
                            tensor           virialScaled)
{

    cudaCheckError();

    int blockSize  = GMX_LINCS_CUDA_TPB;
    int blockCount = (nConstraintsThreads + blockSize - 1)/blockSize;

    /*KernelLaunchConfig config;
       config.blockSize[0]     = blockSize;
       config.blockSize[1]     = 0;
       config.blockSize[2]     = 0;
       config.gridSize[0]      = (nConstraintsThreads + blockSize - 1)/blockSize;
       config.sharedMemorySize = blockSize*DIM*sizeof(float);
       config.stream           = stream;

       auto kernelPtr = lincs_kernel;
       const auto kernelArgs = prepareGpuKernelArguments(kernelPtr, config,
            &nConstraintsThreads, &xDevice, &xpDevice,
            &nIter, &nOrder,
            &constraintsDevice, &constraintsR0Device,
            &coupledConstraintsCountsDevice, &coupledConstraintsIdxesDevice,
            &massFactorsDevice, &matrixADevice,
            &pbcAiuc,
            &mlambdaDevice,
            &invmassDevice,
            &updateVelocities, &vDevice, &invdt);
       launchGpuKernel(lincs_kernel, config, nullptr, "lincs", kernelArgs);*/

    lincs_kernel
    <<< blockCount, blockSize, blockSize*DIM*sizeof(float), stream>>>
    (nConstraintsThreads, xDevice, xpDevice,
     nIter, nOrder,
     constraintsDevice, constraintsR0Device,
     coupledConstraintsCountsDevice, coupledConstraintsIdxesDevice,
     massFactorsDevice, matrixADevice,
     pbcAiuc,
     mlambdaDevice,
     invmassDevice,
     updateVelocities, vDevice, invdt);

    cudaCheckError();


    if (bCalcVir)
    {
        hipMemcpy(virialScaledDevice, virialScaled, DIM*DIM*sizeof(real), hipMemcpyHostToDevice);

        /*config.sharedMemorySize = 0;
           kernelPtr = lincs_virial_kernel;
           kernelArgs = prepareGpuKernelArguments(kernelPtr, config,
            &nConstraintsThreads, &xDevice,
            &virialScaledDevice, &mlambdaDevice,
            &constraintsDevice, &constraintsR0Device,
            &pbcAiuc);
           launchGpuKernel(lincs_virial_kernel, config, nullptr, "lincs-virial", kernelArgs);*/

        lincs_virial_kernel
        <<< blockCount, blockSize, 0, stream>>>
        (nConstraintsThreads, xDevice, virialScaledDevice, mlambdaDevice, constraintsDevice, constraintsR0Device, pbcAiuc);

        cudaCheckError();

        hipMemcpy(virialScaled, virialScaledDevice, DIM*DIM*sizeof(real), hipMemcpyDeviceToHost);

        cudaCheckError();
    }

    hipStreamSynchronize(stream);
    cudaCheckError();

    return;
}

/*! \brief Helper function to go through constraints recurrently
 *
 *  Counts the total number constraints, connected to an atom (including those, connected through other constraints).
 */
inline int countCoupled(int a, std::vector<int> *spaceNeeded,
                        std::vector<std::vector<std::tuple<int, int, int> > > *atomsAdjacencyList)

{
    int c2, a2, sign;
    int counted = 0;
    for (unsigned i = 0; i < atomsAdjacencyList->at(a).size(); i++)
    {
        std::tie(a2, c2, sign) = atomsAdjacencyList->at(a).at(i);
        if (spaceNeeded->at(c2) == -1)
        {
            spaceNeeded->at(c2) = 0; // To indicate we've been here
            counted            += 1 + countCoupled(a2, spaceNeeded, atomsAdjacencyList);
        }
    }
    return counted;
}

/*! \brief
 * Update data-structures (e.g. after NB search step).
 *
 * Updates the constraints data and copies it to the GPU. Should be
 * called if the particles were sorted, redistributed between domains, etc.
 * This version uses common data formats so it can be called from anywhere
 * in the code. Does not recycle the data preparation routines from the CPU
 * version. Works only with simple case when all the constraints in idef are
 * are handled by a single GPU. Triangles are not handled as special case.
 *
 * Information about constraints is taken from:
 *     idef.il[F_CONSTR].iatoms  --- type (T) of constraint and two atom indexes (i1, i2)
 *     idef.iparams[T].constr.dA --- target length for constraint of type T
 * From t_mdatom, the code takes:
 *     md.invmass  --- array of inverse square root of masses for each atom in the system.
 *
 * \param[in] idef  Local topology data to get information on constraints from.
 * \param[in] md    Atoms data to get atom masses from.
 */
void LincsCuda::Impl::set(const t_idef    &idef,
                          const t_mdatoms &md)
{

    int blockSize = GMX_LINCS_CUDA_TPB;

    //t_idef idef = top.idef;
    t_iatom  *iatoms      = idef.il[F_CONSTR].iatoms;
    const int nConstraint = idef.il[F_CONSTR].nr/3;
    // Constructing adjacency list --- usefull intermediate structure
    std::vector<std::vector<std::tuple<int, int, int> > > atomsAdjacencyList(nAtom);
    for (int c = 0; c < nConstraint; c++)
    {
        int a1     = iatoms[3*c + 1];
        int a2     = iatoms[3*c + 2];

        // Each constraint will be represented as a tuple, containing index of the second constrained atom,
        // index of the constraint and a sign that indicates the order of atoms in which they are listed.
        // Sign is needed to compute the mass factors.
        atomsAdjacencyList.at(a1).push_back(std::make_tuple(a2, c, +1));
        atomsAdjacencyList.at(a2).push_back(std::make_tuple(a1, c, -1));
    }

    // Compute, how many coupled constraints are in front of each constraint.
    // Needed to introduce splits in data so that all coupled constraints will be computed in a single GPU block.
    // The position 'c' of the vector spaceNeeded should have the number of constraints that are coupled to a constraint
    // 'c' and are after 'c' in the vector. Only first index of the connected group of the constraints is needed later in the
    // code, hence the spaceNeeded vetor is also used to keep track if the constrain was already counted.
    std::vector<int> spaceNeeded;
    spaceNeeded.resize(nConstraint, -1);
    std::fill(spaceNeeded.begin(), spaceNeeded.end(), -1);
    for (int c = 0; c < nConstraint; c++)
    {
        int a1     = iatoms[3*c + 1];
        int a2     = iatoms[3*c + 2];
        if (spaceNeeded.at(c) == -1)
        {
            spaceNeeded.at(c) = countCoupled(a1, &spaceNeeded, &atomsAdjacencyList) +
                countCoupled(a2, &spaceNeeded, &atomsAdjacencyList);
        }

        // Constraint 'c' is counted twice, but it should be excluded altogether. Hence '-2'.
        /*spaceNeeded.at(c) = atomsAdjacencyList.at(a1).size() + atomsAdjacencyList.at(a2).size() - 2;

           // Only space needed in front of the constraint should be counted.
           while (spaceNeeded.at(c) > 0 && c + 1 < nConstraint)
           {
            c++;
            spaceNeeded.at(c) = spaceNeeded.at(c-1) - 1;
           }*/
    }

    // Map of splits in the constraints data. For each 'old' constraint index gives 'new' which
    // takes into account the empty spaces which might be needed in the end of each thread block.
    std::vector<int> splitMap;
    splitMap.resize(nConstraint, -1);
    int              currentMapIndex = 0;
    for (int c = 0; c < nConstraint; c++)
    {
        if (currentMapIndex / blockSize != (currentMapIndex + spaceNeeded.at(c)) / blockSize)
        {
            currentMapIndex = ((currentMapIndex/blockSize) + 1) * blockSize;
        }
        splitMap.at(c) = currentMapIndex;
        currentMapIndex++;
    }
    nConstraintsThreads = currentMapIndex + blockSize - currentMapIndex % blockSize;


    // Initialize constraints and their target indexes taking into account the splits in the
    // data arrays.
    int2 pair;
    pair.x = -1;
    pair.y = -1;
    constraintsHost.resize(nConstraintsThreads, pair);
    std::fill(constraintsHost.begin(), constraintsHost.end(), pair);
    constraintsR0Host.resize(nConstraintsThreads, 0.0);
    std::fill(constraintsR0Host.begin(), constraintsR0Host.end(), 0.0);
    for (int c = 0; c < nConstraint; c++)
    {
        int  a1     = iatoms[3*c + 1];
        int  a2     = iatoms[3*c + 2];
        int  type   = iatoms[3*c];

        int2 pair;
        pair.x = a1;
        pair.y = a2;
        constraintsHost.at(splitMap.at(c))   = pair;
        constraintsR0Host.at(splitMap.at(c)) = idef.iparams[type].constr.dA;

    }

    // The adjacency list of constraints (i.e. the list of coupled constraints for each constraint).
    // We map a single thread to a single constraint, hence each thread 'c' will be using one element from
    // coupledConstraintsCountsHost array, which is the number of constraints coupled to the constraint 'c'.
    // The coupled constraints indexes are placed into the coupledConstraintsIdxesHost array. Latter is organized
    // as a one-dimensional array to ensure good memory alignment. It is addressed as [c + i*nConstraintsThreads],
    // where 'i' goes from zero to the number of constraints coupled to 'c'. 'nConstraintsThreads' is the width of
    // the array --- a number, greater then total number of constraints, taking into account the splits in the
    // constraints array due to the GPU block borders. This number can be adjusted to improve memory access pattern.
    // Mass factors are saved in a similar data structure.
    int              maxCoupledConstraints = 0;
    for (int c = 0; c < nConstraint; c++)
    {
        int a1     = iatoms[3*c + 1];
        int a2     = iatoms[3*c + 2];

        // Constraint 'c' is counted twice, but it should be excluded altogether. Hence '-2'.
        int nCoupedConstraints = atomsAdjacencyList.at(a1).size() + atomsAdjacencyList.at(a2).size() - 2;

        if (nCoupedConstraints > maxCoupledConstraints)
        {
            maxCoupledConstraints = nCoupedConstraints;
        }
    }

    coupledConstraintsCountsHost.resize(nConstraintsThreads, 0);
    coupledConstraintsIdxesHost.resize(maxCoupledConstraints*nConstraintsThreads, -1);
    massFactorsHost.resize(maxCoupledConstraints*nConstraintsThreads, -1);

    for (int c1 = 0; c1 < nConstraint; c1++)
    {
        coupledConstraintsCountsHost.at(splitMap.at(c1))  = 0;
        int c1a1     = iatoms[3*c1 + 1];
        int c1a2     = iatoms[3*c1 + 2];
        int c2;
        int c2a1;
        int c2a2;

        int sign;

        c2a1 = c1a1;
        for (unsigned j = 0; j < atomsAdjacencyList.at(c1a1).size(); j++)
        {

            std::tie(c2a2, c2, sign) = atomsAdjacencyList.at(c1a1).at(j);

            if (c1 != c2)
            {
                int index = nConstraintsThreads*coupledConstraintsCountsHost.at(splitMap.at(c1)) + splitMap.at(c1);

                coupledConstraintsIdxesHost.at(index) = splitMap.at(c2);

                int  center = c1a1;

                real sqrtmu1 = 1.0/sqrt(md.invmass[c1a1] + md.invmass[c1a2]);
                real sqrtmu2 = 1.0/sqrt(md.invmass[c2a1] + md.invmass[c2a2]);

                massFactorsHost.at(index) = -sign*md.invmass[center]*sqrtmu1*sqrtmu2;

                coupledConstraintsCountsHost.at(splitMap.at(c1))++;

            }
        }

        c2a1 = c1a2;
        for (unsigned j = 0; j < atomsAdjacencyList.at(c1a2).size(); j++)
        {

            std::tie(c2a2, c2, sign) = atomsAdjacencyList.at(c1a2).at(j);

            if (c1 != c2)
            {
                int index = nConstraintsThreads*coupledConstraintsCountsHost.at(splitMap.at(c1)) + splitMap.at(c1);

                coupledConstraintsIdxesHost.at(index) = splitMap.at(c2);

                int  center = c1a2;

                real sqrtmu1 = 1.0/sqrt(md.invmass[c1a1] + md.invmass[c1a2]);
                real sqrtmu2 = 1.0/sqrt(md.invmass[c2a1] + md.invmass[c2a2]);

                massFactorsHost.at(index) = sign*md.invmass[center]*sqrtmu1*sqrtmu2;

                coupledConstraintsCountsHost.at(splitMap.at(c1))++;

            }
        }
    }

    if (nConstraint > maxConstraintsNumberSoFar)
    {

        if (maxConstraintsNumberSoFar > 0)
        {
            hipFree(invmassDevice);
            hipFree(mlambdaDevice);

            hipFree(constraintsDevice);
            hipFree(constraintsR0Device);

            hipFree(coupledConstraintsCountsDevice);
            hipFree(coupledConstraintsIdxesDevice);
            hipFree(massFactorsDevice);
            hipFree(matrixADevice);

        }
        maxConstraintsNumberSoFar = nConstraint;

        hipMalloc(&invmassDevice, nAtom*sizeof(real));
        hipMalloc(&mlambdaDevice, nConstraintsThreads*sizeof(float));

        hipMalloc(&constraintsDevice, nConstraintsThreads*sizeof(int2));
        hipMalloc(&constraintsR0Device, nConstraintsThreads*sizeof(float));

        hipMalloc(&coupledConstraintsCountsDevice, nConstraintsThreads*sizeof(int));
        hipMalloc(&coupledConstraintsIdxesDevice, maxCoupledConstraints*nConstraintsThreads*sizeof(int));
        hipMalloc(&massFactorsDevice, maxCoupledConstraints*nConstraintsThreads*sizeof(float));
        hipMalloc(&matrixADevice, maxCoupledConstraints*nConstraintsThreads*sizeof(float));

    }

    hipMemcpy(constraintsDevice, constraintsHost.data(), nConstraintsThreads*sizeof(int2), hipMemcpyHostToDevice);
    hipMemcpy(constraintsR0Device, constraintsR0Host.data(), nConstraintsThreads*sizeof(float), hipMemcpyHostToDevice);

    hipMemcpy(coupledConstraintsCountsDevice, coupledConstraintsCountsHost.data(),
               nConstraintsThreads*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(coupledConstraintsIdxesDevice, coupledConstraintsIdxesHost.data(),
               maxCoupledConstraints*nConstraintsThreads*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(massFactorsDevice, massFactorsHost.data(), maxCoupledConstraints*nConstraintsThreads*sizeof(float), hipMemcpyHostToDevice);

    cudaCheckError();

    GMX_ASSERT(md.invmass != nullptr, "Masses of attoms should be specified.\n");
    hipMemcpy(invmassDevice, md.invmass, nAtom*sizeof(real), hipMemcpyHostToDevice);

    cudaCheckError();

}

/*! \brief
 * Update PBC data.
 *
 * Converts pbc data from t_pbc into the PbcAiuc format and stores the latter.
 *
 * \param[in] *pbc The PBC data in t_pbc format.
 */
void LincsCuda::Impl::setPbc(t_pbc *pbc)
{
    setPbcAiuc(pbc->ndim_ePBC, pbc->box, &pbcAiuc);
}

/*! \brief
 * Copy coordinates and velocities from provided CPU location to GPU.
 *
 * Copies the coordinates before the integration step (x), coordinates
 * after the integration step (xp) and velocities (v) from the provided
 * CPU location to GPU. The data are assumed to be in float3/fvec format
 * (single precision).
 *
 * \param[in] *x  CPU pointer where coordinates should be copied from.
 * \param[in] *xp CPU pointer where coordinates should be copied from.
 * \param[in] *v  CPU pointer where velocities should be copied from.
 */
void LincsCuda::Impl::copyCoordinatesToGpu(const rvec * x, const rvec * xp, const rvec * v)
{
    hipMemcpy(xDevice, x, nAtom*sizeof(float3), hipMemcpyHostToDevice);
    cudaCheckError();
    hipMemcpy(xpDevice, xp, nAtom*sizeof(float3), hipMemcpyHostToDevice);
    cudaCheckError();
    if (v != nullptr)
    {
        hipMemcpy(vDevice, v, nAtom*sizeof(float3), hipMemcpyHostToDevice);
    }

    cudaCheckError();
    hipDeviceSynchronize();
}

/*! \brief
 * Copy coordinates from GPU to provided CPU location.
 *
 * Copies the constrained coordinates to the provided location. The coordinates
 * are assumed to be in float3/fvec format (single precision).
 *
 * \param[out] *xp CPU pointer where coordinates should be copied to.
 */
void LincsCuda::Impl::copyCoordinatesFromGpu(rvec * xp)
{
    hipMemcpy(xp, xpDevice, nAtom*sizeof(float3), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    cudaCheckError();
}

/*! \brief
 * Copy velocities from GPU to provided CPU location.
 *
 * The velocities are assumed to be in float3/fvec format (single precision).
 *
 * \param[in] *v  Pointer to velocities data.
 */
void LincsCuda::Impl::copyVelocitiesFromGpu(rvec * v)
{
    hipMemcpy(v, vDevice, nAtom*sizeof(float3), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    cudaCheckError();
}

/*! \brief
 * Set the internal GPU-memory x, xprime and v pointers.
 *
 * Data is not copied. The data are assumed to be in float3/fvec format
 * (float3 is used internally, but the data layout should be identical).
 *
 * \param[in] *xDevice  Pointer to the coordinates before integrator update (on GPU)
 * \param[in] *xpDevice Pointer to the coordinates after integrator update, before update (on GPU)
 * \param[in] *vDevice  Pointer to the velocities before integrator update (on GPU)
 */
void LincsCuda::Impl::setXVPointers(rvec * xDevice, rvec * xpDevice, rvec * vDevice)
{
    this->xDevice  = (float3*)xDevice;
    this->xpDevice = (float3*)xpDevice;
    this->vDevice  = (float3*)vDevice;
}


LincsCuda::LincsCuda(int nAtom,
                     int nIter,
                     int nOrder)
    : impl_(new Impl(nAtom, nIter, nOrder))
{
}

LincsCuda::~LincsCuda() = default;

void LincsCuda::apply(bool       updateVelocities,
                      real       invdt,
                      gmx_bool   bCalcVir,
                      tensor     virialScaled)
{
    impl_->apply(updateVelocities,
                 invdt,
                 bCalcVir,
                 virialScaled);
}

void LincsCuda::setPbc(t_pbc *pbc)
{
    impl_->setPbc(pbc);
}

void LincsCuda::set(const t_idef    &idef,
                    const t_mdatoms &md)
{
    impl_->set(idef, md);
}

void LincsCuda::copyCoordinatesToGpu(const rvec *x, const rvec *xp, const rvec *v)
{
    impl_->copyCoordinatesToGpu(x, xp, v);
}

void LincsCuda::copyCoordinatesFromGpu(rvec * xp)
{
    impl_->copyCoordinatesFromGpu(xp);
}

void LincsCuda::copyVelocitiesFromGpu(rvec * v)
{
    impl_->copyVelocitiesFromGpu(v);
}

void LincsCuda::setXVPointers(rvec * xDevice, rvec * xpDevice, rvec * vDevice)
{
    impl_->setXVPointers(xDevice, xpDevice, vDevice);
}
