#include "hip/hip_runtime.h"
/*
 * This file is part of the GROMACS molecular simulation package.
 *
 * Copyright (c) 2016,2017,2018, by the GROMACS development team, led by
 * Mark Abraham, David van der Spoel, Berk Hess, and Erik Lindahl,
 * and including many others, as listed in the AUTHORS file in the
 * top-level source directory and at http://www.gromacs.org.
 *
 * GROMACS is free software; you can redistribute it and/or
 * modify it under the terms of the GNU Lesser General Public License
 * as published by the Free Software Foundation; either version 2.1
 * of the License, or (at your option) any later version.
 *
 * GROMACS is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
 * Lesser General Public License for more details.
 *
 * You should have received a copy of the GNU Lesser General Public
 * License along with GROMACS; if not, see
 * http://www.gnu.org/licenses, or write to the Free Software Foundation,
 * Inc., 51 Franklin Street, Fifth Floor, Boston, MA  02110-1301  USA.
 *
 * If you want to redistribute modifications to GROMACS, please
 * consider that scientific software is very special. Version
 * control is crucial - bugs must be traceable. We will be happy to
 * consider code for inclusion in the official distribution, but
 * derived work must not be called official GROMACS. Details are found
 * in the README & COPYING files - if they are missing, get the
 * official version at http://www.gromacs.org.
 *
 * To help us fund GROMACS development, we humbly ask that you cite
 * the research papers on the package. Check out http://www.gromacs.org.
 */

/*! \internal \file
 * \brief This file contains internal CUDA function implementations
 * for performing the PME calculations on GPU.
 *
 * \author Aleksei Iupinov <a.yupinov@gmail.com>
 */

#include "gmxpre.h"

#include <cmath>

/* The rest */
#include "pme.h"

#include "gromacs/gpu_utils/cudautils.cuh"
#include "gromacs/gpu_utils/devicebuffer.h"
#include "gromacs/gpu_utils/pmalloc_cuda.h"
#include "gromacs/utility/gmxassert.h"
#include "gromacs/utility/smalloc.h"

#include "pme.cuh"
#include "pme-3dfft.cuh"
#include "pme-grid.h"

int pme_gpu_get_atom_data_alignment(const PmeGpu *pmeGpu)
{
    const int order = pmeGpu->common->pme_order;
    GMX_ASSERT(order > 0, "Invalid PME order");
    return PME_ATOM_DATA_ALIGNMENT;
}

int pme_gpu_get_atoms_per_warp(const PmeGpu *pmeGpu)
{
    const int order = pmeGpu->common->pme_order;
    GMX_ASSERT(order > 0, "Invalid PME order");
    return PME_SPREADGATHER_ATOMS_PER_WARP;
}

void pme_gpu_synchronize(const PmeGpu *pmeGpu)
{
    hipError_t stat = hipStreamSynchronize(pmeGpu->archSpecific->pmeStream);
    CU_RET_ERR(stat, "Failed to synchronize the PME GPU stream!");
}

void pme_gpu_alloc_energy_virial(const PmeGpu *pmeGpu)
{
    const size_t energyAndVirialSize = c_virialAndEnergyCount * sizeof(float);
    hipError_t  stat                = hipMalloc((void **)&pmeGpu->kernelParams->constants.d_virialAndEnergy, energyAndVirialSize);
    CU_RET_ERR(stat, "hipMalloc failed on PME energy and virial");
    pmalloc((void **)&pmeGpu->staging.h_virialAndEnergy, energyAndVirialSize);
}

void pme_gpu_free_energy_virial(PmeGpu *pmeGpu)
{
    hipError_t stat = hipFree(pmeGpu->kernelParams->constants.d_virialAndEnergy);
    CU_RET_ERR(stat, "hipFree failed on PME energy and virial");
    pmeGpu->kernelParams->constants.d_virialAndEnergy = nullptr;
    pfree(pmeGpu->staging.h_virialAndEnergy);
    pmeGpu->staging.h_virialAndEnergy = nullptr;
}

void pme_gpu_clear_energy_virial(const PmeGpu *pmeGpu)
{
    clearDeviceBufferAsync(&pmeGpu->kernelParams->constants.d_virialAndEnergy, 0,
                           c_virialAndEnergyCount, pmeGpu->archSpecific->pmeStream);
}

void pme_gpu_realloc_and_copy_bspline_values(const PmeGpu *pmeGpu)
{
    const int splineValuesOffset[DIM] = {
        0,
        pmeGpu->kernelParams->grid.realGridSize[XX],
        pmeGpu->kernelParams->grid.realGridSize[XX] + pmeGpu->kernelParams->grid.realGridSize[YY]
    };
    memcpy((void *)&pmeGpu->kernelParams->grid.splineValuesOffset, &splineValuesOffset, sizeof(splineValuesOffset));

    const int newSplineValuesSize = pmeGpu->kernelParams->grid.realGridSize[XX] +
        pmeGpu->kernelParams->grid.realGridSize[YY] +
        pmeGpu->kernelParams->grid.realGridSize[ZZ];
    const bool shouldRealloc = (newSplineValuesSize > pmeGpu->archSpecific->splineValuesSize);
    reallocateDeviceBuffer(&pmeGpu->kernelParams->grid.d_splineModuli, newSplineValuesSize,
                           &pmeGpu->archSpecific->splineValuesSize, &pmeGpu->archSpecific->splineValuesSizeAlloc, pmeGpu->archSpecific->pmeStream);
    if (shouldRealloc)
    {
        /* Reallocate the host buffer */
        pfree(pmeGpu->staging.h_splineModuli);
        pmalloc((void **)&pmeGpu->staging.h_splineModuli, newSplineValuesSize * sizeof(float));
    }
    for (int i = 0; i < DIM; i++)
    {
        memcpy(pmeGpu->staging.h_splineModuli + splineValuesOffset[i], pmeGpu->common->bsp_mod[i].data(), pmeGpu->common->bsp_mod[i].size() * sizeof(float));
    }
    /* TODO: pin original buffer instead! */
    copyToDeviceBuffer(&pmeGpu->kernelParams->grid.d_splineModuli, pmeGpu->staging.h_splineModuli,
                       0, newSplineValuesSize,
                       pmeGpu->archSpecific->pmeStream, pmeGpu->settings.transferKind, nullptr);
}

void pme_gpu_free_bspline_values(const PmeGpu *pmeGpu)
{
    pfree(pmeGpu->staging.h_splineModuli);
    freeDeviceBuffer(&pmeGpu->kernelParams->grid.d_splineModuli);
}

void pme_gpu_realloc_forces(PmeGpu *pmeGpu)
{
    const size_t newForcesSize = pmeGpu->nAtomsAlloc * DIM;
    GMX_ASSERT(newForcesSize > 0, "Bad number of atoms in PME GPU");
    reallocateDeviceBuffer(&pmeGpu->kernelParams->atoms.d_forces, newForcesSize,
                           &pmeGpu->archSpecific->forcesSize, &pmeGpu->archSpecific->forcesSizeAlloc, pmeGpu->archSpecific->pmeStream);
    pmeGpu->staging.h_forces.reserve(pmeGpu->nAtomsAlloc);
    pmeGpu->staging.h_forces.resize(pmeGpu->kernelParams->atoms.nAtoms);
}

void pme_gpu_free_forces(const PmeGpu *pmeGpu)
{
    freeDeviceBuffer(&pmeGpu->kernelParams->atoms.d_forces);
}

void pme_gpu_copy_input_forces(PmeGpu *pmeGpu)
{
    GMX_ASSERT(pmeGpu->kernelParams->atoms.nAtoms > 0, "Bad number of atoms in PME GPU");
    float *h_forcesFloat = reinterpret_cast<float *>(pmeGpu->staging.h_forces.data());
    copyToDeviceBuffer(&pmeGpu->kernelParams->atoms.d_forces, h_forcesFloat,
                       0, DIM * pmeGpu->kernelParams->atoms.nAtoms,
                       pmeGpu->archSpecific->pmeStream, pmeGpu->settings.transferKind, nullptr);
}

void pme_gpu_copy_output_forces(PmeGpu *pmeGpu)
{
    GMX_ASSERT(pmeGpu->kernelParams->atoms.nAtoms > 0, "Bad number of atoms in PME GPU");
    float *h_forcesFloat = reinterpret_cast<float *>(pmeGpu->staging.h_forces.data());
    copyFromDeviceBuffer(h_forcesFloat, &pmeGpu->kernelParams->atoms.d_forces,
                         0, DIM * pmeGpu->kernelParams->atoms.nAtoms,
                         pmeGpu->archSpecific->pmeStream, pmeGpu->settings.transferKind, nullptr);
}

void pme_gpu_realloc_coordinates(const PmeGpu *pmeGpu)
{
    const size_t newCoordinatesSize = pmeGpu->nAtomsAlloc * DIM;
    GMX_ASSERT(newCoordinatesSize > 0, "Bad number of atoms in PME GPU");
    reallocateDeviceBuffer(&pmeGpu->kernelParams->atoms.d_coordinates, newCoordinatesSize,
                           &pmeGpu->archSpecific->coordinatesSize, &pmeGpu->archSpecific->coordinatesSizeAlloc, pmeGpu->archSpecific->pmeStream);
    if (c_usePadding)
    {
        const size_t paddingIndex = DIM * pmeGpu->kernelParams->atoms.nAtoms;
        const size_t paddingCount = DIM * pmeGpu->nAtomsAlloc - paddingIndex;
        if (paddingCount > 0)
        {
            clearDeviceBufferAsync(&pmeGpu->kernelParams->atoms.d_coordinates, paddingIndex,
                                   paddingCount, pmeGpu->archSpecific->pmeStream);
        }
    }
}

void pme_gpu_copy_input_coordinates(const PmeGpu *pmeGpu, const rvec *h_coordinates)
{
    GMX_ASSERT(h_coordinates, "Bad host-side coordinate buffer in PME GPU");
#if GMX_DOUBLE
    GMX_RELEASE_ASSERT(false, "Only single precision is supported");
    GMX_UNUSED_VALUE(h_coordinates);
#else
    const float *h_coordinatesFloat = reinterpret_cast<const float *>(h_coordinates);
    copyToDeviceBuffer(&pmeGpu->kernelParams->atoms.d_coordinates, h_coordinatesFloat,
                       0, pmeGpu->kernelParams->atoms.nAtoms * DIM,
                       pmeGpu->archSpecific->pmeStream, pmeGpu->settings.transferKind, nullptr);
#endif
}

void pme_gpu_free_coordinates(const PmeGpu *pmeGpu)
{
    freeDeviceBuffer(&pmeGpu->kernelParams->atoms.d_coordinates);
}

void pme_gpu_realloc_and_copy_input_coefficients(const PmeGpu *pmeGpu, const float *h_coefficients)
{
    GMX_ASSERT(h_coefficients, "Bad host-side charge buffer in PME GPU");
    const size_t newCoefficientsSize = pmeGpu->nAtomsAlloc;
    GMX_ASSERT(newCoefficientsSize > 0, "Bad number of atoms in PME GPU");
    reallocateDeviceBuffer(&pmeGpu->kernelParams->atoms.d_coefficients, newCoefficientsSize,
                           &pmeGpu->archSpecific->coefficientsSize, &pmeGpu->archSpecific->coefficientsSizeAlloc, pmeGpu->archSpecific->pmeStream);
    copyToDeviceBuffer(&pmeGpu->kernelParams->atoms.d_coefficients, const_cast<float *>(h_coefficients),
                       0, pmeGpu->kernelParams->atoms.nAtoms,
                       pmeGpu->archSpecific->pmeStream, pmeGpu->settings.transferKind, nullptr);
    if (c_usePadding)
    {
        const size_t paddingIndex = pmeGpu->kernelParams->atoms.nAtoms;
        const size_t paddingCount = pmeGpu->nAtomsAlloc - paddingIndex;
        if (paddingCount > 0)
        {
            clearDeviceBufferAsync(&pmeGpu->kernelParams->atoms.d_coefficients, paddingIndex,
                                   paddingCount, pmeGpu->archSpecific->pmeStream);
        }
    }
}

void pme_gpu_free_coefficients(const PmeGpu *pmeGpu)
{
    freeDeviceBuffer(&pmeGpu->kernelParams->atoms.d_coefficients);
}

void pme_gpu_realloc_spline_data(const PmeGpu *pmeGpu)
{
    const int    order             = pmeGpu->common->pme_order;
    const int    alignment         = pme_gpu_get_atoms_per_warp(pmeGpu);
    const size_t nAtomsPadded      = ((pmeGpu->nAtomsAlloc + alignment - 1) / alignment) * alignment;
    const int    newSplineDataSize = DIM * order * nAtomsPadded;
    GMX_ASSERT(newSplineDataSize > 0, "Bad number of atoms in PME GPU");
    /* Two arrays of the same size */
    const bool shouldRealloc        = (newSplineDataSize > pmeGpu->archSpecific->splineDataSize);
    int        currentSizeTemp      = pmeGpu->archSpecific->splineDataSize;
    int        currentSizeTempAlloc = pmeGpu->archSpecific->splineDataSizeAlloc;
    reallocateDeviceBuffer(&pmeGpu->kernelParams->atoms.d_theta, newSplineDataSize,
                           &currentSizeTemp, &currentSizeTempAlloc, pmeGpu->archSpecific->pmeStream);
    reallocateDeviceBuffer(&pmeGpu->kernelParams->atoms.d_dtheta, newSplineDataSize,
                           &pmeGpu->archSpecific->splineDataSize, &pmeGpu->archSpecific->splineDataSizeAlloc, pmeGpu->archSpecific->pmeStream);
    // the host side reallocation
    if (shouldRealloc)
    {
        pfree(pmeGpu->staging.h_theta);
        pmalloc((void **)&pmeGpu->staging.h_theta, newSplineDataSize * sizeof(float));
        pfree(pmeGpu->staging.h_dtheta);
        pmalloc((void **)&pmeGpu->staging.h_dtheta, newSplineDataSize * sizeof(float));
    }
}

void pme_gpu_free_spline_data(const PmeGpu *pmeGpu)
{
    /* Two arrays of the same size */
    freeDeviceBuffer(&pmeGpu->kernelParams->atoms.d_theta);
    freeDeviceBuffer(&pmeGpu->kernelParams->atoms.d_dtheta);
    pfree(pmeGpu->staging.h_theta);
    pfree(pmeGpu->staging.h_dtheta);
}

void pme_gpu_realloc_grid_indices(const PmeGpu *pmeGpu)
{
    const size_t newIndicesSize = DIM * pmeGpu->nAtomsAlloc;
    GMX_ASSERT(newIndicesSize > 0, "Bad number of atoms in PME GPU");
    reallocateDeviceBuffer(&pmeGpu->kernelParams->atoms.d_gridlineIndices, newIndicesSize,
                           &pmeGpu->archSpecific->gridlineIndicesSize, &pmeGpu->archSpecific->gridlineIndicesSizeAlloc, pmeGpu->archSpecific->pmeStream);
    pfree(pmeGpu->staging.h_gridlineIndices);
    pmalloc((void **)&pmeGpu->staging.h_gridlineIndices, newIndicesSize * sizeof(int));
}

void pme_gpu_free_grid_indices(const PmeGpu *pmeGpu)
{
    freeDeviceBuffer(&pmeGpu->kernelParams->atoms.d_gridlineIndices);
    pfree(pmeGpu->staging.h_gridlineIndices);
}

void pme_gpu_realloc_grids(PmeGpu *pmeGpu)
{
    auto     *kernelParamsPtr = pmeGpu->kernelParams.get();
    const int newRealGridSize = kernelParamsPtr->grid.realGridSizePadded[XX] *
        kernelParamsPtr->grid.realGridSizePadded[YY] *
        kernelParamsPtr->grid.realGridSizePadded[ZZ];
    const int newComplexGridSize = kernelParamsPtr->grid.complexGridSizePadded[XX] *
        kernelParamsPtr->grid.complexGridSizePadded[YY] *
        kernelParamsPtr->grid.complexGridSizePadded[ZZ] * 2;
    // Multiplied by 2 because we count complex grid size for complex numbers, but all allocations/pointers are float
    if (pmeGpu->archSpecific->performOutOfPlaceFFT)
    {
        /* 2 separate grids */
        reallocateDeviceBuffer(&kernelParamsPtr->grid.d_fourierGrid, newComplexGridSize,
                               &pmeGpu->archSpecific->complexGridSize, &pmeGpu->archSpecific->complexGridSizeAlloc, pmeGpu->archSpecific->pmeStream);
        reallocateDeviceBuffer(&kernelParamsPtr->grid.d_realGrid, newRealGridSize,
                               &pmeGpu->archSpecific->realGridSize, &pmeGpu->archSpecific->realGridSizeAlloc, pmeGpu->archSpecific->pmeStream);
    }
    else
    {
        /* A single buffer so that any grid will fit */
        const int newGridsSize = std::max(newRealGridSize, newComplexGridSize);
        reallocateDeviceBuffer(&kernelParamsPtr->grid.d_realGrid, newGridsSize,
                               &pmeGpu->archSpecific->realGridSize, &pmeGpu->archSpecific->realGridSizeAlloc, pmeGpu->archSpecific->pmeStream);
        kernelParamsPtr->grid.d_fourierGrid   = kernelParamsPtr->grid.d_realGrid;
        pmeGpu->archSpecific->complexGridSize = pmeGpu->archSpecific->realGridSize;
        // the size might get used later for copying the grid
    }
}

void pme_gpu_free_grids(const PmeGpu *pmeGpu)
{
    if (pmeGpu->archSpecific->performOutOfPlaceFFT)
    {
        freeDeviceBuffer(&pmeGpu->kernelParams->grid.d_fourierGrid);
    }
    freeDeviceBuffer(&pmeGpu->kernelParams->grid.d_realGrid);
}

void pme_gpu_clear_grids(const PmeGpu *pmeGpu)
{
    clearDeviceBufferAsync(&pmeGpu->kernelParams->grid.d_realGrid, 0,
                           pmeGpu->archSpecific->realGridSize, pmeGpu->archSpecific->pmeStream);
}

void pme_gpu_realloc_and_copy_fract_shifts(PmeGpu *pmeGpu)
{
    pme_gpu_free_fract_shifts(pmeGpu);

    auto        *kernelParamsPtr = pmeGpu->kernelParams.get();

    const int    nx                  = kernelParamsPtr->grid.realGridSize[XX];
    const int    ny                  = kernelParamsPtr->grid.realGridSize[YY];
    const int    nz                  = kernelParamsPtr->grid.realGridSize[ZZ];
    const int    cellCount           = c_pmeNeighborUnitcellCount;
    const int    gridDataOffset[DIM] = {0, cellCount * nx, cellCount * (nx + ny)};

    memcpy(kernelParamsPtr->grid.tablesOffsets, &gridDataOffset, sizeof(gridDataOffset));

    const int    newFractShiftsSize  = cellCount * (nx + ny + nz);

    initParamLookupTable(kernelParamsPtr->grid.d_fractShiftsTable,
                         kernelParamsPtr->fractShiftsTableTexture,
                         pmeGpu->common->fsh.data(),
                         newFractShiftsSize,
                         pmeGpu->deviceInfo);

    initParamLookupTable(kernelParamsPtr->grid.d_gridlineIndicesTable,
                         kernelParamsPtr->gridlineIndicesTableTexture,
                         pmeGpu->common->nn.data(),
                         newFractShiftsSize,
                         pmeGpu->deviceInfo);
}

void pme_gpu_free_fract_shifts(const PmeGpu *pmeGpu)
{
    auto *kernelParamsPtr = pmeGpu->kernelParams.get();
    destroyParamLookupTable(kernelParamsPtr->grid.d_fractShiftsTable,
                            kernelParamsPtr->fractShiftsTableTexture,
                            pmeGpu->deviceInfo);
    destroyParamLookupTable(kernelParamsPtr->grid.d_gridlineIndicesTable,
                            kernelParamsPtr->gridlineIndicesTableTexture,
                            pmeGpu->deviceInfo);
}

bool pme_gpu_stream_query(const PmeGpu *pmeGpu)
{
    return haveStreamTasksCompleted(pmeGpu->archSpecific->pmeStream);
}

void pme_gpu_copy_input_gather_grid(const PmeGpu *pmeGpu, float *h_grid)
{
    copyToDeviceBuffer(&pmeGpu->kernelParams->grid.d_realGrid, h_grid,
                       0, pmeGpu->archSpecific->realGridSize,
                       pmeGpu->archSpecific->pmeStream, pmeGpu->settings.transferKind, nullptr);
}

void pme_gpu_copy_output_spread_grid(const PmeGpu *pmeGpu, float *h_grid)
{
    copyFromDeviceBuffer(h_grid, &pmeGpu->kernelParams->grid.d_realGrid,
                         0, pmeGpu->archSpecific->realGridSize,
                         pmeGpu->archSpecific->pmeStream, pmeGpu->settings.transferKind, nullptr);
    hipError_t  stat = hipEventRecord(pmeGpu->archSpecific->syncSpreadGridD2H, pmeGpu->archSpecific->pmeStream);
    CU_RET_ERR(stat, "PME spread grid sync event record failure");
}

void pme_gpu_copy_output_spread_atom_data(const PmeGpu *pmeGpu)
{
    const int    alignment        = pme_gpu_get_atoms_per_warp(pmeGpu);
    const size_t nAtomsPadded     = ((pmeGpu->nAtomsAlloc + alignment - 1) / alignment) * alignment;
    const size_t splinesCount     = DIM * nAtomsPadded * pmeGpu->common->pme_order;
    auto        *kernelParamsPtr  = pmeGpu->kernelParams.get();
    copyFromDeviceBuffer(pmeGpu->staging.h_dtheta, &kernelParamsPtr->atoms.d_dtheta,
                         0, splinesCount,
                         pmeGpu->archSpecific->pmeStream, pmeGpu->settings.transferKind, nullptr);
    copyFromDeviceBuffer(pmeGpu->staging.h_theta, &kernelParamsPtr->atoms.d_theta,
                         0, splinesCount,
                         pmeGpu->archSpecific->pmeStream, pmeGpu->settings.transferKind, nullptr);
    copyFromDeviceBuffer(pmeGpu->staging.h_gridlineIndices, &kernelParamsPtr->atoms.d_gridlineIndices,
                         0, kernelParamsPtr->atoms.nAtoms * DIM,
                         pmeGpu->archSpecific->pmeStream, pmeGpu->settings.transferKind, nullptr);
}

void pme_gpu_copy_input_gather_atom_data(const PmeGpu *pmeGpu)
{
    const int    alignment       = pme_gpu_get_atoms_per_warp(pmeGpu);
    const size_t nAtomsPadded    = ((pmeGpu->nAtomsAlloc + alignment - 1) / alignment) * alignment;
    const size_t splinesCount    = DIM * nAtomsPadded * pmeGpu->common->pme_order;
    auto        *kernelParamsPtr = pmeGpu->kernelParams.get();
    if (c_usePadding)
    {
        // TODO: could clear only the padding and not the whole thing, but this is a test-exclusive code anyway
        clearDeviceBufferAsync(&kernelParamsPtr->atoms.d_gridlineIndices, 0,
                               pmeGpu->nAtomsAlloc * DIM, pmeGpu->archSpecific->pmeStream);
        clearDeviceBufferAsync(&kernelParamsPtr->atoms.d_dtheta, 0,
                               pmeGpu->nAtomsAlloc * pmeGpu->common->pme_order * DIM, pmeGpu->archSpecific->pmeStream);
        clearDeviceBufferAsync(&kernelParamsPtr->atoms.d_theta, 0,
                               pmeGpu->nAtomsAlloc * pmeGpu->common->pme_order * DIM, pmeGpu->archSpecific->pmeStream);
    }
    copyToDeviceBuffer(&kernelParamsPtr->atoms.d_dtheta, pmeGpu->staging.h_dtheta,
                       0, splinesCount,
                       pmeGpu->archSpecific->pmeStream, pmeGpu->settings.transferKind, nullptr);
    copyToDeviceBuffer(&kernelParamsPtr->atoms.d_theta, pmeGpu->staging.h_theta,
                       0, splinesCount,
                       pmeGpu->archSpecific->pmeStream, pmeGpu->settings.transferKind, nullptr);
    copyToDeviceBuffer(&kernelParamsPtr->atoms.d_gridlineIndices, pmeGpu->staging.h_gridlineIndices,
                       0, kernelParamsPtr->atoms.nAtoms * DIM,
                       pmeGpu->archSpecific->pmeStream, pmeGpu->settings.transferKind, nullptr);
}

void pme_gpu_sync_spread_grid(const PmeGpu *pmeGpu)
{
    hipError_t stat = hipEventSynchronize(pmeGpu->archSpecific->syncSpreadGridD2H);
    CU_RET_ERR(stat, "Error while waiting for the PME GPU spread grid to be copied to the host");
}

void pme_gpu_init_internal(PmeGpu *pmeGpu)
{
    /* Allocate the target-specific structures */
    pmeGpu->archSpecific.reset(new PmeGpuSpecific());
    pmeGpu->kernelParams.reset(new PmeGpuKernelParams());

    pmeGpu->archSpecific->performOutOfPlaceFFT = true;
    /* This should give better performance, according to the cuFFT documentation.
     * The performance seems to be the same though.
     * TODO: PME could also try to pick up nice grid sizes (with factors of 2, 3, 5, 7).
     */

    /* WARNING: CUDA timings are incorrect with multiple streams.
     *          This is the main reason why they are disabled by default.
     */
    // TODO: Consider turning on by default when we can detect nr of streams.
    pmeGpu->archSpecific->useTiming = (getenv("GMX_ENABLE_GPU_TIMING") != nullptr);

    // Prepare to use the device that this PME task was assigned earlier.
    CU_RET_ERR(hipSetDevice(pmeGpu->deviceInfo->id), "Switching to PME CUDA device");

    pmeGpu->maxGridWidthX = pmeGpu->deviceInfo->prop.maxGridSize[0];

    /* Creating a PME CUDA stream */
    hipError_t stat;
    int         highest_priority, lowest_priority;
    stat = hipDeviceGetStreamPriorityRange(&lowest_priority, &highest_priority);
    CU_RET_ERR(stat, "PME hipDeviceGetStreamPriorityRange failed");
    stat = hipStreamCreateWithPriority(&pmeGpu->archSpecific->pmeStream,
                                        hipStreamDefault, //hipStreamNonBlocking,
                                        highest_priority);
    CU_RET_ERR(stat, "hipStreamCreateWithPriority on the PME stream failed");
}

void pme_gpu_destroy_specific(const PmeGpu *pmeGpu)
{
    /* Destroy the CUDA stream */
    hipError_t stat = hipStreamDestroy(pmeGpu->archSpecific->pmeStream);
    CU_RET_ERR(stat, "PME hipStreamDestroy error");
}

void pme_gpu_init_sync_events(const PmeGpu *pmeGpu)
{
    const auto  eventFlags = hipEventDisableTiming;
    CU_RET_ERR(hipEventCreateWithFlags(&pmeGpu->archSpecific->syncSpreadGridD2H, eventFlags), "hipEventCreate on syncSpreadGridD2H failed");
}

void pme_gpu_destroy_sync_events(const PmeGpu *pmeGpu)
{
    CU_RET_ERR(hipEventDestroy(pmeGpu->archSpecific->syncSpreadGridD2H), "hipEventDestroy failed on syncSpreadGridD2H");
}

void pme_gpu_reinit_3dfft(const PmeGpu *pmeGpu)
{
    if (pme_gpu_performs_FFT(pmeGpu))
    {
        pmeGpu->archSpecific->fftSetup.resize(0);
        for (int i = 0; i < pmeGpu->common->ngrids; i++)
        {
            pmeGpu->archSpecific->fftSetup.push_back(std::unique_ptr<GpuParallel3dFft>(new GpuParallel3dFft(pmeGpu)));
        }
    }
}

void pme_gpu_destroy_3dfft(const PmeGpu *pmeGpu)
{
    pmeGpu->archSpecific->fftSetup.resize(0);
}

int getSplineParamFullIndex(int order, int splineIndex, int dimIndex, int warpIndex, int atomWarpIndex)
{
    if (order != 4)
    {
        throw order;
    }
    constexpr int fixedOrder = 4;
    GMX_UNUSED_VALUE(fixedOrder);
    const int     indexBase  = getSplineParamIndexBase<fixedOrder>(warpIndex, atomWarpIndex);
    return getSplineParamIndex<fixedOrder>(indexBase, dimIndex, splineIndex);
}
