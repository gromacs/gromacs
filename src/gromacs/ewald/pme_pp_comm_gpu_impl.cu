/*
 * This file is part of the GROMACS molecular simulation package.
 *
 * Copyright (c) 2019, by the GROMACS development team, led by
 * Mark Abraham, David van der Spoel, Berk Hess, and Erik Lindahl,
 * and including many others, as listed in the AUTHORS file in the
 * top-level source directory and at http://www.gromacs.org.
 *
 * GROMACS is free software; you can redistribute it and/or
 * modify it under the terms of the GNU Lesser General Public License
 * as published by the Free Software Foundation; either version 2.1
 * of the License, or (at your option) any later version.
 *
 * GROMACS is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
 * Lesser General Public License for more details.
 *
 * You should have received a copy of the GNU Lesser General Public
 * License along with GROMACS; if not, see
 * http://www.gnu.org/licenses, or write to the Free Software Foundation,
 * Inc., 51 Franklin Street, Fifth Floor, Boston, MA  02110-1301  USA.
 *
 * If you want to redistribute modifications to GROMACS, please
 * consider that scientific software is very special. Version
 * control is crucial - bugs must be traceable. We will be happy to
 * consider code for inclusion in the official distribution, but
 * derived work must not be called official GROMACS. Details are found
 * in the README & COPYING files - if they are missing, get the
 * official version at http://www.gromacs.org.
 *
 * To help us fund GROMACS development, we humbly ask that you cite
 * the research papers on the package. Check out http://www.gromacs.org.
 */
/*! \internal \file
 *
 * \brief Implements PME-PP communication using CUDA
 *
 *
 * \author Alan Gray <alang@nvidia.com>
 *
 * \ingroup module_ewald
 */
#include "gmxpre.h"

#include "pme_pp_comm_gpu_impl.h"

#include "config.h"

#include "gromacs/gpu_utils/cudautils.cuh"
#include "gromacs/gpu_utils/devicebuffer.h"
#include "gromacs/gpu_utils/gpueventsynchronizer.cuh"
#include "gromacs/utility/gmxmpi.h"

namespace gmx
{

PmePpCommGpu::Impl::Impl(MPI_Comm comm, int pmeRank) : comm_(comm), pmeRank_(pmeRank)
{
    GMX_RELEASE_ASSERT(
            GMX_THREAD_MPI,
            "PME-PP GPU Communication is currently only supported with thread-MPI enabled");
    hipStreamCreate(&pmePpCommStream_);
}

PmePpCommGpu::Impl::~Impl() = default;

void PmePpCommGpu::Impl::reinit(int size)
{
    // This rank will access PME rank memory directly, so needs to receive the remote PME buffer addresses.
#if GMX_MPI
    MPI_Recv(&remotePmeXBuffer_, sizeof(void**), MPI_BYTE, pmeRank_, 0, comm_, MPI_STATUS_IGNORE);
    MPI_Recv(&remotePmeFBuffer_, sizeof(void**), MPI_BYTE, pmeRank_, 0, comm_, MPI_STATUS_IGNORE);

    // Reallocate buffer used for staging PME force on GPU
    reallocateDeviceBuffer(&d_pmeForces_, size, &d_pmeForcesSize_, &d_pmeForcesSizeAlloc_, nullptr);
#else
    GMX_UNUSED_VALUE(size);
#endif
    return;
}

// TODO make this asynchronous by splitting into this into
// launchRecvForceFromPmeCudaDirect() and sycnRecvForceFromPmeCudaDirect()
void PmePpCommGpu::Impl::receiveForceFromPmeCudaDirect(void* recvPtr, int recvSize, bool receivePmeForceToGpu)
{
#if GMX_MPI
    // Receive event from PME task and add to stream, to ensure pull of data doesn't
    // occur before PME force calc is completed
    GpuEventSynchronizer* pmeSync;
    MPI_Recv(&pmeSync, sizeof(GpuEventSynchronizer*), MPI_BYTE, pmeRank_, 0, comm_, MPI_STATUS_IGNORE);
    pmeSync->enqueueWaitEvent(pmePpCommStream_);

    // Pull force data from remote GPU
    void*       pmeForcePtr = receivePmeForceToGpu ? static_cast<void*>(d_pmeForces_) : recvPtr;
    hipError_t stat = hipMemcpyAsync(pmeForcePtr, remotePmeFBuffer_, recvSize * DIM * sizeof(float),
                                       hipMemcpyDefault, pmePpCommStream_);
    CU_RET_ERR(stat, "hipMemcpyAsync on Recv from PME CUDA direct data transfer failed");

    if (receivePmeForceToGpu)
    {
        // Record event to be enqueued in the GPU local buffer operations, to
        // satisfy dependency on receiving the PME force data before
        // reducing it with the other force contributions.
        forcesReadySynchronizer_.markEvent(pmePpCommStream_);
    }
    else
    {
        // Ensure CPU waits for PME forces to be copied before reducing
        // them with other forces on the CPU
        hipStreamSynchronize(pmePpCommStream_);
    }
#else
    GMX_UNUSED_VALUE(recvPtr);
    GMX_UNUSED_VALUE(recvSize);
    GMX_UNUSED_VALUE(receivePmeForceToGpu);
#endif
}

void PmePpCommGpu::Impl::sendCoordinatesToPmeCudaDirect(void* sendPtr,
                                                        int   sendSize,
                                                        bool gmx_unused sendPmeCoordinatesFromGpu,
                                                        GpuEventSynchronizer* coordinatesReadyOnDeviceEvent)
{
#if GMX_MPI
    // ensure stream waits until coordinate data is available on device
    coordinatesReadyOnDeviceEvent->enqueueWaitEvent(pmePpCommStream_);

    hipError_t stat = hipMemcpyAsync(remotePmeXBuffer_, sendPtr, sendSize * DIM * sizeof(float),
                                       hipMemcpyDefault, pmePpCommStream_);
    CU_RET_ERR(stat, "hipMemcpyAsync on Send to PME CUDA direct data transfer failed");

    // Record and send event to allow PME task to sync to above transfer before commencing force calculations
    pmeCoordinatesSynchronizer_.markEvent(pmePpCommStream_);
    GpuEventSynchronizer* pmeSync = &pmeCoordinatesSynchronizer_;
    MPI_Send(&pmeSync, sizeof(GpuEventSynchronizer*), MPI_BYTE, pmeRank_, 0, comm_);
#else
    GMX_UNUSED_VALUE(sendPtr);
    GMX_UNUSED_VALUE(sendSize);
    GMX_UNUSED_VALUE(sendPmeCoordinatesFromGpu);
    GMX_UNUSED_VALUE(coordinatesReadyOnDeviceEvent);
#endif
}
void* PmePpCommGpu::Impl::getGpuForceStagingPtr()
{
    return static_cast<void*>(d_pmeForces_);
}

void* PmePpCommGpu::Impl::getForcesReadySynchronizer()
{
    return static_cast<void*>(&forcesReadySynchronizer_);
}

PmePpCommGpu::PmePpCommGpu(MPI_Comm comm, int pmeRank) : impl_(new Impl(comm, pmeRank)) {}

PmePpCommGpu::~PmePpCommGpu() = default;

void PmePpCommGpu::reinit(int size)
{
    impl_->reinit(size);
}

void PmePpCommGpu::receiveForceFromPmeCudaDirect(void* recvPtr, int recvSize, bool receivePmeForceToGpu)
{
    impl_->receiveForceFromPmeCudaDirect(recvPtr, recvSize, receivePmeForceToGpu);
}

void PmePpCommGpu::sendCoordinatesToPmeCudaDirect(void*                 sendPtr,
                                                  int                   sendSize,
                                                  bool                  sendPmeCoordinatesFromGpu,
                                                  GpuEventSynchronizer* coordinatesReadyOnDeviceEvent)
{
    impl_->sendCoordinatesToPmeCudaDirect(sendPtr, sendSize, sendPmeCoordinatesFromGpu,
                                          coordinatesReadyOnDeviceEvent);
}

void* PmePpCommGpu::getGpuForceStagingPtr()
{
    return impl_->getGpuForceStagingPtr();
}

void* PmePpCommGpu::getForcesReadySynchronizer()
{
    return impl_->getForcesReadySynchronizer();
}

} // namespace gmx
