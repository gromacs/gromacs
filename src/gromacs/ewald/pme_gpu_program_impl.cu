#include "hip/hip_runtime.h"
/*
 * This file is part of the GROMACS molecular simulation package.
 *
 * Copyright 2018- The GROMACS Authors
 * and the project initiators Erik Lindahl, Berk Hess and David van der Spoel.
 * Consult the AUTHORS/COPYING files and https://www.gromacs.org for details.
 *
 * GROMACS is free software; you can redistribute it and/or
 * modify it under the terms of the GNU Lesser General Public License
 * as published by the Free Software Foundation; either version 2.1
 * of the License, or (at your option) any later version.
 *
 * GROMACS is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
 * Lesser General Public License for more details.
 *
 * You should have received a copy of the GNU Lesser General Public
 * License along with GROMACS; if not, see
 * https://www.gnu.org/licenses, or write to the Free Software Foundation,
 * Inc., 51 Franklin Street, Fifth Floor, Boston, MA  02110-1301  USA.
 *
 * If you want to redistribute modifications to GROMACS, please
 * consider that scientific software is very special. Version
 * control is crucial - bugs must be traceable. We will be happy to
 * consider code for inclusion in the official distribution, but
 * derived work must not be called official GROMACS. Details are found
 * in the README & COPYING files - if they are missing, get the
 * official version at https://www.gromacs.org.
 *
 * To help us fund GROMACS development, we humbly ask that you cite
 * the research papers on the package. Check out https://www.gromacs.org.
 */

/*! \internal \file
 * \brief
 * Implements PmeGpuProgramImpl, which stores permanent PME GPU context-derived data,
 * such as (compiled) kernel handles.
 *
 * \author Aleksei Iupinov <a.yupinov@gmail.com>
 * \ingroup module_ewald
 */
#include "gmxpre.h"

#include "pme_gpu_program_impl.h"

#include "pme_gpu_constants.h"
#include "pme_gpu_internal.h" // for GridOrdering enum
#include "pme_gpu_types_host.h"

// PME interpolation order
constexpr int c_pmeOrder = 4;
// These hardcoded spread/gather parameters refer to not-implemented PME GPU 2D decomposition in X/Y
constexpr bool c_wrapX = true;
constexpr bool c_wrapY = true;

constexpr int c_stateA = 0;
constexpr int c_stateB = 1;

//! PME CUDA kernels forward declarations. Kernels are documented in their respective files.
template<int order, bool computeSplines, bool spreadCharges, bool wrapX, bool wrapY, int mode, bool writeGlobal, ThreadsPerAtom threadsPerAtom>
__global__ void pme_spline_and_spread_kernel(PmeGpuKernelParamsBase kernelParams);

// Add extern declarations to inform that there will be a definition
// provided in another translation unit.
// clang-format off
extern template void __global__
pme_spline_and_spread_kernel<c_pmeOrder, true, true, c_wrapX, c_wrapY, 1, true, ThreadsPerAtom::Order>(const PmeGpuKernelParamsBase);
extern template void __global__
pme_spline_and_spread_kernel<c_pmeOrder, true, true, c_wrapX, c_wrapY, 1, true, ThreadsPerAtom::OrderSquared>(const PmeGpuKernelParamsBase);
extern template void __global__
pme_spline_and_spread_kernel<c_pmeOrder, true, false, c_wrapX, c_wrapY, 1, true, ThreadsPerAtom::Order>(const PmeGpuKernelParamsBase);
extern template void __global__
pme_spline_and_spread_kernel<c_pmeOrder, true, false, c_wrapX, c_wrapY, 1, true, ThreadsPerAtom::OrderSquared>(const PmeGpuKernelParamsBase);
extern template __global__ void
pme_spline_and_spread_kernel<c_pmeOrder, false, true, c_wrapX, c_wrapY, 1, true, ThreadsPerAtom::Order>(const PmeGpuKernelParamsBase);
extern template __global__ void
pme_spline_and_spread_kernel<c_pmeOrder, false, true, c_wrapX, c_wrapY, 1, true, ThreadsPerAtom::OrderSquared>(const PmeGpuKernelParamsBase);
extern template __global__ void
pme_spline_and_spread_kernel<c_pmeOrder, true, true, c_wrapX, c_wrapY, 1, false, ThreadsPerAtom::Order>(const PmeGpuKernelParamsBase);
extern template __global__ void
pme_spline_and_spread_kernel<c_pmeOrder, true, true, c_wrapX, c_wrapY, 1, false, ThreadsPerAtom::OrderSquared>(const PmeGpuKernelParamsBase);
extern template __global__ void
pme_spline_and_spread_kernel<c_pmeOrder, true, true, c_wrapX, c_wrapY, 2, true, ThreadsPerAtom::Order>(const PmeGpuKernelParamsBase);
extern template __global__ void
pme_spline_and_spread_kernel<c_pmeOrder, true, true, c_wrapX, c_wrapY, 2, true, ThreadsPerAtom::OrderSquared>(const PmeGpuKernelParamsBase);
extern template __global__ void
pme_spline_and_spread_kernel<c_pmeOrder, true, false, c_wrapX, c_wrapY, 2, true, ThreadsPerAtom::Order>(const PmeGpuKernelParamsBase);
extern template __global__ void
pme_spline_and_spread_kernel<c_pmeOrder, true, false, c_wrapX, c_wrapY, 2, true, ThreadsPerAtom::OrderSquared>(const PmeGpuKernelParamsBase);
extern template __global__ void
pme_spline_and_spread_kernel<c_pmeOrder, false, true, c_wrapX, c_wrapY, 2, true, ThreadsPerAtom::Order>(const PmeGpuKernelParamsBase);
extern template __global__ void
pme_spline_and_spread_kernel<c_pmeOrder, false, true, c_wrapX, c_wrapY, 2, true, ThreadsPerAtom::OrderSquared>(const PmeGpuKernelParamsBase);
extern template __global__ void
pme_spline_and_spread_kernel<c_pmeOrder, true, true, c_wrapX, c_wrapY, 2, false, ThreadsPerAtom::Order>(const PmeGpuKernelParamsBase);
extern template __global__ void
pme_spline_and_spread_kernel<c_pmeOrder, true, true, c_wrapX, c_wrapY, 2, false, ThreadsPerAtom::OrderSquared>(const PmeGpuKernelParamsBase);

template<GridOrdering gridOrdering, bool computeEnergyAndVirial, const int gridIndex> /* It is significantly slower to pass gridIndex as a kernel parameter */
__global__ void pme_solve_kernel(PmeGpuKernelParamsBase kernelParams);

// Add extern declarations to inform that there will be a definition
// provided in another translation unit.
// clang-format off
extern template __global__ void pme_solve_kernel<GridOrdering::XYZ, false, c_stateA>(const PmeGpuKernelParamsBase);
extern template __global__ void pme_solve_kernel<GridOrdering::XYZ, true, c_stateA>(const PmeGpuKernelParamsBase);
extern template __global__ void pme_solve_kernel<GridOrdering::YZX, false, c_stateA>(const PmeGpuKernelParamsBase);
extern template __global__ void pme_solve_kernel<GridOrdering::YZX, true, c_stateA>(const PmeGpuKernelParamsBase);
extern template __global__ void pme_solve_kernel<GridOrdering::XYZ, false, c_stateB>(const PmeGpuKernelParamsBase);
extern template __global__ void pme_solve_kernel<GridOrdering::XYZ, true, c_stateB>(const PmeGpuKernelParamsBase);
extern template __global__ void pme_solve_kernel<GridOrdering::YZX, false, c_stateB>(const PmeGpuKernelParamsBase);
extern template __global__ void pme_solve_kernel<GridOrdering::YZX, true, c_stateB>(const PmeGpuKernelParamsBase);
// clang-format on

template<int order, bool wrapX, bool wrapY, int nGrids, bool readGlobal, ThreadsPerAtom threadsPerAtom>
__global__ void pme_gather_kernel(PmeGpuKernelParamsBase kernelParams);

__global__ void nvshmemSignalKernel(PmeGpuKernelParamsBase kernelParams);

// Add extern declarations to inform that there will be a definition
// provided in another translation unit.
// clang-format off
extern template __global__ void pme_gather_kernel<c_pmeOrder, c_wrapX, c_wrapY, 1, true, ThreadsPerAtom::Order>        (const PmeGpuKernelParamsBase);
extern template __global__ void pme_gather_kernel<c_pmeOrder, c_wrapX, c_wrapY, 1, false, ThreadsPerAtom::Order>       (const PmeGpuKernelParamsBase);
extern template __global__ void pme_gather_kernel<c_pmeOrder, c_wrapX, c_wrapY, 1, true, ThreadsPerAtom::OrderSquared> (const PmeGpuKernelParamsBase);
extern template __global__ void pme_gather_kernel<c_pmeOrder, c_wrapX, c_wrapY, 1, false, ThreadsPerAtom::OrderSquared>(const PmeGpuKernelParamsBase);
extern template __global__ void pme_gather_kernel<c_pmeOrder, c_wrapX, c_wrapY, 2, true, ThreadsPerAtom::Order>          (const PmeGpuKernelParamsBase);
extern template __global__ void pme_gather_kernel<c_pmeOrder, c_wrapX, c_wrapY, 2, false, ThreadsPerAtom::Order>         (const PmeGpuKernelParamsBase);
extern template __global__ void pme_gather_kernel<c_pmeOrder, c_wrapX, c_wrapY, 2, true, ThreadsPerAtom::OrderSquared>   (const PmeGpuKernelParamsBase);
extern template __global__ void pme_gather_kernel<c_pmeOrder, c_wrapX, c_wrapY, 2, false, ThreadsPerAtom::OrderSquared>  (const PmeGpuKernelParamsBase);
// clang-format on

PmeGpuProgramImpl::PmeGpuProgramImpl(const DeviceContext& deviceContext) :
    deviceContext_(deviceContext)
{
    // kernel parameters
    warpSize_             = warp_size;
    spreadWorkGroupSize   = c_spreadMaxThreadsPerBlock;
    solveMaxWorkGroupSize = c_solveMaxThreadsPerBlock;
    gatherWorkGroupSize   = c_gatherMaxThreadsPerBlock;

    /* Not all combinations of the splineAndSpread, spline and Spread kernels are required
     * If only the spline (without the spread) then it does not make sense not to write the data to global memory
     * Similarly the spread kernel (without the spline) implies that we should read the spline data from global memory
     */
    // clang-format off
    splineAndSpreadKernelSingle                       = pme_spline_and_spread_kernel<c_pmeOrder, true, true, c_wrapX, c_wrapY, 1, false, ThreadsPerAtom::OrderSquared>;
    splineAndSpreadKernelThPerAtom4Single             = pme_spline_and_spread_kernel<c_pmeOrder, true, true, c_wrapX, c_wrapY, 1, false, ThreadsPerAtom::Order>;
    splineAndSpreadKernelWriteSplinesSingle           = pme_spline_and_spread_kernel<c_pmeOrder, true, true, c_wrapX, c_wrapY, 1, true, ThreadsPerAtom::OrderSquared>;
    splineAndSpreadKernelWriteSplinesThPerAtom4Single = pme_spline_and_spread_kernel<c_pmeOrder, true, true, c_wrapX, c_wrapY, 1, true, ThreadsPerAtom::Order>;
    splineKernelSingle                                = pme_spline_and_spread_kernel<c_pmeOrder, true, false, c_wrapX, c_wrapY, 1, true, ThreadsPerAtom::OrderSquared>;
    splineKernelThPerAtom4Single                      = pme_spline_and_spread_kernel<c_pmeOrder, true, false, c_wrapX, c_wrapY, 1, true, ThreadsPerAtom::Order>;
    spreadKernelSingle                                = pme_spline_and_spread_kernel<c_pmeOrder, false, true, c_wrapX, c_wrapY, 1, true, ThreadsPerAtom::OrderSquared>;
    spreadKernelThPerAtom4Single                      = pme_spline_and_spread_kernel<c_pmeOrder, false, true, c_wrapX, c_wrapY, 1, true, ThreadsPerAtom::Order>;
    splineAndSpreadKernelDual                         = pme_spline_and_spread_kernel<c_pmeOrder, true, true, c_wrapX, c_wrapY, 2, false, ThreadsPerAtom::OrderSquared>;
    splineAndSpreadKernelThPerAtom4Dual               = pme_spline_and_spread_kernel<c_pmeOrder, true, true, c_wrapX, c_wrapY, 2, false, ThreadsPerAtom::Order>;
    splineAndSpreadKernelWriteSplinesDual             = pme_spline_and_spread_kernel<c_pmeOrder, true, true, c_wrapX, c_wrapY, 2, true, ThreadsPerAtom::OrderSquared>;
    splineAndSpreadKernelWriteSplinesThPerAtom4Dual   = pme_spline_and_spread_kernel<c_pmeOrder, true, true, c_wrapX, c_wrapY, 2, true, ThreadsPerAtom::Order>;
    splineKernelDual                                  = pme_spline_and_spread_kernel<c_pmeOrder, true, false, c_wrapX, c_wrapY, 2, true, ThreadsPerAtom::OrderSquared>;
    splineKernelThPerAtom4Dual                        = pme_spline_and_spread_kernel<c_pmeOrder, true, false, c_wrapX, c_wrapY, 2, true, ThreadsPerAtom::Order>;
    spreadKernelDual                                  = pme_spline_and_spread_kernel<c_pmeOrder, false, true, c_wrapX, c_wrapY, 2, true, ThreadsPerAtom::OrderSquared>;
    spreadKernelThPerAtom4Dual                        = pme_spline_and_spread_kernel<c_pmeOrder, false, true, c_wrapX, c_wrapY, 2, true, ThreadsPerAtom::Order>;
    gatherKernelSingle                                = pme_gather_kernel<c_pmeOrder, c_wrapX, c_wrapY, 1, false, ThreadsPerAtom::OrderSquared>;
    gatherKernelThPerAtom4Single                      = pme_gather_kernel<c_pmeOrder, c_wrapX, c_wrapY, 1, false, ThreadsPerAtom::Order>;
    gatherKernelReadSplinesSingle                     = pme_gather_kernel<c_pmeOrder, c_wrapX, c_wrapY, 1, true, ThreadsPerAtom::OrderSquared>;
    gatherKernelReadSplinesThPerAtom4Single           = pme_gather_kernel<c_pmeOrder, c_wrapX, c_wrapY, 1, true, ThreadsPerAtom::Order>;
    gatherKernelDual                                  = pme_gather_kernel<c_pmeOrder, c_wrapX, c_wrapY, 2, false, ThreadsPerAtom::OrderSquared>;
    gatherKernelThPerAtom4Dual                        = pme_gather_kernel<c_pmeOrder, c_wrapX, c_wrapY, 2, false, ThreadsPerAtom::Order>;
    gatherKernelReadSplinesDual                       = pme_gather_kernel<c_pmeOrder, c_wrapX, c_wrapY, 2, true, ThreadsPerAtom::OrderSquared>;
    gatherKernelReadSplinesThPerAtom4Dual             = pme_gather_kernel<c_pmeOrder, c_wrapX, c_wrapY, 2, true, ThreadsPerAtom::Order>;
    solveXYZKernelA                                   = pme_solve_kernel<GridOrdering::XYZ, false, c_stateA>;
    solveXYZEnergyKernelA                             = pme_solve_kernel<GridOrdering::XYZ, true, c_stateA>;
    solveYZXKernelA                                   = pme_solve_kernel<GridOrdering::YZX, false, c_stateA>;
    solveYZXEnergyKernelA                             = pme_solve_kernel<GridOrdering::YZX, true, c_stateA>;
    solveXYZKernelB                                   = pme_solve_kernel<GridOrdering::XYZ, false, c_stateB>;
    solveXYZEnergyKernelB                             = pme_solve_kernel<GridOrdering::XYZ, true, c_stateB>;
    solveYZXKernelB                                   = pme_solve_kernel<GridOrdering::YZX, false, c_stateB>;
    solveYZXEnergyKernelB                             = pme_solve_kernel<GridOrdering::YZX, true, c_stateB>;

    nvshmemSignalKern = nvshmemSignalKernel;

    // clang-format on
}

PmeGpuProgramImpl::~PmeGpuProgramImpl() = default;
