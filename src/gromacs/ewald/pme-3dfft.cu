/*
 * This file is part of the GROMACS molecular simulation package.
 *
 * Copyright (c) 2016, by the GROMACS development team, led by
 * Mark Abraham, David van der Spoel, Berk Hess, and Erik Lindahl,
 * and including many others, as listed in the AUTHORS file in the
 * top-level source directory and at http://www.gromacs.org.
 *
 * GROMACS is free software; you can redistribute it and/or
 * modify it under the terms of the GNU Lesser General Public License
 * as published by the Free Software Foundation; either version 2.1
 * of the License, or (at your option) any later version.
 *
 * GROMACS is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
 * Lesser General Public License for more details.
 *
 * You should have received a copy of the GNU Lesser General Public
 * License along with GROMACS; if not, see
 * http://www.gnu.org/licenses, or write to the Free Software Foundation,
 * Inc., 51 Franklin Street, Fifth Floor, Boston, MA  02110-1301  USA.
 *
 * If you want to redistribute modifications to GROMACS, please
 * consider that scientific software is very special. Version
 * control is crucial - bugs must be traceable. We will be happy to
 * consider code for inclusion in the official distribution, but
 * derived work must not be called official GROMACS. Details are found
 * in the README & COPYING files - if they are missing, get the
 * official version at http://www.gromacs.org.
 *
 * To help us fund GROMACS development, we humbly ask that you cite
 * the research papers on the package. Check out http://www.gromacs.org.
 */

/*! \internal \file
 *  \brief Implements CUDA FFT routines for PME GPU.
 *
 *  \author Aleksei Iupinov <a.yupinov@gmail.com>
 */

#include "gmxpre.h"

#include <assert.h>

#include "gromacs/gpu_utils/cudautils.cuh"
#include "gromacs/utility/gmxassert.h"
#include "gromacs/utility/smalloc.h"

#include "pme.cuh"
#include "pme-gpu.h"

gmx_parallel_3dfft_gpu_t::gmx_parallel_3dfft_gpu_t(const pme_gpu_t *pmeGPU)
{
    for (int i = 0; i < DIM; i++)
    {
        this->nDataReal[i]   = pmeGPU->kernelParams.grid.localGridSize[i];
        this->sizeComplex[i] = this->sizeReal[i] = pmeGPU->kernelParams.grid.localGridSizePadded[i];
    }
    if (!pmeGPU->archSpecific->bOutOfPlaceFFT)
    {
        GMX_ASSERT(this->sizeComplex[ZZ] % 2 == 0, "Odd inplace cuFFT minor dimension");
    }
    this->sizeComplex[ZZ] /= 2;

    GMX_ASSERT(!pme_gpu_uses_dd(pmeGPU), "FFT decomposition not implemented");

    const int gridSizeComplex = this->sizeComplex[XX] * this->sizeComplex[YY] * this->sizeComplex[ZZ];
    const int gridSizeReal    = this->sizeReal[XX] * this->sizeReal[YY] * this->sizeReal[ZZ];

    memset(this->localOffset, 0, sizeof(this->localOffset)); //!

    this->realGrid = (hipfftReal *)pmeGPU->kernelParams.grid.realGrid;
    assert(this->realGrid);
    this->complexGrid = (hipfftComplex *)pmeGPU->kernelParams.grid.fourierGrid;

    /* Commented code for a simple 3D grid with no padding */
    /*
       result = hipfftPlan3d(&this->planR2C, this->ndataReal[XX], this->ndataReal[YY], this->ndataReal[ZZ], HIPFFT_R2C);
       if (result != HIPFFT_SUCCESS)
       gmx_fatal(FARGS, "hipfftPlan3d R2C error %d\n", result);

       result = hipfftPlan3d(&this->planC2R, this->ndataReal[XX], this->ndataReal[YY], this->ndataReal[ZZ], HIPFFT_C2R);
       if (result != HIPFFT_SUCCESS)
       gmx_fatal(FARGS, "hipfftPlan3d C2R error %d\n", result);
     */

    hipfftResult_t             result;
    const int                 rank = 3, batch = 1;
    result = hipfftPlanMany(&this->planR2C, rank, this->nDataReal,
                           this->sizeReal, 1, gridSizeReal,
                           this->sizeComplex, 1, gridSizeComplex,
                           HIPFFT_R2C,
                           batch);
    if (result != HIPFFT_SUCCESS)
    {
        gmx_fatal(FARGS, "hipfftPlanMany R2C error %d\n", result);
    }

    result = hipfftPlanMany(&this->planC2R, rank, this->nDataReal,
                           this->sizeComplex, 1, gridSizeComplex,
                           this->sizeReal, 1, gridSizeReal,
                           HIPFFT_C2R,
                           batch);
    if (result != HIPFFT_SUCCESS)
    {
        gmx_fatal(FARGS, "hipfftPlanMany C2R error %d\n", result);
    }

    hipStream_t s = pmeGPU->archSpecific->pmeStream;
    assert(s);
    result = hipfftSetStream(this->planR2C, s);
    if (result != HIPFFT_SUCCESS)
    {
        gmx_fatal(FARGS, "hipfftSetStream R2C error %d\n", result);
    }

    result = hipfftSetStream(this->planC2R, s);
    if (result != HIPFFT_SUCCESS)
    {
        gmx_fatal(FARGS, "hipfftSetStream C2R error %d\n", result);
    }
}

gmx_parallel_3dfft_gpu_t::~gmx_parallel_3dfft_gpu_t()
{
    hipfftResult_t result;
    result = hipfftDestroy(this->planR2C);
    if (result != HIPFFT_SUCCESS)
    {
        gmx_fatal(FARGS, "hipfftDestroy R2C error %d\n", result);
    }
    result = hipfftDestroy(this->planC2R);
    if (result != HIPFFT_SUCCESS)
    {
        gmx_fatal(FARGS, "hipfftDestroy C2R error %d\n", result);
    }
}

void gmx_parallel_3dfft_gpu_t::get_real_limits(ivec localNData, ivec localOffset, ivec localSize)
{
    if (localNData)
    {
        memcpy(localNData, this->nDataReal, sizeof(this->nDataReal));
    }
    if (localSize)
    {
        memcpy(localSize, this->sizeReal, sizeof(this->sizeReal));
    }
    if (localOffset)
    {
        memcpy(localOffset, this->localOffset, sizeof(this->localOffset));
    }
}

void gmx_parallel_3dfft_gpu_t::get_complex_limits(ivec localNData, ivec localOffset, ivec localSize)
{
    if (localNData)
    {
        memcpy(localNData, this->nDataReal, sizeof(this->nDataReal));
        localNData[ZZ] = localNData[ZZ] / 2 + 1;
    }
    if (localSize)
    {
        memcpy(localSize, this->sizeComplex, sizeof(this->sizeComplex));
    }
    if (localOffset)
    {
        memcpy(localOffset, this->localOffset, sizeof(this->localOffset));
    }
}

hipfftResult_t gmx_parallel_3dfft_gpu_t::perform_3dfft(gmx_fft_direction dir)
{
    hipfftResult_t result;
    if (dir == GMX_FFT_REAL_TO_COMPLEX)
    {
        result = hipfftExecR2C(this->planR2C, this->realGrid, this->complexGrid);
    }
    else
    {
        result = hipfftExecC2R(this->planC2R, this->complexGrid, this->realGrid);
    }
    return result;
}

void pme_gpu_3dfft(const pme_gpu_t *pmeGPU, gmx_fft_direction dir, int grid_index)
{
    int           timerId = (dir == GMX_FFT_REAL_TO_COMPLEX) ? gtPME_FFT_R2C : gtPME_FFT_C2R;
    pme_gpu_start_timing(pmeGPU, timerId);
    hipfftResult_t result = pmeGPU->archSpecific->pfft_setup_gpu[grid_index]->perform_3dfft(dir);
    pme_gpu_stop_timing(pmeGPU, timerId);
    if (result)
    {
        gmx_fatal(FARGS, "cuFFT %s error %d\n", (dir == GMX_FFT_REAL_TO_COMPLEX) ? "R2C" : "C2R", result);
    }
}
