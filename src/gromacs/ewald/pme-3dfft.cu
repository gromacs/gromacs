/*
 * This file is part of the GROMACS molecular simulation package.
 *
 * Copyright (c) 2016, by the GROMACS development team, led by
 * Mark Abraham, David van der Spoel, Berk Hess, and Erik Lindahl,
 * and including many others, as listed in the AUTHORS file in the
 * top-level source directory and at http://www.gromacs.org.
 *
 * GROMACS is free software; you can redistribute it and/or
 * modify it under the terms of the GNU Lesser General Public License
 * as published by the Free Software Foundation; either version 2.1
 * of the License, or (at your option) any later version.
 *
 * GROMACS is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
 * Lesser General Public License for more details.
 *
 * You should have received a copy of the GNU Lesser General Public
 * License along with GROMACS; if not, see
 * http://www.gnu.org/licenses, or write to the Free Software Foundation,
 * Inc., 51 Franklin Street, Fifth Floor, Boston, MA  02110-1301  USA.
 *
 * If you want to redistribute modifications to GROMACS, please
 * consider that scientific software is very special. Version
 * control is crucial - bugs must be traceable. We will be happy to
 * consider code for inclusion in the official distribution, but
 * derived work must not be called official GROMACS. Details are found
 * in the README & COPYING files - if they are missing, get the
 * official version at http://www.gromacs.org.
 *
 * To help us fund GROMACS development, we humbly ask that you cite
 * the research papers on the package. Check out http://www.gromacs.org.
 */

/*! \internal \file
 *  \brief Implements CUDA FFT routines for PME GPU.
 *
 *  \author Aleksei Iupinov <a.yupinov@gmail.com>
 */

#include "gmxpre.h"

#include "pme-3dfft.cuh"

#include "gromacs/utility/fatalerror.h"
#include "gromacs/utility/gmxassert.h"

#include "pme.cuh"
#include "pme-gpu-types.h"

gmx_parallel_3dfft_gpu_t::gmx_parallel_3dfft_gpu_t(const pme_gpu_t *pmeGPU)
{
    for (int i = 0; i < DIM; i++)
    {
        _nDataReal[i]   = pmeGPU->kernelParams.grid.localGridSize[i];
        _sizeComplex[i] = _sizeReal[i] = pmeGPU->kernelParams.grid.localGridSizePadded[i];
    }
    if (!pmeGPU->archSpecific->bOutOfPlaceFFT)
    {
        GMX_ASSERT(_sizeComplex[ZZ] % 2 == 0, "Odd inplace cuFFT minor dimension");
    }
    _sizeComplex[ZZ] /= 2;

    GMX_ASSERT(!pme_gpu_uses_dd(pmeGPU), "FFT decomposition not implemented");

    const int gridSizeComplex = _sizeComplex[XX] * _sizeComplex[YY] * _sizeComplex[ZZ];
    const int gridSizeReal    = _sizeReal[XX] * _sizeReal[YY] * _sizeReal[ZZ];

    memset(_localOffset, 0, sizeof(_localOffset)); //!

    _realGrid = (hipfftReal *)pmeGPU->kernelParams.grid.realGrid;
    assert(_realGrid);
    _complexGrid = (hipfftComplex *)pmeGPU->kernelParams.grid.fourierGrid;

    /* Commented code for a simple 3D grid with no padding */
    /*
       result = hipfftPlan3d(&_planR2C, _ndataReal[XX], _ndataReal[YY], _ndataReal[ZZ], HIPFFT_R2C);
       if (result != HIPFFT_SUCCESS)
       gmx_fatal(FARGS, "hipfftPlan3d R2C error %d\n", result);

       result = hipfftPlan3d(&_planC2R, _ndataReal[XX], _ndataReal[YY], _ndataReal[ZZ], HIPFFT_C2R);
       if (result != HIPFFT_SUCCESS)
       gmx_fatal(FARGS, "hipfftPlan3d C2R error %d\n", result);
     */

    hipfftResult_t             result;
    const int                 rank = 3, batch = 1;
    result = hipfftPlanMany(&_planR2C, rank, _nDataReal,
                           _sizeReal, 1, gridSizeReal,
                           _sizeComplex, 1, gridSizeComplex,
                           HIPFFT_R2C,
                           batch);
    if (result != HIPFFT_SUCCESS)
    {
        gmx_fatal(FARGS, "hipfftPlanMany R2C error %d\n", result);
    }

    result = hipfftPlanMany(&_planC2R, rank, _nDataReal,
                           _sizeComplex, 1, gridSizeComplex,
                           _sizeReal, 1, gridSizeReal,
                           HIPFFT_C2R,
                           batch);
    if (result != HIPFFT_SUCCESS)
    {
        gmx_fatal(FARGS, "hipfftPlanMany C2R error %d\n", result);
    }

    hipStream_t s = pmeGPU->archSpecific->pmeStream;
    assert(s);
    result = hipfftSetStream(_planR2C, s);
    if (result != HIPFFT_SUCCESS)
    {
        gmx_fatal(FARGS, "hipfftSetStream R2C error %d\n", result);
    }

    result = hipfftSetStream(_planC2R, s);
    if (result != HIPFFT_SUCCESS)
    {
        gmx_fatal(FARGS, "hipfftSetStream C2R error %d\n", result);
    }
}

gmx_parallel_3dfft_gpu_t::~gmx_parallel_3dfft_gpu_t()
{
    hipfftResult_t result;
    result = hipfftDestroy(_planR2C);
    if (result != HIPFFT_SUCCESS)
    {
        gmx_fatal(FARGS, "hipfftDestroy R2C error %d\n", result);
    }
    result = hipfftDestroy(_planC2R);
    if (result != HIPFFT_SUCCESS)
    {
        gmx_fatal(FARGS, "hipfftDestroy C2R error %d\n", result);
    }
}

void gmx_parallel_3dfft_gpu_t::get_real_limits(ivec localNData, ivec localOffset, ivec localSize)
{
    if (localNData)
    {
        memcpy(localNData, _nDataReal, sizeof(_nDataReal));
    }
    if (localSize)
    {
        memcpy(localSize, _sizeReal, sizeof(_sizeReal));
    }
    if (localOffset)
    {
        memcpy(localOffset, _localOffset, sizeof(_localOffset));
    }
}

void gmx_parallel_3dfft_gpu_t::get_complex_limits(ivec localNData, ivec localOffset, ivec localSize)
{
    if (localNData)
    {
        memcpy(localNData, _nDataReal, sizeof(_nDataReal));
        localNData[ZZ] = localNData[ZZ] / 2 + 1;
    }
    if (localSize)
    {
        memcpy(localSize, _sizeComplex, sizeof(_sizeComplex));
    }
    if (localOffset)
    {
        memcpy(localOffset, _localOffset, sizeof(_localOffset));
    }
}

hipfftResult_t gmx_parallel_3dfft_gpu_t::perform_3dfft(gmx_fft_direction dir)
{
    hipfftResult_t result;
    if (dir == GMX_FFT_REAL_TO_COMPLEX)
    {
        result = hipfftExecR2C(_planR2C, _realGrid, _complexGrid);
    }
    else
    {
        result = hipfftExecC2R(_planC2R, _complexGrid, _realGrid);
    }
    return result;
}

void pme_gpu_3dfft(const pme_gpu_t *pmeGPU, gmx_fft_direction dir, int grid_index)
{
    int           timerId = (dir == GMX_FFT_REAL_TO_COMPLEX) ? gtPME_FFT_R2C : gtPME_FFT_C2R;
    pme_gpu_start_timing(pmeGPU, timerId);
    hipfftResult_t result = pmeGPU->archSpecific->pfft_setup_gpu[grid_index]->perform_3dfft(dir);
    pme_gpu_stop_timing(pmeGPU, timerId);
    if (result)
    {
        gmx_fatal(FARGS, "cuFFT %s error %d\n", (dir == GMX_FFT_REAL_TO_COMPLEX) ? "R2C" : "C2R", result);
    }
}
