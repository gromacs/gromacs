/*
 * This file is part of the GROMACS molecular simulation package.
 *
 * Copyright (c) 2016, by the GROMACS development team, led by
 * Mark Abraham, David van der Spoel, Berk Hess, and Erik Lindahl,
 * and including many others, as listed in the AUTHORS file in the
 * top-level source directory and at http://www.gromacs.org.
 *
 * GROMACS is free software; you can redistribute it and/or
 * modify it under the terms of the GNU Lesser General Public License
 * as published by the Free Software Foundation; either version 2.1
 * of the License, or (at your option) any later version.
 *
 * GROMACS is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
 * Lesser General Public License for more details.
 *
 * You should have received a copy of the GNU Lesser General Public
 * License along with GROMACS; if not, see
 * http://www.gnu.org/licenses, or write to the Free Software Foundation,
 * Inc., 51 Franklin Street, Fifth Floor, Boston, MA  02110-1301  USA.
 *
 * If you want to redistribute modifications to GROMACS, please
 * consider that scientific software is very special. Version
 * control is crucial - bugs must be traceable. We will be happy to
 * consider code for inclusion in the official distribution, but
 * derived work must not be called official GROMACS. Details are found
 * in the README & COPYING files - if they are missing, get the
 * official version at http://www.gromacs.org.
 *
 * To help us fund GROMACS development, we humbly ask that you cite
 * the research papers on the package. Check out http://www.gromacs.org.
 */

/*! \internal \file
 *  \brief Implements CUDA FFT routines for PME GPU.
 *
 *  \author Aleksei Iupinov <a.yupinov@gmail.com>
 */

#include "gmxpre.h"

#include <assert.h>

#include <hipfft/hipfft.h>

#include "gromacs/gpu_utils/cudautils.cuh"
#include "gromacs/utility/gmxassert.h"
#include "gromacs/utility/smalloc.h"

#include "pme.cuh"
#include "pme-gpu.h"

struct gmx_parallel_3dfft_gpu
{
    ivec          nDataReal;
    ivec          sizeReal;
    ivec          sizeComplex;

    hipfftHandle   planR2C;
    hipfftHandle   planC2R;
    hipfftReal    *realGrid;
    hipfftComplex *complexGrid;

    /* unused */
    ivec          localOffset;
};

void pme_gpu_init_3dfft_plan(gmx_parallel_3dfft_gpu_t *pfftSetup, const pme_gpu_t *pmeGPU)
{
    hipfftResult_t            result;
    gmx_parallel_3dfft_gpu_t setup;
    snew(setup, 1);
    *pfftSetup = setup;

    for (int i = 0; i < DIM; i++)
    {
        setup->nDataReal[i]   = pmeGPU->kernelParams.grid.localGridSize[i];
        setup->sizeComplex[i] = setup->sizeReal[i] = pmeGPU->kernelParams.grid.localGridSizePadded[i];
    }
    if (!pmeGPU->archSpecific->bOutOfPlaceFFT)
    {
        GMX_ASSERT(setup->sizeComplex[ZZ] % 2 == 0, "Odd inplace cuFFT minor dimension");
    }
    setup->sizeComplex[ZZ] /= 2;

    GMX_ASSERT(!pme_gpu_uses_dd(pmeGPU), "FFT decomposition not implemented");

    const int gridSizeComplex = setup->sizeComplex[XX] * setup->sizeComplex[YY] * setup->sizeComplex[ZZ];
    const int gridSizeReal    = setup->sizeReal[XX] * setup->sizeReal[YY] * setup->sizeReal[ZZ];

    memset(setup->localOffset, 0, sizeof(setup->localOffset)); //!

    setup->realGrid = (hipfftReal *)pmeGPU->kernelParams.grid.realGrid;
    assert(setup->realGrid);
    setup->complexGrid = (hipfftComplex *)pmeGPU->kernelParams.grid.fourierGrid;

    /* Commented code for a simple 3D grid with no padding */
    /*
       result = hipfftPlan3d(&setup->planR2C, setup->ndataReal[XX], setup->ndataReal[YY], setup->ndataReal[ZZ], HIPFFT_R2C);
       if (result != HIPFFT_SUCCESS)
       gmx_fatal(FARGS, "hipfftPlan3d R2C error %d\n", result);

       result = hipfftPlan3d(&setup->planC2R, setup->ndataReal[XX], setup->ndataReal[YY], setup->ndataReal[ZZ], HIPFFT_C2R);
       if (result != HIPFFT_SUCCESS)
       gmx_fatal(FARGS, "hipfftPlan3d C2R error %d\n", result);
     */

    const int rank = 3, batch = 1;
    result = hipfftPlanMany(&setup->planR2C, rank, setup->nDataReal,
                           setup->sizeReal, 1, gridSizeReal,
                           setup->sizeComplex, 1, gridSizeComplex,
                           HIPFFT_R2C,
                           batch);
    if (result != HIPFFT_SUCCESS)
    {
        gmx_fatal(FARGS, "hipfftPlanMany R2C error %d\n", result);
    }

    result = hipfftPlanMany(&setup->planC2R, rank, setup->nDataReal,
                           setup->sizeComplex, 1, gridSizeComplex,
                           setup->sizeReal, 1, gridSizeReal,
                           HIPFFT_C2R,
                           batch);
    if (result != HIPFFT_SUCCESS)
    {
        gmx_fatal(FARGS, "hipfftPlanMany C2R error %d\n", result);
    }

    hipStream_t s = pmeGPU->archSpecific->pmeStream;
    assert(s);
    result = hipfftSetStream(setup->planR2C, s);
    if (result != HIPFFT_SUCCESS)
    {
        gmx_fatal(FARGS, "hipfftSetStream R2C error %d\n", result);
    }

    result = hipfftSetStream(setup->planC2R, s);
    if (result != HIPFFT_SUCCESS)
    {
        gmx_fatal(FARGS, "hipfftSetStream C2R error %d\n", result);
    }
}

void pme_gpu_get_3dfft_real_limits(gmx_parallel_3dfft_gpu_t      setup,
                                   ivec                          localNData,
                                   ivec                          localOffset,
                                   ivec                          localSize)
{
    if (localNData)
    {
        memcpy(localNData, setup->nDataReal, sizeof(setup->nDataReal));
    }
    if (localSize)
    {
        memcpy(localSize, setup->sizeReal, sizeof(setup->sizeReal));
    }
    if (localOffset)
    {
        memcpy(localOffset, setup->localOffset, sizeof(setup->localOffset));
    }
}

void pme_gpu_get_3dfft_complex_limits(const gmx_parallel_3dfft_gpu_t setup,
                                      ivec                           localNData,
                                      ivec                           localOffset,
                                      ivec                           localSize)
{
    if (localNData)
    {
        memcpy(localNData, setup->nDataReal, sizeof(setup->nDataReal));
        localNData[ZZ] = localNData[ZZ] / 2 + 1;
    }
    if (localSize)
    {
        memcpy(localSize, setup->sizeComplex, sizeof(setup->sizeComplex));
    }
    if (localOffset)
    {
        memcpy(localOffset, setup->localOffset, sizeof(setup->localOffset));
    }
}

void pme_gpu_3dfft(const pme_gpu_t        *pmeGPU,
                   gmx_fft_direction       dir,
                   const int               grid_index)
{
    gmx_parallel_3dfft_gpu_t setup = pmeGPU->archSpecific->pfft_setup_gpu[grid_index];

    if (dir == GMX_FFT_REAL_TO_COMPLEX)
    {
        pme_gpu_start_timing(pmeGPU, gtPME_FFT_R2C);
        hipfftResult_t result = hipfftExecR2C(setup->planR2C, setup->realGrid, setup->complexGrid);
        pme_gpu_stop_timing(pmeGPU, gtPME_FFT_R2C);
        if (result)
        {
            gmx_fatal(FARGS, "cufft R2C error %d\n", result);
        }
    }
    else
    {
        pme_gpu_start_timing(pmeGPU, gtPME_FFT_C2R);
        hipfftResult_t result = hipfftExecC2R(setup->planC2R, setup->complexGrid, setup->realGrid);
        pme_gpu_stop_timing(pmeGPU, gtPME_FFT_C2R);
        if (result)
        {
            gmx_fatal(FARGS, "cufft C2R error %d\n", result);
        }
    }
}

void pme_gpu_destroy_3dfft_plan(const gmx_parallel_3dfft_gpu_t &pfftSetup)
{
    if (pfftSetup)
    {
        hipfftResult_t result;

        result = hipfftDestroy(pfftSetup->planR2C);
        if (result != HIPFFT_SUCCESS)
        {
            gmx_fatal(FARGS, "hipfftDestroy R2C error %d\n", result);
        }
        result = hipfftDestroy(pfftSetup->planC2R);
        if (result != HIPFFT_SUCCESS)
        {
            gmx_fatal(FARGS, "hipfftDestroy C2R error %d\n", result);
        }

        sfree(pfftSetup);
    }
}
