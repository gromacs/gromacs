/*
 * This file is part of the GROMACS molecular simulation package.
 *
 * Copyright (c) 2016, by the GROMACS development team, led by
 * Mark Abraham, David van der Spoel, Berk Hess, and Erik Lindahl,
 * and including many others, as listed in the AUTHORS file in the
 * top-level source directory and at http://www.gromacs.org.
 *
 * GROMACS is free software; you can redistribute it and/or
 * modify it under the terms of the GNU Lesser General Public License
 * as published by the Free Software Foundation; either version 2.1
 * of the License, or (at your option) any later version.
 *
 * GROMACS is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
 * Lesser General Public License for more details.
 *
 * You should have received a copy of the GNU Lesser General Public
 * License along with GROMACS; if not, see
 * http://www.gnu.org/licenses, or write to the Free Software Foundation,
 * Inc., 51 Franklin Street, Fifth Floor, Boston, MA  02110-1301  USA.
 *
 * If you want to redistribute modifications to GROMACS, please
 * consider that scientific software is very special. Version
 * control is crucial - bugs must be traceable. We will be happy to
 * consider code for inclusion in the official distribution, but
 * derived work must not be called official GROMACS. Details are found
 * in the README & COPYING files - if they are missing, get the
 * official version at http://www.gromacs.org.
 *
 * To help us fund GROMACS development, we humbly ask that you cite
 * the research papers on the package. Check out http://www.gromacs.org.
 */

/*! \internal \file
 *  \brief Implements CUDA FFT routines for PME GPU.
 *
 *  \author Aleksei Iupinov <a.yupinov@gmail.com>
 */

#include "gmxpre.h"

#include <assert.h>

#include <hipfft/hipfft.h>

#include "gromacs/gpu_utils/cudautils.cuh"
#include "gromacs/utility/gmxassert.h"
#include "gromacs/utility/smalloc.h"

#include "pme.cuh"
#include "pme-gpu.h"

struct gmx_parallel_3dfft_gpu
{
    ivec          ndata_real;
    ivec          size_real;
    ivec          size_complex;

    hipfftHandle   planR2C;
    hipfftHandle   planC2R;
    hipfftReal    *realGrid;
    hipfftComplex *complexGrid;

    /* unused */
    ivec          local_offset;
};

void pme_gpu_init_3dfft(gmx_parallel_3dfft_gpu_t *pfft_setup, ivec ndata, const gmx_pme_t *pme)
{
    hipfftResult_t            result;
    gmx_parallel_3dfft_gpu_t setup;
    snew(setup, 1);

    setup->ndata_real[0] = ndata[XX];
    setup->ndata_real[1] = ndata[YY];
    setup->ndata_real[2] = ndata[ZZ];

    *pfft_setup = setup;

    if (!pme_gpu_uses_dd(pme))
    {
        ndata[XX] = pme->pmegrid_nx;
        ndata[YY] = pme->pmegrid_ny;
        ndata[ZZ] = pme->pmegrid_nz;
    }
    else
    {
        gmx_fatal(FARGS, "FFT decomposition not implemented");
    }

    memcpy(setup->size_real, ndata, sizeof(setup->size_real));

    memcpy(setup->size_complex, setup->size_real, sizeof(setup->size_real));
    GMX_RELEASE_ASSERT(setup->size_complex[ZZ] % 2 == 0, "Odd inplace cuFFT dimension size");
    setup->size_complex[ZZ] /= 2;
    // This is alright because Z includes overlap

    const int gridSizeComplex = setup->size_complex[XX] * setup->size_complex[YY] * setup->size_complex[ZZ];
    const int gridSizeReal    = setup->size_real[XX] * setup->size_real[YY] * setup->size_real[ZZ];

    memset(setup->local_offset, 0, sizeof(setup->local_offset)); //!

    setup->realGrid = (hipfftReal *)pme->gpu->kernelParams.grid.realGrid;
    assert(setup->realGrid);
    setup->complexGrid = (hipfftComplex *)pme->gpu->kernelParams.grid.fourierGrid;

    /* Commented code for a simple 3D grid with no padding */
    /*
       result = hipfftPlan3d(&setup->planR2C, setup->ndata_real[XX], setup->ndata_real[YY], setup->ndata_real[ZZ], HIPFFT_R2C);
       if (result != HIPFFT_SUCCESS)
       gmx_fatal(FARGS, "hipfftPlan3d R2C error %d\n", result);

       result = hipfftPlan3d(&setup->planC2R, setup->ndata_real[XX], setup->ndata_real[YY], setup->ndata_real[ZZ], HIPFFT_C2R);
       if (result != HIPFFT_SUCCESS)
       gmx_fatal(FARGS, "hipfftPlan3d C2R error %d\n", result);
     */

    const int rank = 3, batch = 1;
    result = hipfftPlanMany(&setup->planR2C, rank, setup->ndata_real,
                           setup->size_real, 1, gridSizeReal,
                           setup->size_complex, 1, gridSizeComplex,
                           HIPFFT_R2C,
                           batch);
    if (result != HIPFFT_SUCCESS)
    {
        gmx_fatal(FARGS, "hipfftPlanMany R2C error %d\n", result);
    }

    result = hipfftPlanMany(&setup->planC2R, rank, setup->ndata_real,
                           setup->size_complex, 1, gridSizeComplex,
                           setup->size_real, 1, gridSizeReal,
                           HIPFFT_C2R,
                           batch);
    if (result != HIPFFT_SUCCESS)
    {
        gmx_fatal(FARGS, "hipfftPlanMany C2R error %d\n", result);
    }

    hipStream_t s = pme->gpu->archSpecific->pmeStream;
    assert(s);
    result = hipfftSetStream(setup->planR2C, s);
    if (result != HIPFFT_SUCCESS)
    {
        gmx_fatal(FARGS, "hipfftSetStream R2C error %d\n", result);
    }

    result = hipfftSetStream(setup->planC2R, s);
    if (result != HIPFFT_SUCCESS)
    {
        gmx_fatal(FARGS, "hipfftSetStream C2R error %d\n", result);
    }
}

void pme_gpu_get_3dfft_real_limits(gmx_parallel_3dfft_gpu_t      setup,
                                   ivec                          local_ndata,
                                   ivec                          local_offset,
                                   ivec                          local_size)
{
    if (local_ndata)
    {
        memcpy(local_ndata, setup->ndata_real, sizeof(setup->ndata_real));
    }
    if (local_size)
    {
        memcpy(local_size, setup->size_real, sizeof(setup->size_real));
    }
    if (local_offset)
    {
        memcpy(local_offset, setup->local_offset, sizeof(setup->local_offset));
    }
}

void pme_gpu_get_3dfft_complex_limits(const gmx_parallel_3dfft_gpu_t setup,
                                      ivec                           local_ndata,
                                      ivec                           local_offset,
                                      ivec                           local_size)
{
    if (local_ndata)
    {
        memcpy(local_ndata, setup->ndata_real, sizeof(setup->ndata_real));
        local_ndata[ZZ] = local_ndata[ZZ] / 2 + 1;
    }
    if (local_size)
    {
        memcpy(local_size, setup->size_complex, sizeof(setup->size_complex));
    }
    if (local_offset)
    {
        memcpy(local_offset, setup->local_offset, sizeof(setup->local_offset));
    }
}

void pme_gpu_3dfft(gmx_pme_t        *pme,
                   gmx_fft_direction dir,
                   const int         grid_index)
{
    gmx_parallel_3dfft_gpu_t setup = pme->gpu->archSpecific->pfft_setup_gpu[grid_index];

    if (dir == GMX_FFT_REAL_TO_COMPLEX)
    {
        pme_gpu_start_timing(pme, gtPME_FFT_R2C);
        hipfftResult_t result = hipfftExecR2C(setup->planR2C, setup->realGrid, setup->complexGrid);
        pme_gpu_stop_timing(pme, gtPME_FFT_R2C);
        if (result)
        {
            gmx_fatal(FARGS, "cufft R2C error %d\n", result);
        }
    }
    else
    {
        pme_gpu_start_timing(pme, gtPME_FFT_C2R);
        hipfftResult_t result = hipfftExecC2R(setup->planC2R, setup->complexGrid, setup->realGrid);
        pme_gpu_stop_timing(pme, gtPME_FFT_C2R);
        if (result)
        {
            gmx_fatal(FARGS, "cufft C2R error %d\n", result);
        }
    }
}

void pme_gpu_destroy_3dfft(const gmx_parallel_3dfft_gpu_t &pfft_setup)
{
    if (pfft_setup)
    {
        hipfftResult_t result;

        result = hipfftDestroy(pfft_setup->planR2C);
        if (result != HIPFFT_SUCCESS)
        {
            gmx_fatal(FARGS, "hipfftDestroy R2C error %d\n", result);
        }
        result = hipfftDestroy(pfft_setup->planC2R);
        if (result != HIPFFT_SUCCESS)
        {
            gmx_fatal(FARGS, "hipfftDestroy C2R error %d\n", result);
        }

        sfree(pfft_setup);
    }
}
