#include "hip/hip_runtime.h"
/*
 * This file is part of the GROMACS molecular simulation package.
 *
 * Copyright (c) 2016,2017, by the GROMACS development team, led by
 * Mark Abraham, David van der Spoel, Berk Hess, and Erik Lindahl,
 * and including many others, as listed in the AUTHORS file in the
 * top-level source directory and at http://www.gromacs.org.
 *
 * GROMACS is free software; you can redistribute it and/or
 * modify it under the terms of the GNU Lesser General Public License
 * as published by the Free Software Foundation; either version 2.1
 * of the License, or (at your option) any later version.
 *
 * GROMACS is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
 * Lesser General Public License for more details.
 *
 * You should have received a copy of the GNU Lesser General Public
 * License along with GROMACS; if not, see
 * http://www.gnu.org/licenses, or write to the Free Software Foundation,
 * Inc., 51 Franklin Street, Fifth Floor, Boston, MA  02110-1301  USA.
 *
 * If you want to redistribute modifications to GROMACS, please
 * consider that scientific software is very special. Version
 * control is crucial - bugs must be traceable. We will be happy to
 * consider code for inclusion in the official distribution, but
 * derived work must not be called official GROMACS. Details are found
 * in the README & COPYING files - if they are missing, get the
 * official version at http://www.gromacs.org.
 *
 * To help us fund GROMACS development, we humbly ask that you cite
 * the research papers on the package. Check out http://www.gromacs.org.
 */

/*! \internal \file
 *  \brief Implements PME GPU Fourier grid solving in CUDA.
 *
 *  \author Aleksei Iupinov <a.yupinov@gmail.com>
 */

#include "gmxpre.h"

#include "config.h"

#include "gromacs/gpu_utils/cudautils.cuh"
#include "gromacs/utility/exceptions.h"
#include "gromacs/utility/gmxassert.h"

#include "pme.cuh"
#include "pme-timings.cuh"

// CUDA 6.5 can not compile enum class as a template kernel parameter,
// so we replace it with a duplicate simple enum
#if GMX_CUDA_VERSION >= 7000
using GridOrderingInternal = GridOrdering;
#else
enum GridOrderingInternal
{
    YZX,
    XYZ
};
#endif

/*! \brief
 * PME complex grid solver kernel function.
 *
 * \tparam[in] computeEnergyAndVirial   Tells if the reciprocal energy and virial should be computed.
 * \tparam[in] gridOrdering             Specifies the dimension ordering of the complex grid.
 * \param[in]  kernelParams             Input PME CUDA data in constant memory.
 */
template<
    bool computeEnergyAndVirial,
    GridOrderingInternal gridOrdering
    >
//__launch_bounds__(PME_SOLVE_THREADS_PER_BLOCK, PME_MIN_BLOCKS_PER_MP)
// FIXME: figure out when and why this produces "invalid launch argument"
__global__ void pme_solve_kernel(const struct pme_gpu_cuda_kernel_params_t kernelParams)
{
    /* This kernel supports 2 different dimension orderings: YZX and XYZ */
    int majorDim, middleDim, minorDim;
    switch (gridOrdering)
    {
        case GridOrderingInternal::YZX:
            majorDim  = YY;
            middleDim = ZZ;
            minorDim  = XX;
            break;

        case GridOrderingInternal::XYZ:
            majorDim  = XX;
            middleDim = YY;
            minorDim  = ZZ;
            break;

        default:
            assert(false);
    }

    /* Global memory pointers */
    const float * __restrict__ gm_splineValueMajor    = kernelParams.grid.d_splineModuli + kernelParams.grid.splineValuesOffset[majorDim];
    const float * __restrict__ gm_splineValueMiddle   = kernelParams.grid.d_splineModuli + kernelParams.grid.splineValuesOffset[middleDim];
    const float * __restrict__ gm_splineValueMinor    = kernelParams.grid.d_splineModuli + kernelParams.grid.splineValuesOffset[minorDim];
    float * __restrict__       gm_virialAndEnergy     = kernelParams.constants.d_virialAndEnergy;
    float2 * __restrict__      gm_grid                = (float2 *)kernelParams.grid.d_fourierGrid;

    /* Various grid sizes and indices */
    const int localOffsetMinor = 0, localOffsetMajor = 0, localOffsetMiddle = 0; //unused
    const int localSizeMinor   = kernelParams.grid.complexGridSizePadded[minorDim];
    const int localSizeMiddle  = kernelParams.grid.complexGridSizePadded[middleDim];
    const int localCountMajor  = kernelParams.grid.complexGridSize[majorDim];
    const int localCountMiddle = kernelParams.grid.complexGridSize[middleDim];
    const int localCountMinor  = kernelParams.grid.complexGridSize[minorDim];
    const int nMajor           = kernelParams.grid.realGridSize[majorDim];
    const int nMiddle          = kernelParams.grid.realGridSize[middleDim];
    const int nMinor           = kernelParams.grid.realGridSize[minorDim];
    const int maxkMajor        = (nMajor + 1) / 2;  // X or Y
    const int maxkMiddle       = (nMiddle + 1) / 2; // Y OR Z => only check for !YZX
    const int maxkMinor        = (nMinor + 1) / 2;  // Z or X => only check for YZX

    /* Each thread works on one cell of the Fourier space complex 3D grid (gm_grid).
     * Each block handles PME_SOLVE_THREADS_PER_BLOCK cells -
     * depending on the grid contiguous dimension size,
     * that can range from a part of a single gridline to several complete gridlines.
     */
    const int threadLocalId = (threadIdx.y * blockDim.x) + threadIdx.x;
    const int indexMinor    = blockIdx.x * blockDim.x + threadIdx.x;
    const int indexMiddle   = blockIdx.y * blockDim.y + threadIdx.y;
    const int indexMajor    = blockIdx.z * blockDim.z + threadIdx.z;

    /* Optional outputs */
    float energy = 0.0f;
    float virxx  = 0.0f;
    float virxy  = 0.0f;
    float virxz  = 0.0f;
    float viryy  = 0.0f;
    float viryz  = 0.0f;
    float virzz  = 0.0f;

    if ((indexMajor < localCountMajor) & (indexMiddle < localCountMiddle) & (indexMinor < localCountMinor))
    {
        /* The offset should be equal to the global thread index for coalesced access */
        const int            gridIndex     = (indexMajor * localSizeMiddle + indexMiddle) * localSizeMinor + indexMinor;
        float2 __restrict__ *gm_gridCell   = gm_grid + gridIndex;
        // TODO possibly reuse inlined indexing function from tests

        const int   kMajor = indexMajor + localOffsetMajor;
        /* Checking either X in XYZ, or Y in YZX cases */
        const float mMajor = (kMajor < maxkMajor) ? kMajor : (kMajor - nMajor);

        const int   kMiddle = indexMiddle + localOffsetMiddle;
        float       mMiddle = kMiddle;
        /* Checking Y in XYZ case */
        if (gridOrdering == GridOrderingInternal::XYZ)
        {
            mMiddle = (kMiddle < maxkMiddle) ? kMiddle : (kMiddle - nMiddle);
        }
        const int       kMinor        = localOffsetMinor + indexMinor;
        float           mMinor        = kMinor;
        /* Checking X in YZX case */
        if (gridOrdering == GridOrderingInternal::YZX)
        {
            mMinor = (kMinor < maxkMinor) ? kMinor : (kMinor - nMinor);
        }
        /* We should skip the k-space point (0,0,0) */
        const bool notZeroPoint  = (kMinor > 0) | (kMajor > 0) | (kMiddle > 0);

        float      mX, mY, mZ;
        switch (gridOrdering)
        {
            case GridOrderingInternal::YZX:
                mX = mMinor;
                mY = mMajor;
                mZ = mMiddle;
                break;

            case GridOrderingInternal::XYZ:
                mX = mMajor;
                mY = mMiddle;
                mZ = mMinor;
                break;

            default:
                assert(false);
        }

        /* 0.5 correction factor for the first and last components of a minor dimension */
        float corner_fac = 1.0f;
        switch (gridOrdering)
        {
            case GridOrderingInternal::YZX:
                if ((kMiddle == 0) | (kMiddle == maxkMiddle))
                {
                    corner_fac = 0.5f; //FIXME this actually depends on the FFT minor dim - should probably always take minor? Test with FFTW
                }
                break;

            case GridOrderingInternal::XYZ:
                if ((kMinor == 0) | (kMinor == maxkMinor))
                {
                    corner_fac = 0.5f;
                }
                break;

            default:
                assert(false);
        }

        if (notZeroPoint)
        {
            const float mhxk = mX * kernelParams.step.recipBox[XX][XX];
            const float mhyk = mX * kernelParams.step.recipBox[XX][YY] + mY * kernelParams.step.recipBox[YY][YY];
            const float mhzk = mX * kernelParams.step.recipBox[XX][ZZ] + mY * kernelParams.step.recipBox[YY][ZZ] + mZ * kernelParams.step.recipBox[ZZ][ZZ];

            const float m2k        = mhxk * mhxk + mhyk * mhyk + mhzk * mhzk;
            assert(m2k != 0.0f);
            float       denom = m2k * float(M_PI) * kernelParams.step.boxVolume * gm_splineValueMajor[kMajor] * gm_splineValueMiddle[kMiddle] * gm_splineValueMinor[kMinor];
            assert(!isnan(denom));
            assert(denom != 0.0f);
            const float   tmp1   = expf(-kernelParams.grid.ewaldFactor * m2k);
            const float   etermk = kernelParams.constants.elFactor * tmp1 / denom;

            float2        gridValue    = *gm_gridCell;
            const float2  oldGridValue = gridValue;
            gridValue.x   *= etermk;
            gridValue.y   *= etermk;
            *gm_gridCell   = gridValue;

            if (computeEnergyAndVirial)
            {
                const float tmp1k = 2.0f * (gridValue.x * oldGridValue.x + gridValue.y * oldGridValue.y);

                float       vfactor = (kernelParams.grid.ewaldFactor + 1.0f / m2k) * 2.0f;
                float       ets2    = corner_fac * tmp1k;
                energy = ets2;

                float ets2vf  = ets2 * vfactor;

                virxx   = ets2vf * mhxk * mhxk - ets2;
                virxy   = ets2vf * mhxk * mhyk;
                virxz   = ets2vf * mhxk * mhzk;
                viryy   = ets2vf * mhyk * mhyk - ets2;
                viryz   = ets2vf * mhyk * mhzk;
                virzz   = ets2vf * mhzk * mhzk - ets2;
            }
        }
    }

    if (computeEnergyAndVirial)
    {
        /* The energy and virial reduction */

        /* TODO: there should be a shuffle reduction variant here as well! */
        const int        blockSize = PME_SOLVE_ENERVIR_THREADS_PER_BLOCK;
        __shared__ float sm_virialAndEnergy[c_virialAndEnergyCount * blockSize];

        /* A 7-thread reduction in shared memory inspired by reduce_force_j_generic */
        if (threadLocalId < blockSize)
        {
            sm_virialAndEnergy[threadLocalId + 0 * blockSize] = virxx;
            sm_virialAndEnergy[threadLocalId + 1 * blockSize] = viryy;
            sm_virialAndEnergy[threadLocalId + 2 * blockSize] = virzz;
            sm_virialAndEnergy[threadLocalId + 3 * blockSize] = virxy;
            sm_virialAndEnergy[threadLocalId + 4 * blockSize] = virxz;
            sm_virialAndEnergy[threadLocalId + 5 * blockSize] = viryz;
            sm_virialAndEnergy[threadLocalId + 6 * blockSize] = energy;
        }
        __syncthreads();

        /* Reducing every component to fit into warp_size */
        for (int s = blockSize >> 1; s >= warp_size; s >>= 1)
        {
#pragma unroll
            for (int i = 0; i < c_virialAndEnergyCount; i++)
            {
                if (threadLocalId < s) // TODO: split per threads?
                {
                    sm_virialAndEnergy[i * blockSize + threadLocalId] += sm_virialAndEnergy[i * blockSize + threadLocalId + s];
                }
            }
            __syncthreads();
        }

        const int threadsPerComponent    = warp_size / c_virialAndEnergyCount; // this is also the stride, will be 32 / 7 = 4
        const int contributionsPerThread = warp_size / threadsPerComponent;    // will be 32 / 4 = 8
        if (threadLocalId < c_virialAndEnergyCount * threadsPerComponent)
        {
            const int componentIndex        = threadLocalId / threadsPerComponent;
            const int threadComponentOffset = threadLocalId - componentIndex * threadsPerComponent;

            float     sum = 0.0f;
#pragma unroll
            for (int j = 0; j < contributionsPerThread; j++)
            {
                sum += sm_virialAndEnergy[componentIndex * blockSize + j * threadsPerComponent + threadComponentOffset];
            }
            atomicAdd(gm_virialAndEnergy + componentIndex, sum);
        }

        /* A naive reduction for debugging purposes */
        /*
           if (threadLocalId < blockSize)
           {
            sm_virialAndEnergy[sizing * threadLocalId + 0] = virxx;
            sm_virialAndEnergy[sizing * threadLocalId + 1] = viryy;
            sm_virialAndEnergy[sizing * threadLocalId + 2] = virzz;
            sm_virialAndEnergy[sizing * threadLocalId + 3] = virxy;
            sm_virialAndEnergy[sizing * threadLocalId + 4] = virxz;
            sm_virialAndEnergy[sizing * threadLocalId + 5] = viryz;
            sm_virialAndEnergy[sizing * threadLocalId + 6] = energy;
           }
           __syncthreads();
           #pragma unroll
           for (unsigned int stride = 1; stride < blockSize; stride <<= 1)
           {
            if ((threadLocalId % (stride << 1) == 0))
            {
           #pragma unroll
                for (int i = 0; i < sizing; i++)
                    sm_virialAndEnergy[sizing * threadLocalId + i] += sm_virialAndEnergy[sizing * (threadLocalId + stride) + i];
            }
            __syncthreads();
           }
           if (threadLocalId < sizing)
           {
            atomicAdd(virialAndEnergyGlobal + threadLocalId, sm_virialAndEnergy[threadLocalId]);
           }
         */
    }
}

void pme_gpu_solve(const pme_gpu_t *pmeGpu, t_complex *h_grid,
                   bool computeEnergyAndVirial, GridOrdering gridOrdering)
{
    /* do recip sum over local cells in grid */

    const bool   copyInputAndOutputGrid = pme_gpu_is_testing(pmeGpu) || !pme_gpu_performs_FFT(pmeGpu);

    hipStream_t stream          = pmeGpu->archSpecific->pmeStream;
    const auto  *kernelParamsPtr = pmeGpu->kernelParams.get();

    float2      *d_grid = (float2 *)kernelParamsPtr->grid.d_fourierGrid;
    if (copyInputAndOutputGrid)
    {
        cu_copy_H2D_async(d_grid, h_grid, pmeGpu->archSpecific->complexGridSize * sizeof(float), stream);
    }

    int majorDim = -1, middleDim = -1, minorDim = -1;
    switch (gridOrdering)
    {
        case GridOrdering::YZX:
            majorDim  = YY;
            middleDim = ZZ;
            minorDim  = XX;
            break;

        case GridOrdering::XYZ:
            majorDim  = XX;
            middleDim = YY;
            minorDim  = ZZ;
            break;

        default:
            GMX_ASSERT(false, "Implement grid ordering here and below for the kernel launch");
    }

    const int   maxBlockSize      = computeEnergyAndVirial ? PME_SOLVE_ENERVIR_THREADS_PER_BLOCK : PME_SOLVE_THREADS_PER_BLOCK;
    const int   gridLineSize      = pmeGpu->kernelParams->grid.complexGridSizePadded[minorDim];
    const int   gridLinesPerBlock = max(maxBlockSize / gridLineSize, 1);
    const int   blocksPerGridLine = (gridLineSize + maxBlockSize - 1) / maxBlockSize; // rounded up
    // Z-dimension is too small in CUDA limitations (64 on CC30?), so instead of major-middle-minor sizing we do minor-middle-major
    // FIXME: maybe that is why YZX fails occasionally with launch bounds?
    dim3 threads((maxBlockSize + gridLinesPerBlock - 1) / gridLinesPerBlock, gridLinesPerBlock);
    GMX_ASSERT((int)(threads.x * threads.y * threads.z) >= maxBlockSize, "Wrong PME GPU solve launch parameters");
    dim3 blocks(blocksPerGridLine,
                (pmeGpu->kernelParams->grid.complexGridSize[middleDim] + gridLinesPerBlock - 1) / gridLinesPerBlock, // rounded up middle dimension block number
                pmeGpu->kernelParams->grid.complexGridSize[majorDim]);

    pme_gpu_start_timing(pmeGpu, gtPME_SOLVE);
    if (gridOrdering == GridOrdering::YZX)
    {
        if (computeEnergyAndVirial)
        {
            pme_solve_kernel<true, GridOrderingInternal::YZX> <<< blocks, threads, 0, stream>>> (*kernelParamsPtr);
        }
        else
        {
            pme_solve_kernel<false, GridOrderingInternal::YZX> <<< blocks, threads, 0, stream>>> (*kernelParamsPtr);
        }
    }
    else if (gridOrdering == GridOrdering::XYZ)
    {
        if (computeEnergyAndVirial)
        {
            pme_solve_kernel<true, GridOrderingInternal::XYZ> <<< blocks, threads, 0, stream>>> (*kernelParamsPtr);
        }
        else
        {
            pme_solve_kernel<false, GridOrderingInternal::XYZ> <<< blocks, threads, 0, stream>>> (*kernelParamsPtr);
        }
    }
    CU_LAUNCH_ERR("pme_solve_kernel");
    pme_gpu_stop_timing(pmeGpu, gtPME_SOLVE);

    if (computeEnergyAndVirial)
    {
        cu_copy_D2H_async(pmeGpu->staging.h_virialAndEnergy, kernelParamsPtr->constants.d_virialAndEnergy,
                          c_virialAndEnergyCount * sizeof(float), stream);
        hipError_t stat = hipEventRecord(pmeGpu->archSpecific->syncEnerVirD2H, stream);
        CU_RET_ERR(stat, "PME solve energy/virial event record failure");
    }

    if (copyInputAndOutputGrid)
    {
        cu_copy_D2H_async(h_grid, d_grid, pmeGpu->archSpecific->complexGridSize * sizeof(float), stream);
        hipError_t stat = hipEventRecord(pmeGpu->archSpecific->syncSolveGridD2H, stream);
        CU_RET_ERR(stat, "PME solve grid sync event record failure");
    }
}
