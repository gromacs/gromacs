#include "hip/hip_runtime.h"
/*
 * This file is part of the GROMACS molecular simulation package.
 *
 * Copyright (c) 2018, by the GROMACS development team, led by
 * Mark Abraham, David van der Spoel, Berk Hess, and Erik Lindahl,
 * and including many others, as listed in the AUTHORS file in the
 * top-level source directory and at http://www.gromacs.org.
 *
 * GROMACS is free software; you can redistribute it and/or
 * modify it under the terms of the GNU Lesser General Public License
 * as published by the Free Software Foundation; either version 2.1
 * of the License, or (at your option) any later version.
 *
 * GROMACS is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
 * Lesser General Public License for more details.
 *
 * You should have received a copy of the GNU Lesser General Public
 * License along with GROMACS; if not, see
 * http://www.gnu.org/licenses, or write to the Free Software Foundation,
 * Inc., 51 Franklin Street, Fifth Floor, Boston, MA  02110-1301  USA.
 *
 * If you want to redistribute modifications to GROMACS, please
 * consider that scientific software is very special. Version
 * control is crucial - bugs must be traceable. We will be happy to
 * consider code for inclusion in the official distribution, but
 * derived work must not be called official GROMACS. Details are found
 * in the README & COPYING files - if they are missing, get the
 * official version at http://www.gromacs.org.
 *
 * To help us fund GROMACS development, we humbly ask that you cite
 * the research papers on the package. Check out http://www.gromacs.org.
 */
/*! \internal \file
 *
 * \brief Implements GPU bonded lists for CUDA
 *
 * \author Berk Hess <hess@kth.se>
 * \author Szilárd Páll <pall.szilard@gmail.com>
 * \author Mark Abraham <mark.j.abraham@gmail.com>
 *
 * \ingroup module_listed-forces
 */

#include "gmxpre.h"

#include "gpubonded-impl.h"

#include "gromacs/gpu_utils/cudautils.cuh"
#include "gromacs/gpu_utils/devicebuffer.h"
#include "gromacs/gpu_utils/gpu_vec.cuh"
#include "gromacs/gpu_utils/gputraits.cuh"
#include "gromacs/gpu_utils/hostallocator.h"
#include "gromacs/listed-forces/gpubonded.h"
#include "gromacs/mdtypes/enerdata.h"
#include "gromacs/topology/forcefieldparameters.h"
#include "gromacs/topology/idef.h"

struct t_forcerec;

namespace gmx
{

// ---- GpuBonded::Impl

GpuBonded::Impl::Impl(const gmx_ffparams_t &ffparams,
                      void                 *streamPtr)
{
    stream = *static_cast<CommandStream*>(streamPtr);

    allocateDeviceBuffer(&forceparamsDevice, ffparams.numTypes(), nullptr);
    // TODO can this be Async?
    copyToDeviceBuffer(&forceparamsDevice, ffparams.iparams.data(),
                       0, ffparams.numTypes(),
                       stream, GpuApiCallBehavior::Sync, nullptr);
    vtot.resize(F_NRE);
    allocateDeviceBuffer(&vtotDevice, F_NRE, nullptr);
    clearDeviceBufferAsync(&vtotDevice, 0, F_NRE, stream);

    for (int ftype = 0; ftype < F_NRE; ftype++)
    {
        iListsDevice[ftype].nr     = 0;
        iListsDevice[ftype].iatoms = nullptr;
        iListsDevice[ftype].nalloc = 0;
    }
}

GpuBonded::Impl::~Impl()
{
    for (int ftype : ftypesOnGpu)
    {
        if (iListsDevice[ftype].iatoms)
        {
            freeDeviceBuffer(&iListsDevice[ftype].iatoms);
            iListsDevice[ftype].iatoms = nullptr;
        }
    }

    freeDeviceBuffer(&forceparamsDevice);
    freeDeviceBuffer(&vtotDevice);
}

//! Return whether function type \p ftype in \p idef has perturbed interactions
static bool ftypeHasPerturbedEntries(const t_idef  &idef,
                                     int            ftype)
{
    GMX_ASSERT(idef.ilsort == ilsortNO_FE || idef.ilsort == ilsortFE_SORTED,
               "Perturbed interations should be sorted here");

    const t_ilist &ilist = idef.il[ftype];

    return (idef.ilsort != ilsortNO_FE && ilist.nr_nonperturbed != ilist.nr);
}

//! Converts \p src with atom indices in state order to \p dest in nbnxn order
static void convertIlistToNbnxnOrder(const t_ilist       &src,
                                     HostInteractionList *dest,
                                     int                  numAtomsPerInteraction,
                                     ArrayRef<const int>  nbnxnAtomOrder)
{
    GMX_ASSERT(src.size() == 0 || !nbnxnAtomOrder.empty(), "We need the nbnxn atom order");

    dest->iatoms.resize(src.size());

    for (int i = 0; i < src.size(); i += 1 + numAtomsPerInteraction)
    {
        dest->iatoms[i] = src.iatoms[i];
        for (int a = 0; a < numAtomsPerInteraction; a++)
        {
            dest->iatoms[i + 1 + a] = nbnxnAtomOrder[src.iatoms[i + 1 + a]];
        }
    }
}

//! Divides bonded interactions over threads and GPU
void
GpuBonded::Impl::updateAfterSearch(ArrayRef<const int>  nbnxnAtomOrder,
                                   const t_idef        &idef,
                                   void                *xqDevicePtr,
                                   void                *forceDevicePtr,
                                   void                *fshiftDevicePtr)
{
    haveInteractions_ = false;

    for (int ftype : ftypesOnGpu)
    {
        auto &iList = iLists[ftype];

        /* Perturbation is not implemented in the GPU bonded kernels.
         * But instead of doing all interactions on the CPU, we can
         * still easily handle the types that have no perturbed
         * interactions on the GPU. */
        if (idef.il[ftype].nr > 0 && !ftypeHasPerturbedEntries(idef, ftype))
        {
            haveInteractions_ = true;

            convertIlistToNbnxnOrder(idef.il[ftype],
                                     &iList,
                                     NRAL(ftype), nbnxnAtomOrder);
        }
        else
        {
            iList.iatoms.clear();
        }

        // Update the device if necessary. This can leave some
        // allocations on the device when the host size decreases to
        // zero, which is OK, since we deallocate everything at the
        // end.
        if (iList.size() > 0)
        {
            t_ilist &iListDevice = iListsDevice[ftype];

            reallocateDeviceBuffer(&iListDevice.iatoms, iList.size(), &iListDevice.nr, &iListDevice.nalloc, nullptr);

            copyToDeviceBuffer(&iListDevice.iatoms, iList.iatoms.data(),
                               0, iList.size(),
                               stream, GpuApiCallBehavior::Async, nullptr);
        }
    }

    xqDevice     = static_cast<float4 *>(xqDevicePtr);
    forceDevice  = static_cast<fvec *>(forceDevicePtr);
    fshiftDevice = static_cast<fvec *>(fshiftDevicePtr);
}

bool
GpuBonded::Impl::haveInteractions() const
{
    return haveInteractions_;
}

void
GpuBonded::Impl::transferEnergies(gmx_enerdata_t *enerd)
{
    GMX_ASSERT(haveInteractions_, "Cannot launch bonded kernels without work");

    float        *vtot_h   = vtot.data();
    copyFromDeviceBuffer(vtot_h, &vtotDevice,
                         0, F_NRE,
                         stream, GpuApiCallBehavior::Async, nullptr);
    hipError_t stat = hipStreamSynchronize(stream);
    CU_RET_ERR(stat, "D2H transfer failed");

    for (int ftype : ftypesOnGpu)
    {
        if (ftype != F_LJ14 && ftype != F_COUL14)
        {
            enerd->term[ftype] += vtot[ftype];
        }
    }

    // Note: We do not support energy groups here
    gmx_grppairener_t *grppener = &enerd->grpp;
    GMX_RELEASE_ASSERT(grppener->nener == 1, "No energy group support for bondeds on the GPU");
    grppener->ener[egLJ14][0]   += vtot[F_LJ14];
    grppener->ener[egCOUL14][0] += vtot[F_COUL14];
}

void
GpuBonded::Impl::clearEnergies()
{
    clearDeviceBufferAsync(&vtotDevice, 0, F_NRE, stream);
}

// ---- GpuBonded

GpuBonded::GpuBonded(const gmx_ffparams_t &ffparams,
                     void                 *streamPtr)
    : impl_(new Impl(ffparams, streamPtr))
{
}

GpuBonded::~GpuBonded() = default;

void
GpuBonded::updateAfterSearch(ArrayRef<const int>  nbnxnAtomOrder,
                             const t_idef        &idef,
                             void                *xqDevice,
                             void                *forceDevice,
                             void                *fshiftDevice)
{
    impl_->updateAfterSearch(nbnxnAtomOrder, idef, xqDevice, forceDevice, fshiftDevice);
}

bool
GpuBonded::haveInteractions() const
{
    return impl_->haveInteractions();
}

void
GpuBonded::transferEnergies(gmx_enerdata_t *enerd)
{
    impl_->transferEnergies(enerd);
}

void
GpuBonded::clearEnergies()
{
    impl_->clearEnergies();
}

}   // namespace gmx
