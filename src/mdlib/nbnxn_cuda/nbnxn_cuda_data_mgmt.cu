#include "hip/hip_runtime.h"
/* -*- mode: c; tab-width: 4; indent-tabs-mode: nil; c-basic-offset: 4; c-file-style: "stroustrup"; -*-
 *
 *
 *                This source code is part of
 *
 *                 G   R   O   M   A   C   S
 *
 *          GROningen MAchine for Chemical Simulations
 *
 * Written by David van der Spoel, Erik Lindahl, Berk Hess, and others.
 * Copyright (c) 1991-2000, University of Groningen, The Netherlands.
 * Copyright (c) 2001-2012, The GROMACS development team,
 * check out http://www.gromacs.org for more information.
 *
 * This program is free software; you can redistribute it and/or
 * modify it under the terms of the GNU General Public License
 * as published by the Free Software Foundation; either version 2
 * of the License, or (at your option) any later version.
 *
 * If you want to redistribute modifications, please consider that
 * scientific software is very special. Version control is crucial -
 * bugs must be traceable. We will be happy to consider code for
 * inclusion in the official distribution, but derived work must not
 * be called official GROMACS. Details are found in the README & COPYING
 * files - if they are missing, get the official version at www.gromacs.org.
 *
 * To help us fund GROMACS development, we humbly ask that you cite
 * the papers on the package - you can find them in the top README file.
 *
 * For more info, check our website at http://www.gromacs.org
 *
 * And Hey:
 * Gallium Rubidium Oxygen Manganese Argon Carbon Silicon
 */

#include <stdlib.h>
#include <stdio.h>

#include "config.h"
#include "gmx_fatal.h"
#include "smalloc.h"
#include "tables.h"
#include "types/nb_verlet.h"
#include "types/interaction_const.h"
#include "types/ishift.h"
#include "types/enums.h"
#include "types/force_flags.h"


#include "nbnxn_cuda_types.h"
#include "cudautils.cuh"
#include "nbnxn_cuda_data_mgmt.h"
#include "pmalloc_cuda.h"

#define USE_CUDA_EVENT_BLOCKING_SYNC FALSE  /* makes the CPU thread block */
/* coulomb force talble size chosen such that it fits along the non-bonded 
   parameters in the texture cache */
#define EWALD_COULOMB_FORCE_TABLE_SIZE (1536)

#define MY_PI               (3.1415926535897932384626433832795)
#define TWO_OVER_SQRT_PI    (2.0/sqrt(MY_PI))
    
#define TIME_GPU_TRANSFERS 1

#define NUM_NB_KERNELS 12

static void nbnxn_cuda_clear_e_fshift(nbnxn_cuda_ptr_t /*cu_nb*/);

/*! v1 nonbonded kernel names with names with mangling. */
static const char * const nb_k1_names[NUM_NB_KERNELS] = 
{
    "_Z12k_nbnxn_rf_111cu_atomdata10cu_nbparam8cu_plisti",
    "_Z15k_nbnxn_ewald_111cu_atomdata10cu_nbparam8cu_plisti",
    "_Z16k_nbnxn_cutoff_111cu_atomdata10cu_nbparam8cu_plisti",
    "_Z17k_nbnxn_rf_ener_111cu_atomdata10cu_nbparam8cu_plisti",
    "_Z18k_nbnxn_rf_prune_111cu_atomdata10cu_nbparam8cu_plisti",
    "_Z20k_nbnxn_ewald_ener_111cu_atomdata10cu_nbparam8cu_plisti",
    "_Z21k_nbnxn_ewald_prune_111cu_atomdata10cu_nbparam8cu_plisti",
    "_Z21k_nbnxn_cutoff_ener_111cu_atomdata10cu_nbparam8cu_plisti",
    "_Z22k_nbnxn_cutoff_prune_111cu_atomdata10cu_nbparam8cu_plisti",
    "_Z23k_nbnxn_rf_ener_prune_111cu_atomdata10cu_nbparam8cu_plisti",
    "_Z26k_nbnxn_ewald_ener_prune_111cu_atomdata10cu_nbparam8cu_plisti",
    "_Z27k_nbnxn_cutoff_ener_prune_111cu_atomdata10cu_nbparam8cu_plisti"
};

/*! v2 nonbonded kernel names with names with mangling. */
static const char * const nb_k2_names[NUM_NB_KERNELS] = 
{
    "_Z12k_nbnxn_rf_211cu_atomdata10cu_nbparam8cu_plisti",
    "_Z15k_nbnxn_ewald_211cu_atomdata10cu_nbparam8cu_plisti",
    "_Z16k_nbnxn_cutoff_211cu_atomdata10cu_nbparam8cu_plisti",
    "_Z17k_nbnxn_rf_ener_211cu_atomdata10cu_nbparam8cu_plisti",
    "_Z18k_nbnxn_rf_prune_211cu_atomdata10cu_nbparam8cu_plisti",
    "_Z20k_nbnxn_ewald_ener_211cu_atomdata10cu_nbparam8cu_plisti",
    "_Z21k_nbnxn_ewald_prune_211cu_atomdata10cu_nbparam8cu_plisti",
    "_Z21k_nbnxn_cutoff_ener_211cu_atomdata10cu_nbparam8cu_plisti",
    "_Z22k_nbnxn_cutoff_prune_211cu_atomdata10cu_nbparam8cu_plisti",
    "_Z23k_nbnxn_rf_ener_prune_211cu_atomdata10cu_nbparam8cu_plisti",
    "_Z26k_nbnxn_ewald_ener_prune_211cu_atomdata10cu_nbparam8cu_plisti",
    "_Z27k_nbnxn_cutoff_ener_prune_211cu_atomdata10cu_nbparam8cu_plisti"
};

/*! Dummy kernel used for sanity check. */
__device__ __global__ void k_empty_test(){}

/*! Tabulates the Ewald Coulomb force and initializes the size/scale 
    and the table GPU array. If called with an already allocated table,
    it just re-uploads the table.
 */
static void init_ewald_coulomb_force_table(cu_nbparam_t *nbp)
{
    float       *ftmp, *coul_tab;
    int         tabsize;
    double      tabscale;
    hipError_t stat;

    tabsize     = EWALD_COULOMB_FORCE_TABLE_SIZE;
    /* Subtract 2 iso 1 to avoid access out of range due to rounding */
    tabscale    = (tabsize - 2) / sqrt(nbp->rcoulomb_sq);

    pmalloc((void**)&ftmp, tabsize*sizeof(*ftmp));

    table_spline3_fill_ewald_lr(ftmp, NULL, tabsize, tableformatF,
                                1/tabscale, nbp->ewald_beta);

    /* If the table pointer == NULL the table is generated the first time =>
       the array pointer will be saved to nbparam and the texture is bound.
     */
    coul_tab = nbp->coulomb_tab;
    if (coul_tab == NULL)
    {
        stat = hipMalloc((void **)&coul_tab, tabsize*sizeof(*coul_tab));
        CU_RET_ERR(stat, "hipMalloc failed on coul_tab");

        nbp->coulomb_tab = coul_tab;
        cu_bind_texture("tex_coulomb_tab", coul_tab, tabsize*sizeof(*coul_tab));
    }

    cu_copy_H2D(coul_tab, ftmp, tabsize*sizeof(*coul_tab));

    nbp->coulomb_tab_size     = tabsize;
    nbp->coulomb_tab_scale    = tabscale;

    pfree(ftmp);
}


/*! Initilizes the atomdata structure first time, it only gets filled at 
    pair-search. */
static void init_atomdata_first(cu_atomdata_t *ad, int ntypes)
{
    hipError_t stat;

    ad->ntypes  = ntypes;
    stat = hipMalloc((void**)&ad->shift_vec, SHIFTS*sizeof(*ad->shift_vec));
    CU_RET_ERR(stat, "hipMalloc failed on ad->shift_vec"); 
    ad->shift_vec_uploaded = FALSE;

    stat = hipMalloc((void**)&ad->fshift, SHIFTS*sizeof(*ad->fshift));
    CU_RET_ERR(stat, "hipMalloc failed on ad->fshift");

    stat = hipMalloc((void**)&ad->e_lj, sizeof(*ad->e_lj));
    CU_RET_ERR(stat, "hipMalloc failed on ad->e_lj");
    stat = hipMalloc((void**)&ad->e_el, sizeof(*ad->e_el));
    CU_RET_ERR(stat, "hipMalloc failed on ad->e_el");

    /* initilize to NULL poiters to data that is not allocated here and will
       need reallocation in nbnxn_cuda_init_atomdata */
    ad->xq = NULL;
    ad->f  = NULL;

    /* size -1 indicates that the repective array hasn't been initialized yet */
    ad->natoms = -1;
    ad->nalloc = -1;
}

/*! Initilizes the nonbonded parameter data structure. */
static void init_nbparam(cu_nbparam_t *nbp,
                           const interaction_const_t *ic,
                           const nonbonded_verlet_t *nbv)
{  
    hipError_t stat;
    int         ntypes, nnbfp; 

    ntypes  = nbv->grp[0].nbat->ntype;
    
    nbp->ewald_beta = ic->ewaldcoeff;
    nbp->sh_ewald   = ic->sh_ewald;
    nbp->epsfac     = ic->epsfac;
    nbp->two_k_rf   = 2.0 * ic->k_rf;
    nbp->c_rf       = ic->c_rf;
    nbp->rvdw_sq    = ic->rvdw * ic->rvdw;
    nbp->rcoulomb_sq= ic->rcoulomb * ic->rcoulomb;
    nbp->rlist_sq   = ic->rlist * ic->rlist;
    nbp->sh_invrc6  = ic->sh_invrc6;

    if (ic->eeltype == eelCUT)
    {
        nbp->eeltype = cu_eelCUT;
    }
    else if (EEL_RF(ic->eeltype))
    {                
        nbp->eeltype = cu_eelRF;
    }
    else if ((EEL_PME(ic->eeltype) || ic->eeltype==eelEWALD))
    {
        nbp->eeltype = cu_eelEWALD;
    }
    else 
    {
        gmx_fatal(FARGS, "The requested electrostatics type is not implemented in the CUDA GPU accelerated kernels!");
    }

    /* generate table for PME */
    if (nbp->eeltype == cu_eelEWALD)
    {
        nbp->coulomb_tab = NULL;
        init_ewald_coulomb_force_table(nbp);
    }

    nnbfp = 2*ntypes*ntypes;
    stat = hipMalloc((void **)&nbp->nbfp, nnbfp*sizeof(*nbp->nbfp));
    CU_RET_ERR(stat, "hipMalloc failed on nbp->nbfp"); 
    cu_copy_H2D(nbp->nbfp, nbv->grp[0].nbat->nbfp, nnbfp*sizeof(*nbp->nbfp));
    cu_bind_texture("tex_nbfp", nbp->nbfp, nnbfp*sizeof(*nbp->nbfp));
}

void reset_gpu_rlist_ewaldtab(nbnxn_cuda_ptr_t cu_nb,
                              const interaction_const_t *ic)
{
    cu_nbparam_t *nbp = cu_nb->nbparam;

    nbp->rlist_sq       = ic->rlist * ic->rlist;
    nbp->rcoulomb_sq    = ic->rcoulomb * ic->rcoulomb;
    nbp->ewald_beta     = ic->ewaldcoeff;

    init_ewald_coulomb_force_table(cu_nb->nbparam);
}

/*! Initilizes the pair list data structure. */
static void init_plist(cu_plist_t *pl)
{
    /* initilize to NULL poiters to data that is not allocated here and will
       need reallocation in nbnxn_cuda_init_pairlist */
    pl->sci     = NULL;
    pl->cj4     = NULL;
    pl->excl    = NULL;
    
    /* size -1 indicates that the repective array hasn't been initialized yet */
    pl->na_c        = -1;
    pl->nsci        = -1;
    pl->sci_nalloc  = -1;
    pl->ncj4        = -1;
    pl->cj4_nalloc  = -1;
    pl->nexcl       = -1;
    pl->excl_nalloc = -1;
    pl->do_prune    = FALSE;
}

/*! Initilizes the timer data structure. */
static void init_timers(cu_timers_t *t, gmx_bool bDomDec)
{
    hipError_t stat;
    int eventflags = ( USE_CUDA_EVENT_BLOCKING_SYNC ? hipEventBlockingSync: hipEventDefault );

    stat = hipEventCreateWithFlags(&(t->start_atdat), eventflags);
    CU_RET_ERR(stat, "hipEventCreate on start_atdat failed");
    stat = hipEventCreateWithFlags(&(t->stop_atdat), eventflags);
    CU_RET_ERR(stat, "hipEventCreate on stop_atdat failed");

    /* The non-local counters/stream (second in the array) are needed only with DD. */
    for (int i = 0; i <= bDomDec ? 1 : 0; i++)
    {
        stat = hipEventCreateWithFlags(&(t->start_nb_k[i]), eventflags);
        CU_RET_ERR(stat, "hipEventCreate on start_nb_k failed");
        stat = hipEventCreateWithFlags(&(t->stop_nb_k[i]), eventflags);
        CU_RET_ERR(stat, "hipEventCreate on stop_nb_k failed");


        stat = hipEventCreateWithFlags(&(t->start_pl_h2d[i]), eventflags);
        CU_RET_ERR(stat, "hipEventCreate on start_pl_h2d failed");
        stat = hipEventCreateWithFlags(&(t->stop_pl_h2d[i]), eventflags);
        CU_RET_ERR(stat, "hipEventCreate on stop_pl_h2d failed");

        stat = hipEventCreateWithFlags(&(t->start_nb_h2d[i]), eventflags);
        CU_RET_ERR(stat, "hipEventCreate on start_nb_h2d failed");
        stat = hipEventCreateWithFlags(&(t->stop_nb_h2d[i]), eventflags);
        CU_RET_ERR(stat, "hipEventCreate on stop_nb_h2d failed");

        stat = hipEventCreateWithFlags(&(t->start_nb_d2h[i]), eventflags);
        CU_RET_ERR(stat, "hipEventCreate on start_nb_d2h failed");
        stat = hipEventCreateWithFlags(&(t->stop_nb_d2h[i]), eventflags);
        CU_RET_ERR(stat, "hipEventCreate on stop_nb_d2h failed");
    }
}

/*! Initilizes the timings data structure. */
static void init_timings(wallclock_gpu_t *t)
{
    int i, j;

    t->nb_h2d_t = 0.0;
    t->nb_d2h_t = 0.0;
    t->nb_c    = 0;
    t->pl_h2d_t = 0.0;
    t->pl_h2d_c = 0;
    for (i = 0; i < 2; i++)
    {
        for(j = 0; j < 2; j++)
        {
            t->ktime[i][j].t = 0.0;
            t->ktime[i][j].c = 0;
        }
    }
}

void nbnxn_cuda_init(FILE *fplog,
                     nbnxn_cuda_ptr_t *p_cu_nb,
                     gmx_bool bDomDec)
{
    hipError_t stat;
    nbnxn_cuda_ptr_t  nb;
    char sbuf[STRLEN];


    if (p_cu_nb == NULL) return;

    snew(nb, 1);
    snew(nb->dev_info, 1);
    snew(nb->atdat, 1);
    snew(nb->nbparam, 1);
    snew(nb->plist[eintLocal], 1);
    if (bDomDec)
    {
        snew(nb->plist[eintNonlocal], 1);
    }

    nb->dd_run = bDomDec;

    /* CUDA event timers don't work with multiple streams so 
       we have to disable timing with DD */
    nb->do_time = (!bDomDec && (getenv("GMX_DISABLE_CUDA_TIMING") == NULL));
    snew(nb->timers, 1);
    snew(nb->timings, 1);

    /* init nbst */
    pmalloc((void**)&nb->nbst.e_lj, sizeof(*nb->nbst.e_lj));
    pmalloc((void**)&nb->nbst.e_el, sizeof(*nb->nbst.e_el));
    pmalloc((void**)&nb->nbst.fshift, SHIFTS * sizeof(*nb->nbst.fshift));

    init_plist(nb->plist[eintLocal]);

    /* local/non-local GPU streams */
    stat = hipStreamCreate(&nb->stream[eintLocal]);
    CU_RET_ERR(stat, "hipStreamCreate on stream[eintLocal] failed");
    if (bDomDec)
    {
        init_plist(nb->plist[eintNonlocal]);
        stat = hipStreamCreate(&nb->stream[eintNonlocal]);
        CU_RET_ERR(stat, "hipStreamCreate on stream[eintNonlocal] failed");
    }

    /* init events for sychronization (timing disabled for performance reasons!) */
    stat = hipEventCreateWithFlags(&nb->nonlocal_done, hipEventDisableTiming);
    CU_RET_ERR(stat, "hipEventCreate on nonlocal_done failed");
    stat = hipEventCreateWithFlags(&nb->misc_ops_done, hipEventDisableTiming);
    CU_RET_ERR(stat, "hipEventCreate on misc_ops_one failed");

    /* init device info */
    /* FIXME this should not be done here! */
    stat = hipGetDevice(&nb->dev_info->dev_id);
    CU_RET_ERR(stat, "hipGetDevice failed");
    stat = hipGetDeviceProperties(&nb->dev_info->dev_prop, nb->dev_info->dev_id);
    CU_RET_ERR(stat, "hipGetDeviceProperties failed");

    /* If ECC is enabled hipStreamSynchronize introduces huge idling so we'll 
       switch to the (atmittedly fragile) memory polling waiting to preserve 
       performance. Event-timing also needs to be disabled. */

    gmx_bool bStreamSync    = getenv("GMX_CUDA_STREAMSYNC") != NULL;
    gmx_bool bNoStreamSync  = getenv("GMX_NO_CUDA_STREAMSYNC") != NULL;
    if (bStreamSync && bNoStreamSync)
    {
        gmx_fatal(FARGS, "Conflicting environment variables: both GMX_CUDA_STREAMSYNC and GMX_NO_CUDA_STREAMSYNC defined");
    }

    /* if we can't yield while poll-waiting, we need to use hipStreamSynchronize */
    if (nb->dev_info->dev_prop.ECCEnabled == 1)
    {
        if (bStreamSync)
        {
            nb->use_stream_sync = TRUE;
            nb->do_time         = TRUE;

            sprintf(sbuf, "NOTE: hipStreamSynchronize-based waiting forced by GMX_CUDA_STREAMSYNC, but ECC is turned\n"
                    "      on which generally causes considerable performance loss");
            fprintf(stderr, "\n%s\n", sbuf);
            if (fplog)
            {
                fprintf(fplog, "\n%s\n", sbuf);
            }
        }
        else
        {
#ifdef CANT_CUTHREAD_YIELD
            nb->use_stream_sync = TRUE;
            nb->do_time         = TRUE;

            sprintf(sbuf, "Can't do thread yield as sleep(0)/Sleep(0) (posix/win) is not available.\n"
                        "         Will use the standard hipStreamSynchronize-based waiting.\n"
                        "         Note that with ECC enabled this causes considerable performance loss.");
            gmx_warning("%s\n", sbuf);
            if (fplog)
            {
                fprintf(fplog, "\n%s\n", sbuf);
            }
#else
            nb->use_stream_sync = FALSE;
            nb->do_time         = FALSE;

            sprintf(sbuf, "NOTE: running on a GPU with ECC on; will not use hipStreamSynchronize-based\n"
                    "      waiting as it generally causes performance loss when used with ECC.");
            fprintf(stderr, "\n%s\n", sbuf);
            if (fplog)
            {
                fprintf(fplog, "\n%s\n", sbuf);
            }

#endif
        }
    }
    else
    {
        if (bNoStreamSync)
        {
            nb->use_stream_sync = FALSE;
            nb->do_time         = FALSE;

            sprintf(sbuf, "NOTE: running on a GPU with no ECC, but hipStreamSynchronize-based waiting\n"
                    "forced off by GMX_NO_CUDA_STREAMSYNC");
            fprintf(stderr, "\n%s\n", sbuf);
            if (fplog)
            {
                fprintf(fplog, "\n%s\n", sbuf);
            }
        }
        else
        {
            /* no/off ECC, hipStreamSynchronize not turned off by env. var. */
            nb->use_stream_sync = TRUE;
            nb->do_time         = TRUE;
        }
    }

    if (nb->do_time)
    {
        init_timers(nb->timers, bDomDec);
        init_timings(nb->timings);
    }

    *p_cu_nb = nb;

    if (debug)
    {
        fprintf(debug, "Initialized CUDA data structures.\n");
    }

    /* k_nbnxn_*_1 48/16 kB Shared/L1 */
    for (int i = 0; i < NUM_NB_KERNELS; i++)
    {
        stat = hipFuncSetCacheConfig(reinterpret_cast<const void*>(nb_k1_names[i]),  hipFuncCachePreferShared);
        CU_RET_ERR(stat, "hipFuncSetCacheConfig failed");
    }

    /* k_nbnxn_*_2 16/48 kB Shared/L1 */
    for (int i = 0; i < NUM_NB_KERNELS; i++)
    {
        stat = hipFuncSetCacheConfig(reinterpret_cast<const void*>(nb_k2_names[i]), hipFuncCachePreferL1);
        CU_RET_ERR(stat, "hipFuncSetCacheConfig failed");
    }

    /* TODO: move this to gpu_utils module */
    k_empty_test<<<1, 512>>>();
    CU_LAUNCH_ERR_SYNC("dummy test kernel");
}

void nbnxn_cuda_init_const(nbnxn_cuda_ptr_t cu_nb,
                           const interaction_const_t *ic,
                           const nonbonded_verlet_t *nbv)
{
    init_atomdata_first(cu_nb->atdat, nbv->grp[0].nbat->ntype);
    init_nbparam(cu_nb->nbparam, ic, nbv);

    /* clear energy and shift force outputs */
    nbnxn_cuda_clear_e_fshift(cu_nb);
}

void nbnxn_cuda_init_pairlist(nbnxn_cuda_ptr_t cu_nb,
                              const nbnxn_pairlist_t *h_plist,
                              int iloc)
{
    char         sbuf[STRLEN];
    hipError_t  stat;
    gmx_bool     do_time    = cu_nb->do_time;
    hipStream_t stream     = cu_nb->stream[iloc];
    cu_plist_t  *d_plist    = cu_nb->plist[iloc];

    if (d_plist->na_c < 0)
    {
        d_plist->na_c = h_plist->na_ci;
    }
    else
    {
        if (d_plist->na_c != h_plist->na_ci)
        {
            sprintf(sbuf, "In cu_init_plist: the #atoms per cell has changed (from %d to %d)",
                    d_plist->na_c, h_plist->na_ci);
            gmx_incons(sbuf);
        }
    }

    if (do_time)
    {
        stat = hipEventRecord(cu_nb->timers->start_pl_h2d[iloc], stream);
        CU_RET_ERR(stat, "hipEventRecord failed");
    }

    cu_realloc_buffered((void **)&d_plist->sci, h_plist->sci, sizeof(*d_plist->sci),
                         &d_plist->nsci, &d_plist->sci_nalloc,
                         h_plist->nsci,
                         stream, TRUE);

    cu_realloc_buffered((void **)&d_plist->cj4, h_plist->cj4, sizeof(*d_plist->cj4),
                         &d_plist->ncj4, &d_plist->cj4_nalloc,
                         h_plist->ncj4,
                         stream, TRUE);

    cu_realloc_buffered((void **)&d_plist->excl, h_plist->excl, sizeof(*d_plist->excl),
                         &d_plist->nexcl, &d_plist->excl_nalloc,
                         h_plist->nexcl,
                         stream, TRUE);

    if (do_time)
    {
        stat = hipEventRecord(cu_nb->timers->stop_pl_h2d[iloc], stream);
        CU_RET_ERR(stat, "hipEventRecord failed");
    }

    /* need to prune the pair list during the next step */
    d_plist->do_prune = TRUE;
}

void nbnxn_cuda_upload_shiftvec(nbnxn_cuda_ptr_t cu_nb,
                                const nbnxn_atomdata_t *nbatom)
{
    cu_atomdata_t *adat = cu_nb->atdat;
    hipStream_t  ls    = cu_nb->stream[eintLocal];

    /* only if we have a dynamic box */
    if (nbatom->dynamic_box || !adat->shift_vec_uploaded)
    {
        cu_copy_H2D_async(adat->shift_vec, nbatom->shift_vec, 
                          SHIFTS * sizeof(*adat->shift_vec), ls);
        adat->shift_vec_uploaded = TRUE;
    }
}

/*! Clears the first natoms_clear elements of the GPU nonbonded force output array. */
static void nbnxn_cuda_clear_f(nbnxn_cuda_ptr_t cu_nb, int natoms_clear)
{
    hipError_t   stat;
    cu_atomdata_t *adat = cu_nb->atdat;
    hipStream_t  ls    = cu_nb->stream[eintLocal];

    stat = hipMemsetAsync(adat->f, 0, natoms_clear * sizeof(*adat->f), ls);
    CU_RET_ERR(stat, "hipMemsetAsync on f falied");
}

/*! Clears nonbonded shift force output array and energy outputs on the GPU. */
static void nbnxn_cuda_clear_e_fshift(nbnxn_cuda_ptr_t cu_nb)
{
    hipError_t   stat;
    cu_atomdata_t *adat = cu_nb->atdat;
    hipStream_t  ls    = cu_nb->stream[eintLocal];

    stat = hipMemsetAsync(adat->fshift, 0, SHIFTS * sizeof(*adat->fshift), ls);
    CU_RET_ERR(stat, "hipMemsetAsync on fshift falied");
    stat = hipMemsetAsync(adat->e_lj, 0, sizeof(*adat->e_lj), ls);
    CU_RET_ERR(stat, "hipMemsetAsync on e_lj falied");
    stat = hipMemsetAsync(adat->e_el, 0, sizeof(*adat->e_el), ls);
    CU_RET_ERR(stat, "hipMemsetAsync on e_el falied");
}

void nbnxn_cuda_clear_outputs(nbnxn_cuda_ptr_t cu_nb, int flags)
{
    nbnxn_cuda_clear_f(cu_nb, cu_nb->atdat->natoms + 1);
    /* clear shift force array and energies if the outputs were 
       used in the current step */
    if (flags & GMX_FORCE_VIRIAL)
    {
        nbnxn_cuda_clear_e_fshift(cu_nb);
    }
}

/* TODO: add gmx over_alloc call */
void nbnxn_cuda_init_atomdata(nbnxn_cuda_ptr_t cu_nb,
                              const nbnxn_atomdata_t *nbat)
{
    hipError_t   stat;
    int           nalloc, natoms;
    gmx_bool      realloced;
    gmx_bool      do_time   = cu_nb->do_time;
    cu_timers_t   *timers   = cu_nb->timers;
    cu_atomdata_t *d_atdat  = cu_nb->atdat;
    hipStream_t  ls        = cu_nb->stream[eintLocal];

    natoms = nbat->natoms;
    realloced = FALSE;

    if (do_time)
    {
        /* time async copy */
        stat = hipEventRecord(timers->start_atdat, ls);
        CU_RET_ERR(stat, "hipEventRecord failed");
    }

    /* need to reallocate if we have to copy more atoms than the amount of space
       available and only allocate if we haven't initilzed yet, i.e d_atdat->natoms == -1 */
    /* Add one element to f for possible signalling */
    if (natoms + 1 > d_atdat->nalloc)
    {
        nalloc = (natoms + 1) * 1.2 + 100;
    
        /* free up first if the arrays have already been initialized */
        if (d_atdat->nalloc != -1)
        {
            cu_free_buffered(d_atdat->f, &d_atdat->natoms, &d_atdat->nalloc);
            cu_free_buffered(d_atdat->xq);
            cu_free_buffered(d_atdat->atom_types);
        }
        
        /* Add one element for possible signalling */
        stat = hipMalloc((void **)&d_atdat->f, nalloc*sizeof(*d_atdat->f));
        CU_RET_ERR(stat, "hipMalloc failed on d_atdat->f");
        stat = hipMalloc((void **)&d_atdat->xq, nalloc*sizeof(*d_atdat->xq));
        CU_RET_ERR(stat, "hipMalloc failed on d_atdat->xq");

        stat = hipMalloc((void **)&d_atdat->atom_types, nalloc*sizeof(*d_atdat->atom_types));
        CU_RET_ERR(stat, "hipMalloc failed on d_atdat->atom_types");

        d_atdat->nalloc = nalloc;
        realloced = TRUE;
    }

    d_atdat->natoms = natoms;
    d_atdat->natoms_local = nbat->natoms_local;

    /* need to clear GPU f output if realloc happened */
    if (realloced)
    {
        nbnxn_cuda_clear_f(cu_nb, nalloc);
    }

    cu_copy_H2D_async(d_atdat->atom_types, nbat->type,
                      natoms*sizeof(*d_atdat->atom_types), ls);

    if (do_time)
    {
        stat = hipEventRecord(timers->stop_atdat, ls);
        CU_RET_ERR(stat, "hipEventRecord failed");
    }
}

void nbnxn_cuda_free(FILE *fplog, nbnxn_cuda_ptr_t cu_nb, gmx_bool bDomDec)
{
    hipError_t     stat;
    cu_atomdata_t   *atdat;
    cu_nbparam_t    *nbparam;
    cu_plist_t      *plist, *plist_nl;
    cu_timers_t     *timers;

    if (cu_nb == NULL) return;

    atdat       = cu_nb->atdat;
    nbparam     = cu_nb->nbparam;
    plist       = cu_nb->plist[eintLocal];
    plist_nl    = cu_nb->plist[eintNonlocal];
    timers      = cu_nb->timers;

    if (nbparam->eeltype == cu_eelEWALD)
    {
        cu_unbind_texture("tex_coulomb_tab");
        cu_free_buffered(nbparam->coulomb_tab, &nbparam->coulomb_tab_size);
    }

    stat = hipEventDestroy(cu_nb->nonlocal_done);
    CU_RET_ERR(stat, "hipEventDestroy failed on timers->nonlocal_done");
    stat = hipEventDestroy(cu_nb->misc_ops_done);
    CU_RET_ERR(stat, "hipEventDestroy failed on timers->misc_ops_done");

    if (cu_nb->do_time)
    {
        stat = hipEventDestroy(timers->start_atdat);
        CU_RET_ERR(stat, "hipEventDestroy failed on timers->start_atdat");
        stat = hipEventDestroy(timers->stop_atdat);
        CU_RET_ERR(stat, "hipEventDestroy failed on timers->stop_atdat");

        /* The non-local counters/stream (second in the array) are needed only with DD. */
        for (int i = 0; i <= bDomDec ? 1 : 0; i++)
        {
            stat = hipEventDestroy(timers->start_nb_k[i]);
            CU_RET_ERR(stat, "hipEventDestroy failed on timers->start_nb_k");
            stat = hipEventDestroy(timers->stop_nb_k[i]);
            CU_RET_ERR(stat, "hipEventDestroy failed on timers->stop_nb_k");

            stat = hipEventDestroy(timers->start_pl_h2d[i]);
            CU_RET_ERR(stat, "hipEventDestroy failed on timers->start_pl_h2d");
            stat = hipEventDestroy(timers->stop_pl_h2d[i]);
            CU_RET_ERR(stat, "hipEventDestroy failed on timers->stop_pl_h2d");

            stat = hipStreamDestroy(cu_nb->stream[i]);
            CU_RET_ERR(stat, "hipStreamDestroy failed on stream");

            stat = hipEventDestroy(timers->start_nb_h2d[i]);
            CU_RET_ERR(stat, "hipEventDestroy failed on timers->start_nb_h2d");
            stat = hipEventDestroy(timers->stop_nb_h2d[i]);
            CU_RET_ERR(stat, "hipEventDestroy failed on timers->stop_nb_h2d");

            stat = hipEventDestroy(timers->start_nb_d2h[i]);
            CU_RET_ERR(stat, "hipEventDestroy failed on timers->start_nb_d2h");
            stat = hipEventDestroy(timers->stop_nb_d2h[i]);
            CU_RET_ERR(stat, "hipEventDestroy failed on timers->stop_nb_d2h");
        }
    }

    cu_unbind_texture("tex_nbfp");
    cu_free_buffered(nbparam->nbfp);

    stat = hipFree(atdat->shift_vec);
    CU_RET_ERR(stat, "hipEventDestroy failed on atdat->shift_vec");
    stat = hipFree(atdat->fshift);
    CU_RET_ERR(stat, "hipEventDestroy failed on atdat->fshift");

    stat = hipFree(atdat->e_lj);
    CU_RET_ERR(stat, "hipEventDestroy failed on atdat->e_lj");
    stat = hipFree(atdat->e_el);
    CU_RET_ERR(stat, "hipEventDestroy failed on atdat->e_el");

    cu_free_buffered(atdat->f, &atdat->natoms, &atdat->nalloc);
    cu_free_buffered(atdat->xq);
    cu_free_buffered(atdat->atom_types, &atdat->ntypes);

    cu_free_buffered(plist->sci, &plist->nsci, &plist->sci_nalloc);
    cu_free_buffered(plist->cj4, &plist->ncj4, &plist->cj4_nalloc);
    cu_free_buffered(plist->excl, &plist->nexcl, &plist->excl_nalloc);
    if (bDomDec)
    {
        cu_free_buffered(plist_nl->sci, &plist_nl->nsci, &plist_nl->sci_nalloc);
        cu_free_buffered(plist_nl->cj4, &plist_nl->ncj4, &plist_nl->cj4_nalloc);
        cu_free_buffered(plist_nl->excl, &plist_nl->nexcl, &plist->excl_nalloc);
    }

    stat = hipDeviceReset();
    CU_RET_ERR(stat, "hipDeviceReset failed");

    if (debug)
    {
        fprintf(debug, "Cleaned up CUDA data structures.\n");
    }
}

void cu_synchstream_atdat(nbnxn_cuda_ptr_t cu_nb, int iloc)
{
    hipError_t stat;
    hipStream_t stream = cu_nb->stream[iloc];

    stat = hipStreamWaitEvent(stream, cu_nb->timers->stop_atdat, 0);
    CU_RET_ERR(stat, "hipStreamWaitEvent failed");
}

wallclock_gpu_t * nbnxn_cuda_get_timings(nbnxn_cuda_ptr_t cu_nb)
{
    return (cu_nb != NULL && cu_nb->do_time) ? cu_nb->timings : NULL;
}

void nbnxn_cuda_reset_timings(nbnxn_cuda_ptr_t cu_nb)
{
    if (cu_nb->do_time)
    {
        init_timings(cu_nb->timings);
    }
}

int nbnxn_cuda_min_ci_balanced(nbnxn_cuda_ptr_t cu_nb)
{
    return cu_nb != NULL ? 
        GPU_MIN_CI_BALANCED_FACTOR*cu_nb->dev_info->dev_prop.multiProcessorCount : 0;
}


/****** FIXME Old stuff, mostly deprecated, remove before release  *****/

/* Upload asynchronously to the GPU the coordinate+charge array.
 * XXX not used  
 */
void cu_move_xq(nbnxn_cuda_ptr_t cu_nb, const nbnxn_atomdata_t *nbat, int aloc)
{
    int iloc = -1; 

    /* determine interaction locality from atom locality 
       (needed for indexing timers/streams) */
    if (LOCAL_A(aloc))
    {
        iloc = eintLocal;
    }
    else if (NONLOCAL_A(aloc))
    {
        iloc = eintNonlocal;
    }
    else 
    {
        gmx_incons("Invalid atom locality passed (valid here is only local or nonlocal)");
    }

    cu_atomdata_t   *d_nbat = cu_nb->atdat;
    hipStream_t    stream  = cu_nb->stream[iloc];

    cu_copy_H2D_async(d_nbat->xq, nbat->x,
                        d_nbat->natoms * sizeof(*d_nbat->xq), stream);
}

/*! Waits until the atom data gets copied to the GPU and times the transfer.
 *  XXX not used  
 */
void cu_wait_atdat(nbnxn_cuda_ptr_t cu_nb)
{
    float t;
    cu_wait_event_time(cu_nb->timers->stop_atdat, cu_nb->timers->start_atdat, &t);
    cu_nb->timings->pl_h2d_t += t;
}
